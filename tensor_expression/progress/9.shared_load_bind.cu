@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float32), float32, [268435456], []),
             B: Buffer(B_2: Pointer(float32), float32, [268435456], []),
             C: Buffer(C_2: Pointer(float32), float32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float32, [16384, 16384], []), B_1: B_3: Buffer(B_2, float32, [16384, 16384], []), C_1: C_3: Buffer(C_2, float32, [16384, 16384], [])} {
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128;
  allocate(C.local: Pointer(local float32), float32, [64]), storage_scope = local;
  allocate(A.shared: Pointer(shared float32), float32, [2048]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float32), float32, [2048]), storage_scope = shared;
  allocate(A.shared.local: Pointer(local float32), float32, [8]), storage_scope = local;
  allocate(B.shared.local: Pointer(local float32), float32, [8]), storage_scope = local;
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
  attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 16;
  attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 16 {
    for (ii.c.init: int32, 0, 8) {
      for (jj.c.init: int32, 0, 8) {
        C.local_1: Buffer(C.local, float32, [64], [], scope="local")[((ii.c.init*8) + jj.c.init)] = 0f32
      }
    }
    for (k.outer: int32, 0, 1024) {
      for (ax1.inner: int32, 0, 8) {
        A.shared_1: Buffer(A.shared, float32, [2048], [], scope="shared")[(((threadIdx.x*128) + (threadIdx.y*8)) + ax1.inner)] = A[(((((k.outer*262144) + (threadIdx.x*16384)) + (blockIdx.y*128)) + (threadIdx.y*8)) + ax1.inner)]
      }
      for (ax1.inner_1: int32, 0, 8) {
        B.shared_1: Buffer(B.shared, float32, [2048], [], scope="shared")[(((threadIdx.x*128) + (threadIdx.y*8)) + ax1.inner_1)] = B[(((((k.outer*262144) + (threadIdx.x*16384)) + (blockIdx.x*128)) + (threadIdx.y*8)) + ax1.inner_1)]
      }
      for (k.inner: int32, 0, 16) {
        for (ax1: int32, 0, 8) {
          A.shared.local_1: Buffer(A.shared.local, float32, [8], [], scope="local", align=32)[ax1] = A.shared_1[(((k.inner*128) + (threadIdx.y*8)) + ax1)]
        }
        for (ax1_1: int32, 0, 8) {
          B.shared.local_1: Buffer(B.shared.local, float32, [8], [], scope="local", align=32)[ax1_1] = B.shared_1[(((k.inner*128) + (threadIdx.x*8)) + ax1_1)]
        }
        for (ii.c: int32, 0, 8) {
          for (jj.c: int32, 0, 8) {
            let cse_var_1: int32 = ((ii.c*8) + jj.c)
            C.local_1[cse_var_1] = (C.local_1[cse_var_1] + (A.shared.local_1[jj.c]*B.shared.local_1[ii.c]))
          }
        }
      }
    }
    for (ii.inner.inner: int32, 0, 8) {
      for (jj.inner.inner: int32, 0, 8) {
        C[((((((blockIdx.x*2097152) + (threadIdx.x*131072)) + (ii.inner.inner*16384)) + (blockIdx.y*128)) + (threadIdx.y*8)) + jj.inner.inner)] = C.local_1[((ii.inner.inner*8) + jj.inner.inner)]
      }
    }
  }
}

