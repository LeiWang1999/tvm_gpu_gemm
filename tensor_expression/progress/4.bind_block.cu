@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float32), float32, [268435456], []),
             B: Buffer(B_2: Pointer(float32), float32, [268435456], []),
             C: Buffer(C_2: Pointer(float32), float32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float32, [16384, 16384], []), B_1: B_3: Buffer(B_2, float32, [16384, 16384], []), C_1: C_3: Buffer(C_2, float32, [16384, 16384], [])} {
  allocate(A.shared: Pointer(shared float32), float32, [2097152]), storage_scope = shared;
  allocate(A.shared.local: Pointer(local float32), float32, [2097152]), storage_scope = local;
  allocate(B.shared.local: Pointer(local float32), float32, [2097152]), storage_scope = local;
  allocate(C.local: Pointer(local float32), float32, [16384]), storage_scope = local {
    for (ax0: int32, 0, 16384) {
      for (ax1: int32, 0, 128) {
        A.shared_1: Buffer(A.shared, float32, [2097152], [], scope="shared")[((ax0*128) + ax1)] = A[(((ax0*16384) + (blockIdx.y: int32*128)) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 16384) {
      for (ax1_1: int32, 0, 128) {
        let cse_var_1: int32 = ((ax0_1*128) + ax1_1)
        A.shared.local_1: Buffer(A.shared.local, float32, [2097152], [], scope="local")[cse_var_1] = A.shared_1[cse_var_1]
      }
    }
    for (ax0_2: int32, 0, 16384) {
      for (ax1_2: int32, 0, 128) {
        A.shared_2: Buffer(A.shared, float32, [2097152], [], scope="shared")[((ax0_2*128) + ax1_2)] = B[(((ax0_2*16384) + (blockIdx.x: int32*128)) + ax1_2)]
      }
    }
    for (ax0_3: int32, 0, 16384) {
      for (ax1_3: int32, 0, 128) {
        let cse_var_2: int32 = ((ax0_3*128) + ax1_3)
        B.shared.local_1: Buffer(B.shared.local, float32, [2097152], [], scope="local")[cse_var_2] = A.shared_2[cse_var_2]
      }
    }
    for (ii.c: int32, 0, 128) {
      for (jj.c: int32, 0, 128) {
        C.local_1: Buffer(C.local, float32, [16384], [], scope="local")[((ii.c*128) + jj.c)] = 0f32
        for (k: int32, 0, 16384) {
          let cse_var_4: int32 = (k*128)
          let cse_var_3: int32 = ((ii.c*128) + jj.c)
          C.local_1[cse_var_3] = (C.local_1[cse_var_3] + (A.shared.local_1[(cse_var_4 + jj.c)]*B.shared.local_1[(cse_var_4 + ii.c)]))
        }
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128;
    for (ii.inner: int32, 0, 128) {
      for (jj.inner: int32, 0, 128) {
        C[((((blockIdx.x*2097152) + (ii.inner*16384)) + (blockIdx.y*128)) + jj.inner)] = C.local_1[((ii.inner*128) + jj.inner)]
      }
    }
  }
}

