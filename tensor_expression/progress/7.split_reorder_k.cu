@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float32), float32, [268435456], []),
             B: Buffer(B_2: Pointer(float32), float32, [268435456], []),
             C: Buffer(C_2: Pointer(float32), float32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float32, [16384, 16384], []), B_1: B_3: Buffer(B_2, float32, [16384, 16384], []), C_1: C_3: Buffer(C_2, float32, [16384, 16384], [])} {
  allocate(A.shared: Pointer(shared float32), float32, [131072]), storage_scope = shared;
  allocate(A.shared.local: Pointer(local float32), float32, [131072]), storage_scope = local;
  allocate(B.shared.local: Pointer(local float32), float32, [131072]), storage_scope = local {
    for (ax0: int32, 0, 16384) {
      for (ax1: int32, 0, 8) {
        A.shared_1: Buffer(A.shared, float32, [131072], [], scope="shared")[((ax0*8) + ax1)] = A[((((ax0*16384) + (blockIdx.y: int32*128)) + (threadIdx.y: int32*8)) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 16384) {
      for (ax1_1: int32, 0, 8) {
        let cse_var_1: int32 = ((ax0_1*8) + ax1_1)
        A.shared.local_1: Buffer(A.shared.local, float32, [131072], [], scope="local")[cse_var_1] = A.shared_1[cse_var_1]
      }
    }
    for (ax0_2: int32, 0, 16384) {
      for (ax1_2: int32, 0, 8) {
        A.shared_2: Buffer(A.shared, float32, [131072], [], scope="shared")[((ax0_2*8) + ax1_2)] = B[((((ax0_2*16384) + (blockIdx.x: int32*128)) + (threadIdx.x: int32*8)) + ax1_2)]
      }
    }
    for (ax0_3: int32, 0, 16384) {
      for (ax1_3: int32, 0, 8) {
        let cse_var_2: int32 = ((ax0_3*8) + ax1_3)
        B.shared.local_1: Buffer(B.shared.local, float32, [131072], [], scope="local")[cse_var_2] = A.shared_2[cse_var_2]
      }
    }
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128;
    allocate(C.local: Pointer(local float32), float32, [64]), storage_scope = local;
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
    attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 16;
    attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 16 {
      for (ii.c.init: int32, 0, 8) {
        for (jj.c.init: int32, 0, 8) {
          C.local_1: Buffer(C.local, float32, [64], [], scope="local")[((ii.c.init*8) + jj.c.init)] = 0f32
        }
      }
      for (k.outer: int32, 0, 1024) {
        for (k.inner: int32, 0, 16) {
          for (ii.c: int32, 0, 8) {
            for (jj.c: int32, 0, 8) {
              let cse_var_4: int32 = ((ii.c*8) + jj.c)
              let cse_var_3: int32 = ((k.outer*128) + (k.inner*8))
              C.local_1[cse_var_4] = (C.local_1[cse_var_4] + (A.shared.local_1[(cse_var_3 + jj.c)]*B.shared.local_1[(cse_var_3 + ii.c)]))
            }
          }
        }
      }
      for (ii.inner.inner: int32, 0, 8) {
        for (jj.inner.inner: int32, 0, 8) {
          C[((((((blockIdx.x*2097152) + (threadIdx.x*131072)) + (ii.inner.inner*16384)) + (blockIdx.y*128)) + (threadIdx.y*8)) + jj.inner.inner)] = C.local_1[((ii.inner.inner*8) + jj.inner.inner)]
        }
      }
    }
  }
}

