
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  float C_local[64];
  __shared__ float A_shared[2048];
  __shared__ float B_shared[2048];
  float A_shared_local[8];
  float B_shared_local[8];
  for (int ii_c_init = 0; ii_c_init < 8; ++ii_c_init) {
    for (int jj_c_init = 0; jj_c_init < 8; ++jj_c_init) {
      C_local[((ii_c_init * 8) + jj_c_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 1024; ++k_outer) {
    __syncthreads();
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      A_shared[(((((int)threadIdx.x) * 128) + (((int)threadIdx.y) * 8)) + ax1_inner)] = A[(((((k_outer * 262144) + (((int)threadIdx.x) * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 8)) + ax1_inner)];
    }
    for (int ax1_inner1 = 0; ax1_inner1 < 8; ++ax1_inner1) {
      B_shared[(((((int)threadIdx.x) * 128) + (((int)threadIdx.y) * 8)) + ax1_inner1)] = B[(((((k_outer * 262144) + (((int)threadIdx.x) * 16384)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.y) * 8)) + ax1_inner1)];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 16; ++k_inner) {
      for (int ax1 = 0; ax1 < 8; ++ax1) {
        A_shared_local[ax1] = A_shared[(((k_inner * 128) + (((int)threadIdx.y) * 8)) + ax1)];
      }
      for (int ax11 = 0; ax11 < 8; ++ax11) {
        B_shared_local[ax11] = B_shared[(((k_inner * 128) + (((int)threadIdx.x) * 8)) + ax11)];
      }
      for (int ii_c = 0; ii_c < 8; ++ii_c) {
        for (int jj_c = 0; jj_c < 8; ++jj_c) {
          C_local[((ii_c * 8) + jj_c)] = (C_local[((ii_c * 8) + jj_c)] + (A_shared_local[jj_c] * B_shared_local[ii_c]));
        }
      }
    }
  }
  for (int ii_inner_inner = 0; ii_inner_inner < 8; ++ii_inner_inner) {
    for (int jj_inner_inner = 0; jj_inner_inner < 8; ++jj_inner_inner) {
      C[((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 131072)) + (ii_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 8)) + jj_inner_inner)] = C_local[((ii_inner_inner * 8) + jj_inner_inner)];
    }
  }
}

