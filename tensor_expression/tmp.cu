
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  float C_local[64];
  __shared__ float4 A_shared[512];
  __shared__ float4 B_shared[512];
  float A_shared_local[8];
  float B_shared_local[8];
  for (int ii_c_init = 0; ii_c_init < 4; ++ii_c_init) {
    for (int jj_c_init = 0; jj_c_init < 4; ++jj_c_init) {
      C_local[((ii_c_init * 4) + jj_c_init)] = 0.000000e+00f;
      C_local[(((ii_c_init * 4) + jj_c_init) + 32)] = 0.000000e+00f;
      C_local[(((ii_c_init * 4) + jj_c_init) + 16)] = 0.000000e+00f;
      C_local[(((ii_c_init * 4) + jj_c_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 1024; ++k_outer) {
    __syncthreads();
    for (int ax1_outer = 0; ax1_outer < 2; ++ax1_outer) {
      A_shared[(((((int)threadIdx.y) * 32) + (ax1_outer * 16)) + ((int)threadIdx.x))] = *(float4*)(A + (((((k_outer * 262144) + (((int)threadIdx.y) * 16384)) + (((int)blockIdx.y) * 128)) + (ax1_outer * 64)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax1_outer1 = 0; ax1_outer1 < 2; ++ax1_outer1) {
      B_shared[(((((int)threadIdx.y) * 32) + (ax1_outer1 * 16)) + ((int)threadIdx.x))] = *(float4*)(B + (((((k_outer * 262144) + (((int)threadIdx.y) * 16384)) + (((int)blockIdx.x) * 128)) + (ax1_outer1 * 64)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 16; ++k_inner) {
      *(float4*)(A_shared_local + 0) = A_shared[((k_inner * 32) + ((int)threadIdx.y))];
      *(float4*)(A_shared_local + 4) = A_shared[(((k_inner * 32) + ((int)threadIdx.y)) + 16)];
      *(float4*)(B_shared_local + 0) = B_shared[((k_inner * 32) + ((int)threadIdx.x))];
      *(float4*)(B_shared_local + 4) = B_shared[(((k_inner * 32) + ((int)threadIdx.x)) + 16)];
      for (int ii_c = 0; ii_c < 4; ++ii_c) {
        for (int jj_c = 0; jj_c < 4; ++jj_c) {
          C_local[((ii_c * 4) + jj_c)] = (C_local[((ii_c * 4) + jj_c)] + (A_shared_local[jj_c] * B_shared_local[ii_c]));
          C_local[(((ii_c * 4) + jj_c) + 32)] = (C_local[(((ii_c * 4) + jj_c) + 32)] + (A_shared_local[(jj_c + 4)] * B_shared_local[ii_c]));
          C_local[(((ii_c * 4) + jj_c) + 16)] = (C_local[(((ii_c * 4) + jj_c) + 16)] + (A_shared_local[jj_c] * B_shared_local[(ii_c + 4)]));
          C_local[(((ii_c * 4) + jj_c) + 48)] = (C_local[(((ii_c * 4) + jj_c) + 48)] + (A_shared_local[(jj_c + 4)] * B_shared_local[(ii_c + 4)]));
        }
      }
    }
  }
  for (int ii_inner_inner_inner = 0; ii_inner_inner_inner < 4; ++ii_inner_inner_inner) {
    for (int jj_inner_inner_inner = 0; jj_inner_inner_inner < 4; ++jj_inner_inner_inner) {
      C[((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 65536)) + (ii_inner_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 4)) + jj_inner_inner_inner)] = C_local[((ii_inner_inner_inner * 4) + jj_inner_inner_inner)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 65536)) + (ii_inner_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 4)) + jj_inner_inner_inner) + 64)] = C_local[(((ii_inner_inner_inner * 4) + jj_inner_inner_inner) + 32)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 65536)) + (ii_inner_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 4)) + jj_inner_inner_inner) + 1048576)] = C_local[(((ii_inner_inner_inner * 4) + jj_inner_inner_inner) + 16)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 65536)) + (ii_inner_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 4)) + jj_inner_inner_inner) + 1048640)] = C_local[(((ii_inner_inner_inner * 4) + jj_inner_inner_inner) + 48)];
    }
  }
}

