// GLOBALS: input0:float16[16384, 16384] -> output0:float16[1024, 1024, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 16, 8, 4], "F___output0:D1": [-1, 2, 1, 1], "F___output0:O": [1, 0], "F___output0:S": 1, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[M / 16, N / 16, M % 16, N % 16] =. input0[M, N]", input_dict={"input0": {"dtype": "float16", "shape": [16384, 16384]}, "output0": {"dtype": "float16", "shape": [1024, 1024, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[16384, 16384] -> output0:float16[1024, 1024, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(8) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 32
  // [thread_extent] threadIdx.x = 8
  // [thread_extent] blockIdx.y = 8192
  // [thread_extent] threadIdx.y = 1
  for (int vthread_s = 0; vthread_s < 16; ++vthread_s) {
    ((output0[((((((((((int)blockIdx.y) / 8) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((int)threadIdx.x) / 4) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + ((((int)threadIdx.x) & 3) * 4)))]) = (input0[(((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)))]));
    ((output0[((((((((((((int)blockIdx.y) * 2) + 1) / 16) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((int)threadIdx.x) / 4) * 256)) + ((((((int)blockIdx.y) * 2) + 1) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 16384))]));
    ((output0[((((((((((int)blockIdx.y) / 8) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 1) / 16) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + (((((int)threadIdx.x) * 4) + 1) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 1))]));
    ((output0[((((((((((((int)blockIdx.y) * 2) + 1) / 16) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 1) / 16) * 256)) + ((((((int)blockIdx.y) * 2) + 1) & 15) * 16)) + (((((int)threadIdx.x) * 4) + 1) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 16385))]));
    ((output0[((((((((((int)blockIdx.y) / 8) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 2) / 16) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + (((((int)threadIdx.x) * 4) + 2) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 2))]));
    ((output0[((((((((((((int)blockIdx.y) * 2) + 1) / 16) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 2) / 16) * 256)) + ((((((int)blockIdx.y) * 2) + 1) & 15) * 16)) + (((((int)threadIdx.x) * 4) + 2) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 16386))]));
    ((output0[((((((((((int)blockIdx.y) / 8) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 3) / 16) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + (((((int)threadIdx.x) * 4) + 3) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 3))]));
    ((output0[((((((((((((int)blockIdx.y) * 2) + 1) / 16) * 262144) + (((int)blockIdx.x) * 8192)) + (vthread_s * 512)) + ((((((int)threadIdx.x) * 4) + 3) / 16) * 256)) + ((((((int)blockIdx.y) * 2) + 1) & 15) * 16)) + (((((int)threadIdx.x) * 4) + 3) & 15)))]) = (input0[((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (vthread_s * 32)) + (((int)threadIdx.x) * 4)) + 16387))]));
  }
}

// Saved Perf = 1.256140e-03 sec / run; Step Produced = 473; Planned Steps = 1000;