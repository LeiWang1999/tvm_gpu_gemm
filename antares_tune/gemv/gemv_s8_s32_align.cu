// GLOBALS: input0:int8[18966528, 32], input1:int8[1, 32] -> output0:int32[18966528, 32]
// BACKEND: c-cuda (default)
// CONFIG: {"Moutput0T": 0, "Moutput0:D0": [-1, 2, 16, 7], "Moutput0:D1": [-1, 2, 8, 2], "Moutput0:R0": [-1, 2, 16], "Moutput0:RA": 0, "Moutput0:AL0": 0, "Moutput0:AL1": 1, "Moutput0:S": 3, "Moutput0:U": 1}
// COMPUTE_V1: - einstein_v2("output0[N, M] +=! input0[N, K].cast(`int32`) * input1[K, M].cast(`int32`)", { "input0": {"dtype": "int8", "shape": [18966528, 32]}, "input1": {"dtype": "int8", "shape": [1, 32]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[18966528, 32], input1:int8[1, 32] -> output0:int32[18966528, 32]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(128) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ input1, int* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 84672
  // [thread_extent] threadIdx.x = 128
  int output0_local[56];
  output0_local[(0)] = 0;
  output0_local[(14)] = 0;
  output0_local[(28)] = 0;
  output0_local[(42)] = 0;
  output0_local[(1)] = 0;
  output0_local[(15)] = 0;
  output0_local[(29)] = 0;
  output0_local[(43)] = 0;
  output0_local[(2)] = 0;
  output0_local[(16)] = 0;
  output0_local[(30)] = 0;
  output0_local[(44)] = 0;
  output0_local[(3)] = 0;
  output0_local[(17)] = 0;
  output0_local[(31)] = 0;
  output0_local[(45)] = 0;
  output0_local[(4)] = 0;
  output0_local[(18)] = 0;
  output0_local[(32)] = 0;
  output0_local[(46)] = 0;
  output0_local[(5)] = 0;
  output0_local[(19)] = 0;
  output0_local[(33)] = 0;
  output0_local[(47)] = 0;
  output0_local[(6)] = 0;
  output0_local[(20)] = 0;
  output0_local[(34)] = 0;
  output0_local[(48)] = 0;
  output0_local[(7)] = 0;
  output0_local[(21)] = 0;
  output0_local[(35)] = 0;
  output0_local[(49)] = 0;
  output0_local[(8)] = 0;
  output0_local[(22)] = 0;
  output0_local[(36)] = 0;
  output0_local[(50)] = 0;
  output0_local[(9)] = 0;
  output0_local[(23)] = 0;
  output0_local[(37)] = 0;
  output0_local[(51)] = 0;
  output0_local[(10)] = 0;
  output0_local[(24)] = 0;
  output0_local[(38)] = 0;
  output0_local[(52)] = 0;
  output0_local[(11)] = 0;
  output0_local[(25)] = 0;
  output0_local[(39)] = 0;
  output0_local[(53)] = 0;
  output0_local[(12)] = 0;
  output0_local[(26)] = 0;
  output0_local[(40)] = 0;
  output0_local[(54)] = 0;
  output0_local[(13)] = 0;
  output0_local[(27)] = 0;
  output0_local[(41)] = 0;
  output0_local[(55)] = 0;
  __shared__ char input0_shared[7168];
  // [thread_extent] threadIdx.x = 128
  input0_shared[(((int)threadIdx.x))] = input0[(((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)))];
  input0_shared[((((int)threadIdx.x) + 128))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 128))];
  input0_shared[((((int)threadIdx.x) + 256))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 256))];
  input0_shared[((((int)threadIdx.x) + 384))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 384))];
  input0_shared[((((int)threadIdx.x) + 512))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 512))];
  input0_shared[((((int)threadIdx.x) + 640))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 640))];
  input0_shared[((((int)threadIdx.x) + 768))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 768))];
  input0_shared[((((int)threadIdx.x) + 896))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 896))];
  input0_shared[((((int)threadIdx.x) + 1024))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1024))];
  input0_shared[((((int)threadIdx.x) + 1152))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1152))];
  input0_shared[((((int)threadIdx.x) + 1280))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1280))];
  input0_shared[((((int)threadIdx.x) + 1408))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1408))];
  input0_shared[((((int)threadIdx.x) + 1536))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1536))];
  input0_shared[((((int)threadIdx.x) + 1664))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1664))];
  input0_shared[((((int)threadIdx.x) + 1792))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1792))];
  input0_shared[((((int)threadIdx.x) + 1920))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 1920))];
  input0_shared[((((int)threadIdx.x) + 2048))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2048))];
  input0_shared[((((int)threadIdx.x) + 2176))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2176))];
  input0_shared[((((int)threadIdx.x) + 2304))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2304))];
  input0_shared[((((int)threadIdx.x) + 2432))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2432))];
  input0_shared[((((int)threadIdx.x) + 2560))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2560))];
  input0_shared[((((int)threadIdx.x) + 2688))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2688))];
  input0_shared[((((int)threadIdx.x) + 2816))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2816))];
  input0_shared[((((int)threadIdx.x) + 2944))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 2944))];
  input0_shared[((((int)threadIdx.x) + 3072))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3072))];
  input0_shared[((((int)threadIdx.x) + 3200))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3200))];
  input0_shared[((((int)threadIdx.x) + 3328))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3328))];
  input0_shared[((((int)threadIdx.x) + 3456))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3456))];
  input0_shared[((((int)threadIdx.x) + 3584))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3584))];
  input0_shared[((((int)threadIdx.x) + 3712))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3712))];
  input0_shared[((((int)threadIdx.x) + 3840))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3840))];
  input0_shared[((((int)threadIdx.x) + 3968))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 3968))];
  input0_shared[((((int)threadIdx.x) + 4096))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4096))];
  input0_shared[((((int)threadIdx.x) + 4224))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4224))];
  input0_shared[((((int)threadIdx.x) + 4352))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4352))];
  input0_shared[((((int)threadIdx.x) + 4480))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4480))];
  input0_shared[((((int)threadIdx.x) + 4608))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4608))];
  input0_shared[((((int)threadIdx.x) + 4736))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4736))];
  input0_shared[((((int)threadIdx.x) + 4864))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4864))];
  input0_shared[((((int)threadIdx.x) + 4992))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 4992))];
  input0_shared[((((int)threadIdx.x) + 5120))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5120))];
  input0_shared[((((int)threadIdx.x) + 5248))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5248))];
  input0_shared[((((int)threadIdx.x) + 5376))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5376))];
  input0_shared[((((int)threadIdx.x) + 5504))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5504))];
  input0_shared[((((int)threadIdx.x) + 5632))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5632))];
  input0_shared[((((int)threadIdx.x) + 5760))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5760))];
  input0_shared[((((int)threadIdx.x) + 5888))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 5888))];
  input0_shared[((((int)threadIdx.x) + 6016))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6016))];
  input0_shared[((((int)threadIdx.x) + 6144))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6144))];
  input0_shared[((((int)threadIdx.x) + 6272))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6272))];
  input0_shared[((((int)threadIdx.x) + 6400))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6400))];
  input0_shared[((((int)threadIdx.x) + 6528))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6528))];
  input0_shared[((((int)threadIdx.x) + 6656))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6656))];
  input0_shared[((((int)threadIdx.x) + 6784))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6784))];
  input0_shared[((((int)threadIdx.x) + 6912))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 6912))];
  input0_shared[((((int)threadIdx.x) + 7040))] = input0[((((((int)blockIdx.x) * 7168) + ((int)threadIdx.x)) + 7040))];
  __shared__ char input1_shared[33];
  // [thread_extent] threadIdx.x = 128
  if (((int)threadIdx.x) < 32) {
    input1_shared[(((int)threadIdx.x))] = input1[(((int)threadIdx.x))];
  }
  __syncthreads();
  for (int K_inner = 0; K_inner < 32; ++K_inner) {
    output0_local[(0)] = (output0_local[(0)] + (((int)input0_shared[((((((int)threadIdx.x) >> 3) * 224) + K_inner))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(14)] = (output0_local[(14)] + (((int)input0_shared[((((((int)threadIdx.x) >> 3) * 224) + K_inner))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(28)] = (output0_local[(28)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3584))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(42)] = (output0_local[(42)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3584))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(1)] = (output0_local[(1)] + (((int)input0_shared[((((((int)threadIdx.x) >> 3) * 224) + K_inner))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(15)] = (output0_local[(15)] + (((int)input0_shared[((((((int)threadIdx.x) >> 3) * 224) + K_inner))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(29)] = (output0_local[(29)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3584))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(43)] = (output0_local[(43)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3584))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(2)] = (output0_local[(2)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 32))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(16)] = (output0_local[(16)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 32))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(30)] = (output0_local[(30)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3616))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(44)] = (output0_local[(44)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3616))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(3)] = (output0_local[(3)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 32))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(17)] = (output0_local[(17)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 32))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(31)] = (output0_local[(31)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3616))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(45)] = (output0_local[(45)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3616))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(4)] = (output0_local[(4)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 64))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(18)] = (output0_local[(18)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 64))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(32)] = (output0_local[(32)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3648))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(46)] = (output0_local[(46)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3648))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(5)] = (output0_local[(5)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 64))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(19)] = (output0_local[(19)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 64))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(33)] = (output0_local[(33)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3648))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(47)] = (output0_local[(47)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3648))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(6)] = (output0_local[(6)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 96))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(20)] = (output0_local[(20)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 96))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(34)] = (output0_local[(34)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3680))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(48)] = (output0_local[(48)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3680))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(7)] = (output0_local[(7)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 96))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(21)] = (output0_local[(21)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 96))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(35)] = (output0_local[(35)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3680))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(49)] = (output0_local[(49)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3680))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(8)] = (output0_local[(8)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 128))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(22)] = (output0_local[(22)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 128))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(36)] = (output0_local[(36)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3712))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(50)] = (output0_local[(50)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3712))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(9)] = (output0_local[(9)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 128))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(23)] = (output0_local[(23)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 128))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(37)] = (output0_local[(37)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3712))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(51)] = (output0_local[(51)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3712))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(10)] = (output0_local[(10)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 160))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(24)] = (output0_local[(24)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 160))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(38)] = (output0_local[(38)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3744))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(52)] = (output0_local[(52)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3744))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(11)] = (output0_local[(11)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 160))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(25)] = (output0_local[(25)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 160))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(39)] = (output0_local[(39)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3744))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(53)] = (output0_local[(53)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3744))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(12)] = (output0_local[(12)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 192))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(26)] = (output0_local[(26)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 192))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(40)] = (output0_local[(40)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3776))]) * ((int)input1_shared[(((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)))])));
    output0_local[(54)] = (output0_local[(54)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3776))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 16))])));
    output0_local[(13)] = (output0_local[(13)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 192))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(27)] = (output0_local[(27)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 192))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
    output0_local[(41)] = (output0_local[(41)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3776))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 1))])));
    output0_local[(55)] = (output0_local[(55)] + (((int)input0_shared[(((((((int)threadIdx.x) >> 3) * 224) + K_inner) + 3776))]) * ((int)input1_shared[((((K_inner * 33) + ((((int)threadIdx.x) & 7) * 2)) + 17))])));
  }
  for (int N_inner = 0; N_inner < 7; ++N_inner) {
    for (int M_inner = 0; M_inner < 2; ++M_inner) {
      output0[((((((((int)blockIdx.x) * 7168) + ((((int)threadIdx.x) >> 3) * 224)) + (N_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + M_inner))] = output0_local[(((N_inner * 2) + M_inner))];
      output0[(((((((((int)blockIdx.x) * 7168) + ((((int)threadIdx.x) >> 3) * 224)) + (N_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + M_inner) + 16))] = output0_local[((((N_inner * 2) + M_inner) + 14))];
      output0[(((((((((int)blockIdx.x) * 7168) + ((((int)threadIdx.x) >> 3) * 224)) + (N_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + M_inner) + 3584))] = output0_local[((((N_inner * 2) + M_inner) + 28))];
      output0[(((((((((int)blockIdx.x) * 7168) + ((((int)threadIdx.x) >> 3) * 224)) + (N_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + M_inner) + 3600))] = output0_local[((((N_inner * 2) + M_inner) + 42))];
    }
  }
}

// Saved Perf = 4.736940e-03 sec / run; Step Produced = 984; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.