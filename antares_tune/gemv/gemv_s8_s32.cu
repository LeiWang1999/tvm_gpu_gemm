// GLOBALS: input0:int8[18966528, 25], input1:int8[1, 25] -> output0:int32[18966528, 25]
// BACKEND: c-cuda (default)
// CONFIG: {"Moutput0T": 0, "Moutput0:D0": [-1, 1, 128, 6], "Moutput0:D1": [-1, 5, 1, 5], "Moutput0:R0": [-1, 1, 5], "Moutput0:RA": 0, "Moutput0:AL0": 1, "Moutput0:AL1": 1, "Moutput0:S": 2, "Moutput0:U": 1}
// COMPUTE_V1: - einstein_v2("output0[N, M] +=! input0[N, K].cast(`int32`) * input1[K, M].cast(`int32`)", { "input0": {"dtype": "int8", "shape": [18966528, 25]}, "input1": {"dtype": "int8", "shape": [1, 25]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[18966528, 25], input1:int8[1, 25] -> output0:int32[18966528, 25]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(128) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ input1, int* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 24696
  // [thread_extent] threadIdx.x = 128
  int output0_local[150];
  for (int N_c_inner_init = 0; N_c_inner_init < 6; ++N_c_inner_init) {
    output0_local[((N_c_inner_init * 5))] = 0;
    output0_local[(((N_c_inner_init * 5) + 30))] = 0;
    output0_local[(((N_c_inner_init * 5) + 60))] = 0;
    output0_local[(((N_c_inner_init * 5) + 90))] = 0;
    output0_local[(((N_c_inner_init * 5) + 120))] = 0;
    output0_local[(((N_c_inner_init * 5) + 1))] = 0;
    output0_local[(((N_c_inner_init * 5) + 31))] = 0;
    output0_local[(((N_c_inner_init * 5) + 61))] = 0;
    output0_local[(((N_c_inner_init * 5) + 91))] = 0;
    output0_local[(((N_c_inner_init * 5) + 121))] = 0;
    output0_local[(((N_c_inner_init * 5) + 2))] = 0;
    output0_local[(((N_c_inner_init * 5) + 32))] = 0;
    output0_local[(((N_c_inner_init * 5) + 62))] = 0;
    output0_local[(((N_c_inner_init * 5) + 92))] = 0;
    output0_local[(((N_c_inner_init * 5) + 122))] = 0;
    output0_local[(((N_c_inner_init * 5) + 3))] = 0;
    output0_local[(((N_c_inner_init * 5) + 33))] = 0;
    output0_local[(((N_c_inner_init * 5) + 63))] = 0;
    output0_local[(((N_c_inner_init * 5) + 93))] = 0;
    output0_local[(((N_c_inner_init * 5) + 123))] = 0;
    output0_local[(((N_c_inner_init * 5) + 4))] = 0;
    output0_local[(((N_c_inner_init * 5) + 34))] = 0;
    output0_local[(((N_c_inner_init * 5) + 64))] = 0;
    output0_local[(((N_c_inner_init * 5) + 94))] = 0;
    output0_local[(((N_c_inner_init * 5) + 124))] = 0;
  }
  for (int K_outer_outer = 0; K_outer_outer < 5; ++K_outer_outer) {
    __shared__ char input0_shared[3840];
  // [thread_extent] threadIdx.x = 128
    __syncthreads();
    input0_shared[(((int)threadIdx.x))] = input0[(((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)))];
    input0_shared[((((int)threadIdx.x) + 128))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 128) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 256))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 256) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 384))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 384) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 512))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 512) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    input0_shared[((((int)threadIdx.x) + 640))] = input0[((((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 3200))];
    input0_shared[((((int)threadIdx.x) + 768))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 768) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 896))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 896) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1024))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1024) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1152))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1152) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1280))] = input0[((((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 6400))];
    input0_shared[((((int)threadIdx.x) + 1408))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1408) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1536))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1536) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1664))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1664) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1792))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 1792) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    input0_shared[((((int)threadIdx.x) + 1920))] = input0[((((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 9600))];
    input0_shared[((((int)threadIdx.x) + 2048))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2048) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2176))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2176) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2304))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2304) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2432))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2432) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2560))] = input0[((((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 12800))];
    input0_shared[((((int)threadIdx.x) + 2688))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2688) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2816))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2816) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 2944))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 2944) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 3072))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 3072) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    input0_shared[((((int)threadIdx.x) + 3200))] = input0[((((((((int)blockIdx.x) * 19200) + ((((int)threadIdx.x) / 5) * 25)) + (K_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 16000))];
    input0_shared[((((int)threadIdx.x) + 3328))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 3328) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5)))];
    input0_shared[((((int)threadIdx.x) + 3456))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 3456) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5)))];
    input0_shared[((((int)threadIdx.x) + 3584))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 3584) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5)))];
    input0_shared[((((int)threadIdx.x) + 3712))] = input0[(((((((int)blockIdx.x) * 19200) + (((((int)threadIdx.x) + 3712) / 5) * 25)) + (K_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5)))];
    __shared__ char input1_shared[125];
  // [thread_extent] threadIdx.x = 128
    if (((int)threadIdx.x) < 125) {
      if (((K_outer_outer * 5) + (((int)threadIdx.x) / 25)) < 1) {
        input1_shared[(((int)threadIdx.x))] = input1[(((K_outer_outer * 125) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int K_inner = 0; K_inner < 5; ++K_inner) {
      for (int N_c_inner = 0; N_c_inner < 6; ++N_c_inner) {
        output0_local[((N_c_inner * 5))] = (output0_local[((N_c_inner * 5))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[((K_inner * 25))])));
        output0_local[(((N_c_inner * 5) + 30))] = (output0_local[(((N_c_inner * 5) + 30))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 5))])));
        output0_local[(((N_c_inner * 5) + 60))] = (output0_local[(((N_c_inner * 5) + 60))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 10))])));
        output0_local[(((N_c_inner * 5) + 90))] = (output0_local[(((N_c_inner * 5) + 90))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 15))])));
        output0_local[(((N_c_inner * 5) + 120))] = (output0_local[(((N_c_inner * 5) + 120))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 20))])));
        output0_local[(((N_c_inner * 5) + 1))] = (output0_local[(((N_c_inner * 5) + 1))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 1))])));
        output0_local[(((N_c_inner * 5) + 31))] = (output0_local[(((N_c_inner * 5) + 31))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 6))])));
        output0_local[(((N_c_inner * 5) + 61))] = (output0_local[(((N_c_inner * 5) + 61))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 11))])));
        output0_local[(((N_c_inner * 5) + 91))] = (output0_local[(((N_c_inner * 5) + 91))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 16))])));
        output0_local[(((N_c_inner * 5) + 121))] = (output0_local[(((N_c_inner * 5) + 121))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 21))])));
        output0_local[(((N_c_inner * 5) + 2))] = (output0_local[(((N_c_inner * 5) + 2))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 2))])));
        output0_local[(((N_c_inner * 5) + 32))] = (output0_local[(((N_c_inner * 5) + 32))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 7))])));
        output0_local[(((N_c_inner * 5) + 62))] = (output0_local[(((N_c_inner * 5) + 62))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 12))])));
        output0_local[(((N_c_inner * 5) + 92))] = (output0_local[(((N_c_inner * 5) + 92))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 17))])));
        output0_local[(((N_c_inner * 5) + 122))] = (output0_local[(((N_c_inner * 5) + 122))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 22))])));
        output0_local[(((N_c_inner * 5) + 3))] = (output0_local[(((N_c_inner * 5) + 3))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 3))])));
        output0_local[(((N_c_inner * 5) + 33))] = (output0_local[(((N_c_inner * 5) + 33))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 8))])));
        output0_local[(((N_c_inner * 5) + 63))] = (output0_local[(((N_c_inner * 5) + 63))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 13))])));
        output0_local[(((N_c_inner * 5) + 93))] = (output0_local[(((N_c_inner * 5) + 93))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 18))])));
        output0_local[(((N_c_inner * 5) + 123))] = (output0_local[(((N_c_inner * 5) + 123))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 23))])));
        output0_local[(((N_c_inner * 5) + 4))] = (output0_local[(((N_c_inner * 5) + 4))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 4))])));
        output0_local[(((N_c_inner * 5) + 34))] = (output0_local[(((N_c_inner * 5) + 34))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 9))])));
        output0_local[(((N_c_inner * 5) + 64))] = (output0_local[(((N_c_inner * 5) + 64))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 14))])));
        output0_local[(((N_c_inner * 5) + 94))] = (output0_local[(((N_c_inner * 5) + 94))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 19))])));
        output0_local[(((N_c_inner * 5) + 124))] = (output0_local[(((N_c_inner * 5) + 124))] + (((int)input0_shared[((((((int)threadIdx.x) * 30) + (N_c_inner * 5)) + K_inner))]) * ((int)input1_shared[(((K_inner * 25) + 24))])));
      }
    }
  }
  for (int N_inner = 0; N_inner < 6; ++N_inner) {
    for (int M_inner = 0; M_inner < 5; ++M_inner) {
      output0[(((((((int)blockIdx.x) * 19200) + (((int)threadIdx.x) * 150)) + (N_inner * 25)) + M_inner))] = output0_local[(((N_inner * 5) + M_inner))];
      output0[((((((((int)blockIdx.x) * 19200) + (((int)threadIdx.x) * 150)) + (N_inner * 25)) + M_inner) + 5))] = output0_local[((((N_inner * 5) + M_inner) + 30))];
      output0[((((((((int)blockIdx.x) * 19200) + (((int)threadIdx.x) * 150)) + (N_inner * 25)) + M_inner) + 10))] = output0_local[((((N_inner * 5) + M_inner) + 60))];
      output0[((((((((int)blockIdx.x) * 19200) + (((int)threadIdx.x) * 150)) + (N_inner * 25)) + M_inner) + 15))] = output0_local[((((N_inner * 5) + M_inner) + 90))];
      output0[((((((((int)blockIdx.x) * 19200) + (((int)threadIdx.x) * 150)) + (N_inner * 25)) + M_inner) + 20))] = output0_local[((((N_inner * 5) + M_inner) + 120))];
    }
  }
}

// Saved Perf = 3.298590e-04 sec / run; Step Produced = 95; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.