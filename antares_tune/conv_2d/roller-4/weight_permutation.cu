// GLOBALS: input0:float32[384, 1024] -> output0:float32[24, 64, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 4, 2, 2], "F___output0:D1": [-1, 2, 32, 1], "F___output0:O": [0, 1], "F___output0:S": 4, "F___output0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[M // 16, N // 16, M % 16, N % 16] =. input0[M, N]", input_dict={"input0": {"dtype": "float32", "shape": [384, 1024]}, "output0": {"dtype": "float32", "shape": [24, 64, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float32[384, 1024] -> output0:float32[24, 64, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(64) void template_op_kernel0(float* __restrict__ input0, float* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 24
  // [thread_extent] threadIdx.x = 2
  // [thread_extent] blockIdx.y = 16
  // [thread_extent] threadIdx.y = 32
  ((output0[((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)))]) = (input0[(((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 16))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 1024))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 64))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 4096))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 80))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 5120))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 128))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 8192))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 144))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 9216))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 192))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 12288))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 208))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 13312))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 512))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 32))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 528))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 1056))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 576))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 4128))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 592))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 5152))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 640))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 8224))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 656))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 9248))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 704))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 12320))]));
  ((output0[(((((((((int)blockIdx.x) * 16384) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + (((int)threadIdx.x) * 32)) + (((int)threadIdx.y) & 15)) + 720))]) = (input0[((((((((int)blockIdx.x) * 16384) + (((int)threadIdx.x) * 2048)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 13344))]));
}

// Saved Perf = 5.031400e-06 sec / run; Step Produced = 821; Planned Steps = 1000;