// GLOBALS: input0:int8[128, 42, 42, 1024] -> output0:int8[42, 42, 8, 24, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 1, 1, 1], "F___output0:D1": [-1, 1, 1, 1], "F___output0:D2": [-1, 1, 8, 1], "F___output0:D3": [-1, 32], "F___output0:O": [3, 0, 1, 2], "F___output0:S": 1, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "int8", "shape": [128, 42, 42, 1024]}, "output0": {"dtype": "int8", "shape": [42, 42, 8, 24, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[128, 42, 42, 1024] -> output0:int8[42, 42, 8, 24, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(8) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 42
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 42
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 16
  // [thread_extent] threadIdx.z = 8
  for (int vthread_s = 0; vthread_s < 32; ++vthread_s) {
    for (int vthread_s1 = 0; vthread_s1 < 32; ++vthread_s1) {
      ((output0[((((((((((int)blockIdx.y) * 2064384) + (((int)blockIdx.x) * 49152)) + ((((((int)blockIdx.z) * 8) + ((int)threadIdx.z)) / 16) * 6144)) + (vthread_s * 512)) + ((vthread_s1 / 16) * 256)) + ((((((int)blockIdx.z) * 8) + ((int)threadIdx.z)) & 15) * 16)) + (vthread_s1 & 15)))]) = (input0[(((((((((int)blockIdx.z) * 14450688) + (((int)threadIdx.z) * 1806336)) + (((int)blockIdx.y) * 43008)) + (((int)blockIdx.x) * 1024)) + (vthread_s * 32)) + vthread_s1))]));
    }
  }
}

// Saved Perf = 1.732690e-03 sec / run; Step Produced = 777; Planned Steps = 1000;