// GLOBALS: input0:float16[128, 42, 42, 1024] -> output0:float16[8, 42, 42, 24, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 3, 1, 1], "F___output0:D1": [-1, 2, 32, 1], "F___output0:D2": [-1, 2, 1, 2], "F___output0:D3": [-1, 64], "F___output0:O": [1, 3, 0, 2], "F___output0:S": 2, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "float16", "shape": [128, 42, 42, 1024]}, "output0": {"dtype": "float16", "shape": [8, 42, 42, 24, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1024] -> output0:float16[8, 42, 42, 24, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(32) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 14
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 16
  // [thread_extent] threadIdx.y = 32
  // [thread_extent] blockIdx.z = 2
  // [thread_extent] threadIdx.z = 1
  for (int vthread_s = 0; vthread_s < 64; ++vthread_s) {
    ((output0[((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s & 15) * 16)) + (((int)threadIdx.y) & 15)))]) = (input0[((((((vthread_s * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s & 15) * 16)) + (((int)threadIdx.y) & 15)) + 512))]) = (input0[(((((((vthread_s * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 32))]));
  }
  for (int vthread_s1 = 0; vthread_s1 < 64; ++vthread_s1) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s1 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s1 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 24576))]) = (input0[(((((((vthread_s1 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115605504))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s1 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s1 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 25088))]) = (input0[(((((((vthread_s1 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115605536))]));
  }
  for (int vthread_s2 = 0; vthread_s2 < 64; ++vthread_s2) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s2 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s2 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 258048))]) = (input0[(((((((vthread_s2 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 1024))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s2 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s2 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 258560))]) = (input0[(((((((vthread_s2 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 1056))]));
  }
  for (int vthread_s3 = 0; vthread_s3 < 64; ++vthread_s3) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s3 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s3 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 282624))]) = (input0[(((((((vthread_s3 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115606528))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s3 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s3 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 283136))]) = (input0[(((((((vthread_s3 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115606560))]));
  }
  for (int vthread_s4 = 0; vthread_s4 < 64; ++vthread_s4) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s4 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s4 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 516096))]) = (input0[(((((((vthread_s4 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 2048))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s4 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s4 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 516608))]) = (input0[(((((((vthread_s4 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 2080))]));
  }
  for (int vthread_s5 = 0; vthread_s5 < 64; ++vthread_s5) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s5 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s5 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 540672))]) = (input0[(((((((vthread_s5 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115607552))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s5 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s5 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 541184))]) = (input0[(((((((vthread_s5 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115607584))]));
  }
  for (int vthread_s6 = 0; vthread_s6 < 64; ++vthread_s6) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s6 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s6 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 10838016))]) = (input0[(((((((vthread_s6 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 43008))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s6 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s6 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 10838528))]) = (input0[(((((((vthread_s6 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 43040))]));
  }
  for (int vthread_s7 = 0; vthread_s7 < 64; ++vthread_s7) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s7 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s7 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 10862592))]) = (input0[(((((((vthread_s7 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115648512))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s7 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s7 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 10863104))]) = (input0[(((((((vthread_s7 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115648544))]));
  }
  for (int vthread_s8 = 0; vthread_s8 < 64; ++vthread_s8) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s8 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s8 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11096064))]) = (input0[(((((((vthread_s8 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 44032))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s8 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s8 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11096576))]) = (input0[(((((((vthread_s8 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 44064))]));
  }
  for (int vthread_s9 = 0; vthread_s9 < 64; ++vthread_s9) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s9 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s9 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11120640))]) = (input0[(((((((vthread_s9 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115649536))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s9 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s9 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11121152))]) = (input0[(((((((vthread_s9 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115649568))]));
  }
  for (int vthread_s10 = 0; vthread_s10 < 64; ++vthread_s10) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s10 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s10 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11354112))]) = (input0[(((((((vthread_s10 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 45056))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s10 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s10 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11354624))]) = (input0[(((((((vthread_s10 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 45088))]));
  }
  for (int vthread_s11 = 0; vthread_s11 < 64; ++vthread_s11) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s11 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s11 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11378688))]) = (input0[(((((((vthread_s11 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115650560))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s11 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s11 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 11379200))]) = (input0[(((((((vthread_s11 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115650592))]));
  }
  for (int vthread_s12 = 0; vthread_s12 < 64; ++vthread_s12) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s12 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s12 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21676032))]) = (input0[(((((((vthread_s12 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 86016))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s12 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s12 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21676544))]) = (input0[(((((((vthread_s12 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 86048))]));
  }
  for (int vthread_s13 = 0; vthread_s13 < 64; ++vthread_s13) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s13 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s13 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21700608))]) = (input0[(((((((vthread_s13 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115691520))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s13 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s13 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21701120))]) = (input0[(((((((vthread_s13 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115691552))]));
  }
  for (int vthread_s14 = 0; vthread_s14 < 64; ++vthread_s14) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s14 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s14 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21934080))]) = (input0[(((((((vthread_s14 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 87040))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s14 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s14 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21934592))]) = (input0[(((((((vthread_s14 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 87072))]));
  }
  for (int vthread_s15 = 0; vthread_s15 < 64; ++vthread_s15) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s15 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s15 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21958656))]) = (input0[(((((((vthread_s15 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115692544))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s15 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s15 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 21959168))]) = (input0[(((((((vthread_s15 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115692576))]));
  }
  for (int vthread_s16 = 0; vthread_s16 < 64; ++vthread_s16) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s16 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s16 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 22192128))]) = (input0[(((((((vthread_s16 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 88064))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s16 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s16 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 22192640))]) = (input0[(((((((vthread_s16 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 88096))]));
  }
  for (int vthread_s17 = 0; vthread_s17 < 64; ++vthread_s17) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s17 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s17 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 22216704))]) = (input0[(((((((vthread_s17 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115693568))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s17 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s17 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 22217216))]) = (input0[(((((((vthread_s17 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115693600))]));
  }
  for (int vthread_s18 = 0; vthread_s18 < 64; ++vthread_s18) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s18 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s18 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32514048))]) = (input0[(((((((vthread_s18 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 129024))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s18 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s18 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32514560))]) = (input0[(((((((vthread_s18 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 129056))]));
  }
  for (int vthread_s19 = 0; vthread_s19 < 64; ++vthread_s19) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s19 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s19 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32538624))]) = (input0[(((((((vthread_s19 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115734528))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s19 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s19 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32539136))]) = (input0[(((((((vthread_s19 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115734560))]));
  }
  for (int vthread_s20 = 0; vthread_s20 < 64; ++vthread_s20) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s20 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s20 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32772096))]) = (input0[(((((((vthread_s20 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 130048))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s20 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s20 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32772608))]) = (input0[(((((((vthread_s20 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 130080))]));
  }
  for (int vthread_s21 = 0; vthread_s21 < 64; ++vthread_s21) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s21 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s21 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32796672))]) = (input0[(((((((vthread_s21 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115735552))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s21 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s21 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 32797184))]) = (input0[(((((((vthread_s21 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115735584))]));
  }
  for (int vthread_s22 = 0; vthread_s22 < 64; ++vthread_s22) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s22 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s22 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 33030144))]) = (input0[(((((((vthread_s22 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 131072))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s22 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s22 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 33030656))]) = (input0[(((((((vthread_s22 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 131104))]));
  }
  for (int vthread_s23 = 0; vthread_s23 < 64; ++vthread_s23) {
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s23 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s23 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 33054720))]) = (input0[(((((((vthread_s23 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115736576))]));
    ((output0[(((((((((((int)blockIdx.z) * 43352064) + (((int)blockIdx.x) * 774144)) + ((vthread_s23 / 16) * 6144)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.y) / 16) * 256)) + ((vthread_s23 & 15) * 16)) + (((int)threadIdx.y) & 15)) + 33055232))]) = (input0[(((((((vthread_s23 * 1806336) + (((int)blockIdx.z) * 172032)) + (((int)blockIdx.x) * 3072)) + (((int)blockIdx.y) * 64)) + ((int)threadIdx.y)) + 115736608))]));
  }
}

// Saved Perf = 1.590430e-04 sec / run; Step Produced = 661; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.