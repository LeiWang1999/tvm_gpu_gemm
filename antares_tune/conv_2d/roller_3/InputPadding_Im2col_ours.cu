// GLOBALS: input0:float16[128, 42, 42, 1008] -> output0:float16[225792, 1008]
// BACKEND: c-cuda (default)
// CONFIG: {"Foutput0:D0": [-1, 1, 1, 1], "Foutput0:D1": [-1, 9, 14, 8], "Foutput0:O": [1, 0], "Foutput0:S": 3, "Foutput0:R": 1}
// COMPUTE_V1: - _N, _CI, _H, _W, _CO, _KH, _KW, _SH, _SW, _PH, _PW = 128, 1008, 42, 42, 336, 1, 1, 1, 1, 0, 0;_HO, _WO = (_H - _KH + _PH * 2) // _SH + 1, (_W - _KW + _PW * 2) // _SW + 1;_PHI, _PWI = _H + _PH * 2, _W + _PW * 2; _GM, _GN, _GK = _N * _HO * _WO, _CO, _CI * _KH * _KW; einstein_v2(f"temp0[N, PHI, PWI, C] = input0[N, -{_PH} + PHI, -{_PW} + PWI, C].when([-{_PH} + PHI >= 0, -{_PH} + PHI < {_H}, -{_PW} + PWI >= 0, -{_PW} + PWI < {_W}], const(0.0).cast(`float16`)) where PHI in {_PHI}, PWI in {_PWI};output0[GM, GK] = temp0[GM // ({_HO} * {_WO}), {_SH} * ((GM % ({_HO} * {_WO})) // {_WO}) + (GK // {_CI}) // {_KW}, {_SW} * ((GM % ({_HO} * {_WO})) % {_WO}) + (GK // {_CI}) % {_KW}, GK % {_CI}] where GM in {_GM}, GK in {_GK}", input_dict={"input0": {"dtype": "float16", "shape": [_N, _H, _W, _CI]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> output0:float16[225792, 1008]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(14) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 225792
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 1
  // [thread_extent] threadIdx.y = 14
  output0[(((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)))] = input0[((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 1))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 1))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 2))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 2))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 3))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 3))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 4))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 4))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 5))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 5))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 6))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 6))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 7))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 7))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 112))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 112))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 113))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 113))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 114))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 114))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 115))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 115))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 116))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 116))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 117))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 117))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 118))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 118))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 119))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 119))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 224))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 224))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 225))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 225))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 226))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 226))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 227))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 227))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 228))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 228))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 229))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 229))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 230))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 230))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 231))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 231))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 336))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 336))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 337))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 337))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 338))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 338))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 339))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 339))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 340))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 340))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 341))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 341))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 342))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 342))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 343))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 343))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 448))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 448))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 449))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 449))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 450))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 450))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 451))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 451))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 452))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 452))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 453))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 453))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 454))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 454))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 455))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 455))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 560))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 560))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 561))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 561))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 562))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 562))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 563))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 563))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 564))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 564))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 565))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 565))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 566))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 566))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 567))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 567))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 672))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 672))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 673))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 673))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 674))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 674))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 675))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 675))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 676))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 676))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 677))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 677))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 678))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 678))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 679))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 679))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 784))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 784))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 785))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 785))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 786))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 786))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 787))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 787))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 788))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 788))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 789))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 789))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 790))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 790))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 791))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 791))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 896))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 896))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 897))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 897))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 898))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 898))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 899))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 899))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 900))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 900))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 901))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 901))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 902))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 902))];
  output0[((((((int)blockIdx.x) * 1008) + (((int)threadIdx.y) * 8)) + 903))] = input0[(((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + (((int)threadIdx.y) * 8)) + 903))];
}

// Saved Perf = 1.065600e-03 sec / run; Step Produced = 1413; Planned Steps = 10000;