// GLOBALS: input0:float16[128, 42, 42, 1024], input1:float16[384, 1, 1, 1024] -> output0:float16[128, 42, 42, 384]
// BACKEND: c-cuda (default)
// CONFIG: {"Moutput0T": 0, "Moutput0:D0": [-1, 4, 2, 2], "Moutput0:D1": [-1, 1, 2, 1], "Moutput0:D2": [-1, 1, 2, 1], "Moutput0:D3": [-1, 3, 16, 4], "Moutput0:R0": [-1, 2, 1], "Moutput0:R1": [-1, 1, 1], "Moutput0:R2": [-1, 1, 1], "Moutput0:RA": 0, "Moutput0:AL0": 0, "Moutput0:AL1": 0, "Moutput0:S": 2, "Moutput0:U": 1}
// COMPUTE_V1: - einstein_v2("output0[N, HO, WO, F] +=! input0[N, HO + KH, WO + KW, C] * input1[F, KH, KW, C] where HO in 42, WO in 42", { "input0": {"dtype": "float16", "shape": [128, 42, 42, 1024]}, "input1": {"dtype": "float16", "shape": [384, 1, 1, 1024]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1024], input1:float16[384, 1, 1, 1024] -> output0:float16[128, 42, 42, 384]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(128) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ input1, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 7056
  // [thread_extent] threadIdx.x = 128
  half output0_local[96];
  for (int N_c_inner_init = 0; N_c_inner_init < 2; ++N_c_inner_init) {
    for (int F_c_inner_init = 0; F_c_inner_init < 4; ++F_c_inner_init) {
      output0_local[(((N_c_inner_init * 4) + F_c_inner_init))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 8))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 16))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 24))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 32))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 40))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 48))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 56))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 64))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 72))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 80))] = __float2half_rn(0.000000e+00f);
      output0_local[((((N_c_inner_init * 4) + F_c_inner_init) + 88))] = __float2half_rn(0.000000e+00f);
    }
  }
  for (int C_outer_outer = 0; C_outer_outer < 512; ++C_outer_outer) {
    __shared__ half input0_shared[128];
  // [thread_extent] threadIdx.x = 128
    __syncthreads();
    input0_shared[(((int)threadIdx.x))] = input0[((((((((((((int)blockIdx.x) / 882) * 28901376) + ((((int)threadIdx.x) >> 3) * 1806336)) + (((((int)blockIdx.x) % 882) / 42) * 86016)) + (((((int)threadIdx.x) & 7) >> 2) * 43008)) + (((((int)blockIdx.x) % 42) >> 1) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 1024)) + (C_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    __shared__ half input1_shared[384];
  // [thread_extent] threadIdx.x = 128
    input1_shared[(((int)threadIdx.x))] = input1[((((((((int)blockIdx.x) & 1) * 196608) + ((((int)threadIdx.x) >> 1) * 1024)) + (C_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    input1_shared[((((int)threadIdx.x) + 128))] = input1[(((((((((int)blockIdx.x) & 1) * 196608) + ((((int)threadIdx.x) >> 1) * 1024)) + (C_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 65536))];
    input1_shared[((((int)threadIdx.x) + 256))] = input1[(((((((((int)blockIdx.x) & 1) * 196608) + ((((int)threadIdx.x) >> 1) * 1024)) + (C_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 131072))];
    __syncthreads();
    for (int C_inner = 0; C_inner < 2; ++C_inner) {
      for (int N_c_inner = 0; N_c_inner < 2; ++N_c_inner) {
        for (int F_c_inner = 0; F_c_inner < 4; ++F_c_inner) {
          output0_local[(((N_c_inner * 4) + F_c_inner))] = (output0_local[(((N_c_inner * 4) + F_c_inner))] + (input0_shared[((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner))] * input1_shared[(((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 8))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 8))] + (input0_shared[((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 128))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 16))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 16))] + (input0_shared[((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 256))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 24))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 24))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 32))] * input1_shared[(((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 32))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 32))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 32))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 128))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 40))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 40))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 32))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 256))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 48))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 48))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 64))] * input1_shared[(((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 56))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 56))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 64))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 128))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 64))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 64))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 64))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 256))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 72))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 72))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 96))] * input1_shared[(((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 80))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 80))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 96))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 128))]));
          output0_local[((((N_c_inner * 4) + F_c_inner) + 88))] = (output0_local[((((N_c_inner * 4) + F_c_inner) + 88))] + (input0_shared[(((((((((int)threadIdx.x) >> 6) * 16) + (N_c_inner * 8)) + (((((int)threadIdx.x) & 63) >> 4) * 2)) + C_inner) + 96))] * input1_shared[((((((((int)threadIdx.x) & 15) * 8) + (F_c_inner * 2)) + C_inner) + 256))]));
        }
      }
    }
  }
  for (int N_inner = 0; N_inner < 2; ++N_inner) {
    for (int F_inner = 0; F_inner < 4; ++F_inner) {
      output0[((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner))] = output0_local[(((N_inner * 4) + F_inner))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 64))] = output0_local[((((N_inner * 4) + F_inner) + 8))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 128))] = output0_local[((((N_inner * 4) + F_inner) + 16))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 2709504))] = output0_local[((((N_inner * 4) + F_inner) + 24))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 2709568))] = output0_local[((((N_inner * 4) + F_inner) + 32))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 2709632))] = output0_local[((((N_inner * 4) + F_inner) + 40))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 5419008))] = output0_local[((((N_inner * 4) + F_inner) + 48))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 5419072))] = output0_local[((((N_inner * 4) + F_inner) + 56))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 5419136))] = output0_local[((((N_inner * 4) + F_inner) + 64))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 8128512))] = output0_local[((((N_inner * 4) + F_inner) + 72))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 8128576))] = output0_local[((((N_inner * 4) + F_inner) + 80))];
      output0[(((((((((((((((int)blockIdx.x) / 882) * 10838016) + ((((int)threadIdx.x) >> 6) * 1354752)) + (N_inner * 677376)) + (((((int)blockIdx.x) % 882) / 42) * 32256)) + (((((int)threadIdx.x) & 63) >> 5) * 16128)) + (((((int)blockIdx.x) % 42) >> 1) * 768)) + (((((int)threadIdx.x) & 31) >> 4) * 384)) + ((((int)blockIdx.x) & 1) * 192)) + ((((int)threadIdx.x) & 15) * 4)) + F_inner) + 8128640))] = output0_local[((((N_inner * 4) + F_inner) + 88))];
    }
  }
}

// Saved Perf = 1.648990e-02 sec / run; Step Produced = 69; Planned Steps = 1000;