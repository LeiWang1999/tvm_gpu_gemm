#[version = "0.0.5"]
@main = primfn(a: handle, w: handle, conv: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [128, 42, 42, 1024], []),
             W: Buffer(W_1: Pointer(global float16), float16, [384, 1, 1, 1024], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [225792, 384], [])}
  buffer_map = {a: A, w: W, conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    data_im2col_shared = alloc_buffer(float16[14112, 64, 16, 16])
    data_im2col_shared_warp = alloc_buffer(float16[14112, 64, 16, 16])
    weight_flatten_shared = alloc_buffer(float16[24, 64, 16, 16])
    weight_flatten_shared_warp = alloc_buffer(float16[24, 64, 16, 16])
    Conv_warp = alloc_buffer(float16[14112, 24, 16, 16])
    for (x_0_0: int32, 0, 882) "thread_binding" {
      for (y_0_0: int32, 0, 3) "thread_binding" {
        for (x_0_1: int32, 0, 4) "thread_binding" {
          for (y_0_1: int32, 0, 2) "thread_binding" {
            for (x_0_2_init: int32, 0, 4) {
              for (y_0_2_init: int32, 0, 4) {
                for (x_1_init: int32, 0, 16) {
                  for (y_1_init: int32, 0, 16) {
                    block([225792, 384], "Conv_init") as [v_x, v_y] {
                      bind(v_x, ((((x_0_0*256) + (x_0_1*64)) + (x_0_2_init*16)) + x_1_init))
                      bind(v_y, ((((y_0_0*128) + (y_0_1*64)) + (y_0_2_init*16)) + y_1_init))
                      tir.reads([])
                      tir.writes([Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)]])
                      Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)] = 0f16
                  }
                }
              }
            }
            for (k_0_0: int32, 0, 32) {
              for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 4) "thread_binding" {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 2) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 8) "vectorized" {
                        block([225792, 1024], "data_im2col_shared") as [v0, v1] {
                          bind(v0, (((x_0_0*256) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                          bind(v1, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                          tir.reads([A[floordiv(v0, 1764), (floordiv(v1, 1024) + floordiv(floormod(v0, 1764), 42)), floormod(v0, 42), floormod(v1, 1024)]])
                          tir.writes([data_im2col_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
                          data_im2col_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = @tir.if_then_else(((((0 <= ((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1)))) && (((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1))) < 42)) && (0 <= ((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))))) && (((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))) < 42)), A[floordiv(v0, 1764), (((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1))) - 0), (((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))) - 0), floormod(v1, 1024)], 0f16, dtype=float16)
                      }
                    }
                  }
                }
              }
              for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 4) "thread_binding" {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 2) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 2) {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 8) "vectorized" {
                        block([384, 1024], "weight_flatten_shared") as [v0_1, v1_1] {
                          bind(v0_1, (((y_0_0*128) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                          bind(v1_1, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                          tir.reads([W[v0_1, floordiv(v1_1, 1024), 0, floormod(v1_1, 1024)]])
                          tir.writes([weight_flatten_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
                          weight_flatten_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = W[v0_1, floordiv(floordiv(v1_1, 1024), 1), floormod(floordiv(v1_1, 1024), 1), floormod(v1_1, 1024)]
                      }
                    }
                  }
                }
              }
              for (k_0_1: int32, 0, 2) {
                for (ax0_0: int32, 0, 4) {
                  for (ax1_0: int32, 0, 1) {
                    for (ax0_1: int32, 0, 16) {
                      for (ax1_1: int32, 0, 16) {
                        block([225792, 1024], "data_im2col_shared_warp") as [v0_2, v1_2] {
                          bind(v0_2, ((((x_0_0*256) + (x_0_1*64)) + (ax0_0*16)) + ax0_1))
                          bind(v1_2, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0*16)) + ax1_1))
                          tir.reads([data_im2col_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                          tir.writes([data_im2col_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                          data_im2col_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = data_im2col_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                      }
                    }
                  }
                }
                for (ax0_0_1: int32, 0, 4) {
                  for (ax1_0_1: int32, 0, 1) {
                    for (ax0_1_1: int32, 0, 16) {
                      for (ax1_1_1: int32, 0, 16) {
                        block([384, 1024], "weight_flatten_shared_warp") as [v0_3, v1_3] {
                          bind(v0_3, ((((y_0_0*128) + (y_0_1*64)) + (ax0_0_1*16)) + ax0_1_1))
                          bind(v1_3, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0_1*16)) + ax1_1_1))
                          tir.reads([weight_flatten_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                          tir.writes([weight_flatten_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                          weight_flatten_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = weight_flatten_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                      }
                    }
                  }
                }
                for (x_0_2: int32, 0, 4) {
                  for (y_0_2: int32, 0, 4) {
                    for (x_1: int32, 0, 16) {
                      for (y_1: int32, 0, 16) {
                        for (k_1: int32, 0, 16) {
                          block([225792, 384, tir.reduce_axis(0, 1024)], "Conv_update") as [v_x_1, v_y_1, v_k] {
                            bind(v_x_1, ((((x_0_0*256) + (x_0_1*64)) + (x_0_2*16)) + x_1))
                            bind(v_y_1, ((((y_0_0*128) + (y_0_1*64)) + (y_0_2*16)) + y_1))
                            bind(v_k, (((k_0_0*32) + (k_0_1*16)) + k_1))
                            tir.reads([Conv_warp[floordiv(v_x_1, 16), floordiv(v_y_1, 16), floormod(v_x_1, 16), floormod(v_y_1, 16)], data_im2col_shared_warp[floordiv(v_x_1, 16), floordiv(v_k, 16), floormod(v_x_1, 16), floormod(v_k, 16)], weight_flatten_shared_warp[floordiv(v_y_1, 16), floordiv(v_k, 16), floormod(v_y_1, 16), floormod(v_k, 16)]])
                            tir.writes([Conv_warp[floordiv(v_x_1, 16), floordiv(v_y_1, 16), floormod(v_x_1, 16), floormod(v_y_1, 16)]])
                            Conv_warp[floordiv(v_x_1, 16), floordiv(v_y_1, 16), floormod(v_x_1, 16), floormod(v_y_1, 16)] = (Conv_warp[floordiv(v_x_1, 16), floordiv(v_y_1, 16), floormod(v_x_1, 16), floormod(v_y_1, 16)] + (data_im2col_shared_warp[floordiv(v_x_1, 16), floordiv(v_k, 16), floormod(v_x_1, 16), floormod(v_k, 16)]*weight_flatten_shared_warp[floordiv(v_y_1, 16), floordiv(v_k, 16), floormod(v_y_1, 16), floormod(v_k, 16)]))
                        }
                      }
                    }
                  }
                }
              }
            }
            for (ax0_0_2: int32, 0, 4) {
              for (ax1_0_2: int32, 0, 4) {
                for (ax0_1_2: int32, 0, 16) {
                  for (ax1_1_2: int32, 0, 16) {
                    block([225792, 384], "Conv_warp") as [v0_4, v1_4] {
                      bind(v0_4, ((((x_0_0*256) + (x_0_1*64)) + (ax0_0_2*16)) + ax0_1_2))
                      bind(v1_4, ((((y_0_0*128) + (y_0_1*64)) + (ax1_0_2*16)) + ax1_1_2))
                      tir.reads([Conv_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                      tir.writes([Conv[v0_4, v1_4]])
                      Conv[v0_4, v1_4] = Conv_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}