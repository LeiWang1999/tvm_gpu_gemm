#[version = "0.0.5"]
@main = primfn(a: handle, w: handle, conv: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [128, 42, 42, 1024], []),
             W: Buffer(W_1: Pointer(global float16), float16, [384, 1, 1, 1024], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [225792, 384], [])}
  buffer_map = {a: A, w: W, conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    data_im2col_shared = alloc_buffer(float16[14112, 64, 16, 16])
    data_im2col_shared_warp = alloc_buffer(float16[14112, 64, 16, 16])
    weight_flatten_shared = alloc_buffer(float16[24, 64, 16, 16])
    weight_flatten_shared_warp = alloc_buffer(float16[24, 64, 16, 16])
    Conv_warp = alloc_buffer(float16[14112, 24, 16, 16])
     {
      for (ax0: int32, 0, 225792) {
        for (ax1: int32, 0, 1024) {
          block([225792, 1024], "data_im2col_shared") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([A[floordiv(v0, 1764), (floordiv(v1, 1024) + floordiv(floormod(v0, 1764), 42)), floormod(v0, 42), floormod(v1, 1024)]])
            tir.writes([data_im2col_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            data_im2col_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = @tir.if_then_else(((((0 <= ((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1)))) && (((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1))) < 42)) && (0 <= ((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))))) && (((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))) < 42)), A[floordiv(v0, 1764), (((1*floordiv(floormod(v0, 1764), 42)) + (1*floordiv(floordiv(v1, 1024), 1))) - 0), (((1*floormod(floormod(v0, 1764), 42)) + (1*floormod(floordiv(v1, 1024), 1))) - 0), floormod(v1, 1024)], 0f16, dtype=float16)
        }
      }
      for (ax0_1: int32, 0, 225792) {
        for (ax1_1: int32, 0, 1024) {
          block([225792, 1024], "data_im2col_shared_warp") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([data_im2col_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            tir.writes([data_im2col_shared_warp[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            data_im2col_shared_warp[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = data_im2col_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]
        }
      }
      for (ax0_2: int32, 0, 384) {
        for (ax1_2: int32, 0, 1024) {
          block([384, 1024], "weight_flatten_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([W[v0_2, floordiv(v1_2, 1024), 0, floormod(v1_2, 1024)]])
            tir.writes([weight_flatten_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
            weight_flatten_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = W[v0_2, floordiv(floordiv(v1_2, 1024), 1), floormod(floordiv(v1_2, 1024), 1), floormod(v1_2, 1024)]
        }
      }
      for (ax0_3: int32, 0, 384) {
        for (ax1_3: int32, 0, 1024) {
          block([384, 1024], "weight_flatten_shared_warp") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([weight_flatten_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
            tir.writes([weight_flatten_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
            weight_flatten_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = weight_flatten_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
        }
      }
      for (x_0_0: int32, 0, 882) {
        for (y_0_0: int32, 0, 3) {
          for (x_0_1: int32, 0, 4) {
            for (y_0_1: int32, 0, 2) {
              for (k_0_0: int32, 0, 32) {
                for (k_0_1: int32, 0, 2) {
                  for (x_0_2: int32, 0, 4) {
                    for (y_0_2: int32, 0, 4) {
                      for (x_1: int32, 0, 16) {
                        for (y_1: int32, 0, 16) {
                          for (k_1: int32, 0, 16) {
                            block([225792, 384, tir.reduce_axis(0, 1024)], "Conv") as [v_x, v_y, v_k] {
                              bind(v_x, ((((x_0_0*256) + (x_0_1*64)) + (x_0_2*16)) + x_1))
                              bind(v_y, ((((y_0_0*128) + (y_0_1*64)) + (y_0_2*16)) + y_1))
                              bind(v_k, (((k_0_0*32) + (k_0_1*16)) + k_1))
                              tir.reads([data_im2col_shared_warp[floordiv(v_x, 16), floordiv(v_k, 16), floormod(v_x, 16), floormod(v_k, 16)], weight_flatten_shared_warp[floordiv(v_y, 16), floordiv(v_k, 16), floormod(v_y, 16), floormod(v_k, 16)]])
                              tir.writes([Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)]])
                              with init() {
                                Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)] = 0f16
                              }
                              Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)] = (Conv_warp[floordiv(v_x, 16), floordiv(v_y, 16), floormod(v_x, 16), floormod(v_y, 16)] + (data_im2col_shared_warp[floordiv(v_x, 16), floordiv(v_k, 16), floormod(v_x, 16), floormod(v_k, 16)]*weight_flatten_shared_warp[floordiv(v_y, 16), floordiv(v_k, 16), floormod(v_y, 16), floormod(v_k, 16)]))
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_4: int32, 0, 225792) {
        for (ax1_4: int32, 0, 384) {
          block([225792, 384], "Conv_warp") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([Conv_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
            tir.writes([Conv[v0_4, v1_4]])
            Conv[v0_4, v1_4] = Conv_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}