// GLOBALS: input0:float16[128, 42, 42, 1008] -> output0:float16[128, 42, 42, 42]
// BACKEND: c-cuda (default)
// CONFIG: {"Foutput0:D0": [-1, 1, 1, 1], "Foutput0:D1": [-1, 1, 1, 1], "Foutput0:D2": [-1, 1, 3, 1], "Foutput0:D3": [-1, 1], "Foutput0:O": [3, 1, 2, 0], "Foutput0:S": 0, "Foutput0:R": 1}
// COMPUTE_V1: - _N, _CI, _H, _W, _CO, _KH, _KW, _SH, _SW, _PH, _PW = 128, 1008, 42, 42, 336, 1, 1, 1, 1, 0, 0; _PHI, _PWI = _H + _PH * 2, _W + _PW * 2; einstein_v2(f"output0[N, C, PHI, PWI] = input0[N, C, -{_PH} + PHI, -{_PW} + PWI].when([-{_PH} + PHI >= 0, -{_PH} + PHI < {_H}, -{_PW} + PWI >= 0, -{_PW} + PWI < {_W}], const(0.0).cast(`float16`)) where PHI in {_PHI}, PWI in {_PWI}", input_dict={"input0": {"dtype": "float16", "shape": [_N, _H, _W, _CI]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> output0:float16[128, 42, 42, 42]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(3) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 128
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 42
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 14
  // [thread_extent] threadIdx.z = 3
  for (int vthread_s = 0; vthread_s < 42; ++vthread_s) {
    output0[((((((((int)blockIdx.x) * 74088) + (((int)blockIdx.y) * 1764)) + (((int)blockIdx.z) * 126)) + (((int)threadIdx.z) * 42)) + vthread_s))] = input0[((((((((int)blockIdx.x) * 1778112) + (((int)blockIdx.y) * 42336)) + (((int)blockIdx.z) * 3024)) + (((int)threadIdx.z) * 1008)) + vthread_s))];
  }
}

// Saved Perf = 1.140860e-04 sec / run; Step Produced = 260; Planned Steps = 1000;