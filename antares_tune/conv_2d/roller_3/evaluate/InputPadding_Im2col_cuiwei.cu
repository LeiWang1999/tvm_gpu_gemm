// GLOBALS: input0:float16[128, 42, 42, 1008] -> temp0:float16[225792, 1008]
// BACKEND: c-cuda (default)
// CONFIG: null
// COMPUTE_V1: - N, C, F = 128, 1008, 336; HI = WI = 42; KW = KH = 1; SH = SW = 1; PH = PW = 0; HO = (HI - KH + PH * 2) // SH + 1; WO = (WI - KW + PW * 2) // SW + 1; einstein_v2(f"temp0[I, K] = input0[I / alter(`HOWO:{HO * WO}`), (I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`)), (I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`)), K % alter(`C:{C}`)].when([I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`) >= 0, I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`) < alter(`HI:{HI}`), I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`) >= 0, I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`) < alter(`WI:{WI}`)], const(0.0).cast(`float16`)) where I in I:{N * HO * WO}, K in K:{KH * KW * C}", { "input0": {"dtype": "float16", "shape": [f"N:{N}", f"HI:{HI}", f"WI:{WI}", f"C:{C}"]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> temp0:float16[225792, 1008]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(1) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ temp0) {
  // [thread_extent] blockIdx.x = 225792
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 1008
  // [thread_extent] threadIdx.y = 1
  temp0[(((((int)blockIdx.x) * 1008) + ((int)blockIdx.y)))] = input0[((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) / 42) % 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + ((int)blockIdx.y)))];
}
