// GLOBALS: input0:float16[128, 42, 42, 1008] -> output0:float16[225792, 1008]
// BACKEND: c-cuda (default)
// CONFIG: null
// COMPUTE_V1: - _N, _CI, _H, _W, _CO, _KH, _KW, _SH, _SW, _PH, _PW = 128, 1008, 42, 42, 336, 1, 1, 1, 1, 0, 0;_HO, _WO = (_H - _KH + _PH * 2) // _SH + 1, (_W - _KW + _PW * 2) // _SW + 1;_PHI, _PWI = _H + _PH * 2, _W + _PW * 2; _GM, _GN, _GK = _N * _HO * _WO, _CO, _CI * _KH * _KW; einstein_v2(f"temp0[N, PHI, PWI, C] = input0[N, -{_PH} + PHI, -{_PW} + PWI, C].when([-{_PH} + PHI >= 0, -{_PH} + PHI < {_H}, -{_PW} + PWI >= 0, -{_PW} + PWI < {_W}], const(0.0).cast(`float16`)) where PHI in {_PHI}, PWI in {_PWI};output0[GM, GK] = temp0[GM // ({_HO} * {_WO}), {_SH} * ((GM % ({_HO} * {_WO})) // {_WO}) + (GK // {_CI}) // {_KW}, {_SW} * ((GM % ({_HO} * {_WO})) % {_WO}) + (GK // {_CI}) % {_KW}, GK % {_CI}] where GM in {_GM}, GK in {_GK}", input_dict={"input0": {"dtype": "float16", "shape": [_N, _H, _W, _CI]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> output0:float16[225792, 1008]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(1) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 225792
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 1008
  // [thread_extent] threadIdx.y = 1
  output0[(((((int)blockIdx.x) * 1008) + ((int)blockIdx.y)))] = input0[((((((((int)blockIdx.x) / 1764) * 1778112) + (((((int)blockIdx.x) % 1764) / 42) * 42336)) + ((((int)blockIdx.x) % 42) * 1008)) + ((int)blockIdx.y)))];
}
