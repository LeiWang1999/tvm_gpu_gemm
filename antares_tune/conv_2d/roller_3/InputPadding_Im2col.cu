// GLOBALS: input0:float16[128, 42, 42, 1008] -> temp0:float16[225792, 1008]
// BACKEND: c-cuda (default)
// CONFIG: {"Ftemp0:D0": [-1, 3, 2, 1], "Ftemp0:D1": [-1, 2, 504, 1], "Ftemp0:O": [1, 0], "Ftemp0:S": 3, "Ftemp0:R": 0}
// COMPUTE_V1: - N, C, F = 128, 1008, 336; HI = WI = 42; KW = KH = 1; SH = SW = 1; PH = PW = 0; HO = (HI - KH + PH * 2) // SH + 1; WO = (WI - KW + PW * 2) // SW + 1; einstein_v2(f"temp0[I, K] = input0[I / alter(`HOWO:{HO * WO}`), (I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`)), (I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`)), K % alter(`C:{C}`)].when([I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`) >= 0, I / alter(`WO:{WO}`) % alter(`HO:{HO}`) * alter(`SH:{SH}`) + K / alter(`KWC:{KW * C}`) - alter(`PH:{PH}`) < alter(`HI:{HI}`), I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`) >= 0, I % alter(`WO:{WO}`) * alter(`SW:{SW}`) + K / alter(`C:{C}`) % alter(`KW:{KW}`) - alter(`PW:{PW}`) < alter(`WI:{WI}`)], const(0.0).cast(`float16`)) where I in I:{N * HO * WO}, K in K:{KH * KW * C}", { "input0": {"dtype": "float16", "shape": [f"N:{N}", f"HI:{HI}", f"WI:{WI}", f"C:{C}"]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> temp0:float16[225792, 1008]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(1008) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ temp0) {
  // [thread_extent] blockIdx.x = 37632
  // [thread_extent] threadIdx.x = 2
  // [thread_extent] blockIdx.y = 1
  // [thread_extent] threadIdx.y = 504
  temp0[((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)))] = input0[((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) / 1764) * 1778112) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) / 42) % 42) * 42336)) + ((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) % 42) * 1008)) + ((int)threadIdx.y)))];
  temp0[(((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)) + 504))] = input0[(((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) / 1764) * 1778112) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) / 42) % 42) * 42336)) + ((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) % 42) * 1008)) + ((int)threadIdx.y)) + 504))];
  temp0[(((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)) + 2016))] = input0[(((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) / 1764) * 1778112) + ((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) / 42) % 42) * 42336)) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) % 42) * 1008)) + ((int)threadIdx.y)))];
  temp0[(((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)) + 2520))] = input0[((((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) / 1764) * 1778112) + ((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) / 42) % 42) * 42336)) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 2) % 42) * 1008)) + ((int)threadIdx.y)) + 504))];
  temp0[(((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)) + 4032))] = input0[(((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) / 1764) * 1778112) + ((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) / 42) % 42) * 42336)) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) % 42) * 1008)) + ((int)threadIdx.y)))];
  temp0[(((((((int)blockIdx.x) * 6048) + (((int)threadIdx.x) * 1008)) + ((int)threadIdx.y)) + 4536))] = input0[((((((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) / 1764) * 1778112) + ((((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) / 42) % 42) * 42336)) + (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) + 4) % 42) * 1008)) + ((int)threadIdx.y)) + 504))];
}

// Saved Perf = 1.066720e-03 sec / run; Step Produced = 491; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.