// GLOBALS: input0:float16[128, 42, 42, 1008] -> output0:float16[128, 42, 42, 1024]
// BACKEND: c-cuda (default)
// CONFIG: {"Foutput0:D0": [-1, 2, 1, 1], "Foutput0:D1": [-1, 6, 1, 1], "Foutput0:D2": [-1, 1, 2, 1], "Foutput0:D3": [-1, 256], "Foutput0:O": [3, 0, 1, 2], "Foutput0:S": 1, "Foutput0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[N, H, W, C] = input0[N, H, W, C].when([N < 128, H < 42, W < 42, C < 1008], const(0.0).cast(`float16`)) where N in 128, H in 42, W in 42, C in 1024 ", input_dict={"input0": {"dtype": "float16", "shape": [128, 42, 42, 1008]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 42, 42, 1008] -> output0:float16[128, 42, 42, 1024]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(2) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 64
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 7
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 21
  // [thread_extent] threadIdx.z = 2
  for (int vthread_s = 0; vthread_s < 256; ++vthread_s) {
    output0[((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s))] = input0[((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s))];
  }
  for (int vthread_s1 = 0; vthread_s1 < 256; ++vthread_s1) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s1) + 256))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s1) + 256))];
  }
  for (int vthread_s2 = 0; vthread_s2 < 256; ++vthread_s2) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s2) + 512))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s2) + 512))];
  }
  for (int vthread_s3 = 0; vthread_s3 < 256; ++vthread_s3) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s3) + 768))] = ((vthread_s3 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s3) + 768))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s4 = 0; vthread_s4 < 256; ++vthread_s4) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s4) + 1806336))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s4) + 1778112))];
  }
  for (int vthread_s5 = 0; vthread_s5 < 256; ++vthread_s5) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s5) + 1806592))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s5) + 1778368))];
  }
  for (int vthread_s6 = 0; vthread_s6 < 256; ++vthread_s6) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s6) + 1806848))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s6) + 1778624))];
  }
  for (int vthread_s7 = 0; vthread_s7 < 256; ++vthread_s7) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s7) + 1807104))] = ((vthread_s7 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s7) + 1778880))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s8 = 0; vthread_s8 < 256; ++vthread_s8) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s8) + 43008))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s8) + 42336))];
  }
  for (int vthread_s9 = 0; vthread_s9 < 256; ++vthread_s9) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s9) + 43264))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s9) + 42592))];
  }
  for (int vthread_s10 = 0; vthread_s10 < 256; ++vthread_s10) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s10) + 43520))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s10) + 42848))];
  }
  for (int vthread_s11 = 0; vthread_s11 < 256; ++vthread_s11) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s11) + 43776))] = ((vthread_s11 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s11) + 43104))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s12 = 0; vthread_s12 < 256; ++vthread_s12) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s12) + 1849344))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s12) + 1820448))];
  }
  for (int vthread_s13 = 0; vthread_s13 < 256; ++vthread_s13) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s13) + 1849600))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s13) + 1820704))];
  }
  for (int vthread_s14 = 0; vthread_s14 < 256; ++vthread_s14) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s14) + 1849856))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s14) + 1820960))];
  }
  for (int vthread_s15 = 0; vthread_s15 < 256; ++vthread_s15) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s15) + 1850112))] = ((vthread_s15 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s15) + 1821216))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s16 = 0; vthread_s16 < 256; ++vthread_s16) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s16) + 86016))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s16) + 84672))];
  }
  for (int vthread_s17 = 0; vthread_s17 < 256; ++vthread_s17) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s17) + 86272))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s17) + 84928))];
  }
  for (int vthread_s18 = 0; vthread_s18 < 256; ++vthread_s18) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s18) + 86528))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s18) + 85184))];
  }
  for (int vthread_s19 = 0; vthread_s19 < 256; ++vthread_s19) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s19) + 86784))] = ((vthread_s19 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s19) + 85440))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s20 = 0; vthread_s20 < 256; ++vthread_s20) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s20) + 1892352))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s20) + 1862784))];
  }
  for (int vthread_s21 = 0; vthread_s21 < 256; ++vthread_s21) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s21) + 1892608))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s21) + 1863040))];
  }
  for (int vthread_s22 = 0; vthread_s22 < 256; ++vthread_s22) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s22) + 1892864))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s22) + 1863296))];
  }
  for (int vthread_s23 = 0; vthread_s23 < 256; ++vthread_s23) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s23) + 1893120))] = ((vthread_s23 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s23) + 1863552))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s24 = 0; vthread_s24 < 256; ++vthread_s24) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s24) + 129024))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s24) + 127008))];
  }
  for (int vthread_s25 = 0; vthread_s25 < 256; ++vthread_s25) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s25) + 129280))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s25) + 127264))];
  }
  for (int vthread_s26 = 0; vthread_s26 < 256; ++vthread_s26) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s26) + 129536))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s26) + 127520))];
  }
  for (int vthread_s27 = 0; vthread_s27 < 256; ++vthread_s27) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s27) + 129792))] = ((vthread_s27 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s27) + 127776))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s28 = 0; vthread_s28 < 256; ++vthread_s28) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s28) + 1935360))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s28) + 1905120))];
  }
  for (int vthread_s29 = 0; vthread_s29 < 256; ++vthread_s29) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s29) + 1935616))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s29) + 1905376))];
  }
  for (int vthread_s30 = 0; vthread_s30 < 256; ++vthread_s30) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s30) + 1935872))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s30) + 1905632))];
  }
  for (int vthread_s31 = 0; vthread_s31 < 256; ++vthread_s31) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s31) + 1936128))] = ((vthread_s31 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s31) + 1905888))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s32 = 0; vthread_s32 < 256; ++vthread_s32) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s32) + 172032))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s32) + 169344))];
  }
  for (int vthread_s33 = 0; vthread_s33 < 256; ++vthread_s33) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s33) + 172288))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s33) + 169600))];
  }
  for (int vthread_s34 = 0; vthread_s34 < 256; ++vthread_s34) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s34) + 172544))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s34) + 169856))];
  }
  for (int vthread_s35 = 0; vthread_s35 < 256; ++vthread_s35) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s35) + 172800))] = ((vthread_s35 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s35) + 170112))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s36 = 0; vthread_s36 < 256; ++vthread_s36) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s36) + 1978368))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s36) + 1947456))];
  }
  for (int vthread_s37 = 0; vthread_s37 < 256; ++vthread_s37) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s37) + 1978624))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s37) + 1947712))];
  }
  for (int vthread_s38 = 0; vthread_s38 < 256; ++vthread_s38) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s38) + 1978880))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s38) + 1947968))];
  }
  for (int vthread_s39 = 0; vthread_s39 < 256; ++vthread_s39) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s39) + 1979136))] = ((vthread_s39 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s39) + 1948224))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s40 = 0; vthread_s40 < 256; ++vthread_s40) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s40) + 215040))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s40) + 211680))];
  }
  for (int vthread_s41 = 0; vthread_s41 < 256; ++vthread_s41) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s41) + 215296))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s41) + 211936))];
  }
  for (int vthread_s42 = 0; vthread_s42 < 256; ++vthread_s42) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s42) + 215552))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s42) + 212192))];
  }
  for (int vthread_s43 = 0; vthread_s43 < 256; ++vthread_s43) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s43) + 215808))] = ((vthread_s43 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s43) + 212448))] : __float2half_rn(0.000000e+00f));
  }
  for (int vthread_s44 = 0; vthread_s44 < 256; ++vthread_s44) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s44) + 2021376))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s44) + 1989792))];
  }
  for (int vthread_s45 = 0; vthread_s45 < 256; ++vthread_s45) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s45) + 2021632))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s45) + 1990048))];
  }
  for (int vthread_s46 = 0; vthread_s46 < 256; ++vthread_s46) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s46) + 2021888))] = input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s46) + 1990304))];
  }
  for (int vthread_s47 = 0; vthread_s47 < 256; ++vthread_s47) {
    output0[(((((((((int)blockIdx.x) * 3612672) + (((int)blockIdx.y) * 258048)) + (((int)blockIdx.z) * 2048)) + (((int)threadIdx.z) * 1024)) + vthread_s47) + 2022144))] = ((vthread_s47 < 240) ? input0[(((((((((int)blockIdx.x) * 3556224) + (((int)blockIdx.y) * 254016)) + (((int)blockIdx.z) * 2016)) + (((int)threadIdx.z) * 1008)) + vthread_s47) + 1990560))] : __float2half_rn(0.000000e+00f));
  }
}

// Saved Perf = 3.275290e-03 sec / run; Step Produced = 841; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.