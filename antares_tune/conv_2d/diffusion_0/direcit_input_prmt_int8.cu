// GLOBALS: input0:int8[128, 32, 32, 1280] -> output0:int8[8, 32, 32, 32, 80, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 1, 4, 1], "F___output0:D1": [-1, 1, 1, 2], "F___output0:D2": [-1, 32, 40, 1], "F___output0:D3": [-1, 1], "F___output0:O": [2, 3, 1, 0], "F___output0:S": 2, "F___output0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "int8", "shape": [128, 32, 32, 1280]}, "output0": {"dtype": "int8", "shape": [8, 32, 32, 32, 80, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[128, 32, 32, 1280] -> output0:int8[8, 32, 32, 32, 80, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(160) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 32
  // [thread_extent] threadIdx.x = 4
  // [thread_extent] blockIdx.y = 16
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 1
  // [thread_extent] threadIdx.z = 40
  for (int vthread_s = 0; vthread_s < 32; ++vthread_s) {
    ((output0[((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s * 40) + ((int)threadIdx.z)) & 15)))]) = (input0[((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s * 40)) + ((int)threadIdx.z)))]));
  }
  for (int vthread_s1 = 0; vthread_s1 < 32; ++vthread_s1) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s1 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s1 * 40) + ((int)threadIdx.z)) & 15)) + 41943040))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s1 * 40)) + ((int)threadIdx.z)) + 40960))]));
  }
  for (int vthread_s2 = 0; vthread_s2 < 32; ++vthread_s2) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s2 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s2 * 40) + ((int)threadIdx.z)) & 15)) + 83886080))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s2 * 40)) + ((int)threadIdx.z)) + 81920))]));
  }
  for (int vthread_s3 = 0; vthread_s3 < 32; ++vthread_s3) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s3 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s3 * 40) + ((int)threadIdx.z)) & 15)) + 125829120))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s3 * 40)) + ((int)threadIdx.z)) + 122880))]));
  }
  for (int vthread_s4 = 0; vthread_s4 < 32; ++vthread_s4) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s4 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s4 * 40) + ((int)threadIdx.z)) & 15)) + 167772160))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s4 * 40)) + ((int)threadIdx.z)) + 163840))]));
  }
  for (int vthread_s5 = 0; vthread_s5 < 32; ++vthread_s5) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s5 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s5 * 40) + ((int)threadIdx.z)) & 15)) + 209715200))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s5 * 40)) + ((int)threadIdx.z)) + 204800))]));
  }
  for (int vthread_s6 = 0; vthread_s6 < 32; ++vthread_s6) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s6 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s6 * 40) + ((int)threadIdx.z)) & 15)) + 251658240))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s6 * 40)) + ((int)threadIdx.z)) + 245760))]));
  }
  for (int vthread_s7 = 0; vthread_s7 < 32; ++vthread_s7) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s7 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s7 * 40) + ((int)threadIdx.z)) & 15)) + 293601280))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s7 * 40)) + ((int)threadIdx.z)) + 286720))]));
  }
  for (int vthread_s8 = 0; vthread_s8 < 32; ++vthread_s8) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s8 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s8 * 40) + ((int)threadIdx.z)) & 15)) + 1310720))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s8 * 40)) + ((int)threadIdx.z)) + 1280))]));
  }
  for (int vthread_s9 = 0; vthread_s9 < 32; ++vthread_s9) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s9 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s9 * 40) + ((int)threadIdx.z)) & 15)) + 43253760))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s9 * 40)) + ((int)threadIdx.z)) + 42240))]));
  }
  for (int vthread_s10 = 0; vthread_s10 < 32; ++vthread_s10) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s10 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s10 * 40) + ((int)threadIdx.z)) & 15)) + 85196800))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s10 * 40)) + ((int)threadIdx.z)) + 83200))]));
  }
  for (int vthread_s11 = 0; vthread_s11 < 32; ++vthread_s11) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s11 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s11 * 40) + ((int)threadIdx.z)) & 15)) + 127139840))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s11 * 40)) + ((int)threadIdx.z)) + 124160))]));
  }
  for (int vthread_s12 = 0; vthread_s12 < 32; ++vthread_s12) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s12 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s12 * 40) + ((int)threadIdx.z)) & 15)) + 169082880))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s12 * 40)) + ((int)threadIdx.z)) + 165120))]));
  }
  for (int vthread_s13 = 0; vthread_s13 < 32; ++vthread_s13) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s13 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s13 * 40) + ((int)threadIdx.z)) & 15)) + 211025920))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s13 * 40)) + ((int)threadIdx.z)) + 206080))]));
  }
  for (int vthread_s14 = 0; vthread_s14 < 32; ++vthread_s14) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s14 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s14 * 40) + ((int)threadIdx.z)) & 15)) + 252968960))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s14 * 40)) + ((int)threadIdx.z)) + 247040))]));
  }
  for (int vthread_s15 = 0; vthread_s15 < 32; ++vthread_s15) {
    ((output0[(((((((((int)blockIdx.y) * 2621440) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) / 16) * 40960)) + ((((vthread_s15 * 40) + ((int)threadIdx.z)) / 16) * 1280)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) * 16)) + (((vthread_s15 * 40) + ((int)threadIdx.z)) & 15)) + 294912000))]) = (input0[(((((((((int)blockIdx.x) * 5242880) + (((int)threadIdx.x) * 1310720)) + (((int)blockIdx.y) * 2560)) + (vthread_s15 * 40)) + ((int)threadIdx.z)) + 288000))]));
  }
}

// Saved Perf = 9.504470e-05 sec / run; Step Produced = 593; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.