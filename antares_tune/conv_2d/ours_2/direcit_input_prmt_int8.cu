// GLOBALS: input0:int8[128, 56, 56, 128] -> output0:int8[8, 56, 56, 32, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 1, 16, 1], "F___output0:D1": [-1, 1, 1, 1], "F___output0:D2": [-1, 4, 2, 1], "F___output0:D3": [-1, 2], "F___output0:O": [3, 0, 2, 1], "F___output0:S": 2, "F___output0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "int8", "shape": [128, 56, 56, 128]}, "output0": {"dtype": "int8", "shape": [8, 56, 56, 32, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[128, 56, 56, 128] -> output0:int8[8, 56, 56, 32, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(32) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 8
  // [thread_extent] threadIdx.x = 16
  // [thread_extent] blockIdx.y = 8
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 7
  // [thread_extent] threadIdx.z = 2
  for (int vthread_s = 0; vthread_s < 64; ++vthread_s) {
    ((output0[((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((vthread_s / 8) * 8192)) + (((int)blockIdx.x) * 256)) + ((vthread_s & 7) * 32)) + ((int)threadIdx.x)))]) = (input0[(((((((vthread_s * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))]));
    ((output0[((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((((vthread_s * 2) + 1) / 16) * 8192)) + (((int)blockIdx.x) * 256)) + ((((vthread_s * 2) + 1) & 15) * 16)) + ((int)threadIdx.x)))]) = (input0[((((((((vthread_s * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 401408))]));
  }
  for (int vthread_s1 = 0; vthread_s1 < 64; ++vthread_s1) {
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((vthread_s1 / 8) * 8192)) + (((int)blockIdx.x) * 256)) + ((vthread_s1 & 7) * 32)) + ((int)threadIdx.x)) + 917504))]) = (input0[((((((((vthread_s1 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 256))]));
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((((vthread_s1 * 2) + 1) / 16) * 8192)) + (((int)blockIdx.x) * 256)) + ((((vthread_s1 * 2) + 1) & 15) * 16)) + ((int)threadIdx.x)) + 917504))]) = (input0[((((((((vthread_s1 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 401664))]));
  }
  for (int vthread_s2 = 0; vthread_s2 < 64; ++vthread_s2) {
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((vthread_s2 / 8) * 8192)) + (((int)blockIdx.x) * 256)) + ((vthread_s2 & 7) * 32)) + ((int)threadIdx.x)) + 1835008))]) = (input0[((((((((vthread_s2 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 512))]));
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((((vthread_s2 * 2) + 1) / 16) * 8192)) + (((int)blockIdx.x) * 256)) + ((((vthread_s2 * 2) + 1) & 15) * 16)) + ((int)threadIdx.x)) + 1835008))]) = (input0[((((((((vthread_s2 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 401920))]));
  }
  for (int vthread_s3 = 0; vthread_s3 < 64; ++vthread_s3) {
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((vthread_s3 / 8) * 8192)) + (((int)blockIdx.x) * 256)) + ((vthread_s3 & 7) * 32)) + ((int)threadIdx.x)) + 2752512))]) = (input0[((((((((vthread_s3 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 768))]));
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.z) * 3670016)) + (((int)threadIdx.z) * 458752)) + ((((vthread_s3 * 2) + 1) / 16) * 8192)) + (((int)blockIdx.x) * 256)) + ((((vthread_s3 * 2) + 1) & 15) * 16)) + ((int)threadIdx.x)) + 2752512))]) = (input0[((((((((vthread_s3 * 802816) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.z) * 1024)) + (((int)threadIdx.z) * 128)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 402176))]));
  }
}

// Saved Perf = 2.077610e-05 sec / run; Step Produced = 996; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.