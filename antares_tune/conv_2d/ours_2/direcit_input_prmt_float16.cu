// GLOBALS: input0:float16[128, 56, 56, 128] -> output0:float16[8, 56, 56, 32, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 1, 1, 1], "F___output0:D1": [-1, 1, 1, 1], "F___output0:D2": [-1, 2, 32, 1], "F___output0:D3": [-1, 1], "F___output0:O": [2, 1, 0, 3], "F___output0:S": 1, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "float16", "shape": [128, 56, 56, 128]}, "output0": {"dtype": "float16", "shape": [8, 56, 56, 32, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 56, 56, 128] -> output0:float16[8, 56, 56, 32, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(32) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 56
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 8
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 2
  // [thread_extent] threadIdx.z = 32
  for (int vthread_s = 0; vthread_s < 128; ++vthread_s) {
    ((output0[((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.x) * 458752)) + ((vthread_s / 16) * 8192)) + (((int)blockIdx.z) * 1024)) + ((((int)threadIdx.z) / 16) * 256)) + ((vthread_s & 15) * 16)) + (((int)threadIdx.z) & 15)))]) = (input0[((((((vthread_s * 401408) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.x) * 128)) + (((int)blockIdx.z) * 64)) + ((int)threadIdx.z)))]));
    ((output0[(((((((((((int)blockIdx.y) * 25690112) + (((int)blockIdx.x) * 458752)) + ((vthread_s / 16) * 8192)) + (((int)blockIdx.z) * 1024)) + ((((int)threadIdx.z) / 16) * 256)) + ((vthread_s & 15) * 16)) + (((int)threadIdx.z) & 15)) + 512))]) = (input0[(((((((vthread_s * 401408) + (((int)blockIdx.y) * 7168)) + (((int)blockIdx.x) * 128)) + (((int)blockIdx.z) * 64)) + ((int)threadIdx.z)) + 32))]));
  }
}

// Saved Perf = 3.758740e-05 sec / run; Step Produced = 832; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.