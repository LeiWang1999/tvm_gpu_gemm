// GLOBALS: input0:int8[128, 56, 56, 256] -> output0:int8[8, 56, 56, 16, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 1, 1, 2], "F___output0:D1": [-1, 2, 16, 1], "F___output0:D2": [-1, 1, 1, 1], "F___output0:D3": [-1, 16], "F___output0:O": [2, 3, 0, 1], "F___output0:S": 1, "F___output0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[H, W, N // 16, C // 16, N % 16, C % 16] =. input0[N, H, W, C]", input_dict={"input0": {"dtype": "int8", "shape": [128, 56, 56, 256]}, "output0": {"dtype": "int8", "shape": [8, 56, 56, 16, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[128, 56, 56, 256] -> output0:int8[8, 56, 56, 16, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(16) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 4
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.y = 16
  // [thread_extent] blockIdx.z = 56
  // [thread_extent] threadIdx.z = 1
  for (int vthread_s = 0; vthread_s < 16; ++vthread_s) {
    for (int vthread_s1 = 0; vthread_s1 < 16; ++vthread_s1) {
      ((output0[(((((((((int)blockIdx.x) * 25690112) + (((int)blockIdx.z) * 229376)) + (((int)blockIdx.y) * 8192)) + (vthread_s * 256)) + (((int)threadIdx.y) * 16)) + vthread_s1))]) = (input0[(((((((((int)blockIdx.y) * 25690112) + (((int)threadIdx.y) * 802816)) + (((int)blockIdx.x) * 28672)) + (((int)blockIdx.z) * 256)) + (vthread_s * 16)) + vthread_s1))]));
    }
  }
  for (int vthread_s2 = 0; vthread_s2 < 16; ++vthread_s2) {
    for (int vthread_s3 = 0; vthread_s3 < 16; ++vthread_s3) {
      ((output0[((((((((((int)blockIdx.x) * 25690112) + (((int)blockIdx.z) * 229376)) + (((int)blockIdx.y) * 8192)) + (vthread_s2 * 256)) + (((int)threadIdx.y) * 16)) + vthread_s3) + 12845056))]) = (input0[((((((((((int)blockIdx.y) * 25690112) + (((int)threadIdx.y) * 802816)) + (((int)blockIdx.x) * 28672)) + (((int)blockIdx.z) * 256)) + (vthread_s2 * 16)) + vthread_s3) + 14336))]));
    }
  }
  for (int vthread_s4 = 0; vthread_s4 < 16; ++vthread_s4) {
    for (int vthread_s5 = 0; vthread_s5 < 16; ++vthread_s5) {
      ((output0[((((((((((int)blockIdx.x) * 25690112) + (((int)blockIdx.z) * 229376)) + (((int)blockIdx.y) * 8192)) + (vthread_s4 * 256)) + (((int)threadIdx.y) * 16)) + vthread_s5) + 4096))]) = (input0[((((((((((int)blockIdx.y) * 25690112) + (((int)threadIdx.y) * 802816)) + (((int)blockIdx.x) * 28672)) + (((int)blockIdx.z) * 256)) + (vthread_s4 * 16)) + vthread_s5) + 12845056))]));
    }
  }
  for (int vthread_s6 = 0; vthread_s6 < 16; ++vthread_s6) {
    for (int vthread_s7 = 0; vthread_s7 < 16; ++vthread_s7) {
      ((output0[((((((((((int)blockIdx.x) * 25690112) + (((int)blockIdx.z) * 229376)) + (((int)blockIdx.y) * 8192)) + (vthread_s6 * 256)) + (((int)threadIdx.y) * 16)) + vthread_s7) + 12849152))]) = (input0[((((((((((int)blockIdx.y) * 25690112) + (((int)threadIdx.y) * 802816)) + (((int)blockIdx.x) * 28672)) + (((int)blockIdx.z) * 256)) + (vthread_s6 * 16)) + vthread_s7) + 12859392))]));
    }
  }
}

// Saved Perf = 1.160520e-04 sec / run; Step Produced = 685; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.