// GLOBALS: input0:float16[128, 28, 28, 128] -> output0:float16[128, 28, 28, 128]
// BACKEND: c-cuda (default)
// CONFIG: {"Foutput0:D0": [-1, 1, 1, 2], "Foutput0:D1": [-1, 1, 1, 1], "Foutput0:D2": [-1, 1, 2, 1], "Foutput0:D3": [-1, 2], "Foutput0:O": [2, 3, 0, 1], "Foutput0:S": 2, "Foutput0:R": 1}
// COMPUTE_V1: - _N, _CI, _H, _W, _CO, _KH, _KW, _SH, _SW, _PH, _PW = 128, 128, 28, 28, 128, 3, 3, 1, 1, 0, 0;_HO, _WO = (_H - _KH + _PH * 2) // _SH + 1, (_W - _KW + _PW * 2) // _SW + 1;_PHI, _PWI = _H + _PH * 2, _W + _PW * 2; _GM, _GN, _GK = _N * _HO * _WO, _CO, _CI * _KH * _KW; einstein_v2(f"output0[N, PHI, PWI, C] = input0[N, -{_PH} + PHI, -{_PW} + PWI, C].when([-{_PH} + PHI >= 0, -{_PH} + PHI < {_H}, -{_PW} + PWI >= 0, -{_PW} + PWI < {_W}], const(0.0).cast(`float16`)) where PHI in {_PHI}, PWI in {_PWI};", input_dict={"input0": {"dtype": "float16", "shape": [_N, _H, _W, _CI]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float16[128, 28, 28, 128] -> output0:float16[128, 28, 28, 128]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(2) void template_op_kernel0(half* __restrict__ input0, half* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 64
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 28
  // [thread_extent] threadIdx.y = 1
  // [thread_extent] blockIdx.z = 14
  // [thread_extent] threadIdx.z = 2
  for (int vthread_s = 0; vthread_s < 64; ++vthread_s) {
    output0[((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s * 2)))] = input0[((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s * 2)))];
    output0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s * 2)) + 1))] = input0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s * 2)) + 1))];
  }
  for (int vthread_s1 = 0; vthread_s1 < 64; ++vthread_s1) {
    output0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s1 * 2)) + 100352))] = input0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s1 * 2)) + 100352))];
    output0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s1 * 2)) + 100353))] = input0[(((((((((int)blockIdx.x) * 200704) + (((int)blockIdx.y) * 3584)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (vthread_s1 * 2)) + 100353))];
  }
}

// Saved Perf = 1.918960e-04 sec / run; Step Produced = 832; Planned Steps = 1000;
// Antares Tuning Completed in 1000 steps.