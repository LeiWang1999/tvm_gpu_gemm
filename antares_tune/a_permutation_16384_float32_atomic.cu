// GLOBALS: input0:float32[16384, 16384] -> output0:float32[1024, 2048, 16, 8]
// BACKEND: c-cuda (default)
// CONFIG: {"Foutput0:D0": [-1, 1, 1, 1], "Foutput0:D1": [-1, 64, 4, 1], "Foutput0:D2": [-1, 1, 2, 2], "Foutput0:D3": [-1, 2], "Foutput0:O": [3, 0, 2, 1], "Foutput0:S": 3, "Foutput0:R": 0}
// COMPUTE_V1: - einstein_v2("output0[MM, NN, M, N] = input0[MM * 16 + M, NN * 8 + N] where MM in 1024, NN in 2048, M in 16, N in 8", input_dict={"input0": {"dtype": "float32", "shape": [16384, 16384]}, "output0": {"dtype": "float32", "shape": [1024, 2048, 16, 8]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float32[16384, 16384] -> output0:float32[1024, 2048, 16, 8]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(8) void template_op_kernel0(float* __restrict__ input0, float* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 1024
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 8
  // [thread_extent] threadIdx.y = 4
  // [thread_extent] blockIdx.z = 4
  // [thread_extent] threadIdx.z = 2
  for (int vthread_s = 0; vthread_s < 64; ++vthread_s) {
    output0[(((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)))] = input0[(((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 1))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 1))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 2))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 2))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 3))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 3))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 4))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 4))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 5))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 5))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 6))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 6))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 7))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 7))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 8))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16384))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 9))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16385))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 10))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16386))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 11))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16387))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 12))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16388))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 13))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16389))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 14))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16390))];
    output0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.y) * 32768)) + (vthread_s * 512)) + (((int)threadIdx.y) * 128)) + (((int)blockIdx.z) * 32)) + (((int)threadIdx.z) * 16)) + 15))] = input0[((((((((((int)blockIdx.x) * 262144) + (((int)blockIdx.z) * 65536)) + (((int)threadIdx.z) * 32768)) + (((int)blockIdx.y) * 2048)) + (vthread_s * 32)) + (((int)threadIdx.y) * 8)) + 16391))];
  }
}

// Saved Perf = 2.612440e-03 sec / run; Step Produced = 463; Planned Steps = 1000;