// GLOBALS: input0:float32[16384, 16384] -> output0:float32[1024, 2048, 16, 8]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 2, 1, 1], "F___output0:D1": [-1, 1, 1024, 1], "F___output0:O": [1, 0], "F___output0:S": 0, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[M // 16, N // 8, M % 16, N % 8] =. input0[M, N]", input_dict={"input0": {"dtype": "float32", "shape": [16384, 16384]}, "output0": {"dtype": "float32", "shape": [1024, 2048, 16, 8]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:float32[16384, 16384] -> output0:float32[1024, 2048, 16, 8]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(1024) void template_op_kernel0(float* __restrict__ input0, float* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 8192
  // [thread_extent] threadIdx.x = 1
  // [thread_extent] blockIdx.y = 16
  // [thread_extent] threadIdx.y = 1024
  ((output0[(((((((((int)blockIdx.x) / 8) * 262144) + (((int)blockIdx.y) * 16384)) + ((((int)threadIdx.y) / 8) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((int)threadIdx.y) & 7)))]) = (input0[((((((int)blockIdx.x) * 32768) + (((int)blockIdx.y) * 1024)) + ((int)threadIdx.y)))]));
  ((output0[(((((((((((int)blockIdx.x) * 2) + 1) / 16) * 262144) + (((int)blockIdx.y) * 16384)) + ((((int)threadIdx.y) / 8) * 128)) + ((((((int)blockIdx.x) * 2) + 1) & 15) * 8)) + (((int)threadIdx.y) & 7)))]) = (input0[(((((((int)blockIdx.x) * 32768) + (((int)blockIdx.y) * 1024)) + ((int)threadIdx.y)) + 16384))]));
}

// Saved Perf = 2.526300e-03 sec / run; Step Produced = 314; Planned Steps = 1000;