// GLOBALS: input0:int8[16384, 16384] -> output0:int8[1024, 1024, 16, 16]
// BACKEND: c-cuda (default)
// CONFIG: {"F___output0:D0": [-1, 2, 2, 2], "F___output0:D1": [-1, 32, 128, 1], "F___output0:O": [0, 1], "F___output0:S": 4, "F___output0:R": 1}
// COMPUTE_V1: - einstein_v2("output0[M / 16, N / 16, M % 16, N % 16] =. input0[M, N]", input_dict={"input0": {"dtype": "int8", "shape": [16384, 16384]}, "output0": {"dtype": "int8", "shape": [1024, 1024, 16, 16]}})


// ---------------------------------------------------------------------------
// LOCAL: template_op_kernel0 -- input0:int8[16384, 16384] -> output0:int8[1024, 1024, 16, 16]

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef __CUDA_COMMON_MACRO__
#define __CUDA_COMMON_MACRO__

#if (__CUDA_ARCH__ >= 600)
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) { return a > b ? a : b; }
__forceinline__ __device__ __half hmin(const __half &a, const __half &b) { return a < b ? a : b; }
#endif

#endif


extern "C" __global__ __launch_bounds__(256) void template_op_kernel0(char* __restrict__ input0, char* __restrict__ output0) {
  // [thread_extent] blockIdx.x = 2048
  // [thread_extent] threadIdx.x = 2
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.y = 128
  for (int vthread_s = 0; vthread_s < 32; ++vthread_s) {
    ((output0[((((((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) / 16) * 262144) + (((int)blockIdx.y) * 65536)) + (vthread_s * 2048)) + ((((int)threadIdx.y) / 16) * 256)) + ((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) & 15) * 16)) + (((int)threadIdx.y) & 15)))]) = (input0[((((((((int)blockIdx.x) * 131072) + (((int)threadIdx.x) * 32768)) + (((int)blockIdx.y) * 4096)) + (vthread_s * 128)) + ((int)threadIdx.y)))]));
    ((output0[(((((((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 1) / 16) * 262144) + (((int)blockIdx.y) * 65536)) + (vthread_s * 2048)) + ((((int)threadIdx.y) / 16) * 256)) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 1) & 15) * 16)) + (((int)threadIdx.y) & 15)))]) = (input0[(((((((((int)blockIdx.x) * 131072) + (((int)threadIdx.x) * 32768)) + (((int)blockIdx.y) * 4096)) + (vthread_s * 128)) + ((int)threadIdx.y)) + 16384))]));
    ((output0[(((((((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 4) / 16) * 262144) + (((int)blockIdx.y) * 65536)) + (vthread_s * 2048)) + ((((int)threadIdx.y) / 16) * 256)) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 4) & 15) * 16)) + (((int)threadIdx.y) & 15)))]) = (input0[(((((((((int)blockIdx.x) * 131072) + (((int)threadIdx.x) * 32768)) + (((int)blockIdx.y) * 4096)) + (vthread_s * 128)) + ((int)threadIdx.y)) + 65536))]));
    ((output0[(((((((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 5) / 16) * 262144) + (((int)blockIdx.y) * 65536)) + (vthread_s * 2048)) + ((((int)threadIdx.y) / 16) * 256)) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + 5) & 15) * 16)) + (((int)threadIdx.y) & 15)))]) = (input0[(((((((((int)blockIdx.x) * 131072) + (((int)threadIdx.x) * 32768)) + (((int)blockIdx.y) * 4096)) + (vthread_s * 128)) + ((int)threadIdx.y)) + 81920))]));
  }
}

// Saved Perf = 7.146940e-04 sec / run; Step Produced = 30; Planned Steps = 1000;