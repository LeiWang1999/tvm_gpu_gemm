#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [512, 512, 16, 16], []),
             B: Buffer(B_1: Pointer(global float16), float16, [512, 512, 16, 16], []),
             C: Buffer(C_1: Pointer(global float16), float16, [512, 512, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[512, 512, 16, 16])
    A_shared_warp = alloc_buffer(float16[512, 512, 32, 8])
    B_shared = alloc_buffer(float16[512, 512, 16, 16])
    B_shared_warp = alloc_buffer(float16[512, 512, 32, 8])
    C_warp = alloc_buffer(float16[512, 512, 32, 8])
    for (ii_0: int32, 0, 64) "thread_binding" {
      for (jj_0_0: int32, 0, 2) "thread_binding" {
        for (jj_0_1: int32, 0, 16) "thread_binding" {
          for (ii_1: int32, 0, 1) "thread_binding" {
            for (jj_1: int32, 0, 4) "thread_binding" {
              for (ii_2_init: int32, 0, 8) {
                for (jj_2_init: int32, 0, 4) {
                  block([512, 512, 1, 1], "B_init_o") as [vii, vjj, vi_o, vj_o] {
                    bind(vii, (((ii_0*8) + (ii_1*8)) + ii_2_init))
                    bind(vjj, ((((jj_0_0*256) + (jj_0_1*16)) + (jj_1*4)) + jj_2_init))
                    bind(vi_o, 0)
                    bind(vj_o, 0)
                    tir.reads([])
                    tir.writes([C_warp[vii, vjj, 0:32, 0:8]])
                    C_warp_1 = match_buffer(C_warp[vii, vjj, 0:32, 0:8])
                    attr [IterVar(tx: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_fill(8, C_warp_2: Pointer(warp float16), elem_offset: int32, dtype=float16)
                }
              }
              for (kk_0: int32, 0, 256) {
                for (ax0_ax1_ax2_ax3_fused_0: int32, 0, 1) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2: int32, 0, 4) {
                      for (ax0_ax1_ax2_ax3_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4: int32, 0, 8) "vectorized" {
                          block([512, 512, 16, 16], "A_shared") as [v0, v1, v2, v3] {
                            bind(v0, ((ii_0*8) + floordiv((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512)))
                            bind(v1, ((kk_0*2) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512), 256)))
                            bind(v2, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 256), 16))
                            bind(v3, floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 16))
                            tir.reads([A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 8)), ((floordiv(v2, 8)*8) + floormod(v3, 8))]])
                            tir.writes([A_shared[v0, v1, v2, v3]])
                            A_shared[v0, v1, v2, v3] = A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 8)), ((floordiv(v2, 8)*8) + floormod(v3, 8))]
                        }
                      }
                    }
                  }
                }
                for (ax0_ax1_ax2_ax3_fused_0_1: int32, 0, 1) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2_1: int32, 0, 8) {
                      for (ax0_ax1_ax2_ax3_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4_1: int32, 0, 8) "vectorized" {
                          block([512, 512, 16, 16], "B_shared") as [v0_1, v1_1, v2_1, v3_1] {
                            bind(v0_1, ((kk_0*2) + floordiv((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 4096)))
                            bind(v1_1, (((jj_0_0*256) + (jj_0_1*16)) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 4096), 256)))
                            bind(v2_1, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 256), 16))
                            bind(v3_1, floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 16))
                            tir.reads([B[v0_1, v1_1, ((floormod(v2_1, 8)*2) + floordiv(v3_1, 8)), ((floordiv(v2_1, 8)*8) + floormod(v3_1, 8))]])
                            tir.writes([B_shared[v0_1, v1_1, v2_1, v3_1]])
                            B_shared[v0_1, v1_1, v2_1, v3_1] = B[v0_1, v1_1, ((floormod(v2_1, 8)*2) + floordiv(v3_1, 8)), ((floordiv(v2_1, 8)*8) + floormod(v3_1, 8))]
                        }
                      }
                    }
                  }
                }
                for (kk_1: int32, 0, 2) {
                  for (ax0: int32, 0, 8) {
                    block([512, 512, 1, 1], "A_shared_warp_o") as [v0_2, v1_2, v2_o, v3_o] {
                      bind(v0_2, ((ii_0*8) + ax0))
                      bind(v1_2, ((kk_0*2) + kk_1))
                      bind(v2_o, 0)
                      bind(v3_o, 0)
                      tir.reads([A_shared[v0_2, v1_2, 0:16, 0:16]])
                      tir.writes([A_shared_warp[v0_2, v1_2, 0:32, 0:8]])
                      warp = match_buffer(A_shared_warp[v0_2, v1_2, 0:32, 0:8])
                      shared = match_buffer(A_shared[v0_2, v1_2, 0:16, 0:16])
                      attr [IterVar(tx_1: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                      @tir.ptx_ldmatrix(False, 4, ".b16", warp_1: Pointer(warp float16), (elem_offset_1: int32 + (8*tx_1)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_1: Pointer(shared float16), elem_offset_2: int32, (shared_s0: int32*16), 1, dtype=handle), (8*tx_1), dtype=float16)
                  }
                  for (ax0_1: int32, 0, 4) {
                    block([512, 512, 1, 1], "B_shared_warp_o") as [v0_3, v1_3, v2_o_1, v3_o_1] {
                      bind(v0_3, ((kk_0*2) + kk_1))
                      bind(v1_3, ((((jj_0_0*256) + (jj_0_1*16)) + (jj_1*4)) + ax0_1))
                      bind(v2_o_1, 0)
                      bind(v3_o_1, 0)
                      tir.reads([B_shared[v0_3, v1_3, 0:16, 0:16]])
                      tir.writes([B_shared_warp[v0_3, v1_3, 0:32, 0:8]])
                      warp_2 = match_buffer(B_shared_warp[v0_3, v1_3, 0:32, 0:8])
                      shared_2 = match_buffer(B_shared[v0_3, v1_3, 0:16, 0:16])
                      attr [IterVar(tx_2: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                      @tir.ptx_ldmatrix(True, 4, ".b16", warp_3: Pointer(warp float16), (elem_offset_3: int32 + (8*tx_2)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_3: Pointer(shared float16), elem_offset_4: int32, (shared_s0_1: int32*16), 1, dtype=handle), (8*tx_2), dtype=float16)
                  }
                  for (ii_2: int32, 0, 8) {
                    for (jj_2: int32, 0, 4) {
                      block([512, 512, tir.reduce_axis(0, 512), 1, 1, tir.reduce_axis(0, 1)], "B_update_o") as [vii_1, vjj_1, vkk, vi_o_1, vj_o_1, vk_o] {
                        bind(vii_1, (((ii_0*8) + (ii_1*8)) + ii_2))
                        bind(vjj_1, ((((jj_0_0*256) + (jj_0_1*16)) + (jj_1*4)) + jj_2))
                        bind(vkk, ((kk_0*2) + kk_1))
                        bind(vi_o_1, 0)
                        bind(vj_o_1, 0)
                        bind(vk_o, 0)
                        tir.reads([C_warp[vii_1, vjj_1, 0:32, 0:8], A_shared_warp[vii_1, vkk, 0:32, 0:8], B_shared_warp[vkk, vjj_1, 0:32, 0:8]])
                        tir.writes([C_warp[vii_1, vjj_1, 0:32, 0:8]])
                        A_2 = match_buffer(A_shared_warp[vii_1, vkk, 0:32, 0:8])
                        B_2 = match_buffer(B_shared_warp[vkk, vjj_1, 0:32, 0:8])
                        C_2 = match_buffer(C_warp[vii_1, vjj_1, 0:32, 0:8])
                        attr [IterVar(tx_3: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32 {
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3: Pointer(warp float16), (elem_offset_5: int32 + (tx_3*8)), B_3: Pointer(warp float16), (elem_offset_6: int32 + (tx_3*8)), C_3: Pointer(warp float16), (elem_offset_7: int32 + (tx_3*8)), False, dtype=float16)
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3, (elem_offset_5 + (tx_3*8)), B_3, ((elem_offset_6 + (tx_3*8)) + floordiv(8, 2)), C_3, ((elem_offset_7 + (tx_3*8)) + floordiv(8, 2)), False, dtype=float16)
                        }
                    }
                  }
                }
              }
              for (ax0_2: int32, 0, 8) {
                for (ax1: int32, 0, 4) {
                  block([512, 512, 1, 1], "C_warp_o") as [v0_4, v1_4, v2_o_2, v3_o_2] {
                    bind(v0_4, ((ii_0*8) + ax0_2))
                    bind(v1_4, ((((jj_0_0*256) + (jj_0_1*16)) + (jj_1*4)) + ax1))
                    bind(v2_o_2, 0)
                    bind(v3_o_2, 0)
                    tir.reads([C_warp[v0_4, v1_4, 0:32, 0:8]])
                    tir.writes([C[v0_4, v1_4, 0:16, 0:16]])
                    C_warp_3 = match_buffer(C_warp[v0_4, v1_4, 0:32, 0:8])
                    C_4 = match_buffer(C[v0_4, v1_4, 0:16, 0:16])
                    attr [IterVar(tx_4: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_store(16, 16, @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), C_5: Pointer(global float16), elem_offset_8: int32, (C_s0: int32*16), 2, dtype=handle), C_warp_4: Pointer(warp float16), elem_offset_9: int32, C_s0, dtype=float16)
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}