#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [8192, 8192], []),
             B: Buffer(B_1: Pointer(global float16), float16, [8192, 8192], []),
             C: Buffer(C_1: Pointer(global float16), float16, [8192, 8192], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(float16[8192, 8192])
    A_global_shared = alloc_buffer(float16[8192, 8192])
    A_global_shared_warp = alloc_buffer(float16[8192, 8192])
    B_global = alloc_buffer(float16[8192, 8192])
    B_global_shared = alloc_buffer(float16[8192, 8192])
    B_global_shared_warp = alloc_buffer(float16[8192, 8192])
    C_warp = alloc_buffer(float16[8192, 8192])
     {
      for (ax0: int32, 0, 8192) {
        for (ax1: int32, 0, 8192) {
          block([8192, 8192], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[v0, v1]])
            B_global[v0, v1] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 8192) {
        for (ax1_1: int32, 0, 8192) {
          block([8192, 8192], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[v0_1, v1_1]])
            A_global[v0_1, v1_1] = A[v0_1, v1_1]
        }
      }
      for (ax0_2: int32, 0, 8192) {
        for (ax1_2: int32, 0, 8192) {
          block([8192, 8192], "A_global_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([A_global[v0_2, v1_2]])
            tir.writes([A_global_shared[v0_2, v1_2]])
            A_global_shared[v0_2, v1_2] = A_global[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 8192) {
        for (ax1_3: int32, 0, 8192) {
          block([8192, 8192], "A_global_shared_warp") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([A_global_shared[v0_3, v1_3]])
            tir.writes([A_global_shared_warp[v0_3, v1_3]])
            A_global_shared_warp[v0_3, v1_3] = A_global_shared[v0_3, v1_3]
        }
      }
      for (ax0_4: int32, 0, 8192) {
        for (ax1_4: int32, 0, 8192) {
          block([8192, 8192], "B_global_shared") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([B_global[v0_4, v1_4]])
            tir.writes([B_global_shared[v0_4, v1_4]])
            B_global_shared[v0_4, v1_4] = B_global[v0_4, v1_4]
        }
      }
      for (ax0_5: int32, 0, 8192) {
        for (ax1_5: int32, 0, 8192) {
          block([8192, 8192], "B_global_shared_warp") as [v0_5, v1_5] {
            bind(v0_5, ax0_5)
            bind(v1_5, ax1_5)
            tir.reads([B_global_shared[v0_5, v1_5]])
            tir.writes([B_global_shared_warp[v0_5, v1_5]])
            B_global_shared_warp[v0_5, v1_5] = B_global_shared[v0_5, v1_5]
        }
      }
      for (i: int32, 0, 8192) {
        for (j: int32, 0, 8192) {
          for (k: int32, 0, 8192) {
            block([8192, 8192, tir.reduce_axis(0, 8192)], "B") as [vi, vj, vk] {
              bind(vi, i)
              bind(vj, j)
              bind(vk, k)
              tir.reads([A_global_shared_warp[vi, vk], B_global_shared_warp[vk, vj]])
              tir.writes([C_warp[vi, vj]])
              with init() {
                C_warp[vi, vj] = 0f16
              }
              C_warp[vi, vj] = (C_warp[vi, vj] + (A_global_shared_warp[vi, vk]*B_global_shared_warp[vk, vj]))
          }
        }
      }
      for (ax0_6: int32, 0, 8192) {
        for (ax1_6: int32, 0, 8192) {
          block([8192, 8192], "C_warp") as [v0_6, v1_6] {
            bind(v0_6, ax0_6)
            bind(v1_6, ax1_6)
            tir.reads([C_warp[v0_6, v1_6]])
            tir.writes([C[v0_6, v1_6]])
            C[v0_6, v1_6] = C_warp[v0_6, v1_6]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}