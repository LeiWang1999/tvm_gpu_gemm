#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [8192, 8192], []),
             B: Buffer(B_1: Pointer(global float16), float16, [8192, 8192], []),
             C: Buffer(C_1: Pointer(global float16), float16, [8192, 8192], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    for (i: int32, 0, 8192) {
      for (j: int32, 0, 8192) {
        for (k: int32, 0, 8192) {
          block([8192, 8192, tir.reduce_axis(0, 8192)], "B") as [vi, vj, vk] {
            bind(vi, i)
            bind(vj, j)
            bind(vk, k)
            tir.reads([A[vi, vk], B[vk, vj]])
            tir.writes([C[vi, vj]])
            with init() {
              C[vi, vj] = 0f16
            }
            C[vi, vj] = (C[vi, vj] + (A[vi, vk]*B[vk, vj]))
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}