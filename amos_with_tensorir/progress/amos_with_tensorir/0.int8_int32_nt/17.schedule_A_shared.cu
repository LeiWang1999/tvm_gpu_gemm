#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    C_global = alloc_buffer(int32[1024, 1024, 16, 16])
    C_global_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 256) "thread_binding" {
        for (j_0_0: int32, 0, 64) "thread_binding" {
          for (i_0_1: int32, 0, 2) "thread_binding" {
            for (j_0_1: int32, 0, 2) "thread_binding" {
              for (k_0_0: int32, 0, 512) {
                for (ax0_ax1_fused_0: int32, 0, 2) "thread_binding" {
                  for (ax0_ax1_fused_1: int32, 0, 2) "thread_binding" {
                    for (ax0_ax1_fused_2: int32, 0, 1) {
                      for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_fused_4: int32, 0, 16) "vectorized" {
                          block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
                            bind(v0_2, ((i_0_0*64) + floordiv((((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*512)) + (ax0_ax1_fused_3*16)) + ax0_ax1_fused_4), 32)))
                            bind(v1_2, ((k_0_0*32) + floormod((((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*512)) + (ax0_ax1_fused_3*16)) + ax0_ax1_fused_4), 32)))
                            tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                        }
                      }
                    }
                  }
                }
                for (ax0_2: int32, 0, 256) {
                  for (ax1_2: int32, 0, 32) {
                    block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
                      bind(v0_3, ((j_0_0*256) + ax0_2))
                      bind(v1_3, ((k_0_0*32) + ax1_2))
                      tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                      tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                      B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                  }
                }
                for (k_0_1: int32, 0, 2) {
                  for (ax0_0: int32, 0, 2) {
                    for (ax1_0: int32, 0, 1) {
                      for (ax0_1_1: int32, 0, 16) {
                        for (ax1_1_1: int32, 0, 16) {
                          block([16384, 16384], "A_global_shared_wmma.matrix_a") as [v0_4, v1_4] {
                            bind(v0_4, ((((i_0_0*64) + (i_0_1*32)) + (ax0_0*16)) + ax0_1_1))
                            bind(v1_4, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0*16)) + ax1_1_1))
                            tir.reads([A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            tir.writes([A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                        }
                      }
                    }
                  }
                  for (ax0_0_1: int32, 0, 8) {
                    for (ax1_0_1: int32, 0, 1) {
                      for (ax0_1_2: int32, 0, 16) {
                        for (ax1_1_2: int32, 0, 16) {
                          block([16384, 16384], "B_global_shared_wmma.matrix_b") as [v0_5, v1_5] {
                            bind(v0_5, ((((j_0_0*256) + (j_0_1*128)) + (ax0_0_1*16)) + ax0_1_2))
                            bind(v1_5, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0_1*16)) + ax1_1_2))
                            tir.reads([B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                            tir.writes([B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                            B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                        }
                      }
                    }
                  }
                  for (i_0_2: int32, 0, 2) {
                    for (j_0_2: int32, 0, 8) {
                      for (i_1: int32, 0, 16) {
                        for (j_1: int32, 0, 16) {
                          for (k_1: int32, 0, 16) {
                            block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                              bind(vi, ((((i_0_0*64) + (i_0_1*32)) + (i_0_2*16)) + i_1))
                              bind(vj, ((((j_0_0*256) + (j_0_1*128)) + (j_0_2*16)) + j_1))
                              bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                              tir.reads([A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)], B_global_shared_wmma.matrix_b[floordiv(vj, 16), floordiv(vk, 16), floormod(vj, 16), floormod(vk, 16)]])
                              tir.writes([C_global_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                              with init() {
                                C_global_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = 0
                              }
                              C_global_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = (C_global_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] + (cast(int32, A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)])*cast(int32, B_global_shared_wmma.matrix_b[floordiv(vj, 16), floordiv(vk, 16), floormod(vj, 16), floormod(vk, 16)])))
                          }
                        }
                      }
                    }
                  }
                }
              }
              for (ax0_0_2: int32, 0, 2) {
                for (ax1_0_2: int32, 0, 8) {
                  for (ax0_1_3: int32, 0, 16) {
                    for (ax1_1_3: int32, 0, 16) {
                      block([16384, 16384], "C_global_wmma.accumulator") as [v0_6, v1_6] {
                        bind(v0_6, ((((i_0_0*64) + (i_0_1*32)) + (ax0_0_2*16)) + ax0_1_3))
                        bind(v1_6, ((((j_0_0*256) + (j_0_1*128)) + (ax1_0_2*16)) + ax1_1_3))
                        tir.reads([C_global_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                        tir.writes([C_global[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                        C_global[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)] = C_global_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "C_global") as [v0_7, v1_7] {
            bind(v0_7, ax0_3)
            bind(v1_7, ax1_3)
            tir.reads([C_global[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)]])
            tir.writes([C[v0_7, v1_7]])
            C[v0_7, v1_7] = C_global[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}