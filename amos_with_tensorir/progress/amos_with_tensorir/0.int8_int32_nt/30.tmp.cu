#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < 16; ++ax0_ax1_fused_1) {
    *(int4*)(A_global + ((((((int)blockIdx.x) * 262144) + (((int)threadIdx.y) * 8192)) + (((int)threadIdx.x) * 256)) + (ax0_ax1_fused_1 * 16))) = *(int4*)(A + ((((((int)blockIdx.x) * 262144) + (ax0_ax1_fused_1 * 16384)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ A_global, signed char* __restrict__ B_global, int* __restrict__ C_global) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_global_wmma_accumulator[16];
  __shared__ signed char A_global_shared[2048];
  __shared__ signed char B_global_shared[8192];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> B_global_shared_wmma_matrix_b[8];
  for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 8; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(C_global_wmma_accumulator[((i_0_2_init * 8) + j_0_2_init)], 0.000000e+00f);
    }
  }
  for (int k_0_0 = 0; k_0_0 < 512; ++k_0_0) {
    __syncthreads();
    *(int4*)(A_global_shared + ((((((int)threadIdx.y) * 1024) + (((int)threadIdx.z) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + ((((int)threadIdx.x) >> 1) * 16))) = *(int4*)(A_global + ((((((((int)blockIdx.x) * 1048576) + (((int)threadIdx.y) * 524288)) + (((int)threadIdx.z) * 262144)) + (k_0_0 * 512)) + ((((int)threadIdx.x) & 1) * 256)) + ((((int)threadIdx.x) >> 1) * 16)));
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {
      *(int4*)(B_global_shared + (((((((int)threadIdx.y) * 4096) + (((int)threadIdx.z) * 2048)) + (ax0_ax1_fused_2 * 512)) + ((((int)threadIdx.x) & 1) * 256)) + ((((int)threadIdx.x) >> 1) * 16))) = *(int4*)(B_global + (((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 2097152)) + (((int)threadIdx.z) * 1048576)) + (ax0_ax1_fused_2 * 262144)) + (k_0_0 * 512)) + ((((int)threadIdx.x) & 1) * 256)) + ((((int)threadIdx.x) >> 1) * 16)));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a[ax0_0], (&(A_global_shared[(((((int)threadIdx.y) * 1024) + (ax0_0 * 512)) + (k_0_1 * 256))])), 16);
      }
      for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1) {
        nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b[ax0_0_1], (&(B_global_shared[(((((int)threadIdx.z) * 4096) + (ax0_0_1 * 512)) + (k_0_1 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 8; ++j_0_2) {
          nvcuda::wmma::mma_sync(C_global_wmma_accumulator[((i_0_2 * 8) + j_0_2)], A_global_shared_wmma_matrix_a[i_0_2], B_global_shared_wmma_matrix_b[j_0_2], C_global_wmma_accumulator[((i_0_2 * 8) + j_0_2)]);
        }
      }
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {
    for (int ax1_0 = 0; ax1_0 < 8; ++ax1_0) {
      nvcuda::wmma::store_matrix_sync((&(C_global[((((((((int)blockIdx.x) * 1048576) + (((int)threadIdx.y) * 524288)) + (ax0_0_2 * 262144)) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.z) * 2048)) + (ax1_0 * 256))])), C_global_wmma_accumulator[((ax0_0_2 * 8) + ax1_0)], 16, nvcuda::wmma::mem_row_major);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < 16; ++ax0_ax1_fused_1) {
    *(int4*)(B_global + ((((((int)blockIdx.x) * 262144) + (((int)threadIdx.y) * 8192)) + (((int)threadIdx.x) * 256)) + (ax0_ax1_fused_1 * 16))) = *(int4*)(B + ((((((int)blockIdx.x) * 262144) + (ax0_ax1_fused_1 * 16384)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel3(int* __restrict__ C, int* __restrict__ C_global) {
  for (int ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < 64; ++ax0_ax1_fused_1) {
    *(int4*)(C + ((((((int)blockIdx.x) * 262144) + (ax0_ax1_fused_1 * 4096)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 4))) = *(int4*)(C_global + ((((((int)blockIdx.x) * 262144) + (((((ax0_ax1_fused_1 * 256) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) >> 2)) & 1023) * 256)) + ((ax0_ax1_fused_1 >> 2) * 16)) + ((((int)threadIdx.x) & 3) * 4)));
  }
}

