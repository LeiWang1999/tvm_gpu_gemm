@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float32), float32, [268435456], []),
             B: Buffer(B_2: Pointer(float32), float32, [268435456], []),
             C: Buffer(C_2: Pointer(float32), float32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float32, [16384, 16384], []), B_1: B_3: Buffer(B_2, float32, [16384, 16384], []), C_1: C_3: Buffer(C_2, float32, [16384, 16384], [])} {
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
  allocate(C.local: Pointer(local float32), float32, [64]), storage_scope = local;
  allocate(A.shared: Pointer(shared float32), float32, [2048]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float32), float32, [2048]), storage_scope = shared;
  allocate(A.shared.local: Pointer(local float32), float32, [8]), storage_scope = local;
  allocate(B.shared.local: Pointer(local float32), float32, [8]), storage_scope = local;
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128;
  attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 16;
  attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 16 {
    for (ii.c.init: int32, 0, 4) {
      for (jj.c.init: int32, 0, 4) {
        let cse_var_1: int32 = ((ii.c.init*4) + jj.c.init)
         {
          C.local_1: Buffer(C.local, float32, [8192], [], scope="local", align=64)[cse_var_1] = 0f32
          C.local_1[(cse_var_1 + 32)] = 0f32
          C.local_1[(cse_var_1 + 16)] = 0f32
          C.local_1[(cse_var_1 + 48)] = 0f32
        }
      }
    }
    for (k.outer: int32, 0, 1024) {
      for (ax1.outer: int32, 0, 2) {
        let cse_var_2: int32 = (ax1.outer*64)
        A.shared_1: Buffer(A.shared, float32, [2048], [], scope="shared")[ramp((((threadIdx.y*128) + cse_var_2) + (threadIdx.x*4)), 1, 4)] = A[ramp((((((k.outer*262144) + (threadIdx.y*16384)) + (blockIdx.x*128)) + cse_var_2) + (threadIdx.x*4)), 1, 4)]
      }
      for (ax0: int32, 0, 16) {
        for (ax1: int32, 0, 128) {
          B.shared_1: Buffer(B.shared, float32, [2048], [], scope="shared")[((ax0*128) + ax1)] = B[((((k.outer*262144) + (ax0*16384)) + (blockIdx.y*128)) + ax1)]
        }
      }
      for (ax1_1: int32, 0, 4) {
        A.shared.local_1: Buffer(A.shared.local, float32, [16], [], scope="local", align=16)[ax1_1] = A.shared_1[((threadIdx.x*4) + ax1_1)]
        A.shared.local_1[(ax1_1 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_1) + 64)]
      }
      for (ax1_2: int32, 0, 4) {
        B.shared.local_1: Buffer(B.shared.local, float32, [16], [], scope="local", align=16)[ax1_2] = B.shared_1[((threadIdx.y*4) + ax1_2)]
        B.shared.local_1[(ax1_2 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_2) + 64)]
      }
      for (ii.c: int32, 0, 4) {
        for (jj.c: int32, 0, 4) {
          let cse_var_8: int32 = (jj.c + 4)
          let cse_var_7: int32 = (ii.c + 4)
          let cse_var_6: int32 = ((ii.c*4) + jj.c)
          let cse_var_5: int32 = (cse_var_6 + 48)
          let cse_var_4: int32 = (cse_var_6 + 32)
          let cse_var_3: int32 = (cse_var_6 + 16)
           {
            C.local_1[cse_var_6] = (C.local_1[cse_var_6] + (A.shared.local_1[jj.c]*B.shared.local_1[ii.c]))
            C.local_1[cse_var_4] = (C.local_1[cse_var_4] + (A.shared.local_1[jj.c]*B.shared.local_1[cse_var_7]))
            C.local_1[cse_var_3] = (C.local_1[cse_var_3] + (A.shared.local_1[cse_var_8]*B.shared.local_1[ii.c]))
            C.local_1[cse_var_5] = (C.local_1[cse_var_5] + (A.shared.local_1[cse_var_8]*B.shared.local_1[cse_var_7]))
          }
        }
      }
      for (ax1_3: int32, 0, 4) {
        A.shared.local_1[ax1_3] = A.shared_1[(((threadIdx.x*4) + ax1_3) + 128)]
        A.shared.local_1[(ax1_3 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_3) + 192)]
      }
      for (ax1_4: int32, 0, 4) {
        B.shared.local_1[ax1_4] = B.shared_1[(((threadIdx.y*4) + ax1_4) + 128)]
        B.shared.local_1[(ax1_4 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_4) + 192)]
      }
      for (ii.c_1: int32, 0, 4) {
        for (jj.c_1: int32, 0, 4) {
          let cse_var_14: int32 = (jj.c_1 + 4)
          let cse_var_13: int32 = (ii.c_1 + 4)
          let cse_var_12: int32 = ((ii.c_1*4) + jj.c_1)
          let cse_var_11: int32 = (cse_var_12 + 48)
          let cse_var_10: int32 = (cse_var_12 + 32)
          let cse_var_9: int32 = (cse_var_12 + 16)
           {
            C.local_1[cse_var_12] = (C.local_1[cse_var_12] + (A.shared.local_1[jj.c_1]*B.shared.local_1[ii.c_1]))
            C.local_1[cse_var_10] = (C.local_1[cse_var_10] + (A.shared.local_1[jj.c_1]*B.shared.local_1[cse_var_13]))
            C.local_1[cse_var_9] = (C.local_1[cse_var_9] + (A.shared.local_1[cse_var_14]*B.shared.local_1[ii.c_1]))
            C.local_1[cse_var_11] = (C.local_1[cse_var_11] + (A.shared.local_1[cse_var_14]*B.shared.local_1[cse_var_13]))
          }
        }
      }
      for (ax1_5: int32, 0, 4) {
        A.shared.local_1[ax1_5] = A.shared_1[(((threadIdx.x*4) + ax1_5) + 256)]
        A.shared.local_1[(ax1_5 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_5) + 320)]
      }
      for (ax1_6: int32, 0, 4) {
        B.shared.local_1[ax1_6] = B.shared_1[(((threadIdx.y*4) + ax1_6) + 256)]
        B.shared.local_1[(ax1_6 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_6) + 320)]
      }
      for (ii.c_2: int32, 0, 4) {
        for (jj.c_2: int32, 0, 4) {
          let cse_var_20: int32 = (jj.c_2 + 4)
          let cse_var_19: int32 = (ii.c_2 + 4)
          let cse_var_18: int32 = ((ii.c_2*4) + jj.c_2)
          let cse_var_17: int32 = (cse_var_18 + 48)
          let cse_var_16: int32 = (cse_var_18 + 32)
          let cse_var_15: int32 = (cse_var_18 + 16)
           {
            C.local_1[cse_var_18] = (C.local_1[cse_var_18] + (A.shared.local_1[jj.c_2]*B.shared.local_1[ii.c_2]))
            C.local_1[cse_var_16] = (C.local_1[cse_var_16] + (A.shared.local_1[jj.c_2]*B.shared.local_1[cse_var_19]))
            C.local_1[cse_var_15] = (C.local_1[cse_var_15] + (A.shared.local_1[cse_var_20]*B.shared.local_1[ii.c_2]))
            C.local_1[cse_var_17] = (C.local_1[cse_var_17] + (A.shared.local_1[cse_var_20]*B.shared.local_1[cse_var_19]))
          }
        }
      }
      for (ax1_7: int32, 0, 4) {
        A.shared.local_1[ax1_7] = A.shared_1[(((threadIdx.x*4) + ax1_7) + 384)]
        A.shared.local_1[(ax1_7 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_7) + 448)]
      }
      for (ax1_8: int32, 0, 4) {
        B.shared.local_1[ax1_8] = B.shared_1[(((threadIdx.y*4) + ax1_8) + 384)]
        B.shared.local_1[(ax1_8 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_8) + 448)]
      }
      for (ii.c_3: int32, 0, 4) {
        for (jj.c_3: int32, 0, 4) {
          let cse_var_26: int32 = (jj.c_3 + 4)
          let cse_var_25: int32 = (ii.c_3 + 4)
          let cse_var_24: int32 = ((ii.c_3*4) + jj.c_3)
          let cse_var_23: int32 = (cse_var_24 + 48)
          let cse_var_22: int32 = (cse_var_24 + 32)
          let cse_var_21: int32 = (cse_var_24 + 16)
           {
            C.local_1[cse_var_24] = (C.local_1[cse_var_24] + (A.shared.local_1[jj.c_3]*B.shared.local_1[ii.c_3]))
            C.local_1[cse_var_22] = (C.local_1[cse_var_22] + (A.shared.local_1[jj.c_3]*B.shared.local_1[cse_var_25]))
            C.local_1[cse_var_21] = (C.local_1[cse_var_21] + (A.shared.local_1[cse_var_26]*B.shared.local_1[ii.c_3]))
            C.local_1[cse_var_23] = (C.local_1[cse_var_23] + (A.shared.local_1[cse_var_26]*B.shared.local_1[cse_var_25]))
          }
        }
      }
      for (ax1_9: int32, 0, 4) {
        A.shared.local_1[ax1_9] = A.shared_1[(((threadIdx.x*4) + ax1_9) + 512)]
        A.shared.local_1[(ax1_9 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_9) + 576)]
      }
      for (ax1_10: int32, 0, 4) {
        B.shared.local_1[ax1_10] = B.shared_1[(((threadIdx.y*4) + ax1_10) + 512)]
        B.shared.local_1[(ax1_10 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_10) + 576)]
      }
      for (ii.c_4: int32, 0, 4) {
        for (jj.c_4: int32, 0, 4) {
          let cse_var_32: int32 = (jj.c_4 + 4)
          let cse_var_31: int32 = (ii.c_4 + 4)
          let cse_var_30: int32 = ((ii.c_4*4) + jj.c_4)
          let cse_var_29: int32 = (cse_var_30 + 48)
          let cse_var_28: int32 = (cse_var_30 + 32)
          let cse_var_27: int32 = (cse_var_30 + 16)
           {
            C.local_1[cse_var_30] = (C.local_1[cse_var_30] + (A.shared.local_1[jj.c_4]*B.shared.local_1[ii.c_4]))
            C.local_1[cse_var_28] = (C.local_1[cse_var_28] + (A.shared.local_1[jj.c_4]*B.shared.local_1[cse_var_31]))
            C.local_1[cse_var_27] = (C.local_1[cse_var_27] + (A.shared.local_1[cse_var_32]*B.shared.local_1[ii.c_4]))
            C.local_1[cse_var_29] = (C.local_1[cse_var_29] + (A.shared.local_1[cse_var_32]*B.shared.local_1[cse_var_31]))
          }
        }
      }
      for (ax1_11: int32, 0, 4) {
        A.shared.local_1[ax1_11] = A.shared_1[(((threadIdx.x*4) + ax1_11) + 640)]
        A.shared.local_1[(ax1_11 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_11) + 704)]
      }
      for (ax1_12: int32, 0, 4) {
        B.shared.local_1[ax1_12] = B.shared_1[(((threadIdx.y*4) + ax1_12) + 640)]
        B.shared.local_1[(ax1_12 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_12) + 704)]
      }
      for (ii.c_5: int32, 0, 4) {
        for (jj.c_5: int32, 0, 4) {
          let cse_var_38: int32 = (jj.c_5 + 4)
          let cse_var_37: int32 = (ii.c_5 + 4)
          let cse_var_36: int32 = ((ii.c_5*4) + jj.c_5)
          let cse_var_35: int32 = (cse_var_36 + 48)
          let cse_var_34: int32 = (cse_var_36 + 32)
          let cse_var_33: int32 = (cse_var_36 + 16)
           {
            C.local_1[cse_var_36] = (C.local_1[cse_var_36] + (A.shared.local_1[jj.c_5]*B.shared.local_1[ii.c_5]))
            C.local_1[cse_var_34] = (C.local_1[cse_var_34] + (A.shared.local_1[jj.c_5]*B.shared.local_1[cse_var_37]))
            C.local_1[cse_var_33] = (C.local_1[cse_var_33] + (A.shared.local_1[cse_var_38]*B.shared.local_1[ii.c_5]))
            C.local_1[cse_var_35] = (C.local_1[cse_var_35] + (A.shared.local_1[cse_var_38]*B.shared.local_1[cse_var_37]))
          }
        }
      }
      for (ax1_13: int32, 0, 4) {
        A.shared.local_1[ax1_13] = A.shared_1[(((threadIdx.x*4) + ax1_13) + 768)]
        A.shared.local_1[(ax1_13 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_13) + 832)]
      }
      for (ax1_14: int32, 0, 4) {
        B.shared.local_1[ax1_14] = B.shared_1[(((threadIdx.y*4) + ax1_14) + 768)]
        B.shared.local_1[(ax1_14 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_14) + 832)]
      }
      for (ii.c_6: int32, 0, 4) {
        for (jj.c_6: int32, 0, 4) {
          let cse_var_44: int32 = (jj.c_6 + 4)
          let cse_var_43: int32 = (ii.c_6 + 4)
          let cse_var_42: int32 = ((ii.c_6*4) + jj.c_6)
          let cse_var_41: int32 = (cse_var_42 + 48)
          let cse_var_40: int32 = (cse_var_42 + 32)
          let cse_var_39: int32 = (cse_var_42 + 16)
           {
            C.local_1[cse_var_42] = (C.local_1[cse_var_42] + (A.shared.local_1[jj.c_6]*B.shared.local_1[ii.c_6]))
            C.local_1[cse_var_40] = (C.local_1[cse_var_40] + (A.shared.local_1[jj.c_6]*B.shared.local_1[cse_var_43]))
            C.local_1[cse_var_39] = (C.local_1[cse_var_39] + (A.shared.local_1[cse_var_44]*B.shared.local_1[ii.c_6]))
            C.local_1[cse_var_41] = (C.local_1[cse_var_41] + (A.shared.local_1[cse_var_44]*B.shared.local_1[cse_var_43]))
          }
        }
      }
      for (ax1_15: int32, 0, 4) {
        A.shared.local_1[ax1_15] = A.shared_1[(((threadIdx.x*4) + ax1_15) + 896)]
        A.shared.local_1[(ax1_15 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_15) + 960)]
      }
      for (ax1_16: int32, 0, 4) {
        B.shared.local_1[ax1_16] = B.shared_1[(((threadIdx.y*4) + ax1_16) + 896)]
        B.shared.local_1[(ax1_16 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_16) + 960)]
      }
      for (ii.c_7: int32, 0, 4) {
        for (jj.c_7: int32, 0, 4) {
          let cse_var_50: int32 = (jj.c_7 + 4)
          let cse_var_49: int32 = (ii.c_7 + 4)
          let cse_var_48: int32 = ((ii.c_7*4) + jj.c_7)
          let cse_var_47: int32 = (cse_var_48 + 48)
          let cse_var_46: int32 = (cse_var_48 + 32)
          let cse_var_45: int32 = (cse_var_48 + 16)
           {
            C.local_1[cse_var_48] = (C.local_1[cse_var_48] + (A.shared.local_1[jj.c_7]*B.shared.local_1[ii.c_7]))
            C.local_1[cse_var_46] = (C.local_1[cse_var_46] + (A.shared.local_1[jj.c_7]*B.shared.local_1[cse_var_49]))
            C.local_1[cse_var_45] = (C.local_1[cse_var_45] + (A.shared.local_1[cse_var_50]*B.shared.local_1[ii.c_7]))
            C.local_1[cse_var_47] = (C.local_1[cse_var_47] + (A.shared.local_1[cse_var_50]*B.shared.local_1[cse_var_49]))
          }
        }
      }
      for (ax1_17: int32, 0, 4) {
        A.shared.local_1[ax1_17] = A.shared_1[(((threadIdx.x*4) + ax1_17) + 1024)]
        A.shared.local_1[(ax1_17 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_17) + 1088)]
      }
      for (ax1_18: int32, 0, 4) {
        B.shared.local_1[ax1_18] = B.shared_1[(((threadIdx.y*4) + ax1_18) + 1024)]
        B.shared.local_1[(ax1_18 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_18) + 1088)]
      }
      for (ii.c_8: int32, 0, 4) {
        for (jj.c_8: int32, 0, 4) {
          let cse_var_56: int32 = (jj.c_8 + 4)
          let cse_var_55: int32 = (ii.c_8 + 4)
          let cse_var_54: int32 = ((ii.c_8*4) + jj.c_8)
          let cse_var_53: int32 = (cse_var_54 + 48)
          let cse_var_52: int32 = (cse_var_54 + 32)
          let cse_var_51: int32 = (cse_var_54 + 16)
           {
            C.local_1[cse_var_54] = (C.local_1[cse_var_54] + (A.shared.local_1[jj.c_8]*B.shared.local_1[ii.c_8]))
            C.local_1[cse_var_52] = (C.local_1[cse_var_52] + (A.shared.local_1[jj.c_8]*B.shared.local_1[cse_var_55]))
            C.local_1[cse_var_51] = (C.local_1[cse_var_51] + (A.shared.local_1[cse_var_56]*B.shared.local_1[ii.c_8]))
            C.local_1[cse_var_53] = (C.local_1[cse_var_53] + (A.shared.local_1[cse_var_56]*B.shared.local_1[cse_var_55]))
          }
        }
      }
      for (ax1_19: int32, 0, 4) {
        A.shared.local_1[ax1_19] = A.shared_1[(((threadIdx.x*4) + ax1_19) + 1152)]
        A.shared.local_1[(ax1_19 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_19) + 1216)]
      }
      for (ax1_20: int32, 0, 4) {
        B.shared.local_1[ax1_20] = B.shared_1[(((threadIdx.y*4) + ax1_20) + 1152)]
        B.shared.local_1[(ax1_20 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_20) + 1216)]
      }
      for (ii.c_9: int32, 0, 4) {
        for (jj.c_9: int32, 0, 4) {
          let cse_var_62: int32 = (jj.c_9 + 4)
          let cse_var_61: int32 = (ii.c_9 + 4)
          let cse_var_60: int32 = ((ii.c_9*4) + jj.c_9)
          let cse_var_59: int32 = (cse_var_60 + 48)
          let cse_var_58: int32 = (cse_var_60 + 32)
          let cse_var_57: int32 = (cse_var_60 + 16)
           {
            C.local_1[cse_var_60] = (C.local_1[cse_var_60] + (A.shared.local_1[jj.c_9]*B.shared.local_1[ii.c_9]))
            C.local_1[cse_var_58] = (C.local_1[cse_var_58] + (A.shared.local_1[jj.c_9]*B.shared.local_1[cse_var_61]))
            C.local_1[cse_var_57] = (C.local_1[cse_var_57] + (A.shared.local_1[cse_var_62]*B.shared.local_1[ii.c_9]))
            C.local_1[cse_var_59] = (C.local_1[cse_var_59] + (A.shared.local_1[cse_var_62]*B.shared.local_1[cse_var_61]))
          }
        }
      }
      for (ax1_21: int32, 0, 4) {
        A.shared.local_1[ax1_21] = A.shared_1[(((threadIdx.x*4) + ax1_21) + 1280)]
        A.shared.local_1[(ax1_21 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_21) + 1344)]
      }
      for (ax1_22: int32, 0, 4) {
        B.shared.local_1[ax1_22] = B.shared_1[(((threadIdx.y*4) + ax1_22) + 1280)]
        B.shared.local_1[(ax1_22 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_22) + 1344)]
      }
      for (ii.c_10: int32, 0, 4) {
        for (jj.c_10: int32, 0, 4) {
          let cse_var_68: int32 = (jj.c_10 + 4)
          let cse_var_67: int32 = (ii.c_10 + 4)
          let cse_var_66: int32 = ((ii.c_10*4) + jj.c_10)
          let cse_var_65: int32 = (cse_var_66 + 48)
          let cse_var_64: int32 = (cse_var_66 + 32)
          let cse_var_63: int32 = (cse_var_66 + 16)
           {
            C.local_1[cse_var_66] = (C.local_1[cse_var_66] + (A.shared.local_1[jj.c_10]*B.shared.local_1[ii.c_10]))
            C.local_1[cse_var_64] = (C.local_1[cse_var_64] + (A.shared.local_1[jj.c_10]*B.shared.local_1[cse_var_67]))
            C.local_1[cse_var_63] = (C.local_1[cse_var_63] + (A.shared.local_1[cse_var_68]*B.shared.local_1[ii.c_10]))
            C.local_1[cse_var_65] = (C.local_1[cse_var_65] + (A.shared.local_1[cse_var_68]*B.shared.local_1[cse_var_67]))
          }
        }
      }
      for (ax1_23: int32, 0, 4) {
        A.shared.local_1[ax1_23] = A.shared_1[(((threadIdx.x*4) + ax1_23) + 1408)]
        A.shared.local_1[(ax1_23 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_23) + 1472)]
      }
      for (ax1_24: int32, 0, 4) {
        B.shared.local_1[ax1_24] = B.shared_1[(((threadIdx.y*4) + ax1_24) + 1408)]
        B.shared.local_1[(ax1_24 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_24) + 1472)]
      }
      for (ii.c_11: int32, 0, 4) {
        for (jj.c_11: int32, 0, 4) {
          let cse_var_74: int32 = (jj.c_11 + 4)
          let cse_var_73: int32 = (ii.c_11 + 4)
          let cse_var_72: int32 = ((ii.c_11*4) + jj.c_11)
          let cse_var_71: int32 = (cse_var_72 + 48)
          let cse_var_70: int32 = (cse_var_72 + 32)
          let cse_var_69: int32 = (cse_var_72 + 16)
           {
            C.local_1[cse_var_72] = (C.local_1[cse_var_72] + (A.shared.local_1[jj.c_11]*B.shared.local_1[ii.c_11]))
            C.local_1[cse_var_70] = (C.local_1[cse_var_70] + (A.shared.local_1[jj.c_11]*B.shared.local_1[cse_var_73]))
            C.local_1[cse_var_69] = (C.local_1[cse_var_69] + (A.shared.local_1[cse_var_74]*B.shared.local_1[ii.c_11]))
            C.local_1[cse_var_71] = (C.local_1[cse_var_71] + (A.shared.local_1[cse_var_74]*B.shared.local_1[cse_var_73]))
          }
        }
      }
      for (ax1_25: int32, 0, 4) {
        A.shared.local_1[ax1_25] = A.shared_1[(((threadIdx.x*4) + ax1_25) + 1536)]
        A.shared.local_1[(ax1_25 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_25) + 1600)]
      }
      for (ax1_26: int32, 0, 4) {
        B.shared.local_1[ax1_26] = B.shared_1[(((threadIdx.y*4) + ax1_26) + 1536)]
        B.shared.local_1[(ax1_26 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_26) + 1600)]
      }
      for (ii.c_12: int32, 0, 4) {
        for (jj.c_12: int32, 0, 4) {
          let cse_var_80: int32 = (jj.c_12 + 4)
          let cse_var_79: int32 = (ii.c_12 + 4)
          let cse_var_78: int32 = ((ii.c_12*4) + jj.c_12)
          let cse_var_77: int32 = (cse_var_78 + 48)
          let cse_var_76: int32 = (cse_var_78 + 32)
          let cse_var_75: int32 = (cse_var_78 + 16)
           {
            C.local_1[cse_var_78] = (C.local_1[cse_var_78] + (A.shared.local_1[jj.c_12]*B.shared.local_1[ii.c_12]))
            C.local_1[cse_var_76] = (C.local_1[cse_var_76] + (A.shared.local_1[jj.c_12]*B.shared.local_1[cse_var_79]))
            C.local_1[cse_var_75] = (C.local_1[cse_var_75] + (A.shared.local_1[cse_var_80]*B.shared.local_1[ii.c_12]))
            C.local_1[cse_var_77] = (C.local_1[cse_var_77] + (A.shared.local_1[cse_var_80]*B.shared.local_1[cse_var_79]))
          }
        }
      }
      for (ax1_27: int32, 0, 4) {
        A.shared.local_1[ax1_27] = A.shared_1[(((threadIdx.x*4) + ax1_27) + 1664)]
        A.shared.local_1[(ax1_27 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_27) + 1728)]
      }
      for (ax1_28: int32, 0, 4) {
        B.shared.local_1[ax1_28] = B.shared_1[(((threadIdx.y*4) + ax1_28) + 1664)]
        B.shared.local_1[(ax1_28 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_28) + 1728)]
      }
      for (ii.c_13: int32, 0, 4) {
        for (jj.c_13: int32, 0, 4) {
          let cse_var_86: int32 = (jj.c_13 + 4)
          let cse_var_85: int32 = (ii.c_13 + 4)
          let cse_var_84: int32 = ((ii.c_13*4) + jj.c_13)
          let cse_var_83: int32 = (cse_var_84 + 48)
          let cse_var_82: int32 = (cse_var_84 + 32)
          let cse_var_81: int32 = (cse_var_84 + 16)
           {
            C.local_1[cse_var_84] = (C.local_1[cse_var_84] + (A.shared.local_1[jj.c_13]*B.shared.local_1[ii.c_13]))
            C.local_1[cse_var_82] = (C.local_1[cse_var_82] + (A.shared.local_1[jj.c_13]*B.shared.local_1[cse_var_85]))
            C.local_1[cse_var_81] = (C.local_1[cse_var_81] + (A.shared.local_1[cse_var_86]*B.shared.local_1[ii.c_13]))
            C.local_1[cse_var_83] = (C.local_1[cse_var_83] + (A.shared.local_1[cse_var_86]*B.shared.local_1[cse_var_85]))
          }
        }
      }
      for (ax1_29: int32, 0, 4) {
        A.shared.local_1[ax1_29] = A.shared_1[(((threadIdx.x*4) + ax1_29) + 1792)]
        A.shared.local_1[(ax1_29 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_29) + 1856)]
      }
      for (ax1_30: int32, 0, 4) {
        B.shared.local_1[ax1_30] = B.shared_1[(((threadIdx.y*4) + ax1_30) + 1792)]
        B.shared.local_1[(ax1_30 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_30) + 1856)]
      }
      for (ii.c_14: int32, 0, 4) {
        for (jj.c_14: int32, 0, 4) {
          let cse_var_92: int32 = (jj.c_14 + 4)
          let cse_var_91: int32 = (ii.c_14 + 4)
          let cse_var_90: int32 = ((ii.c_14*4) + jj.c_14)
          let cse_var_89: int32 = (cse_var_90 + 48)
          let cse_var_88: int32 = (cse_var_90 + 32)
          let cse_var_87: int32 = (cse_var_90 + 16)
           {
            C.local_1[cse_var_90] = (C.local_1[cse_var_90] + (A.shared.local_1[jj.c_14]*B.shared.local_1[ii.c_14]))
            C.local_1[cse_var_88] = (C.local_1[cse_var_88] + (A.shared.local_1[jj.c_14]*B.shared.local_1[cse_var_91]))
            C.local_1[cse_var_87] = (C.local_1[cse_var_87] + (A.shared.local_1[cse_var_92]*B.shared.local_1[ii.c_14]))
            C.local_1[cse_var_89] = (C.local_1[cse_var_89] + (A.shared.local_1[cse_var_92]*B.shared.local_1[cse_var_91]))
          }
        }
      }
      for (ax1_31: int32, 0, 4) {
        A.shared.local_1[ax1_31] = A.shared_1[(((threadIdx.x*4) + ax1_31) + 1920)]
        A.shared.local_1[(ax1_31 + 4)] = A.shared_1[(((threadIdx.x*4) + ax1_31) + 1984)]
      }
      for (ax1_32: int32, 0, 4) {
        B.shared.local_1[ax1_32] = B.shared_1[(((threadIdx.y*4) + ax1_32) + 1920)]
        B.shared.local_1[(ax1_32 + 4)] = B.shared_1[(((threadIdx.y*4) + ax1_32) + 1984)]
      }
      for (ii.c_15: int32, 0, 4) {
        for (jj.c_15: int32, 0, 4) {
          let cse_var_98: int32 = (jj.c_15 + 4)
          let cse_var_97: int32 = (ii.c_15 + 4)
          let cse_var_96: int32 = ((ii.c_15*4) + jj.c_15)
          let cse_var_95: int32 = (cse_var_96 + 48)
          let cse_var_94: int32 = (cse_var_96 + 32)
          let cse_var_93: int32 = (cse_var_96 + 16)
           {
            C.local_1[cse_var_96] = (C.local_1[cse_var_96] + (A.shared.local_1[jj.c_15]*B.shared.local_1[ii.c_15]))
            C.local_1[cse_var_94] = (C.local_1[cse_var_94] + (A.shared.local_1[jj.c_15]*B.shared.local_1[cse_var_97]))
            C.local_1[cse_var_93] = (C.local_1[cse_var_93] + (A.shared.local_1[cse_var_98]*B.shared.local_1[ii.c_15]))
            C.local_1[cse_var_95] = (C.local_1[cse_var_95] + (A.shared.local_1[cse_var_98]*B.shared.local_1[cse_var_97]))
          }
        }
      }
    }
    for (ii.inner.inner.inner: int32, 0, 4) {
      for (jj.inner.inner.inner: int32, 0, 4) {
        let cse_var_99: int32 = ((ii.inner.inner.inner*4) + jj.inner.inner.inner)
         {
          C[((((((blockIdx.y*2097152) + (threadIdx.y*65536)) + (ii.inner.inner.inner*16384)) + (blockIdx.x*128)) + (threadIdx.x*4)) + jj.inner.inner.inner)] = C.local_1[cse_var_99]
          C[(((((((blockIdx.y*2097152) + (threadIdx.y*65536)) + (ii.inner.inner.inner*16384)) + (blockIdx.x*128)) + (threadIdx.x*4)) + jj.inner.inner.inner) + 1048576)] = C.local_1[(cse_var_99 + 32)]
          C[(((((((blockIdx.y*2097152) + (threadIdx.y*65536)) + (ii.inner.inner.inner*16384)) + (blockIdx.x*128)) + (threadIdx.x*4)) + jj.inner.inner.inner) + 64)] = C.local_1[(cse_var_99 + 16)]
          C[(((((((blockIdx.y*2097152) + (threadIdx.y*65536)) + (ii.inner.inner.inner*16384)) + (blockIdx.x*128)) + (threadIdx.x*4)) + jj.inner.inner.inner) + 1048640)] = C.local_1[(cse_var_99 + 48)]
        }
      }
    }
  }
}

