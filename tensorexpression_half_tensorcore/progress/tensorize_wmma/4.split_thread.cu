@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [256], []),
             B: Buffer(B_2: Pointer(float16), float16, [256], []),
             C: Buffer(C_2: Pointer(float32), float32, [256], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [16, 16], []), B_1: B_3: Buffer(B_2, float16, [16, 16], []), C_1: C_3: Buffer(C_2, float32, [16, 16], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [256]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [256]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [256]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [256]), storage_scope = wmma.accumulator {
    for (ax0: int32, 0, 16) {
      for (ax1: int32, 0, 16) {
        let cse_var_1: int32 = ((ax0*16) + ax1)
        A.shared_1: Buffer(A.shared, float16, [256], [], scope="shared")[cse_var_1] = A[cse_var_1]
      }
    }
    for (ax0_1: int32, 0, 16) {
      for (ax1_1: int32, 0, 16) {
        let cse_var_2: int32 = ((ax0_1*16) + ax1_1)
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [256], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 16) {
      for (ax1_2: int32, 0, 16) {
        let cse_var_3: int32 = ((ax0_2*16) + ax1_2)
        A.shared_2: Buffer(A.shared, float16, [256], [], scope="shared")[cse_var_3] = B[cse_var_3]
      }
    }
    for (ax0_3: int32, 0, 16) {
      for (ax1_3: int32, 0, 16) {
        let cse_var_4: int32 = ((ax0_3*16) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [256], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    for (ii.c: int32, 0, 16) {
      for (jj.c: int32, 0, 16) {
        C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [256], [], scope="wmma.accumulator")[((ii.c*16) + jj.c)] = 0f32
        for (rk: int32, 0, 16) {
          let cse_var_6: int32 = (ii.c*16)
          let cse_var_5: int32 = (cse_var_6 + jj.c)
          C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(float32, A.shared.wmma.matrix_a_1[(cse_var_6 + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((rk*16) + jj.c)])))
        }
      }
    }
    for (ii.inner.jj.inner.fused.outer.inner: int32, 0, 32) {
      for (ii.inner.jj.inner.fused.inner: int32, 0, 8) {
        let cse_var_7: int32 = ((ii.inner.jj.inner.fused.outer.inner*8) + ii.inner.jj.inner.fused.inner)
        C[cse_var_7] = C.wmma.accumulator_1[cse_var_7]
      }
    }
  }
}

