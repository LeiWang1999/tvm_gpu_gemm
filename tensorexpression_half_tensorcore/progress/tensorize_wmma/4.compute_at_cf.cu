@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [256], []),
             B: Buffer(B_2: Pointer(float16), float16, [256], []),
             C: Buffer(C_2: Pointer(float32), float32, [256], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [16, 16], []), B_1: B_3: Buffer(B_2, float16, [16, 16], []), C_1: C_3: Buffer(C_2, float32, [16, 16], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [1808]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [1808]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [256]), storage_scope = wmma.matrix_b {
    for (ax0: int32, 0, 113) {
      if @tir.likely((((((threadIdx.z: int32*4) + (threadIdx.y: int32*2)) + floordiv(threadIdx.x: int32, 16)) + ax0) < 16), dtype=bool) {
        for (ax1: int32, 0, 16) {
          let cse_var_1: int32 = (ax0*16)
          A.shared_1: Buffer(A.shared, float16, [1808], [], scope="shared")[(cse_var_1 + ax1)] = A[(((((threadIdx.z*64) + (threadIdx.y*32)) + (floordiv(threadIdx.x, 16)*16)) + cse_var_1) + ax1)]
        }
      }
    }
    for (ax0_1: int32, 0, 113) {
      if @tir.likely((((((threadIdx.z*4) + (threadIdx.y*2)) + floordiv(threadIdx.x, 16)) + ax0_1) < 16), dtype=bool) {
        for (ax1_1: int32, 0, 16) {
          let cse_var_2: int32 = ((ax0_1*16) + ax1_1)
          A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [1808], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
        }
      }
    }
    for (ax0_2: int32, 0, 16) {
      for (ax1_2: int32, 0, 16) {
        let cse_var_3: int32 = ((ax0_2*16) + ax1_2)
        A.shared_2: Buffer(A.shared, float16, [256], [], scope="shared")[cse_var_3] = B[cse_var_3]
      }
    }
    for (ax0_3: int32, 0, 16) {
      for (ax1_3: int32, 0, 16) {
        let cse_var_4: int32 = ((ax0_3*16) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [256], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 1;
    allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [1808]), storage_scope = wmma.accumulator;
    attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 1 {
      for (ii.c: int32, 0, 113) {
        for (jj.c: int32, 0, 16) {
          C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [1808], [], scope="wmma.accumulator")[((ii.c*16) + jj.c)] = 0f32
          if @tir.likely((((((threadIdx.z*4) + (threadIdx.y*2)) + floordiv(threadIdx.x, 16)) + ii.c) < 16), dtype=bool) {
            for (rk: int32, 0, 16) {
              let cse_var_6: int32 = (ii.c*16)
              let cse_var_5: int32 = (cse_var_6 + jj.c)
              C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(float32, A.shared.wmma.matrix_a_1[(cse_var_6 + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((rk*16) + jj.c)])))
            }
          }
        }
      }
      for (ii.inner.jj.inner.fused.outer.outer.outer.outer: int32, 0, 8) {
        attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 4;
        attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
        attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
        if @tir.likely((((((ii.inner.jj.inner.fused.outer.outer.outer.outer*256) + (threadIdx.z*64)) + (threadIdx.y*32)) + threadIdx.x) < 256), dtype=bool) {
          for (ii.inner.jj.inner.fused.inner.s: int32, 0, 8) {
            if @tir.likely((floormod(threadIdx.x, 16) < 2), dtype=bool) {
              let cse_var_7: int32 = (ii.inner.jj.inner.fused.outer.outer.outer.outer*256)
              C[(((((cse_var_7 + (threadIdx.z*64)) + (threadIdx.y*32)) + (floordiv(threadIdx.x, 16)*16)) + (floormod(threadIdx.x, 16)*8)) + ii.inner.jj.inner.fused.inner.s)] = C.wmma.accumulator_1[((cse_var_7 + (floormod(threadIdx.x, 16)*8)) + ii.inner.jj.inner.fused.inner.s)]
            }
          }
        }
      }
    }
  }
}

