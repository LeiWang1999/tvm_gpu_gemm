@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [65536], []),
             B: Buffer(B_2: Pointer(float16), float16, [65536], []),
             C: Buffer(C_2: Pointer(float32), float32, [65536], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [256, 256], []), B_1: B_3: Buffer(B_2, float16, [256, 256], []), C_1: C_3: Buffer(C_2, float32, [256, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [65536]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [65536]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [65536]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [65536]), storage_scope = wmma.accumulator {
    for (ax0: int32, 0, 256) {
      for (ax1: int32, 0, 256) {
        let cse_var_1: int32 = ((ax0*256) + ax1)
        A.shared_1: Buffer(A.shared, float16, [65536], [], scope="shared")[cse_var_1] = A[cse_var_1]
      }
    }
    for (ax0_1: int32, 0, 256) {
      for (ax1_1: int32, 0, 256) {
        let cse_var_2: int32 = ((ax0_1*256) + ax1_1)
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [65536], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 256) {
      for (ax1_2: int32, 0, 256) {
        let cse_var_3: int32 = ((ax0_2*256) + ax1_2)
        A.shared_2: Buffer(A.shared, float16, [65536], [], scope="shared")[cse_var_3] = B[cse_var_3]
      }
    }
    for (ax0_3: int32, 0, 256) {
      for (ax1_3: int32, 0, 256) {
        let cse_var_4: int32 = ((ax0_3*256) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [65536], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    for (ii.c: int32, 0, 256) {
      for (jj.c: int32, 0, 256) {
        C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [65536], [], scope="wmma.accumulator")[((ii.c*256) + jj.c)] = 0f32
        for (rk: int32, 0, 256) {
          let cse_var_6: int32 = (ii.c*256)
          let cse_var_5: int32 = (cse_var_6 + jj.c)
          C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(float32, A.shared.wmma.matrix_a_1[(cse_var_6 + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((rk*256) + jj.c)])))
        }
      }
    }
    for (ii: int32, 0, 256) {
      for (jj: int32, 0, 256) {
        let cse_var_7: int32 = ((ii*256) + jj)
        C[cse_var_7] = C.wmma.accumulator_1[cse_var_7]
      }
    }
  }
}

