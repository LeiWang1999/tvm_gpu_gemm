@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [65536], []),
             B: Buffer(B_2: Pointer(float16), float16, [65536], []),
             C: Buffer(C_2: Pointer(float32), float32, [65536], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [256, 256], []), B_1: B_3: Buffer(B_2, float16, [256, 256], []), C_1: C_3: Buffer(C_2, float32, [256, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [16384]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [16384]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [8192]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [2048]), storage_scope = wmma.accumulator {
    for (ax0: int32, 0, 64) {
      for (ax1: int32, 0, 256) {
        let cse_var_1: int32 = (ax0*256)
        A.shared_1: Buffer(A.shared, float16, [16384], [], scope="shared")[(cse_var_1 + ax1)] = A[((((blockIdx.y: int32*32768) + (threadIdx.z: int32*16384)) + cse_var_1) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 64) {
      for (ax1_1: int32, 0, 256) {
        let cse_var_2: int32 = ((ax0_1*256) + ax1_1)
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [16384], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 32) {
      for (ax1_2: int32, 0, 256) {
        let cse_var_3: int32 = (ax0_2*256)
        A.shared_2: Buffer(A.shared, float16, [8192], [], scope="shared")[(cse_var_3 + ax1_2)] = B[((((blockIdx.x: int32*16384) + (threadIdx.y: int32*8192)) + cse_var_3) + ax1_2)]
      }
    }
    for (ax0_3: int32, 0, 32) {
      for (ax1_3: int32, 0, 256) {
        let cse_var_4: int32 = ((ax0_3*256) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [8192], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    for (ii.c: int32, 0, 64) {
      for (jj.c: int32, 0, 32) {
        C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [2048], [], scope="wmma.accumulator")[((ii.c*32) + jj.c)] = 0f32
        for (rk: int32, 0, 256) {
          let cse_var_5: int32 = ((ii.c*32) + jj.c)
          C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(float32, A.shared.wmma.matrix_a_1[((ii.c*256) + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((jj.c*256) + rk)])))
        }
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 2;
    allocate(C.wmma.accumulator.shared: Pointer(shared float32), float32, [8192]), storage_scope = shared;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 4 {
      attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 2;
      attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
      for (ax0.outer.inner: int32, 0, 2) {
        for (ax1.outer.inner: int32, 0, 4) {
          for (ax0.inner: int32, 0, 32) {
            for (ax1.inner: int32, 0, 8) {
              let cse_var_6: int32 = (ax1.outer.inner*8)
              C.wmma.accumulator.shared_1: Buffer(C.wmma.accumulator.shared, float32, [8192], [], scope="shared")[((((((threadIdx.z*4096) + (ax0.outer.inner*2048)) + (ax0.inner*64)) + (threadIdx.y*32)) + cse_var_6) + ax1.inner)] = C.wmma.accumulator_1[((((ax0.outer.inner*1024) + (ax0.inner*32)) + cse_var_6) + ax1.inner)]
            }
          }
        }
      }
      for (ii.inner.jj.inner.fused.outer.outer.outer: int32, 0, 64) {
        attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 2;
        attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
        attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
        C[((((((blockIdx.y*32768) + (ii.inner.jj.inner.fused.outer.outer.outer*512)) + (threadIdx.z*256)) + (blockIdx.x*64)) + (threadIdx.y*32)) + threadIdx.x)] = C.wmma.accumulator.shared_1[((((ii.inner.jj.inner.fused.outer.outer.outer*128) + (threadIdx.z*64)) + (threadIdx.y*32)) + threadIdx.x)]
      }
    }
  }
}

