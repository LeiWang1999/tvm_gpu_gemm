@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [65536], []),
             B: Buffer(B_2: Pointer(float16), float16, [65536], []),
             C: Buffer(C_2: Pointer(float32), float32, [65536], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [256, 256], []), B_1: B_3: Buffer(B_2, float16, [256, 256], []), C_1: C_3: Buffer(C_2, float32, [256, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [32768]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [28928]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [32768]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [14464]), storage_scope = wmma.accumulator {
    for (ax0: int32, 0, 113) {
      for (ax1: int32, 0, 256) {
        let cse_var_1: int32 = (ax0*256)
        A.shared_1: Buffer(A.shared, float16, [28928], [], scope="shared")[(cse_var_1 + ax1)] = A[((((((blockIdx.y: int32*32768) + (threadIdx.z: int32*1024)) + (threadIdx.y: int32*512)) + (floordiv(threadIdx.x: int32, 16)*256)) + cse_var_1) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 113) {
      for (ax1_1: int32, 0, 256) {
        let cse_var_2: int32 = ((ax0_1*256) + ax1_1)
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [28928], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 256) {
      for (ax1_2: int32, 0, 128) {
        A.shared_2: Buffer(A.shared, float16, [32768], [], scope="shared")[((ax0_2*128) + ax1_2)] = B[(((ax0_2*256) + (blockIdx.x: int32*128)) + ax1_2)]
      }
    }
    for (ax0_3: int32, 0, 256) {
      for (ax1_3: int32, 0, 128) {
        let cse_var_3: int32 = ((ax0_3*128) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [32768], [], scope="wmma.matrix_b")[cse_var_3] = A.shared_2[cse_var_3]
      }
    }
    for (ii.c: int32, 0, 113) {
      for (jj.c: int32, 0, 128) {
        C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [14464], [], scope="wmma.accumulator")[((ii.c*128) + jj.c)] = 0f32
        for (rk: int32, 0, 256) {
          let cse_var_4: int32 = ((ii.c*128) + jj.c)
          C.wmma.accumulator_1[cse_var_4] = (C.wmma.accumulator_1[cse_var_4] + (cast(float32, A.shared.wmma.matrix_a_1[((ii.c*256) + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((rk*128) + jj.c)])))
        }
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 2;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 2;
    for (ii.inner.jj.inner.fused.outer.outer.outer.outer: int32, 0, 8) {
      attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 4;
      attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
      attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
      C[ramp((((((((blockIdx.y*32768) + (ii.inner.jj.inner.fused.outer.outer.outer.outer*4096)) + (threadIdx.z*1024)) + (threadIdx.y*512)) + (floordiv(threadIdx.x, 16)*256)) + (blockIdx.x*128)) + (floormod(threadIdx.x, 16)*8)), 1, 8)] = C.wmma.accumulator_1[ramp((((ii.inner.jj.inner.fused.outer.outer.outer.outer*2048) + (threadIdx.x*8)) - (floordiv(threadIdx.x, 16)*128)), 1, 8)]
    }
  }
}

