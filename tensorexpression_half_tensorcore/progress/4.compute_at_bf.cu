@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [65536], []),
             B: Buffer(B_2: Pointer(float16), float16, [65536], []),
             C: Buffer(C_2: Pointer(float32), float32, [65536], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [256, 256], []), B_1: B_3: Buffer(B_2, float16, [256, 256], []), C_1: C_3: Buffer(C_2, float32, [256, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [28928]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float16), float16, [32768]), storage_scope = shared {
    for (ax0: int32, 0, 113) {
      for (ax1: int32, 0, 256) {
        let cse_var_1: int32 = (ax0*256)
        A.shared_1: Buffer(A.shared, float16, [28928], [], scope="shared")[(cse_var_1 + ax1)] = A[((((((blockIdx.y: int32*32768) + (threadIdx.z: int32*1024)) + (threadIdx.y: int32*512)) + (floordiv(threadIdx.x: int32, 16)*256)) + cse_var_1) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 256) {
      for (ax1_1: int32, 0, 128) {
        B.shared_1: Buffer(B.shared, float16, [32768], [], scope="shared")[((ax0_1*128) + ax1_1)] = B[(((ax0_1*256) + (blockIdx.x: int32*128)) + ax1_1)]
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 2;
    allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [14464]), storage_scope = wmma.accumulator;
    allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [1808]), storage_scope = wmma.matrix_a;
    allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [2048]), storage_scope = wmma.matrix_b;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 2 {
      for (ii.c.outer.init: int32, 0, 8) {
        for (jj.c.outer.init: int32, 0, 8) {
          for (ii.c.inner.init: int32, 0, 16) {
            if @tir.likely((((ii.c.outer.init*16) + ii.c.inner.init) < 113), dtype=bool) {
              for (jj.c.inner.init: int32, 0, 16) {
                C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [14464], [], scope="wmma.accumulator")[((((ii.c.outer.init*2048) + (ii.c.inner.init*128)) + (jj.c.outer.init*16)) + jj.c.inner.init)] = 0f32
              }
            }
          }
        }
      }
      for (rk.outer.outer: int32, 0, 16) {
        for (ax0.outer: int32, 0, 8) {
          for (ax0.inner: int32, 0, 16) {
            if @tir.likely((((ax0.outer*16) + ax0.inner) < 113), dtype=bool) {
              for (ax1.inner: int32, 0, 16) {
                if @tir.likely((((((((blockIdx.y*128) + (ax0.outer*16)) + (threadIdx.z*4)) + (threadIdx.y*2)) + floordiv(threadIdx.x, 16)) + ax0.inner) < 256), dtype=bool) {
                  A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [1808], [], scope="wmma.matrix_a")[(((ax0.outer*256) + (ax0.inner*16)) + ax1.inner)] = A.shared_1[((((ax0.outer*4096) + (ax0.inner*256)) + (rk.outer.outer*16)) + ax1.inner)]
                }
              }
            }
          }
        }
        for (ax1.outer: int32, 0, 8) {
          for (ax0.inner_1: int32, 0, 16) {
            for (ax1.inner_1: int32, 0, 16) {
              let cse_var_3: int32 = (ax0.inner_1*128)
              let cse_var_2: int32 = (ax1.outer*16)
              B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [2048], [], scope="wmma.matrix_b")[((cse_var_3 + cse_var_2) + ax1.inner_1)] = B.shared_1[((((rk.outer.outer*2048) + cse_var_3) + cse_var_2) + ax1.inner_1)]
            }
          }
        }
        for (ii.c.outer: int32, 0, 8) {
          for (jj.c.outer: int32, 0, 8) {
            for (ii.c.inner: int32, 0, 16) {
              if @tir.likely((((ii.c.outer*16) + ii.c.inner) < 113), dtype=bool) {
                for (jj.c.inner: int32, 0, 16) {
                  for (rk.inner: int32, 0, 16) {
                    if @tir.likely((((((((blockIdx.y*128) + (ii.c.outer*16)) + (threadIdx.z*4)) + (threadIdx.y*2)) + floordiv(threadIdx.x, 16)) + ii.c.inner) < 256), dtype=bool) {
                      let cse_var_5: int32 = (jj.c.outer*16)
                      let cse_var_4: int32 = ((((ii.c.outer*2048) + (ii.c.inner*128)) + cse_var_5) + jj.c.inner)
                      C.wmma.accumulator_1[cse_var_4] = (C.wmma.accumulator_1[cse_var_4] + (cast(float32, A.shared.wmma.matrix_a_1[(((ii.c.outer*256) + (ii.c.inner*16)) + rk.inner)])*cast(float32, B.shared.wmma.matrix_b_1[(((rk.inner*128) + cse_var_5) + jj.c.inner)])))
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ii.inner.jj.inner.fused.outer.outer.outer.outer: int32, 0, 8) {
        attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 4;
        attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
        attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
        C[ramp((((((((blockIdx.y*32768) + (ii.inner.jj.inner.fused.outer.outer.outer.outer*4096)) + (threadIdx.z*1024)) + (threadIdx.y*512)) + (floordiv(threadIdx.x, 16)*256)) + (blockIdx.x*128)) + (floormod(threadIdx.x, 16)*8)), 1, 8)] = C.wmma.accumulator_1[ramp((((ii.inner.jj.inner.fused.outer.outer.outer.outer*2048) + (threadIdx.x*8)) - (floordiv(threadIdx.x, 16)*128)), 1, 8)]
      }
    }
  }
}

