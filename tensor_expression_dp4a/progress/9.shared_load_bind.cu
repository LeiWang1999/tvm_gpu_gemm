@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
  allocate(C.local: Pointer(local int32), int32, [64]), storage_scope = local;
  allocate(A.shared: Pointer(shared int8), int8, [4096]), storage_scope = shared;
  allocate(B.shared: Pointer(shared int8), int8, [4096]), storage_scope = shared;
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128;
  attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 16;
  attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 16 {
    for (i.c.init: int32, 0, 8) {
      for (j.c.init: int32, 0, 8) {
        C.local_1: Buffer(C.local, int32, [64], [], scope="local")[((i.c.init*8) + j.c.init)] = 0
      }
    }
    for (k.outer: int32, 0, 512) {
      let cse_var_1: int32 = (k.outer*32)
       {
        A.shared_1: Buffer(A.shared, int8, [4096], [], scope="shared")[ramp(((threadIdx.y*256) + (threadIdx.x*16)), 1, 16)] = A[ramp((((((blockIdx.x*2097152) + (threadIdx.y*131072)) + (floordiv(threadIdx.x, 2)*16384)) + cse_var_1) + (floormod(threadIdx.x, 2)*16)), 1, 16)]
        B.shared_1: Buffer(B.shared, int8, [4096], [], scope="shared")[ramp(((threadIdx.y*256) + (threadIdx.x*16)), 1, 16)] = B[ramp((((((blockIdx.y*2097152) + (threadIdx.y*131072)) + (floordiv(threadIdx.x, 2)*16384)) + cse_var_1) + (floormod(threadIdx.x, 2)*16)), 1, 16)]
        for (k.inner: int32, 0, 32) {
          for (i.c: int32, 0, 8) {
            for (j.c: int32, 0, 8) {
              let cse_var_2: int32 = ((i.c*8) + j.c)
              C.local_1[cse_var_2] = (C.local_1[cse_var_2] + (cast(int32, A.shared_1[(((threadIdx.x*256) + (i.c*32)) + k.inner)])*cast(int32, B.shared_1[(((threadIdx.y*256) + (j.c*32)) + k.inner)])))
            }
          }
        }
      }
    }
    for (i.inner.inner: int32, 0, 8) {
      for (j.inner.inner: int32, 0, 8) {
        C[((((((blockIdx.x*2097152) + (threadIdx.x*131072)) + (i.inner.inner*16384)) + (blockIdx.y*128)) + (threadIdx.y*8)) + j.inner.inner)] = C.local_1[((i.inner.inner*8) + j.inner.inner)]
      }
    }
  }
}

