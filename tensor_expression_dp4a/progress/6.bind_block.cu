@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.shared: Pointer(shared int8), int8, [2097152]), storage_scope = shared;
  allocate(A.shared.local: Pointer(local int8), int8, [2097152]), storage_scope = local;
  allocate(B.shared.local: Pointer(local int8), int8, [2097152]), storage_scope = local;
  allocate(C.local: Pointer(local int32), int32, [268435456]), storage_scope = local {
    for (ax0: int32, 0, 128) {
      for (ax1: int32, 0, 16384) {
        let cse_var_1: int32 = (ax0*16384)
        A.shared_1: Buffer(A.shared, int8, [2097152], [], scope="shared")[(cse_var_1 + ax1)] = A[(((blockIdx.y: int32*2097152) + cse_var_1) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 128) {
      for (ax1_1: int32, 0, 16384) {
        let cse_var_2: int32 = ((ax0_1*16384) + ax1_1)
        A.shared.local_1: Buffer(A.shared.local, int8, [2097152], [], scope="local")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 128) {
      for (ax1_2: int32, 0, 16384) {
        let cse_var_3: int32 = (ax0_2*16384)
        A.shared_2: Buffer(A.shared, int8, [2097152], [], scope="shared")[(cse_var_3 + ax1_2)] = B[(((blockIdx.x: int32*2097152) + cse_var_3) + ax1_2)]
      }
    }
    for (ax0_3: int32, 0, 128) {
      for (ax1_3: int32, 0, 16384) {
        let cse_var_4: int32 = ((ax0_3*16384) + ax1_3)
        B.shared.local_1: Buffer(B.shared.local, int8, [2097152], [], scope="local")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 128;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 128 {
      for (ii.c.inner.init: int32, 0, 128) {
        for (jj.c.inner.init: int32, 0, 128) {
          C.local_1: Buffer(C.local, int32, [268435456], [], scope="local")[((ii.c.inner.init*16384) + jj.c.inner.init)] = 0
        }
      }
      for (k.outer: int32, 0, 512) {
        for (ii.c.inner: int32, 0, 128) {
          for (jj.c.inner: int32, 0, 128) {
            for (k.inner: int32, 0, 32) {
              let cse_var_7: int32 = (ii.c.inner*16384)
              let cse_var_6: int32 = (k.outer*32)
              let cse_var_5: int32 = (cse_var_7 + jj.c.inner)
              C.local_1[cse_var_5] = (C.local_1[cse_var_5] + (cast(int32, A.shared.local_1[((((jj.c.inner*16384) + cse_var_6) + k.inner) - (blockIdx.y*2097152))])*cast(int32, B.shared.local_1[(((cse_var_7 + cse_var_6) + k.inner) - (blockIdx.x*2097152))])))
            }
          }
        }
      }
    }
    for (ii: int32, 0, 16384) {
      for (jj: int32, 0, 16384) {
        let cse_var_8: int32 = ((ii*16384) + jj)
        C[cse_var_8] = C.local_1[cse_var_8]
      }
    }
  }
}

