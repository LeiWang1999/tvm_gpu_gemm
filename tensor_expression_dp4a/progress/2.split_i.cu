@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.shared: Pointer(shared int8), int8, [268435456]), storage_scope = shared;
  allocate(B.shared: Pointer(shared int8), int8, [268435456]), storage_scope = shared;
  allocate(C.local: Pointer(local int32), int32, [268435456]), storage_scope = local {
    for (ax0: int32, 0, 16384) {
      for (ax1: int32, 0, 16384) {
        let cse_var_1: int32 = ((ax0*16384) + ax1)
        A.shared_1: Buffer(A.shared, int8, [268435456], [], scope="shared")[cse_var_1] = A[cse_var_1]
      }
    }
    for (ax0_1: int32, 0, 16384) {
      for (ax1_1: int32, 0, 16384) {
        let cse_var_2: int32 = ((ax0_1*16384) + ax1_1)
        B.shared_1: Buffer(B.shared, int8, [268435456], [], scope="shared")[cse_var_2] = B[cse_var_2]
      }
    }
    for (i.c: int32, 0, 16384) {
      for (j.c: int32, 0, 16384) {
        C.local_1: Buffer(C.local, int32, [268435456], [], scope="local")[((i.c*16384) + j.c)] = 0
        for (k: int32, 0, 16384) {
          let cse_var_4: int32 = (i.c*16384)
          let cse_var_3: int32 = (cse_var_4 + j.c)
          C.local_1[cse_var_3] = (C.local_1[cse_var_3] + (cast(int32, A.shared_1[(cse_var_4 + k)])*cast(int32, B.shared_1[((j.c*16384) + k)])))
        }
      }
    }
    for (i.outer: int32, 0, 128) {
      for (i.inner: int32, 0, 128) {
        for (j: int32, 0, 16384) {
          let cse_var_5: int32 = (((i.outer*2097152) + (i.inner*16384)) + j)
          C[cse_var_5] = C.local_1[cse_var_5]
        }
      }
    }
  }
}

