
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  int C_local[64];
  __shared__ signed char A_shared[4096];
  __shared__ signed char B_shared[4096];
  for (int i_c_outer_init = 0; i_c_outer_init < 8; ++i_c_outer_init) {
    for (int j_c_outer_init = 0; j_c_outer_init < 8; ++j_c_outer_init) {
      C_local[((i_c_outer_init * 8) + j_c_outer_init)] = 0;
    }
  }
  for (int k_outer = 0; k_outer < 512; ++k_outer) {
    __syncthreads();
    *(int4*)(A_shared + ((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 16))) = *(int4*)(A + (((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.y) * 131072)) + ((((int)threadIdx.x) >> 1) * 16384)) + (k_outer * 32)) + ((((int)threadIdx.x) & 1) * 16)));
    *(int4*)(B_shared + ((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 16))) = *(int4*)(B + (((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.y) * 131072)) + ((((int)threadIdx.x) >> 1) * 16384)) + (k_outer * 32)) + ((((int)threadIdx.x) & 1) * 16)));
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 8; ++k_inner_outer) {
      for (int i_c_outer = 0; i_c_outer < 8; ++i_c_outer) {
        for (int j_c_outer = 0; j_c_outer < 8; ++j_c_outer) {
          C_local[((i_c_outer * 8) + j_c_outer)] = __dp4a(*(int*)(A_shared + (((((int)threadIdx.x) * 256) + (i_c_outer * 32)) + (k_inner_outer * 4))), *(int*)(B_shared + (((((int)threadIdx.y) * 256) + (j_c_outer * 32)) + (k_inner_outer * 4))), C_local[((i_c_outer * 8) + j_c_outer)]);
        }
      }
    }
  }
  for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
    for (int j_inner_inner = 0; j_inner_inner < 8; ++j_inner_inner) {
      C[((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 131072)) + (i_inner_inner * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 8)) + j_inner_inner)] = C_local[((i_inner_inner * 8) + j_inner_inner)];
    }
  }
}

