#[version = "0.0.5"]
@main = primfn(var_A: handle, var_W: handle, var_Conv: handle) -> ()
  attr = {"global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1, 224, 224, 256], []),
             W: Buffer(W_1: Pointer(global float16), float16, [7, 7, 256, 512], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [1, 48400, 512], [])}
  buffer_map = {var_A: A, var_W: W, var_Conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    data_im2col_global = alloc_buffer(float16[1, 3025, 784, 16, 16])
    data_im2col_global_shared = alloc_buffer(float16[1, 3025, 784, 16, 16])
    data_im2col_global_shared_wmma.matrix_a = alloc_buffer(float16[1, 3025, 784, 16, 16])
    weight_flatten_global = alloc_buffer(float16[784, 32, 16, 16])
    weight_flatten_global_shared = alloc_buffer(float16[784, 32, 16, 16])
    weight_flatten_global_shared_wmma.matrix_b = alloc_buffer(float16[784, 32, 16, 16])
    Conv_wmma.accumulator = alloc_buffer(float16[1, 3025, 32, 16, 16])
     {
      for (ax0: int32, 0, 1) {
        for (ax1: int32, 0, 48400) {
          for (ax2: int32, 0, 12544) {
            block([1, 48400, 12544], "data_im2col_global") as [v0, v1, v2] {
              bind(v0, ax0)
              bind(v1, ax1)
              bind(v2, ax2)
              tir.reads([A[v0, ((floordiv(v2, 1792) + floordiv(v1, 220)) - 1), ((floordiv(floormod(v2, 1792), 256) + floormod(v1, 220)) - 1), floormod(v2, 256)]])
              tir.writes([data_im2col_global[v0, floordiv(v1, 16), floordiv(v2, 16), floormod(v1, 16), floormod(v2, 16)]])
              data_im2col_global[v0, floordiv(v1, 16), floordiv(v2, 16), floormod(v1, 16), floormod(v2, 16)] = @tir.if_then_else(((((1 <= (floordiv(v2, 1792) + floordiv(v1, 220))) && ((floordiv(v2, 1792) + floordiv(v1, 220)) < 225)) && (1 <= (floordiv(floormod(v2, 1792), 256) + floormod(v1, 220)))) && ((floordiv(floormod(v2, 1792), 256) + floormod(v1, 220)) < 225)), A[v0, ((floordiv(v2, 1792) + floordiv(v1, 220)) - 1), ((floordiv(floormod(v2, 1792), 256) + floormod(v1, 220)) - 1), floormod(v2, 256)], 0f16, dtype=float16)
          }
        }
      }
      for (ax0_1: int32, 0, 12544) {
        for (ax1_1: int32, 0, 512) {
          block([12544, 512], "weight_flatten_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([W[floordiv(v0_1, 1792), floordiv(floormod(v0_1, 1792), 256), floormod(v0_1, 256), v1_1]])
            tir.writes([weight_flatten_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            weight_flatten_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = W[floordiv(v0_1, 1792), floordiv(floormod(v0_1, 1792), 256), floormod(v0_1, 256), v1_1]
        }
      }
      for (n: int32, 0, 1) {
        for (x_0_0: int32, 0, 3025) "thread_binding" {
          for (y_0_0_0: int32, 0, 2) "thread_binding" {
            for (y_0_0_1: int32, 0, 16) "thread_binding" {
              for (x_0_1: int32, 0, 1) "thread_binding" {
                for (y_0_1: int32, 0, 1) "thread_binding" {
                  for (x_0_2_init: int32, 0, 1) {
                    for (y_0_2_init: int32, 0, 1) {
                      block([1, 3025, 32], "Conv_init_o") as [v_n, v_x_o, v_y_o] {
                        bind(v_n, n)
                        bind(v_x_o, ((x_0_0 + x_0_1) + x_0_2_init))
                        bind(v_y_o, ((((y_0_0_0*16) + y_0_0_1) + y_0_1) + y_0_2_init))
                        tir.reads([])
                        tir.writes([Conv_wmma.accumulator[v_n, v_x_o, v_y_o, 0:16, 0:16]])
                        C = match_buffer(Conv_wmma.accumulator[v_n, v_x_o, v_y_o, 0:16, 0:16])
                        @tir.tvm_fill_fragment(C_1: Pointer(wmma.accumulator float16), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
                    }
                  }
                  for (k_0_0: int32, 0, 196) {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 1) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                            for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 8) "vectorized" {
                              block([1, 48400, 12544], "data_im2col_global_shared") as [v0_2, v1_2, v2_1] {
                                bind(v0_2, 0)
                                bind(v1_2, ((x_0_0*16) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                                bind(v2_1, (((k_0_0*64) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                                tir.reads([data_im2col_global[v0_2, floordiv(v1_2, 16), floordiv(v2_1, 16), floormod(v1_2, 16), floormod(v2_1, 16)]])
                                tir.writes([data_im2col_global_shared[v0_2, floordiv(v1_2, 16), floordiv(v2_1, 16), floormod(v1_2, 16), floormod(v2_1, 16)]])
                                data_im2col_global_shared[v0_2, floordiv(v1_2, 16), floordiv(v2_1, 16), floormod(v1_2, 16), floormod(v2_1, 16)] = data_im2col_global[v0_2, floordiv(v1_2, 16), floordiv(v2_1, 16), floormod(v1_2, 16), floormod(v2_1, 16)]
                            }
                          }
                        }
                      }
                    }
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 1) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 4) {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                            for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 8) "vectorized" {
                              block([12544, 512], "weight_flatten_global_shared") as [v0_3, v1_3] {
                                bind(v0_3, (((k_0_0*64) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                                bind(v1_3, (((y_0_0_0*256) + (y_0_0_1*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                                tir.reads([weight_flatten_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                                tir.writes([weight_flatten_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                                weight_flatten_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = weight_flatten_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                            }
                          }
                        }
                      }
                    }
                    for (k_0_1: int32, 0, 4) {
                      for (ax0_0: int32, 0, 1) {
                        for (ax1_0: int32, 0, 1) {
                          block([1, 3025, 784], "data_im2col_global_shared_wmma.matrix_a_o") as [v0_4, v1_o, v2_o] {
                            bind(v0_4, 0)
                            bind(v1_o, (x_0_0 + ax0_0))
                            bind(v2_o, (((k_0_0*4) + k_0_1) + ax1_0))
                            tir.reads([data_im2col_global_shared[v0_4, v1_o, v2_o, 0:16, 0:16]])
                            tir.writes([data_im2col_global_shared_wmma.matrix_a[v0_4, v1_o, v2_o, 0:16, 0:16]])
                            A_2 = match_buffer(data_im2col_global_shared[v0_4, v1_o, v2_o, 0:16, 0:16])
                            C_2 = match_buffer(data_im2col_global_shared_wmma.matrix_a[v0_4, v1_o, v2_o, 0:16, 0:16])
                            @tir.tvm_load_matrix_sync(C_3: Pointer(wmma.matrix_a float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), A_3: Pointer(shared float16), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                        }
                      }
                      for (ax0_0_1: int32, 0, 1) {
                        for (ax1_0_1: int32, 0, 1) {
                          block([784, 32], "weight_flatten_global_shared_wmma.matrix_b_o") as [v0_o, v1_o_1] {
                            bind(v0_o, (((k_0_0*4) + k_0_1) + ax0_0_1))
                            bind(v1_o_1, (((y_0_0_0*16) + y_0_0_1) + ax1_0_1))
                            tir.reads([weight_flatten_global_shared[v0_o, v1_o_1, 0:16, 0:16]])
                            tir.writes([weight_flatten_global_shared_wmma.matrix_b[v0_o, v1_o_1, 0:16, 0:16]])
                            A_4 = match_buffer(weight_flatten_global_shared[v0_o, v1_o_1, 0:16, 0:16])
                            C_4 = match_buffer(weight_flatten_global_shared_wmma.matrix_b[v0_o, v1_o_1, 0:16, 0:16])
                            @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_b float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_3: int32, C_s0_2: int32), 16)*floordiv(C_s0_2, 16)) + floordiv(floormod(elem_offset_3, C_s0_2), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), A_5: Pointer(shared float16), elem_offset_4: int32, (A_s0_1: int32*16), 1, dtype=handle), A_s0_1, "row_major", dtype=handle)
                        }
                      }
                      for (x_0_2: int32, 0, 1) {
                        for (y_0_2: int32, 0, 1) {
                          block([1, 3025, 32, tir.reduce_axis(0, 784)], "Conv_update_o") as [v_n_1, v_x_o_1, v_y_o_1, v_k_o] {
                            bind(v_n_1, n)
                            bind(v_x_o_1, ((x_0_0 + x_0_1) + x_0_2))
                            bind(v_y_o_1, ((((y_0_0_0*16) + y_0_0_1) + y_0_1) + y_0_2))
                            bind(v_k_o, ((k_0_0*4) + k_0_1))
                            tir.reads([Conv_wmma.accumulator[v_n_1, v_x_o_1, v_y_o_1, 0:16, 0:16], data_im2col_global_shared_wmma.matrix_a[v_n_1, v_x_o_1, v_k_o, 0:16, 0:16], weight_flatten_global_shared_wmma.matrix_b[v_k_o, v_y_o_1, 0:16, 0:16]])
                            tir.writes([Conv_wmma.accumulator[v_n_1, v_x_o_1, v_y_o_1, 0:16, 0:16]])
                            A_6 = match_buffer(data_im2col_global_shared_wmma.matrix_a[v_n_1, v_x_o_1, v_k_o, 0:16, 0:16])
                            B = match_buffer(weight_flatten_global_shared_wmma.matrix_b[v_k_o, v_y_o_1, 0:16, 0:16])
                            C_6 = match_buffer(Conv_wmma.accumulator[v_n_1, v_x_o_1, v_y_o_1, 0:16, 0:16])
                            @tir.tvm_mma_sync(C_7: Pointer(wmma.accumulator float16), ((floordiv(floordiv(elem_offset_5: int32, C_s0_3: int32), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), A_7: Pointer(wmma.matrix_a float16), ((floordiv(floordiv(elem_offset_6: int32, A_s0_2: int32), 16)*floordiv(A_s0_2, 16)) + floordiv(floormod(elem_offset_6, A_s0_2), 16)), B_1: Pointer(wmma.matrix_b float16), ((floordiv(floordiv(elem_offset_7: int32, B_s0: int32), 16)*floordiv(B_s0, 16)) + floordiv(floormod(elem_offset_7, B_s0), 16)), C_7, ((floordiv(floordiv(elem_offset_5, C_s0_3), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), dtype=handle)
                        }
                      }
                    }
                  }
                  for (ax0_0_2: int32, 0, 1) {
                    for (ax1_0_2: int32, 0, 1) {
                      block([1, 3025, 32], "Conv_wmma.accumulator_o") as [v0_5, v1_o_2, v2_o_1] {
                        bind(v0_5, 0)
                        bind(v1_o_2, (x_0_0 + ax0_0_2))
                        bind(v2_o_1, (((y_0_0_0*16) + y_0_0_1) + ax1_0_2))
                        tir.reads([Conv_wmma.accumulator[v0_5, v1_o_2, v2_o_1, 0:16, 0:16]])
                        tir.writes([Conv[v0_5, (v1_o_2*16):((v1_o_2*16) + 16), (v2_o_1*16):((v2_o_1*16) + 16)]])
                        A_8 = match_buffer(Conv_wmma.accumulator[v0_5, v1_o_2, v2_o_1, 0:16, 0:16])
                        C_8 = match_buffer(Conv[v0_5, (v1_o_2*16):((v1_o_2*16) + 16), (v2_o_1*16):((v2_o_1*16) + 16)])
                        @tir.tvm_store_matrix_sync(A_9: Pointer(wmma.accumulator float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_8: int32, A_s0_3: int32), 16)*floordiv(A_s0_3, 16)) + floordiv(floormod(elem_offset_8, A_s0_3), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), C_9: Pointer(global float16), elem_offset_9: int32, (C_s0_4: int32*16), 2, dtype=handle), C_s0_4, "row_major", dtype=handle)
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}