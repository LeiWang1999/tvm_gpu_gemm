#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [196, 36, 16, 16], []),
             B: Buffer(B_1: Pointer(global float16), float16, [36, 4, 16, 16], []),
             C: Buffer(C_1: Pointer(global float16), float16, [1, 196, 4, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[196, 36, 16, 16])
    A_shared_warp = alloc_buffer(float16[196, 36, 16, 16])
    B_shared = alloc_buffer(float16[36, 4, 16, 16])
    B_shared_warp = alloc_buffer(float16[36, 4, 16, 16])
    C_warp = alloc_buffer(float16[1, 196, 4, 16, 16])
     {
      for (ax0: int32, 0, 36) {
        for (ax1: int32, 0, 4) {
          for (ax2: int32, 0, 16) {
            for (ax3: int32, 0, 16) {
              block([36, 4, 16, 16], "B_shared") as [v0, v1, v2, v3] {
                bind(v0, ax0)
                bind(v1, ax1)
                bind(v2, ax2)
                bind(v3, ax3)
                tir.reads([B[v0, v1, v2, v3]])
                tir.writes([B_shared[v0, v1, v2, v3]])
                B_shared[v0, v1, v2, v3] = B[v0, v1, v2, v3]
            }
          }
        }
      }
      for (ax0_1: int32, 0, 196) {
        for (ax1_1: int32, 0, 36) {
          for (ax2_1: int32, 0, 16) {
            for (ax3_1: int32, 0, 16) {
              block([196, 36, 16, 16], "A_shared") as [v0_1, v1_1, v2_1, v3_1] {
                bind(v0_1, ax0_1)
                bind(v1_1, ax1_1)
                bind(v2_1, ax2_1)
                bind(v3_1, ax3_1)
                tir.reads([A[v0_1, v1_1, v2_1, v3_1]])
                tir.writes([A_shared[v0_1, v1_1, v2_1, v3_1]])
                A_shared[v0_1, v1_1, v2_1, v3_1] = A[v0_1, v1_1, v2_1, v3_1]
            }
          }
        }
      }
      for (ax0_2: int32, 0, 196) {
        for (ax1_2: int32, 0, 36) {
          for (ax2_2: int32, 0, 16) {
            for (ax3_2: int32, 0, 16) {
              block([196, 36, 16, 16], "A_shared_warp") as [v0_2, v1_2, v2_2, v3_2] {
                bind(v0_2, ax0_2)
                bind(v1_2, ax1_2)
                bind(v2_2, ax2_2)
                bind(v3_2, ax3_2)
                tir.reads([A_shared[v0_2, v1_2, v2_2, v3_2]])
                tir.writes([A_shared_warp[v0_2, v1_2, v2_2, v3_2]])
                A_shared_warp[v0_2, v1_2, v2_2, v3_2] = A_shared[v0_2, v1_2, v2_2, v3_2]
            }
          }
        }
      }
      for (ax0_3: int32, 0, 36) {
        for (ax1_3: int32, 0, 4) {
          for (ax2_3: int32, 0, 16) {
            for (ax3_3: int32, 0, 16) {
              block([36, 4, 16, 16], "B_shared_warp") as [v0_3, v1_3, v2_3, v3_3] {
                bind(v0_3, ax0_3)
                bind(v1_3, ax1_3)
                bind(v2_3, ax2_3)
                bind(v3_3, ax3_3)
                tir.reads([B_shared[v0_3, v1_3, v2_3, v3_3]])
                tir.writes([B_shared_warp[v0_3, v1_3, v2_3, v3_3]])
                B_shared_warp[v0_3, v1_3, v2_3, v3_3] = B_shared[v0_3, v1_3, v2_3, v3_3]
            }
          }
        }
      }
      for (sk: int32, 0, 1) "thread_binding" {
        for (ii_0: int32, 0, 49) "thread_binding" {
          for (jj_0: int32, 0, 1) "thread_binding" {
            for (ii_1: int32, 0, 2) "thread_binding" {
              for (jj_1: int32, 0, 2) "thread_binding" {
                for (kk_0: int32, 0, 18) {
                  for (kk_1: int32, 0, 2) {
                    for (ii_2: int32, 0, 2) {
                      for (jj_2: int32, 0, 2) {
                        for (i: int32, 0, 16) {
                          for (j: int32, 0, 16) {
                            for (k: int32, 0, 16) {
                              block([1, 196, 4, tir.reduce_axis(0, 36), 16, 16, tir.reduce_axis(0, 16)], "B") as [vsk, vii, vjj, vkk, vi, vj, vk] {
                                bind(vsk, sk)
                                bind(vii, (((ii_0*4) + (ii_1*2)) + ii_2))
                                bind(vjj, (((jj_0*4) + (jj_1*2)) + jj_2))
                                bind(vkk, ((kk_0*2) + kk_1))
                                bind(vi, i)
                                bind(vj, j)
                                bind(vk, k)
                                tir.reads([A_shared_warp[vii, vkk, vi, vk], B_shared_warp[vkk, vjj, vk, vj]])
                                tir.writes([C_warp[vsk, vii, vjj, vi, vj]])
                                with init() {
                                  C_warp[vsk, vii, vjj, vi, vj] = 0f32
                                }
                                C_warp[vsk, vii, vjj, vi, vj] = (C_warp[vsk, vii, vjj, vi, vj] + (A_shared_warp[vii, vkk, vi, vk]*B_shared_warp[vkk, vjj, vk, vj]))
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_4: int32, 0, 1) {
        for (ax1_4: int32, 0, 196) {
          for (ax2_4: int32, 0, 4) {
            for (ax3_4: int32, 0, 16) {
              for (ax4: int32, 0, 16) {
                block([1, 196, 4, 16, 16], "C_warp") as [v0_4, v1_4, v2_4, v3_4, v4] {
                  bind(v0_4, ax0_4)
                  bind(v1_4, ax1_4)
                  bind(v2_4, ax2_4)
                  bind(v3_4, ax3_4)
                  bind(v4, ax4)
                  tir.reads([C_warp[v0_4, v1_4, v2_4, v3_4, v4]])
                  tir.writes([C[v0_4, v1_4, v2_4, v3_4, v4]])
                  C[v0_4, v1_4, v2_4, v3_4, v4] = C_warp[v0_4, v1_4, v2_4, v3_4, v4]
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}