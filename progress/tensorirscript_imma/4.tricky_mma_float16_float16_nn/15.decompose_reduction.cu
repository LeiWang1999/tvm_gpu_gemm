#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [196, 36, 16, 16], []),
             B: Buffer(B_1: Pointer(global float16), float16, [36, 4, 16, 16], []),
             C: Buffer(C_1: Pointer(global float16), float16, [1, 196, 4, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[196, 36, 16, 16])
    A_shared_warp = alloc_buffer(float16[196, 36, 16, 16])
    B_shared = alloc_buffer(float16[36, 4, 16, 16])
    B_shared_warp = alloc_buffer(float16[36, 4, 16, 16])
    C_warp = alloc_buffer(float16[1, 196, 4, 16, 16])
    for (sk: int32, 0, 1) "thread_binding" {
      for (ii_0: int32, 0, 49) "thread_binding" {
        for (jj_0: int32, 0, 1) "thread_binding" {
          for (ii_1: int32, 0, 2) "thread_binding" {
            for (jj_1: int32, 0, 2) "thread_binding" {
              for (ii_2_init: int32, 0, 2) {
                for (jj_2_init: int32, 0, 2) {
                  for (i_init: int32, 0, 16) {
                    for (j_init: int32, 0, 16) {
                      block([1, 196, 4, 16, 16], "B_init") as [vsk, vii, vjj, vi, vj] {
                        bind(vsk, sk)
                        bind(vii, (((ii_0*4) + (ii_1*2)) + ii_2_init))
                        bind(vjj, (((jj_0*4) + (jj_1*2)) + jj_2_init))
                        bind(vi, i_init)
                        bind(vj, j_init)
                        tir.reads([])
                        tir.writes([C_warp[vsk, vii, vjj, vi, vj]])
                        C_warp[vsk, vii, vjj, vi, vj] = 0f32
                    }
                  }
                }
              }
              for (kk_0: int32, 0, 18) {
                for (ax0_ax1_ax2_ax3_fused_0: int32, 0, 2) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1: int32, 0, 2) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2: int32, 0, 2) {
                      for (ax0_ax1_ax2_ax3_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4: int32, 0, 8) "vectorized" {
                          block([196, 36, 16, 16], "A_shared") as [v0, v1, v2, v3] {
                            bind(v0, ((ii_0*4) + floordiv((((((ax0_ax1_ax2_ax3_fused_0*1024) + (ax0_ax1_ax2_ax3_fused_1*512)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512)))
                            bind(v1, ((kk_0*2) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*1024) + (ax0_ax1_ax2_ax3_fused_1*512)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512), 256)))
                            bind(v2, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*1024) + (ax0_ax1_ax2_ax3_fused_1*512)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 256), 16))
                            bind(v3, floormod((((((ax0_ax1_ax2_ax3_fused_0*1024) + (ax0_ax1_ax2_ax3_fused_1*512)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 16))
                            tir.reads([A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 8)), ((floordiv(v2, 8)*8) + floormod(v3, 8))]])
                            tir.writes([A_shared[v0, v1, v2, v3]])
                            A_shared[v0, v1, v2, v3] = A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 8)), ((floordiv(v2, 8)*8) + floormod(v3, 8))]
                        }
                      }
                    }
                  }
                }
                for (ax0_ax1_ax2_ax3_fused_0_1: int32, 0, 2) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1_1: int32, 0, 2) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2_1: int32, 0, 2) {
                      for (ax0_ax1_ax2_ax3_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4_1: int32, 0, 8) "vectorized" {
                          block([36, 4, 16, 16], "B_shared") as [v0_1, v1_1, v2_1, v3_1] {
                            bind(v0_1, ((kk_0*2) + floordiv((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*512)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 1024)))
                            bind(v1_1, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*512)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 1024), 256))
                            bind(v2_1, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*512)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 256), 16))
                            bind(v3_1, floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*512)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 16))
                            tir.reads([B[v0_1, v1_1, ((floormod(v2_1, 8)*2) + floordiv(v3_1, 8)), ((floordiv(v2_1, 8)*8) + floormod(v3_1, 8))]])
                            tir.writes([B_shared[v0_1, v1_1, v2_1, v3_1]])
                            B_shared[v0_1, v1_1, v2_1, v3_1] = B[v0_1, v1_1, ((floormod(v2_1, 8)*2) + floordiv(v3_1, 8)), ((floordiv(v2_1, 8)*8) + floormod(v3_1, 8))]
                        }
                      }
                    }
                  }
                }
                for (kk_1: int32, 0, 2) {
                  for (ax0: int32, 0, 2) {
                    for (ax1: int32, 0, 16) {
                      for (ax2: int32, 0, 16) {
                        block([196, 36, 16, 16], "A_shared_warp") as [v0_2, v1_2, v2_2, v3_2] {
                          bind(v0_2, (((ii_0*4) + (ii_1*2)) + ax0))
                          bind(v1_2, ((kk_0*2) + kk_1))
                          bind(v2_2, ax1)
                          bind(v3_2, ax2)
                          tir.reads([A_shared[v0_2, v1_2, v2_2, v3_2]])
                          tir.writes([A_shared_warp[v0_2, v1_2, v2_2, v3_2]])
                          A_shared_warp[v0_2, v1_2, v2_2, v3_2] = A_shared[v0_2, v1_2, v2_2, v3_2]
                      }
                    }
                  }
                  for (ax0_1: int32, 0, 2) {
                    for (ax1_1: int32, 0, 16) {
                      for (ax2_1: int32, 0, 16) {
                        block([36, 4, 16, 16], "B_shared_warp") as [v0_3, v1_3, v2_3, v3_3] {
                          bind(v0_3, ((kk_0*2) + kk_1))
                          bind(v1_3, ((jj_1*2) + ax0_1))
                          bind(v2_3, ax1_1)
                          bind(v3_3, ax2_1)
                          tir.reads([B_shared[v0_3, v1_3, v2_3, v3_3]])
                          tir.writes([B_shared_warp[v0_3, v1_3, v2_3, v3_3]])
                          B_shared_warp[v0_3, v1_3, v2_3, v3_3] = B_shared[v0_3, v1_3, v2_3, v3_3]
                      }
                    }
                  }
                  for (ii_2: int32, 0, 2) {
                    for (jj_2: int32, 0, 2) {
                      for (i: int32, 0, 16) {
                        for (j: int32, 0, 16) {
                          for (k: int32, 0, 16) {
                            block([1, 196, 4, tir.reduce_axis(0, 36), 16, 16, tir.reduce_axis(0, 16)], "B_update") as [vsk_1, vii_1, vjj_1, vkk, vi_1, vj_1, vk] {
                              bind(vsk_1, sk)
                              bind(vii_1, (((ii_0*4) + (ii_1*2)) + ii_2))
                              bind(vjj_1, (((jj_0*4) + (jj_1*2)) + jj_2))
                              bind(vkk, ((kk_0*2) + kk_1))
                              bind(vi_1, i)
                              bind(vj_1, j)
                              bind(vk, k)
                              tir.reads([C_warp[vsk_1, vii_1, vjj_1, vi_1, vj_1], A_shared_warp[vii_1, vkk, vi_1, vk], B_shared_warp[vkk, vjj_1, vk, vj_1]])
                              tir.writes([C_warp[vsk_1, vii_1, vjj_1, vi_1, vj_1]])
                              C_warp[vsk_1, vii_1, vjj_1, vi_1, vj_1] = (C_warp[vsk_1, vii_1, vjj_1, vi_1, vj_1] + (A_shared_warp[vii_1, vkk, vi_1, vk]*B_shared_warp[vkk, vjj_1, vk, vj_1]))
                          }
                        }
                      }
                    }
                  }
                }
              }
              for (ax0_2: int32, 0, 2) {
                for (ax1_2: int32, 0, 2) {
                  for (ax2_2: int32, 0, 16) {
                    for (ax3: int32, 0, 16) {
                      block([1, 196, 4, 16, 16], "C_warp") as [v0_4, v1_4, v2_4, v3_4, v4] {
                        bind(v0_4, 0)
                        bind(v1_4, (((ii_0*4) + (ii_1*2)) + ax0_2))
                        bind(v2_4, ((jj_1*2) + ax1_2))
                        bind(v3_4, ax2_2)
                        bind(v4, ax3)
                        tir.reads([C_warp[v0_4, v1_4, v2_4, v3_4, v4]])
                        tir.writes([C[v0_4, v1_4, v2_4, v3_4, v4]])
                        C[v0_4, v1_4, v2_4, v3_4, v4] = C_warp[v0_4, v1_4, v2_4, v3_4, v4]
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}