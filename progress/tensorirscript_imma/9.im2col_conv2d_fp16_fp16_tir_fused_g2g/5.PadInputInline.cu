#[version = "0.0.5"]
@main = primfn(a: handle, w: handle, conv: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1, 56, 56, 64], []),
             W: Buffer(W_1: Pointer(global float16), float16, [3, 3, 64, 64], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [1, 3136, 64], [])}
  buffer_map = {a: A, w: W, conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    data_im2col = alloc_buffer(float16[1, 3136, 576])
    weight_flatten = alloc_buffer(float16[576, 64])
    data_im2colPad = alloc_buffer(float16[1, 3136, 576])
    weight_flattenPad = alloc_buffer(float16[576, 64])
    CPad = alloc_buffer(float16[1, 3136, 64])
    data_im2colPad_shared = alloc_buffer(float16[1, 3136, 576])
    data_im2colPad_shared_wmma.matrix_a = alloc_buffer(float16[1, 3136, 576])
    weight_flattenPad_shared = alloc_buffer(float16[576, 64])
    weight_flattenPad_shared_wmma.matrix_b = alloc_buffer(float16[576, 64])
    CPad_shared = alloc_buffer(float16[1, 3136, 64])
    CPad_shared_wmma.accumulator = alloc_buffer(float16[1, 3136, 64])
     {
      for (n: int32, 0, 1) {
        for (x: int32, 0, 3136) {
          for (y: int32, 0, 576) {
            block([1, 3136, 576], "data_im2col") as [v_n, v_x, v_y] {
              bind(v_n, n)
              bind(v_x, x)
              bind(v_y, y)
              tir.reads([A[v_n, ((floordiv(v_y, 192) + floordiv(v_x, 56)) - 1), ((floordiv(floormod(v_y, 192), 64) + floormod(v_x, 56)) - 1), floormod(v_y, 64)]])
              tir.writes([data_im2col[v_n, v_x, v_y]])
              data_im2col[v_n, v_x, v_y] = @tir.if_then_else(((((1 <= ((1*floordiv(v_x, 56)) + (1*floordiv(floordiv(v_y, 64), 3)))) && (((1*floordiv(v_x, 56)) + (1*floordiv(floordiv(v_y, 64), 3))) < 57)) && (1 <= ((1*floormod(v_x, 56)) + (1*floormod(floordiv(v_y, 64), 3))))) && (((1*floormod(v_x, 56)) + (1*floormod(floordiv(v_y, 64), 3))) < 57)), A[v_n, (((1*floordiv(v_x, 56)) + (1*floordiv(floordiv(v_y, 64), 3))) - 1), (((1*floormod(v_x, 56)) + (1*floormod(floordiv(v_y, 64), 3))) - 1), floormod(v_y, 64)], 0f16, dtype=float16)
          }
        }
      }
      for (x_1: int32, 0, 576) {
        for (y_1: int32, 0, 64) {
          block([576, 64], "weight_flatten") as [v_x_1, v_y_1] {
            bind(v_x_1, x_1)
            bind(v_y_1, y_1)
            tir.reads([W[floordiv(v_x_1, 192), floordiv(floormod(v_x_1, 192), 64), floormod(v_x_1, 64), v_y_1]])
            tir.writes([weight_flatten[v_x_1, v_y_1]])
            weight_flatten[v_x_1, v_y_1] = W[floordiv(floordiv(v_x_1, 64), 3), floormod(floordiv(v_x_1, 64), 3), floormod(v_x_1, 64), v_y_1]
        }
      }
      for (n_1: int32, 0, 1) {
        for (i: int32, 0, 3136) {
          for (k: int32, 0, 576) {
            block([1, 3136, 576], "data_im2colPad") as [vn, vi, vk] {
              bind(vn, n_1)
              bind(vi, i)
              bind(vk, k)
              tir.reads([data_im2col[vn, vi, vk]])
              tir.writes([data_im2colPad[vn, vi, vk]])
              data_im2colPad[vn, vi, vk] = @tir.if_then_else(((vi < 3136) && (vk < 576)), data_im2col[vn, vi, vk], 0f16, dtype=float16)
          }
        }
      }
      for (k_1: int32, 0, 576) {
        for (j: int32, 0, 64) {
          block([576, 64], "weight_flattenPad") as [vk_1, vj] {
            bind(vk_1, k_1)
            bind(vj, j)
            tir.reads([weight_flatten[vk_1, vj]])
            tir.writes([weight_flattenPad[vk_1, vj]])
            weight_flattenPad[vk_1, vj] = @tir.if_then_else(((vk_1 < 576) && (vj < 64)), weight_flatten[vk_1, vj], 0f16, dtype=float16)
        }
      }
      for (ax0: int32, 0, 1) {
        for (ax1: int32, 0, 3136) {
          for (ax2: int32, 0, 576) {
            block([1, 3136, 576], "data_im2colPad_shared") as [v0, v1, v2] {
              bind(v0, ax0)
              bind(v1, ax1)
              bind(v2, ax2)
              tir.reads([data_im2colPad[v0, v1, v2]])
              tir.writes([data_im2colPad_shared[v0, v1, v2]])
              data_im2colPad_shared[v0, v1, v2] = data_im2colPad[v0, v1, v2]
          }
        }
      }
      for (ax0_1: int32, 0, 1) {
        for (ax1_1: int32, 0, 3136) {
          for (ax2_1: int32, 0, 576) {
            block([1, 3136, 576], "data_im2colPad_shared_wmma.matrix_a") as [v0_1, v1_1, v2_1] {
              bind(v0_1, ax0_1)
              bind(v1_1, ax1_1)
              bind(v2_1, ax2_1)
              tir.reads([data_im2colPad_shared[v0_1, v1_1, v2_1]])
              tir.writes([data_im2colPad_shared_wmma.matrix_a[v0_1, v1_1, v2_1]])
              data_im2colPad_shared_wmma.matrix_a[v0_1, v1_1, v2_1] = data_im2colPad_shared[v0_1, v1_1, v2_1]
          }
        }
      }
      for (ax0_2: int32, 0, 576) {
        for (ax1_2: int32, 0, 64) {
          block([576, 64], "weight_flattenPad_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([weight_flattenPad[v0_2, v1_2]])
            tir.writes([weight_flattenPad_shared[v0_2, v1_2]])
            weight_flattenPad_shared[v0_2, v1_2] = weight_flattenPad[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 576) {
        for (ax1_3: int32, 0, 64) {
          block([576, 64], "weight_flattenPad_shared_wmma.matrix_b") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([weight_flattenPad_shared[v0_3, v1_3]])
            tir.writes([weight_flattenPad_shared_wmma.matrix_b[v0_3, v1_3]])
            weight_flattenPad_shared_wmma.matrix_b[v0_3, v1_3] = weight_flattenPad_shared[v0_3, v1_3]
        }
      }
      for (n_2: int32, 0, 1) {
        for (x_2: int32, 0, 3136) {
          for (y_2: int32, 0, 64) {
            for (k_2: int32, 0, 576) {
              block([1, 3136, 64, tir.reduce_axis(0, 576)], "Conv") as [v_n_1, v_x_2, v_y_2, v_k] {
                bind(v_n_1, n_2)
                bind(v_x_2, x_2)
                bind(v_y_2, y_2)
                bind(v_k, k_2)
                tir.reads([data_im2colPad_shared_wmma.matrix_a[v_n_1, v_x_2, v_k], weight_flattenPad_shared_wmma.matrix_b[v_k, v_y_2]])
                tir.writes([CPad_shared_wmma.accumulator[v_n_1, v_x_2, v_y_2]])
                with init() {
                  CPad_shared_wmma.accumulator[v_n_1, v_x_2, v_y_2] = 0f16
                }
                CPad_shared_wmma.accumulator[v_n_1, v_x_2, v_y_2] = (CPad_shared_wmma.accumulator[v_n_1, v_x_2, v_y_2] + (data_im2colPad_shared_wmma.matrix_a[v_n_1, v_x_2, v_k]*weight_flattenPad_shared_wmma.matrix_b[v_k, v_y_2]))
            }
          }
        }
      }
      for (ax0_4: int32, 0, 1) {
        for (ax1_4: int32, 0, 3136) {
          for (ax2_2: int32, 0, 64) {
            block([1, 3136, 64], "CPad_shared_wmma.accumulator") as [v0_4, v1_4, v2_2] {
              bind(v0_4, ax0_4)
              bind(v1_4, ax1_4)
              bind(v2_2, ax2_2)
              tir.reads([CPad_shared_wmma.accumulator[v0_4, v1_4, v2_2]])
              tir.writes([CPad_shared[v0_4, v1_4, v2_2]])
              CPad_shared[v0_4, v1_4, v2_2] = CPad_shared_wmma.accumulator[v0_4, v1_4, v2_2]
          }
        }
      }
      for (ax0_5: int32, 0, 1) {
        for (ax1_5: int32, 0, 3136) {
          for (ax2_3: int32, 0, 64) {
            block([1, 3136, 64], "CPad_shared") as [v0_5, v1_5, v2_3] {
              bind(v0_5, ax0_5)
              bind(v1_5, ax1_5)
              bind(v2_3, ax2_3)
              tir.reads([CPad_shared[v0_5, v1_5, v2_3]])
              tir.writes([CPad[v0_5, v1_5, v2_3]])
              CPad[v0_5, v1_5, v2_3] = CPad_shared[v0_5, v1_5, v2_3]
          }
        }
      }
      for (n_3: int32, 0, 1) {
        for (i_1: int32, 0, 3136) {
          for (j_1: int32, 0, 64) {
            block([1, 3136, 64], "CPad") as [vn_1, vi_1, vj_1] {
              bind(vn_1, n_3)
              bind(vi_1, i_1)
              bind(vj_1, j_1)
              tir.reads([CPad[vn_1, vi_1, vj_1]])
              tir.writes([Conv[vn_1, vi_1, vj_1]])
              Conv[vn_1, vi_1, vj_1] = CPad[vn_1, vi_1, vj_1]
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}