#[version = "0.0.5"]
@main = primfn(a: handle, w: handle, conv: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1, 56, 56, 64], []),
             W: Buffer(W_1: Pointer(global float16), float16, [3, 3, 64, 64], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [1, 3136, 64], [])}
  buffer_map = {a: A, w: W, conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    CPad = alloc_buffer(float16[1, 3136, 64])
    data_im2colPad_shared = alloc_buffer(float16[1, 3136, 576])
    data_im2colPad_shared_wmma.matrix_a = alloc_buffer(float16[1, 3136, 576])
    weight_flattenPad_shared = alloc_buffer(float16[576, 64])
    weight_flattenPad_shared_wmma.matrix_b = alloc_buffer(float16[576, 64])
    CPad_shared = alloc_buffer(float16[1, 3136, 64])
    CPad_shared_wmma.accumulator = alloc_buffer(float16[1, 3136, 64])
     {
      for (ax0: int32, 0, 1) {
        for (ax1: int32, 0, 3136) {
          for (ax2: int32, 0, 576) {
            block([1, 3136, 576], "data_im2colPad_shared") as [v0, v1, v2] {
              bind(v0, ax0)
              bind(v1, ax1)
              bind(v2, ax2)
              tir.reads([A[v0, ((floordiv(v2, 192) + floordiv(v1, 56)) - 1), ((floordiv(floormod(v2, 192), 64) + floormod(v1, 56)) - 1), floormod(v2, 64)]])
              tir.writes([data_im2colPad_shared[v0, v1, v2]])
              data_im2colPad_shared[v0, v1, v2] = @tir.if_then_else(((v1 < 3136) && (v2 < 576)), @tir.if_then_else(((((1 <= ((1*floordiv(v1, 56)) + (1*floordiv(floordiv(v2, 64), 3)))) && (((1*floordiv(v1, 56)) + (1*floordiv(floordiv(v2, 64), 3))) < 57)) && (1 <= ((1*floormod(v1, 56)) + (1*floormod(floordiv(v2, 64), 3))))) && (((1*floormod(v1, 56)) + (1*floormod(floordiv(v2, 64), 3))) < 57)), A[v0, (((1*floordiv(v1, 56)) + (1*floordiv(floordiv(v2, 64), 3))) - 1), (((1*floormod(v1, 56)) + (1*floormod(floordiv(v2, 64), 3))) - 1), floormod(v2, 64)], 0f16, dtype=float16), 0f16, dtype=float16)
          }
        }
      }
      for (ax0_1: int32, 0, 1) {
        for (ax1_1: int32, 0, 3136) {
          for (ax2_1: int32, 0, 576) {
            block([1, 3136, 576], "data_im2colPad_shared_wmma.matrix_a") as [v0_1, v1_1, v2_1] {
              bind(v0_1, ax0_1)
              bind(v1_1, ax1_1)
              bind(v2_1, ax2_1)
              tir.reads([data_im2colPad_shared[v0_1, v1_1, v2_1]])
              tir.writes([data_im2colPad_shared_wmma.matrix_a[v0_1, v1_1, v2_1]])
              data_im2colPad_shared_wmma.matrix_a[v0_1, v1_1, v2_1] = data_im2colPad_shared[v0_1, v1_1, v2_1]
          }
        }
      }
      for (ax0_2: int32, 0, 576) {
        for (ax1_2: int32, 0, 64) {
          block([576, 64], "weight_flattenPad_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([W[floordiv(v0_2, 192), floordiv(floormod(v0_2, 192), 64), floormod(v0_2, 64), v1_2]])
            tir.writes([weight_flattenPad_shared[v0_2, v1_2]])
            weight_flattenPad_shared[v0_2, v1_2] = @tir.if_then_else(((v0_2 < 576) && (v1_2 < 64)), W[floordiv(floordiv(v0_2, 64), 3), floormod(floordiv(v0_2, 64), 3), floormod(v0_2, 64), v1_2], 0f16, dtype=float16)
        }
      }
      for (ax0_3: int32, 0, 576) {
        for (ax1_3: int32, 0, 64) {
          block([576, 64], "weight_flattenPad_shared_wmma.matrix_b") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([weight_flattenPad_shared[v0_3, v1_3]])
            tir.writes([weight_flattenPad_shared_wmma.matrix_b[v0_3, v1_3]])
            weight_flattenPad_shared_wmma.matrix_b[v0_3, v1_3] = weight_flattenPad_shared[v0_3, v1_3]
        }
      }
      for (n: int32, 0, 1) {
        for (x: int32, 0, 3136) {
          for (y: int32, 0, 64) {
            for (k: int32, 0, 576) {
              block([1, 3136, 64, tir.reduce_axis(0, 576)], "Conv") as [v_n, v_x, v_y, v_k] {
                bind(v_n, n)
                bind(v_x, x)
                bind(v_y, y)
                bind(v_k, k)
                tir.reads([data_im2colPad_shared_wmma.matrix_a[v_n, v_x, v_k], weight_flattenPad_shared_wmma.matrix_b[v_k, v_y]])
                tir.writes([CPad_shared_wmma.accumulator[v_n, v_x, v_y]])
                with init() {
                  CPad_shared_wmma.accumulator[v_n, v_x, v_y] = 0f16
                }
                CPad_shared_wmma.accumulator[v_n, v_x, v_y] = (CPad_shared_wmma.accumulator[v_n, v_x, v_y] + (data_im2colPad_shared_wmma.matrix_a[v_n, v_x, v_k]*weight_flattenPad_shared_wmma.matrix_b[v_k, v_y]))
            }
          }
        }
      }
      for (ax0_4: int32, 0, 1) {
        for (ax1_4: int32, 0, 3136) {
          for (ax2_2: int32, 0, 64) {
            block([1, 3136, 64], "CPad_shared_wmma.accumulator") as [v0_4, v1_4, v2_2] {
              bind(v0_4, ax0_4)
              bind(v1_4, ax1_4)
              bind(v2_2, ax2_2)
              tir.reads([CPad_shared_wmma.accumulator[v0_4, v1_4, v2_2]])
              tir.writes([CPad_shared[v0_4, v1_4, v2_2]])
              CPad_shared[v0_4, v1_4, v2_2] = CPad_shared_wmma.accumulator[v0_4, v1_4, v2_2]
          }
        }
      }
      for (ax0_5: int32, 0, 1) {
        for (ax1_5: int32, 0, 3136) {
          for (ax2_3: int32, 0, 64) {
            block([1, 3136, 64], "CPad_shared") as [v0_5, v1_5, v2_3] {
              bind(v0_5, ax0_5)
              bind(v1_5, ax1_5)
              bind(v2_3, ax2_3)
              tir.reads([CPad_shared[v0_5, v1_5, v2_3]])
              tir.writes([CPad[v0_5, v1_5, v2_3]])
              CPad[v0_5, v1_5, v2_3] = CPad_shared[v0_5, v1_5, v2_3]
          }
        }
      }
      for (n_1: int32, 0, 1) {
        for (i: int32, 0, 3136) {
          for (j: int32, 0, 64) {
            block([1, 3136, 64], "CPad") as [vn, vi, vj] {
              bind(vn, n_1)
              bind(vi, i)
              bind(vj, j)
              tir.reads([CPad[vn, vi, vj]])
              tir.writes([Conv[vn, vi, vj]])
              Conv[vn, vi, vj] = CPad[vn, vi, vj]
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}