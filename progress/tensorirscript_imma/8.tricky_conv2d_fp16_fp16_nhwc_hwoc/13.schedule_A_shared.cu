#[version = "0.0.5"]
@main = primfn(var_A: handle, var_W: handle, var_Conv: handle) -> ()
  attr = {"global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16, 14, 14, 16, 16, 16], []),
             W: Buffer(W_1: Pointer(global float16), float16, [3, 3, 16, 32, 16, 16], []),
             Conv: Buffer(Conv_1: Pointer(global float16), float16, [16, 14, 14, 32, 16, 16], [])}
  buffer_map = {var_A: A, var_W: W, var_Conv: Conv} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    Apad = alloc_buffer(float16[16, 16, 16, 16, 16, 16])
    Apad_shared = alloc_buffer(float16[16, 16, 16, 16, 16, 16])
    Apad_shared_wmma.matrix_a = alloc_buffer(float16[16, 16, 16, 16, 16, 16])
    W_shared = alloc_buffer(float16[3, 3, 16, 32, 16, 16])
    W_shared_wmma.matrix_b = alloc_buffer(float16[3, 3, 16, 32, 16, 16])
    Conv_wmma.accumulator = alloc_buffer(float16[16, 14, 14, 32, 16, 16])
     {
      for (ax0: int32, 0, 3) {
        for (ax1: int32, 0, 3) {
          for (ax2: int32, 0, 16) {
            for (ax3: int32, 0, 32) {
              for (ax4: int32, 0, 16) {
                for (ax5: int32, 0, 16) {
                  block([3, 3, 16, 32, 16, 16], "W_shared") as [v0, v1, v2, v3, v4, v5] {
                    bind(v0, ax0)
                    bind(v1, ax1)
                    bind(v2, ax2)
                    bind(v3, ax3)
                    bind(v4, ax4)
                    bind(v5, ax5)
                    tir.reads([W[v0, v1, v2, v3, v4, v5]])
                    tir.writes([W_shared[v0, v1, v2, v3, v4, v5]])
                    W_shared[v0, v1, v2, v3, v4, v5] = W[v0, v1, v2, v3, v4, v5]
                }
              }
            }
          }
        }
      }
      for (n: int32, 0, 16) {
        for (h: int32, 0, 16) {
          for (w: int32, 0, 16) {
            for (i: int32, 0, 16) {
              for (nn: int32, 0, 16) {
                for (ii: int32, 0, 16) {
                  block([16, 16, 16, 16, 16, 16], "Apad_pad_const") as [v_n, v_h, v_w, v_i, v_nn, v_ii] {
                    bind(v_n, n)
                    bind(v_h, h)
                    bind(v_w, w)
                    bind(v_i, i)
                    bind(v_nn, nn)
                    bind(v_ii, ii)
                    tir.reads([])
                    tir.writes([Apad[v_n, v_h, v_w, v_i, v_nn, v_ii]])
                    Apad[v_n, v_h, v_w, v_i, v_nn, v_ii] = 0f16
                }
              }
            }
          }
        }
      }
      for (n_1: int32, 0, 16) {
        for (h_1: int32, 0, 14) {
          for (w_1: int32, 0, 14) {
            for (i_1: int32, 0, 16) {
              for (nn_1: int32, 0, 16) {
                for (ii_1: int32, 0, 16) {
                  block([16, 14, 14, 16, 16, 16], "Apad") as [v_n_1, v_h_1, v_w_1, v_i_1, v_nn_1, v_ii_1] {
                    bind(v_n_1, n_1)
                    bind(v_h_1, h_1)
                    bind(v_w_1, w_1)
                    bind(v_i_1, i_1)
                    bind(v_nn_1, nn_1)
                    bind(v_ii_1, ii_1)
                    tir.reads([A[v_n_1, v_h_1, v_w_1, v_i_1, v_nn_1, v_ii_1]])
                    tir.writes([Apad[v_n_1, (v_h_1 + 1), (v_w_1 + 1), v_i_1, v_nn_1, v_ii_1]])
                    Apad[v_n_1, (v_h_1 + 1), (v_w_1 + 1), v_i_1, v_nn_1, v_ii_1] = A[v_n_1, v_h_1, v_w_1, v_i_1, v_nn_1, v_ii_1]
                }
              }
            }
          }
        }
      }
      for (n_0_0: int32, 0, 2) "thread_binding" {
        for (o_0_0: int32, 0, 4) "thread_binding" {
          for (n_0_1: int32, 0, 4) "thread_binding" {
            for (h_2: int32, 0, 14) {
              for (w_2: int32, 0, 14) {
                for (o_0_1: int32, 0, 2) "thread_binding" {
                  for (ic_0: int32, 0, 8) {
                    for (kh: int32, 0, 3) {
                      for (ax0_1_0: int32, 0, 2) "thread_binding" {
                        for (ax0_0: int32, 0, 4) "thread_binding" {
                          for (ax3_ax4_fused_1: int32, 0, 32) "thread_binding" {
                            for (ax1_1: int32, 0, 3) {
                              for (ax2_1: int32, 0, 2) {
                                for (ax3_ax4_fused_0: int32, 0, 8) {
                                  for (ax0_1_1: int32, 0, 1) {
                                    block([16, 16, 16, 16, 16, 16], "Apad_shared") as [v0_1, v1_1, v2_1, v3_1, v4_1, v5_1] {
                                      bind(v0_1, (((ax0_1_1 + (n_0_0*8)) + (ax0_0*2)) + ax0_1_0))
                                      bind(v1_1, (h_2 + kh))
                                      bind(v2_1, (w_2 + ax1_1))
                                      bind(v3_1, ((ic_0*2) + ax2_1))
                                      bind(v4_1, floordiv(((ax3_ax4_fused_0*32) + ax3_ax4_fused_1), 16))
                                      bind(v5_1, floormod(((ax3_ax4_fused_0*32) + ax3_ax4_fused_1), 16))
                                      tir.reads([Apad[v0_1, v1_1, v2_1, v3_1, v4_1, v5_1]])
                                      tir.writes([Apad_shared[v0_1, v1_1, v2_1, v3_1, v4_1, v5_1]])
                                      Apad_shared[v0_1, v1_1, v2_1, v3_1, v4_1, v5_1] = Apad[v0_1, v1_1, v2_1, v3_1, v4_1, v5_1]
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      for (ic_1: int32, 0, 2) {
                        for (kw: int32, 0, 3) {
                          for (ax0_1: int32, 0, 2) {
                            for (ax1_2: int32, 0, 16) {
                              for (ax2_2: int32, 0, 16) {
                                block([16, 16, 16, 16, 16, 16], "Apad_shared_wmma.matrix_a") as [v0_2, v1_2, v2_2, v3_2, v4_2, v5_2] {
                                  bind(v0_2, (((n_0_0*8) + (n_0_1*2)) + ax0_1))
                                  bind(v1_2, (h_2 + kh))
                                  bind(v2_2, (w_2 + kw))
                                  bind(v3_2, ((ic_0*2) + ic_1))
                                  bind(v4_2, ax1_2)
                                  bind(v5_2, ax2_2)
                                  tir.reads([Apad_shared[v0_2, v1_2, v2_2, v3_2, v4_2, v5_2]])
                                  tir.writes([Apad_shared_wmma.matrix_a[v0_2, v1_2, v2_2, v3_2, v4_2, v5_2]])
                                  Apad_shared_wmma.matrix_a[v0_2, v1_2, v2_2, v3_2, v4_2, v5_2] = Apad_shared[v0_2, v1_2, v2_2, v3_2, v4_2, v5_2]
                              }
                            }
                          }
                          for (ax0_2: int32, 0, 4) {
                            for (ax1_3: int32, 0, 16) {
                              for (ax2_3: int32, 0, 16) {
                                block([3, 3, 16, 32, 16, 16], "W_shared_wmma.matrix_b") as [v0_3, v1_3, v2_3, v3_3, v4_3, v5_3] {
                                  bind(v0_3, kh)
                                  bind(v1_3, kw)
                                  bind(v2_3, ((ic_0*2) + ic_1))
                                  bind(v3_3, (((o_0_0*8) + (o_0_1*4)) + ax0_2))
                                  bind(v4_3, ax1_3)
                                  bind(v5_3, ax2_3)
                                  tir.reads([W_shared[v0_3, v1_3, v2_3, v3_3, v4_3, v5_3]])
                                  tir.writes([W_shared_wmma.matrix_b[v0_3, v1_3, v2_3, v3_3, v4_3, v5_3]])
                                  W_shared_wmma.matrix_b[v0_3, v1_3, v2_3, v3_3, v4_3, v5_3] = W_shared[v0_3, v1_3, v2_3, v3_3, v4_3, v5_3]
                              }
                            }
                          }
                          for (n_1_1: int32, 0, 2) {
                            for (o_1: int32, 0, 4) {
                              for (nn_2: int32, 0, 16) {
                                for (oo: int32, 0, 16) {
                                  for (ii_2: int32, 0, 16) {
                                    block([16, 14, 14, 32, 16, 16, tir.reduce_axis(0, 16), tir.reduce_axis(0, 3), tir.reduce_axis(0, 3), tir.reduce_axis(0, 16)], "Conv") as [v_n_2, v_h_2, v_w_2, v_o, v_nn_2, v_oo, v_ic, v_kh, v_kw, v_ii_2] {
                                      bind(v_n_2, (((n_0_0*8) + (n_0_1*2)) + n_1_1))
                                      bind(v_h_2, h_2)
                                      bind(v_w_2, w_2)
                                      bind(v_o, (((o_0_0*8) + (o_0_1*4)) + o_1))
                                      bind(v_nn_2, nn_2)
                                      bind(v_oo, oo)
                                      bind(v_ic, ((ic_0*2) + ic_1))
                                      bind(v_kh, kh)
                                      bind(v_kw, kw)
                                      bind(v_ii_2, ii_2)
                                      tir.reads([Apad_shared_wmma.matrix_a[v_n_2, (v_h_2 + v_kh), (v_w_2 + v_kw), v_ic, v_nn_2, v_ii_2], W_shared_wmma.matrix_b[v_kh, v_kw, v_ic, v_o, v_ii_2, v_oo]])
                                      tir.writes([Conv_wmma.accumulator[v_n_2, v_h_2, v_w_2, v_o, v_nn_2, v_oo]])
                                      with init() {
                                        Conv_wmma.accumulator[v_n_2, v_h_2, v_w_2, v_o, v_nn_2, v_oo] = 0f16
                                      }
                                      Conv_wmma.accumulator[v_n_2, v_h_2, v_w_2, v_o, v_nn_2, v_oo] = (Conv_wmma.accumulator[v_n_2, v_h_2, v_w_2, v_o, v_nn_2, v_oo] + (Apad_shared_wmma.matrix_a[v_n_2, (v_h_2 + v_kh), (v_w_2 + v_kw), v_ic, v_nn_2, v_ii_2]*W_shared_wmma.matrix_b[v_kh, v_kw, v_ic, v_o, v_ii_2, v_oo]))
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  for (ax0_3: int32, 0, 2) {
                    for (ax1_4: int32, 0, 4) {
                      for (ax2_4: int32, 0, 16) {
                        for (ax3_1: int32, 0, 16) {
                          block([16, 14, 14, 32, 16, 16], "Conv_wmma.accumulator") as [v0_4, v1_4, v2_4, v3_4, v4_4, v5_4] {
                            bind(v0_4, (((n_0_0*8) + (n_0_1*2)) + ax0_3))
                            bind(v1_4, h_2)
                            bind(v2_4, w_2)
                            bind(v3_4, (((o_0_0*8) + (o_0_1*4)) + ax1_4))
                            bind(v4_4, ax2_4)
                            bind(v5_4, ax3_1)
                            tir.reads([Conv_wmma.accumulator[v0_4, v1_4, v2_4, v3_4, v4_4, v5_4]])
                            tir.writes([Conv[v0_4, v1_4, v2_4, v3_4, v4_4, v5_4]])
                            Conv[v0_4, v1_4, v2_4, v3_4, v4_4, v5_4] = Conv_wmma.accumulator[v0_4, v1_4, v2_4, v3_4, v4_4, v5_4]
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}