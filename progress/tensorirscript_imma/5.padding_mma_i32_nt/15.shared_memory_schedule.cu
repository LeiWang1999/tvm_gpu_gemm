#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    PA = alloc_buffer(int32[16384])
    A_shared = alloc_buffer(int8[16384, 16384])
    B_shared = alloc_buffer(int8[16384, 16384])
    for (i_0: int32, 0, 128) "thread_binding" {
      for (j_0: int32, 0, 64) "thread_binding" {
        for (i_1_0: int32, 0, 2) "thread_binding" {
          for (j_1_0: int32, 0, 4) "thread_binding" {
            for (k_0: int32, 0, 256) {
              for (ax0_ax1_fused_0: int32, 0, 131072) {
                for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                    for (ax0_ax1_fused_3: int32, 0, 16) "vectorized" {
                      block([16384, 16384], "A_shared") as [v0, v1] {
                        bind(v0, floordiv(((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*16)) + ax0_ax1_fused_3), 16384))
                        bind(v1, floormod(((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*16)) + ax0_ax1_fused_3), 16384))
                        tir.reads([A[v0, v1]])
                        tir.writes([A_shared[v0, v1]])
                        tir.attrs({"buffer_dim_align": [[0, 0, 32, 0]]})
                        A_shared[v0, v1] = A[v0, v1]
                    }
                  }
                }
              }
              for (ax0_ax1_fused_0_1: int32, 0, 8) {
                for (ax0_ax1_fused_1_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_fused_2_1: int32, 0, 32) "thread_binding" {
                    for (ax0_ax1_fused_3_1: int32, 0, 16) "vectorized" {
                      block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                        bind(v0_1, ((j_0*256) + floordiv(((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*512)) + (ax0_ax1_fused_2_1*16)) + ax0_ax1_fused_3_1), 64)))
                        bind(v1_1, ((k_0*64) + floormod(((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*512)) + (ax0_ax1_fused_2_1*16)) + ax0_ax1_fused_3_1), 64)))
                        tir.reads([B[v0_1, v1_1]])
                        tir.writes([B_shared[v0_1, v1_1]])
                        tir.attrs({"buffer_dim_align": [[0, 0, 32, 0]]})
                        B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                    }
                  }
                }
              }
              for (i_1_1: int32, 0, 64) {
                for (j_1_1: int32, 0, 64) {
                  for (k_1: int32, 0, 64) {
                    block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                      bind(vi, (((i_0*128) + (i_1_0*64)) + i_1_1))
                      bind(vj, (((j_0*256) + (j_1_0*64)) + j_1_1))
                      bind(vk, ((k_0*64) + k_1))
                      tir.reads([A_shared[vi, vk], B_shared[vj, vk]])
                      tir.writes([C[vi, vj]])
                      with init() {
                        C[vi, vj] = 0
                      }
                      C[vi, vj] = (C[vi, vj] + (cast(int32, A_shared[vi, vk])*cast(int32, B_shared[vj, vk])))
                  }
                }
              }
              for (ax0: int32, 0, 16384) {
                for (ax1: int32, 0, 16384) {
                  block([16384, tir.reduce_axis(0, 16384)], "Pre_compute_A") as [vi_1, vk_1] {
                    bind(vi_1, ax0)
                    bind(vk_1, ax1)
                    tir.reads([A_shared[vi_1, vk_1]])
                    tir.writes([PA[vi_1]])
                    with init() {
                      PA[vi_1] = 0
                    }
                    PA[vi_1] = (PA[vi_1] + (1*cast(int32, A_shared[vi_1, vk_1])))
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}