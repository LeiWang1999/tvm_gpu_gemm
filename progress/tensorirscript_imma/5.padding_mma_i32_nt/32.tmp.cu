
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  int C_warp[128];
  __shared__ signed char A_shared[8192];
  __shared__ signed char B_shared[16384];
  signed char A_shared_warp[16];
  signed char B_shared_warp[16];
  for (int i_1_1_0_init = 0; i_1_1_0_init < 4; ++i_1_1_0_init) {
    for (int j_1_1_0_init = 0; j_1_1_0_init < 4; ++j_1_1_0_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[((i_1_1_0_init * 32) + (j_1_1_0_init * 8)) + i] = 0.0;}
;
    }
  }
  for (int k_0 = 0; k_0 < 256; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {
      *(int4*)(A_shared + (((ax0_ax1_fused_0 * 2048) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A + ((((((((int)blockIdx.y) * 2097152) + (ax0_ax1_fused_0 * 524288)) + (((int)threadIdx.y) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + (k_0 * 64)) + ((((int)threadIdx.x) & 3) * 16)));
    }
    for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 8; ++ax0_ax1_fused_0_1) {
      *(int4*)(B_shared + (((ax0_ax1_fused_0_1 * 2048) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B + ((((((((int)blockIdx.x) * 4194304) + (ax0_ax1_fused_0_1 * 524288)) + (((int)threadIdx.y) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + (k_0 * 64)) + ((((int)threadIdx.x) & 3) * 16)));
    }
    __syncthreads();
    for (int i_1_1_0 = 0; i_1_1_0 < 4; ++i_1_1_0) {
      for (int j_1_1_0 = 0; j_1_1_0 < 4; ++j_1_1_0) {
        for (int k_1_0 = 0; k_1_0 < 2; ++k_1_0) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[(((((int)threadIdx.z) * 4096) + (i_1_1_0 * 1024)) + (k_1_0 * 32))])) + (((((int)threadIdx.x) & 15) * 64) + ((((int)threadIdx.x) >> 4) * 16))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
      : "r"(addr)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_shared[(((((int)threadIdx.y) * 4096) + (j_1_1_0 * 1024)) + (k_1_0 * 32))])) + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 7) * 64)) + (((((int)threadIdx.x) & 15) >> 3) * 16))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_shared_warp + 0))[0]), "=r"(((unsigned *)(B_shared_warp + 0))[1]), "=r"(((unsigned *)(B_shared_warp + 0))[2]), "=r"(((unsigned *)(B_shared_warp + 0))[3])
      : "r"(addr)
    );
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[0]), "=r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[1]), "=r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[2]), "=r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[3])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + 0))[0]), "r"(((unsigned *)(B_shared_warp + 0))[1]), "r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[0]), "r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[1]), "r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[2]), "r"(((int *)(C_warp + ((i_1_1_0 * 32) + (j_1_1_0 * 8))))[3]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[0]), "=r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[1]), "=r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[2]), "=r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[3])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + 8))[0]), "r"(((unsigned *)(B_shared_warp + 8))[1]), "r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[0]), "r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[1]), "r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[2]), "r"(((int *)(C_warp + (((i_1_1_0 * 32) + (j_1_1_0 * 8)) + 4)))[3]));
  }
        }
      }
    }
  }
  for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0) {
    for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
      for (int local_id = 0; local_id < 8; ++local_id) {
(&(C[((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.z) * 1048576)) + (ax0_0 * 262144)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.y) * 64)) + (ax1_0 * 16))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16384) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))] = C_warp[((ax0_0 * 32) + (ax1_0 * 8)) + local_id];
}
;
    }
  }
}

