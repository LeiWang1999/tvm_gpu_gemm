#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [256, 256], []),
             B: Buffer(B_1: Pointer(global float16), float16, [256, 256], []),
             C: Buffer(C_1: Pointer(global float16), float16, [256, 256], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[256, 256])
    B_shared = alloc_buffer(float16[256, 256])
    A_shared_warp = alloc_buffer(float16[256, 256])
    B_shared_warp = alloc_buffer(float16[256, 256])
    C_warp = alloc_buffer(float16[256, 256])
    for (i_0: int32, 0, 1) "thread_binding" {
      for (j_0: int32, 0, 2) "thread_binding" {
        for (i_1_0: int32, 0, 2) "thread_binding" {
          for (j_1_0: int32, 0, 4) "thread_binding" {
            for (i_1_1_0_init: int32, 0, 8) {
              for (j_1_1_0_init: int32, 0, 2) {
                for (i_1_1_1_init: int32, 0, 16) {
                  for (j_1_1_1_init: int32, 0, 16) {
                    block([256, 256], "B_init") as [vi, vj] {
                      bind(vi, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0_init*16)) + i_1_1_1_init))
                      bind(vj, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0_init*16)) + j_1_1_1_init))
                      tir.reads([])
                      tir.writes([C_warp[vi, vj]])
                      C_warp[vi, vj] = 0f16
                  }
                }
              }
            }
            for (k_0: int32, 0, 8) {
              for (ax0_ax1_fused_0: int32, 0, 8) {
                for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                    for (ax0_ax1_fused_3: int32, 0, 8) "vectorized" {
                      block([256, 256], "A_shared") as [v0, v1] {
                        bind(v0, floordiv(((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*256)) + (ax0_ax1_fused_2*8)) + ax0_ax1_fused_3), 32))
                        bind(v1, ((k_0*32) + floormod(((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*256)) + (ax0_ax1_fused_2*8)) + ax0_ax1_fused_3), 32)))
                        tir.reads([A[v0, v1]])
                        tir.writes([A_shared[v0, v1]])
                        tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                        A_shared[v0, v1] = A[v0, v1]
                    }
                  }
                }
              }
              for (ax0_ax1_fused_0_1: int32, 0, 4) {
                for (ax0_ax1_fused_1_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_fused_2_1: int32, 0, 32) "thread_binding" {
                    for (ax0_ax1_fused_3_1: int32, 0, 8) "vectorized" {
                      block([256, 256], "B_shared") as [v0_1, v1_1] {
                        bind(v0_1, ((j_0*128) + floordiv(((((ax0_ax1_fused_0_1*1024) + (ax0_ax1_fused_1_1*256)) + (ax0_ax1_fused_2_1*8)) + ax0_ax1_fused_3_1), 32)))
                        bind(v1_1, ((k_0*32) + floormod(((((ax0_ax1_fused_0_1*1024) + (ax0_ax1_fused_1_1*256)) + (ax0_ax1_fused_2_1*8)) + ax0_ax1_fused_3_1), 32)))
                        tir.reads([B[v0_1, v1_1]])
                        tir.writes([B_shared[v0_1, v1_1]])
                        tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                        B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                    }
                  }
                }
              }
              for (i_1_1_0: int32, 0, 8) {
                for (j_1_1_0: int32, 0, 2) {
                  for (k_1_0: int32, 0, 2) {
                    for (ax0: int32, 0, 16) {
                      for (ax1: int32, 0, 16) {
                        block([256, 256], "A_shared_warp") as [v0_2, v1_2] {
                          bind(v0_2, (((i_1_0*128) + (i_1_1_0*16)) + ax0))
                          bind(v1_2, (((k_0*32) + (k_1_0*16)) + ax1))
                          tir.reads([A_shared[v0_2, v1_2]])
                          tir.writes([A_shared_warp[v0_2, v1_2]])
                          A_shared_warp[v0_2, v1_2] = A_shared[v0_2, v1_2]
                      }
                    }
                    for (ax0_1: int32, 0, 16) {
                      for (ax1_1: int32, 0, 16) {
                        block([256, 256], "B_shared_warp") as [v0_3, v1_3] {
                          bind(v0_3, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0*16)) + ax0_1))
                          bind(v1_3, (((k_0*32) + (k_1_0*16)) + ax1_1))
                          tir.reads([B_shared[v0_3, v1_3]])
                          tir.writes([B_shared_warp[v0_3, v1_3]])
                          B_shared_warp[v0_3, v1_3] = B_shared[v0_3, v1_3]
                      }
                    }
                    for (i_1_1_1: int32, 0, 16) {
                      for (j_1_1_1: int32, 0, 16) {
                        for (k_1_1: int32, 0, 16) {
                          block([256, 256, tir.reduce_axis(0, 256)], "B_update") as [vi_1, vj_1, vk] {
                            bind(vi_1, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0*16)) + i_1_1_1))
                            bind(vj_1, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0*16)) + j_1_1_1))
                            bind(vk, (((k_0*32) + (k_1_0*16)) + k_1_1))
                            tir.reads([C_warp[vi_1, vj_1], A_shared_warp[vi_1, vk], B_shared_warp[vj_1, vk]])
                            tir.writes([C_warp[vi_1, vj_1]])
                            C_warp[vi_1, vj_1] = (C_warp[vi_1, vj_1] + (A_shared_warp[vi_1, vk]*B_shared_warp[vj_1, vk]))
                        }
                      }
                    }
                  }
                }
              }
            }
            for (ax0_0: int32, 0, 8) {
              for (ax1_0: int32, 0, 2) {
                for (ax0_1_1: int32, 0, 16) {
                  for (ax1_1_1: int32, 0, 16) {
                    block([256, 256], "C_warp") as [v0_4, v1_4] {
                      bind(v0_4, (((i_1_0*128) + (ax0_0*16)) + ax0_1_1))
                      bind(v1_4, ((((j_0*128) + (j_1_0*32)) + (ax1_0*16)) + ax1_1_1))
                      tir.reads([C_warp[v0_4, v1_4]])
                      tir.writes([C[v0_4, v1_4]])
                      C[v0_4, v1_4] = C_warp[v0_4, v1_4]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}