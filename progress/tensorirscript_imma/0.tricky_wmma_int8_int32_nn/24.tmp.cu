#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_shared[8192];
  __shared__ signed char B_shared[2048];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[16];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> B_shared_wmma_matrix_b[1];
  for (int ii_2_init = 0; ii_2_init < 16; ++ii_2_init) {
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[ii_2_init], 0.000000e+00f);
  }
  for (int kk_0 = 0; kk_0 < 512; ++kk_0) {
    __syncthreads();
    for (int ax0_ax1_ax2_ax3_fused_2 = 0; ax0_ax1_ax2_ax3_fused_2 < 4; ++ax0_ax1_ax2_ax3_fused_2) {
      *(int4*)(A_shared + (((((int)threadIdx.z) * 2048) + (ax0_ax1_ax2_ax3_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A + (((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.z) * 1048576)) + (ax0_ax1_ax2_ax3_fused_2 * 262144)) + (kk_0 * 512)) + (((int)threadIdx.x) * 16)));
    }
    *(int4*)(B_shared + ((((int)threadIdx.z) * 512) + (((int)threadIdx.x) * 16))) = *(int4*)(B + ((((((kk_0 * 524288) + ((((int)threadIdx.z) >> 1) * 262144)) + (((int)blockIdx.z) * 16384)) + (((int)blockIdx.x) * 1024)) + ((((int)threadIdx.z) & 1) * 512)) + (((int)threadIdx.x) * 16)));
    __syncthreads();
    for (int kk_1 = 0; kk_1 < 2; ++kk_1) {
      for (int ax0 = 0; ax0 < 16; ++ax0) {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0], (&(A_shared[((ax0 * 512) + (kk_1 * 256))])), 16);
      }
      nvcuda::wmma::load_matrix_sync(B_shared_wmma_matrix_b[0], (&(B_shared[((kk_1 * 1024) + (((int)threadIdx.z) * 256))])), 16);
      for (int ii_2 = 0; ii_2 < 16; ++ii_2) {
        nvcuda::wmma::mma_sync(C_wmma_accumulator[ii_2], A_shared_wmma_matrix_a[ii_2], B_shared_wmma_matrix_b[0], C_wmma_accumulator[ii_2]);
      }
    }
  }
  for (int ax0_1 = 0; ax0_1 < 16; ++ax0_1) {
    nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 4194304) + (ax0_1 * 262144)) + (((int)blockIdx.z) * 16384)) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.z) * 256))])), C_wmma_accumulator[ax0_1], 16, nvcuda::wmma::mem_row_major);
  }
}

