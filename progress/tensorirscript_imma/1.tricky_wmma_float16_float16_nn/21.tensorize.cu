#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1024, 1024, 16, 16], []),
             B: Buffer(B_1: Pointer(global float16), float16, [1024, 1024, 16, 16], []),
             C: Buffer(C_1: Pointer(global float16), float16, [1024, 1024, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    A_shared_wmma.matrix_a = alloc_buffer(float16[1024, 1024, 16, 16])
    B_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    B_shared_wmma.matrix_b = alloc_buffer(float16[1024, 1024, 16, 16])
    C_wmma.accumulator = alloc_buffer(float16[1024, 1024, 16, 16])
    for (ii_0: int32, 0, 128) "thread_binding" {
      for (jj_0: int32, 0, 64) "thread_binding" {
        for (ii_1: int32, 0, 1) "thread_binding" {
          for (jj_1: int32, 0, 4) "thread_binding" {
            for (ii_2_init: int32, 0, 8) {
              for (jj_2_init: int32, 0, 4) {
                block([1024, 1024, 1, 1], "B_init_o") as [vii, vjj, vi_o, vj_o] {
                  bind(vii, (((ii_0*8) + (ii_1*8)) + ii_2_init))
                  bind(vjj, (((jj_0*16) + (jj_1*4)) + jj_2_init))
                  bind(vi_o, 0)
                  bind(vj_o, 0)
                  tir.reads([])
                  tir.writes([C_wmma.accumulator[vii, vjj, 0:16, 0:16]])
                  C_2 = match_buffer(C_wmma.accumulator[vii, vjj, 0:16, 0:16])
                  @tir.tvm_fill_fragment(C_3: Pointer(wmma.accumulator float16), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
              }
            }
            for (kk_0: int32, 0, 512) {
              for (ax0_ax1_ax2_ax3_fused_0: int32, 0, 1) "thread_binding" {
                for (ax0_ax1_ax2_ax3_fused_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_2: int32, 0, 4) {
                    for (ax0_ax1_ax2_ax3_fused_3: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_ax2_ax3_fused_4: int32, 0, 8) "vectorized" {
                        block([1024, 1024, 16, 16], "A_shared") as [v0, v1, v2, v3] {
                          bind(v0, ((ii_0*8) + floordiv((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512)))
                          bind(v1, ((kk_0*2) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 512), 256)))
                          bind(v2, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 256), 16))
                          bind(v3, floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*1024)) + (ax0_ax1_ax2_ax3_fused_2*256)) + (ax0_ax1_ax2_ax3_fused_3*8)) + ax0_ax1_ax2_ax3_fused_4), 16))
                          tir.reads([A[v0, v1, v2, v3]])
                          tir.writes([A_shared[v0, v1, v2, v3]])
                          A_shared[v0, v1, v2, v3] = A[v0, v1, v2, v3]
                      }
                    }
                  }
                }
              }
              for (ax0_ax1_ax2_ax3_fused_0_1: int32, 0, 1) "thread_binding" {
                for (ax0_ax1_ax2_ax3_fused_1_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_2_1: int32, 0, 8) {
                    for (ax0_ax1_ax2_ax3_fused_3_1: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_ax2_ax3_fused_4_1: int32, 0, 8) "vectorized" {
                        block([1024, 1024, 16, 16], "B_shared") as [v0_1, v1_1, v2_1, v3_1] {
                          bind(v0_1, ((kk_0*2) + floordiv((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 4096)))
                          bind(v1_1, ((jj_0*16) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 4096), 256)))
                          bind(v2_1, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 256), 16))
                          bind(v3_1, floormod((((((ax0_ax1_ax2_ax3_fused_0_1*8192) + (ax0_ax1_ax2_ax3_fused_1_1*2048)) + (ax0_ax1_ax2_ax3_fused_2_1*256)) + (ax0_ax1_ax2_ax3_fused_3_1*8)) + ax0_ax1_ax2_ax3_fused_4_1), 16))
                          tir.reads([B[v0_1, v1_1, v2_1, v3_1]])
                          tir.writes([B_shared[v0_1, v1_1, v2_1, v3_1]])
                          B_shared[v0_1, v1_1, v2_1, v3_1] = B[v0_1, v1_1, v2_1, v3_1]
                      }
                    }
                  }
                }
              }
              for (kk_1: int32, 0, 2) {
                for (ax0: int32, 0, 8) {
                  block([1024, 1024, 1, 1], "A_shared_wmma.matrix_a_o") as [v0_2, v1_2, v2_o, v3_o] {
                    bind(v0_2, ((ii_0*8) + ax0))
                    bind(v1_2, ((kk_0*2) + kk_1))
                    bind(v2_o, 0)
                    bind(v3_o, 0)
                    tir.reads([A_shared[v0_2, v1_2, 0:16, 0:16]])
                    tir.writes([A_shared_wmma.matrix_a[v0_2, v1_2, 0:16, 0:16]])
                    A_2 = match_buffer(A_shared[v0_2, v1_2, 0:16, 0:16])
                    C_4 = match_buffer(A_shared_wmma.matrix_a[v0_2, v1_2, 0:16, 0:16])
                    @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_a float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), A_3: Pointer(shared float16), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                }
                for (ax0_1: int32, 0, 4) {
                  block([1024, 1024, 1, 1], "B_shared_wmma.matrix_b_o") as [v0_3, v1_3, v2_o_1, v3_o_1] {
                    bind(v0_3, ((kk_0*2) + kk_1))
                    bind(v1_3, (((jj_0*16) + (jj_1*4)) + ax0_1))
                    bind(v2_o_1, 0)
                    bind(v3_o_1, 0)
                    tir.reads([B_shared[v0_3, v1_3, 0:16, 0:16]])
                    tir.writes([B_shared_wmma.matrix_b[v0_3, v1_3, 0:16, 0:16]])
                    A_4 = match_buffer(B_shared[v0_3, v1_3, 0:16, 0:16])
                    C_6 = match_buffer(B_shared_wmma.matrix_b[v0_3, v1_3, 0:16, 0:16])
                    @tir.tvm_load_matrix_sync(C_7: Pointer(wmma.matrix_b float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_3: int32, C_s0_2: int32), 16)*floordiv(C_s0_2, 16)) + floordiv(floormod(elem_offset_3, C_s0_2), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), A_5: Pointer(shared float16), elem_offset_4: int32, (A_s0_1: int32*16), 1, dtype=handle), A_s0_1, "row_major", dtype=handle)
                }
                for (ii_2: int32, 0, 8) {
                  for (jj_2: int32, 0, 4) {
                    block([1024, 1024, tir.reduce_axis(0, 1024), 1, 1, tir.reduce_axis(0, 1)], "B_update_o") as [vii_1, vjj_1, vkk, vi_o_1, vj_o_1, vk_o] {
                      bind(vii_1, (((ii_0*8) + (ii_1*8)) + ii_2))
                      bind(vjj_1, (((jj_0*16) + (jj_1*4)) + jj_2))
                      bind(vkk, ((kk_0*2) + kk_1))
                      bind(vi_o_1, 0)
                      bind(vj_o_1, 0)
                      bind(vk_o, 0)
                      tir.reads([C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16], A_shared_wmma.matrix_a[vii_1, vkk, 0:16, 0:16], B_shared_wmma.matrix_b[vkk, vjj_1, 0:16, 0:16]])
                      tir.writes([C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16]])
                      A_6 = match_buffer(A_shared_wmma.matrix_a[vii_1, vkk, 0:16, 0:16])
                      B_2 = match_buffer(B_shared_wmma.matrix_b[vkk, vjj_1, 0:16, 0:16])
                      C_8 = match_buffer(C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16])
                      @tir.tvm_mma_sync(C_9: Pointer(wmma.accumulator float16), ((floordiv(floordiv(elem_offset_5: int32, C_s0_3: int32), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), A_7: Pointer(wmma.matrix_a float16), ((floordiv(floordiv(elem_offset_6: int32, A_s0_2: int32), 16)*floordiv(A_s0_2, 16)) + floordiv(floormod(elem_offset_6, A_s0_2), 16)), B_3: Pointer(wmma.matrix_b float16), ((floordiv(floordiv(elem_offset_7: int32, B_s0: int32), 16)*floordiv(B_s0, 16)) + floordiv(floormod(elem_offset_7, B_s0), 16)), C_9, ((floordiv(floordiv(elem_offset_5, C_s0_3), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), dtype=handle)
                  }
                }
              }
            }
            for (ax0_2: int32, 0, 8) {
              for (ax1: int32, 0, 4) {
                block([1024, 1024, 1, 1], "C_wmma.accumulator_o") as [v0_4, v1_4, v2_o_2, v3_o_2] {
                  bind(v0_4, ((ii_0*8) + ax0_2))
                  bind(v1_4, (((jj_0*16) + (jj_1*4)) + ax1))
                  bind(v2_o_2, 0)
                  bind(v3_o_2, 0)
                  tir.reads([C_wmma.accumulator[v0_4, v1_4, 0:16, 0:16]])
                  tir.writes([C[v0_4, v1_4, 0:16, 0:16]])
                  A_8 = match_buffer(C_wmma.accumulator[v0_4, v1_4, 0:16, 0:16])
                  C_10 = match_buffer(C[v0_4, v1_4, 0:16, 0:16])
                  @tir.tvm_store_matrix_sync(A_9: Pointer(wmma.accumulator float16), 16, 16, 16, ((floordiv(floordiv(elem_offset_8: int32, A_s0_3: int32), 16)*floordiv(A_s0_3, 16)) + floordiv(floormod(elem_offset_8, A_s0_3), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), C_11: Pointer(global float16), elem_offset_9: int32, (C_s0_4: int32*16), 2, dtype=handle), C_s0_4, "row_major", dtype=handle)
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}