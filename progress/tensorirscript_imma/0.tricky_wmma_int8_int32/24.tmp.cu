#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_shared[16384];
  __shared__ signed char B_shared[4096];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> B_shared_wmma_matrix_b[4];
  for (int ii_2_init = 0; ii_2_init < 4; ++ii_2_init) {
    for (int jj_2_init = 0; jj_2_init < 4; ++jj_2_init) {
      nvcuda::wmma::fill_fragment(C_wmma_accumulator[((ii_2_init * 4) + jj_2_init)], 0.000000e+00f);
    }
  }
  for (int kk_0 = 0; kk_0 < 256; ++kk_0) {
    __syncthreads();
    for (int ax0_ax1_ax2_ax3_fused_2 = 0; ax0_ax1_ax2_ax3_fused_2 < 8; ++ax0_ax1_ax2_ax3_fused_2) {
      *(int4*)(A_shared + (((((int)threadIdx.y) * 4096) + (ax0_ax1_ax2_ax3_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A + ((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + ((ax0_ax1_ax2_ax3_fused_2 >> 1) * 262144)) + (kk_0 * 1024)) + ((ax0_ax1_ax2_ax3_fused_2 & 1) * 512)) + (((int)threadIdx.x) * 16)));
    }
    for (int ax0_ax1_ax2_ax3_fused_2_1 = 0; ax0_ax1_ax2_ax3_fused_2_1 < 2; ++ax0_ax1_ax2_ax3_fused_2_1) {
      *(int4*)(B_shared + (((((int)threadIdx.y) * 1024) + (ax0_ax1_ax2_ax3_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B + ((((((((int)blockIdx.z) * 33554432) + (((int)blockIdx.x) * 1048576)) + (((int)threadIdx.y) * 262144)) + (kk_0 * 1024)) + (ax0_ax1_ax2_ax3_fused_2_1 * 512)) + (((int)threadIdx.x) * 16)));
    }
    __syncthreads();
    for (int kk_1 = 0; kk_1 < 4; ++kk_1) {
      for (int ax0 = 0; ax0 < 4; ++ax0) {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0], (&(A_shared[(((((int)threadIdx.y) * 4096) + (ax0 * 1024)) + (kk_1 * 256))])), 16);
      }
      for (int ax0_1 = 0; ax0_1 < 4; ++ax0_1) {
        nvcuda::wmma::load_matrix_sync(B_shared_wmma_matrix_b[ax0_1], (&(B_shared[((ax0_1 * 1024) + (kk_1 * 256))])), 16);
      }
      for (int ii_2 = 0; ii_2 < 4; ++ii_2) {
        for (int jj_2 = 0; jj_2 < 4; ++jj_2) {
          nvcuda::wmma::mma_sync(C_wmma_accumulator[((ii_2 * 4) + jj_2)], A_shared_wmma_matrix_a[ii_2], B_shared_wmma_matrix_b[jj_2], C_wmma_accumulator[((ii_2 * 4) + jj_2)]);
        }
      }
    }
  }
  for (int ax0_2 = 0; ax0_2 < 4; ++ax0_2) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_2 * 262144)) + (((int)blockIdx.z) * 32768)) + (((int)blockIdx.x) * 1024)) + (ax1 * 256))])), C_wmma_accumulator[((ax0_2 * 4) + ax1)], 16, nvcuda::wmma::mem_row_major);
    }
  }
}

