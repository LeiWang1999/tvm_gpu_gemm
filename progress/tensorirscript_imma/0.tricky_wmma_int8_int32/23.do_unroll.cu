#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [1024, 1024, 16, 16], []),
             B: Buffer(B_1: Pointer(global int8), int8, [1024, 1024, 16, 16], []),
             C: Buffer(C_1: Pointer(global int32), int32, [1024, 1024, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    C_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
    for (ii_0: int32, 0, 64) "thread_binding" {
      for (jj_0_0: int32, 0, 8) "thread_binding" {
        for (jj_0_1: int32, 0, 32) "thread_binding" {
          for (ii_1: int32, 0, 4) "thread_binding" {
            for (jj_1: int32, 0, 1) "thread_binding" {
              for (ii_2_init: int32, 0, 4) {
                for (jj_2_init: int32, 0, 4) {
                  block([1024, 1024, 1, 1], "B_init_o") as [vii, vjj, vi_o, vj_o] {
                    bind(vii, (((ii_0*16) + (ii_1*4)) + ii_2_init))
                    bind(vjj, ((((jj_0_0*128) + (jj_0_1*4)) + (jj_1*4)) + jj_2_init))
                    bind(vi_o, 0)
                    bind(vj_o, 0)
                    tir.reads([])
                    tir.writes([C_wmma.accumulator[vii, vjj, 0:16, 0:16]])
                    C_2 = match_buffer(C_wmma.accumulator[vii, vjj, 0:16, 0:16])
                    @tir.tvm_fill_fragment(C_3: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
                }
              }
              for (kk_0: int32, 0, 256) {
                for (ax0_ax1_ax2_ax3_fused_0: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1: int32, 0, 1) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2: int32, 0, 8) {
                      for (ax0_ax1_ax2_ax3_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4: int32, 0, 16) "vectorized" {
                          block([1024, 1024, 16, 16], "A_shared") as [v0, v1, v2, v3] {
                            bind(v0, ((ii_0*16) + floordiv((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*4096)) + (ax0_ax1_ax2_ax3_fused_2*512)) + (ax0_ax1_ax2_ax3_fused_3*16)) + ax0_ax1_ax2_ax3_fused_4), 1024)))
                            bind(v1, ((kk_0*4) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*4096)) + (ax0_ax1_ax2_ax3_fused_2*512)) + (ax0_ax1_ax2_ax3_fused_3*16)) + ax0_ax1_ax2_ax3_fused_4), 1024), 256)))
                            bind(v2, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*4096)) + (ax0_ax1_ax2_ax3_fused_2*512)) + (ax0_ax1_ax2_ax3_fused_3*16)) + ax0_ax1_ax2_ax3_fused_4), 256), 16))
                            bind(v3, floormod((((((ax0_ax1_ax2_ax3_fused_0*4096) + (ax0_ax1_ax2_ax3_fused_1*4096)) + (ax0_ax1_ax2_ax3_fused_2*512)) + (ax0_ax1_ax2_ax3_fused_3*16)) + ax0_ax1_ax2_ax3_fused_4), 16))
                            tir.reads([A[v0, v1, v2, v3]])
                            tir.writes([A_shared[v0, v1, v2, v3]])
                            A_shared[v0, v1, v2, v3] = A[v0, v1, v2, v3]
                        }
                      }
                    }
                  }
                }
                for (ax0_ax1_ax2_ax3_fused_0_1: int32, 0, 4) "thread_binding" {
                  for (ax0_ax1_ax2_ax3_fused_1_1: int32, 0, 1) "thread_binding" {
                    for (ax0_ax1_ax2_ax3_fused_2_1: int32, 0, 2) {
                      for (ax0_ax1_ax2_ax3_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_ax2_ax3_fused_4_1: int32, 0, 16) "vectorized" {
                          block([1024, 1024, 16, 16], "B_shared") as [v0_1, v1_1, v2_1, v3_1] {
                            bind(v0_1, (((jj_0_0*128) + (jj_0_1*4)) + floordiv((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*1024)) + (ax0_ax1_ax2_ax3_fused_2_1*512)) + (ax0_ax1_ax2_ax3_fused_3_1*16)) + ax0_ax1_ax2_ax3_fused_4_1), 1024)))
                            bind(v1_1, ((kk_0*4) + floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*1024)) + (ax0_ax1_ax2_ax3_fused_2_1*512)) + (ax0_ax1_ax2_ax3_fused_3_1*16)) + ax0_ax1_ax2_ax3_fused_4_1), 1024), 256)))
                            bind(v2_1, floordiv(floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*1024)) + (ax0_ax1_ax2_ax3_fused_2_1*512)) + (ax0_ax1_ax2_ax3_fused_3_1*16)) + ax0_ax1_ax2_ax3_fused_4_1), 256), 16))
                            bind(v3_1, floormod((((((ax0_ax1_ax2_ax3_fused_0_1*1024) + (ax0_ax1_ax2_ax3_fused_1_1*1024)) + (ax0_ax1_ax2_ax3_fused_2_1*512)) + (ax0_ax1_ax2_ax3_fused_3_1*16)) + ax0_ax1_ax2_ax3_fused_4_1), 16))
                            tir.reads([B[v0_1, v1_1, v2_1, v3_1]])
                            tir.writes([B_shared[v0_1, v1_1, v2_1, v3_1]])
                            B_shared[v0_1, v1_1, v2_1, v3_1] = B[v0_1, v1_1, v2_1, v3_1]
                        }
                      }
                    }
                  }
                }
                for (kk_1: int32, 0, 4) {
                  for (ax0: int32, 0, 4) {
                    block([1024, 1024, 1, 1], "A_shared_wmma.matrix_a_o") as [v0_2, v1_2, v2_o, v3_o] {
                      bind(v0_2, (((ii_0*16) + (ii_1*4)) + ax0))
                      bind(v1_2, ((kk_0*4) + kk_1))
                      bind(v2_o, 0)
                      bind(v3_o, 0)
                      tir.reads([A_shared[v0_2, v1_2, 0:16, 0:16]])
                      tir.writes([A_shared_wmma.matrix_a[v0_2, v1_2, 0:16, 0:16]])
                      A_2 = match_buffer(A_shared[v0_2, v1_2, 0:16, 0:16])
                      C_4 = match_buffer(A_shared_wmma.matrix_a[v0_2, v1_2, 0:16, 0:16])
                      @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_a int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_3: Pointer(shared int8), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                  }
                  for (ax0_1: int32, 0, 4) {
                    block([1024, 1024, 1, 1], "B_shared_wmma.matrix_b_o") as [v0_3, v1_3, v2_o_1, v3_o_1] {
                      bind(v0_3, (((jj_0_0*128) + (jj_0_1*4)) + ax0_1))
                      bind(v1_3, ((kk_0*4) + kk_1))
                      bind(v2_o_1, 0)
                      bind(v3_o_1, 0)
                      tir.reads([B_shared[v0_3, v1_3, 0:16, 0:16]])
                      tir.writes([B_shared_wmma.matrix_b[v0_3, v1_3, 0:16, 0:16]])
                      A_4 = match_buffer(B_shared[v0_3, v1_3, 0:16, 0:16])
                      C_6 = match_buffer(B_shared_wmma.matrix_b[v0_3, v1_3, 0:16, 0:16])
                      @tir.tvm_load_matrix_sync(C_7: Pointer(wmma.matrix_b int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_3: int32, C_s0_2: int32), 16)*floordiv(C_s0_2, 16)) + floordiv(floormod(elem_offset_3, C_s0_2), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_5: Pointer(shared int8), elem_offset_4: int32, (A_s0_1: int32*16), 1, dtype=handle), A_s0_1, "col_major", dtype=handle)
                  }
                  for (ii_2: int32, 0, 4) {
                    for (jj_2: int32, 0, 4) {
                      block([1024, 1024, tir.reduce_axis(0, 1024), 1, 1, tir.reduce_axis(0, 1)], "B_update_o") as [vii_1, vjj_1, vkk, vi_o_1, vj_o_1, vk_o] {
                        bind(vii_1, (((ii_0*16) + (ii_1*4)) + ii_2))
                        bind(vjj_1, ((((jj_0_0*128) + (jj_0_1*4)) + (jj_1*4)) + jj_2))
                        bind(vkk, ((kk_0*4) + kk_1))
                        bind(vi_o_1, 0)
                        bind(vj_o_1, 0)
                        bind(vk_o, 0)
                        tir.reads([C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16], A_shared_wmma.matrix_a[vii_1, vkk, 0:16, 0:16], B_shared_wmma.matrix_b[vjj_1, vkk, 0:16, 0:16]])
                        tir.writes([C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16]])
                        A_6 = match_buffer(A_shared_wmma.matrix_a[vii_1, vkk, 0:16, 0:16])
                        B_2 = match_buffer(B_shared_wmma.matrix_b[vjj_1, vkk, 0:16, 0:16])
                        C_8 = match_buffer(C_wmma.accumulator[vii_1, vjj_1, 0:16, 0:16])
                        @tir.tvm_mma_sync(C_9: Pointer(wmma.accumulator int32), ((floordiv(floordiv(elem_offset_5: int32, C_s0_3: int32), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), A_7: Pointer(wmma.matrix_a int8), ((floordiv(floordiv(elem_offset_6: int32, A_s0_2: int32), 16)*floordiv(A_s0_2, 16)) + floordiv(floormod(elem_offset_6, A_s0_2), 16)), B_3: Pointer(wmma.matrix_b int8), ((floordiv(floordiv(elem_offset_7: int32, B_s0: int32), 16)*floordiv(B_s0, 16)) + floordiv(floormod(elem_offset_7, B_s0), 16)), C_9, ((floordiv(floordiv(elem_offset_5, C_s0_3), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), dtype=handle)
                    }
                  }
                }
              }
              for (ax0_2: int32, 0, 4) {
                for (ax1: int32, 0, 4) {
                  block([1024, 1024, 1, 1], "C_wmma.accumulator_o") as [v0_4, v1_4, v2_o_2, v3_o_2] {
                    bind(v0_4, (((ii_0*16) + (ii_1*4)) + ax0_2))
                    bind(v1_4, (((jj_0_0*128) + (jj_0_1*4)) + ax1))
                    bind(v2_o_2, 0)
                    bind(v3_o_2, 0)
                    tir.reads([C_wmma.accumulator[v0_4, v1_4, 0:16, 0:16]])
                    tir.writes([C[v0_4, v1_4, 0:16, 0:16]])
                    A_8 = match_buffer(C_wmma.accumulator[v0_4, v1_4, 0:16, 0:16])
                    C_10 = match_buffer(C[v0_4, v1_4, 0:16, 0:16])
                    @tir.tvm_store_matrix_sync(A_9: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset_8: int32, A_s0_3: int32), 16)*floordiv(A_s0_3, 16)) + floordiv(floormod(elem_offset_8, A_s0_3), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int32), C_11: Pointer(global int32), elem_offset_9: int32, (C_s0_4: int32*16), 2, dtype=handle), C_s0_4, "row_major", dtype=handle)
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}