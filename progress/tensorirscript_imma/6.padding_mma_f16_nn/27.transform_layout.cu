#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float16), float16, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[16384, 16384])
    B_shared = alloc_buffer(float16[16384, 16384])
    A_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    B_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    C_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    for (i_0: int32, 0, 64) "thread_binding" {
      for (j_0: int32, 0, 128) "thread_binding" {
        for (i_1_0: int32, 0, 2) "thread_binding" {
          for (j_1_0: int32, 0, 4) "thread_binding" {
            for (i_1_1_0_init: int32, 0, 8) {
              for (j_1_1_0_init: int32, 0, 2) {
                for (i_1_1_1_init: int32, 0, 16) {
                  for (j_1_1_1_init: int32, 0, 16) {
                    block([16384, 16384], "B_init") as [vi, vj] {
                      bind(vi, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0_init*16)) + i_1_1_1_init))
                      bind(vj, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0_init*16)) + j_1_1_1_init))
                      tir.reads([])
                      tir.writes([C_warp[floordiv(vi, 16), floordiv(vj, 16), ((floormod(vi, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi, 16), 8)*2)) + floormod(vj, 2))]])
                      C_warp[floordiv(vi, 16), floordiv(vj, 16), ((floormod(vi, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi, 16), 8)*2)) + floormod(vj, 2))] = 0f16
                  }
                }
              }
            }
            for (k_0: int32, 0, 512) {
              for (ax0_ax1_fused_0: int32, 0, 4) {
                for (ax0_ax1_fused_1: int32, 0, 2) {
                  for (ax0_ax1_fused_2: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_fused_4: int32, 0, 8) "vectorized" {
                        block([16384, 16384], "A_shared") as [v0, v1] {
                          bind(v0, ((i_0*256) + floordiv((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*1024)) + (ax0_ax1_fused_2*256)) + (ax0_ax1_fused_3*8)) + ax0_ax1_fused_4), 32)))
                          bind(v1, ((k_0*32) + floormod((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*1024)) + (ax0_ax1_fused_2*256)) + (ax0_ax1_fused_3*8)) + ax0_ax1_fused_4), 32)))
                          tir.reads([A[v0, v1]])
                          tir.writes([A_shared[v0, v1]])
                          tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                          A_shared[v0, v1] = A[v0, v1]
                      }
                    }
                  }
                }
              }
              for (ax0_ax1_fused_0_1: int32, 0, 2) {
                for (ax0_ax1_fused_1_1: int32, 0, 2) {
                  for (ax0_ax1_fused_2_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_3_1: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_fused_4_1: int32, 0, 8) "vectorized" {
                        block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                          bind(v0_1, ((k_0*32) + floordiv((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*1024)) + (ax0_ax1_fused_2_1*256)) + (ax0_ax1_fused_3_1*8)) + ax0_ax1_fused_4_1), 128)))
                          bind(v1_1, ((j_0*128) + floormod((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*1024)) + (ax0_ax1_fused_2_1*256)) + (ax0_ax1_fused_3_1*8)) + ax0_ax1_fused_4_1), 128)))
                          tir.reads([B[v0_1, v1_1]])
                          tir.writes([B_shared[v0_1, v1_1]])
                          tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                          B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                      }
                    }
                  }
                }
              }
              for (i_1_1_0: int32, 0, 8) {
                for (j_1_1_0: int32, 0, 2) {
                  for (k_1_0: int32, 0, 2) {
                    for (ax0: int32, 0, 16) {
                      for (ax1: int32, 0, 16) {
                        block([16384, 16384], "A_shared_warp") as [v0_2, v1_2] {
                          bind(v0_2, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0*16)) + ax0))
                          bind(v1_2, (((k_0*32) + (k_1_0*16)) + ax1))
                          tir.reads([A_shared[v0_2, v1_2]])
                          tir.writes([A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 8), 2)), (((floordiv(floormod(v1_2, 16), 8)*4) + (floordiv(floormod(v0_2, 16), 8)*2)) + floormod(v1_2, 2))]])
                          A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 8), 2)), (((floordiv(floormod(v1_2, 16), 8)*4) + (floordiv(floormod(v0_2, 16), 8)*2)) + floormod(v1_2, 2))] = A_shared[v0_2, v1_2]
                      }
                    }
                    for (ax0_1: int32, 0, 16) {
                      for (ax1_1: int32, 0, 16) {
                        block([16384, 16384], "B_shared_warp") as [v0_3, v1_3] {
                          bind(v0_3, (((k_0*32) + (k_1_0*16)) + ax0_1))
                          bind(v1_3, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0*16)) + ax1_1))
                          tir.reads([B_shared[v0_3, v1_3]])
                          tir.writes([B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), ((floormod(v0_3, 8)*4) + floordiv(floormod(v1_3, 8), 2)), (((floordiv(floormod(v1_3, 16), 8)*4) + (floordiv(floormod(v0_3, 16), 8)*2)) + floormod(v1_3, 2))]])
                          B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), ((floormod(v0_3, 8)*4) + floordiv(floormod(v1_3, 8), 2)), (((floordiv(floormod(v1_3, 16), 8)*4) + (floordiv(floormod(v0_3, 16), 8)*2)) + floormod(v1_3, 2))] = B_shared[v0_3, v1_3]
                      }
                    }
                    for (i_1_1_1: int32, 0, 16) {
                      for (j_1_1_1: int32, 0, 16) {
                        for (k_1_1: int32, 0, 16) {
                          block([16384, 16384, tir.reduce_axis(0, 16384)], "B_update") as [vi_1, vj_1, vk] {
                            bind(vi_1, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0*16)) + i_1_1_1))
                            bind(vj_1, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0*16)) + j_1_1_1))
                            bind(vk, (((k_0*32) + (k_1_0*16)) + k_1_1))
                            tir.reads([C_warp[floordiv(vi_1, 16), floordiv(vj_1, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj_1, 2))], A_shared_warp[floordiv(vi_1, 16), floordiv(vk, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vk, 8), 2)), (((floordiv(floormod(vk, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vk, 2))], B_shared_warp[floordiv(vk, 16), floordiv(vj_1, 16), ((floormod(vk, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vk, 16), 8)*2)) + floormod(vj_1, 2))]])
                            tir.writes([C_warp[floordiv(vi_1, 16), floordiv(vj_1, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj_1, 2))]])
                            C_warp[floordiv(vi_1, 16), floordiv(vj_1, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj_1, 2))] = (C_warp[floordiv(vi_1, 16), floordiv(vj_1, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj_1, 2))] + (A_shared_warp[floordiv(vi_1, 16), floordiv(vk, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vk, 8), 2)), (((floordiv(floormod(vk, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vk, 2))]*B_shared_warp[floordiv(vk, 16), floordiv(vj_1, 16), ((floormod(vk, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vk, 16), 8)*2)) + floormod(vj_1, 2))]))
                        }
                      }
                    }
                  }
                }
              }
            }
            for (ax0_0: int32, 0, 8) {
              for (ax1_0: int32, 0, 2) {
                for (ax0_1_1: int32, 0, 16) {
                  for (ax1_1_1: int32, 0, 16) {
                    block([16384, 16384], "C_warp") as [v0_4, v1_4] {
                      bind(v0_4, ((((i_0*256) + (i_1_0*128)) + (ax0_0*16)) + ax0_1_1))
                      bind(v1_4, ((((j_0*128) + (j_1_0*32)) + (ax1_0*16)) + ax1_1_1))
                      tir.reads([C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), ((floormod(v0_4, 8)*4) + floordiv(floormod(v1_4, 8), 2)), (((floordiv(floormod(v1_4, 16), 8)*4) + (floordiv(floormod(v0_4, 16), 8)*2)) + floormod(v1_4, 2))]])
                      tir.writes([C[v0_4, v1_4]])
                      C[v0_4, v1_4] = C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), ((floormod(v0_4, 8)*4) + floordiv(floormod(v1_4, 8), 2)), (((floordiv(floormod(v1_4, 16), 8)*4) + (floordiv(floormod(v0_4, 16), 8)*2)) + floormod(v1_4, 2))]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}