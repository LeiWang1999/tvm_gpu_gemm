#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float16), float16, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    for (i_0: int32, 0, 64) "thread_binding" {
      for (j_0: int32, 0, 128) "thread_binding" {
        for (k_0: int32, 0, 512) {
          for (i_1: int32, 0, 256) {
            for (j_1: int32, 0, 128) {
              for (k_1: int32, 0, 32) {
                block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                  bind(vi, ((i_0*256) + i_1))
                  bind(vj, ((j_0*128) + j_1))
                  bind(vk, ((k_0*32) + k_1))
                  tir.reads([A[vi, vk], B[vk, vj]])
                  tir.writes([C[vi, vj]])
                  with init() {
                    C[vi, vj] = 0f16
                  }
                  C[vi, vj] = (C[vi, vj] + (A[vi, vk]*B[vk, vj]))
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}