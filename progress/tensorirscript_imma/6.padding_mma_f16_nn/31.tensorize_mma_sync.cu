#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float16), float16, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[16384, 16384])
    B_shared = alloc_buffer(float16[16384, 16384])
    A_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    B_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    C_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    for (i_0: int32, 0, 64) "thread_binding" {
      for (j_0: int32, 0, 128) "thread_binding" {
        for (i_1_0: int32, 0, 2) "thread_binding" {
          for (j_1_0: int32, 0, 4) "thread_binding" {
            for (i_1_1_0_init: int32, 0, 8) {
              for (j_1_1_0_init: int32, 0, 2) {
                for (i_1_1_1_init: int32, 0, 16) {
                  for (j_1_1_1_init: int32, 0, 16) {
                    block([16384, 16384], "B_init") as [vi, vj] {
                      bind(vi, ((((i_0*256) + (i_1_0*128)) + (i_1_1_0_init*16)) + i_1_1_1_init))
                      bind(vj, ((((j_0*128) + (j_1_0*32)) + (j_1_1_0_init*16)) + j_1_1_1_init))
                      tir.reads([])
                      tir.writes([C_warp[floordiv(vi, 16), floordiv(vj, 16), ((floormod(vi, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi, 16), 8)*2)) + floormod(vj, 2))]])
                      C_warp[floordiv(vi, 16), floordiv(vj, 16), ((floormod(vi, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi, 16), 8)*2)) + floormod(vj, 2))] = 0f16
                  }
                }
              }
            }
            for (k_0: int32, 0, 512) {
              for (ax0_ax1_fused_0: int32, 0, 4) {
                for (ax0_ax1_fused_1: int32, 0, 2) {
                  for (ax0_ax1_fused_2: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_fused_4: int32, 0, 8) "vectorized" {
                        block([16384, 16384], "A_shared") as [v0, v1] {
                          bind(v0, ((i_0*256) + floordiv((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*1024)) + (ax0_ax1_fused_2*256)) + (ax0_ax1_fused_3*8)) + ax0_ax1_fused_4), 32)))
                          bind(v1, ((k_0*32) + floormod((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*1024)) + (ax0_ax1_fused_2*256)) + (ax0_ax1_fused_3*8)) + ax0_ax1_fused_4), 32)))
                          tir.reads([A[v0, v1]])
                          tir.writes([A_shared[v0, v1]])
                          tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                          A_shared[v0, v1] = A[v0, v1]
                      }
                    }
                  }
                }
              }
              for (ax0_ax1_fused_0_1: int32, 0, 2) {
                for (ax0_ax1_fused_1_1: int32, 0, 2) {
                  for (ax0_ax1_fused_2_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_3_1: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_fused_4_1: int32, 0, 8) "vectorized" {
                        block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                          bind(v0_1, ((k_0*32) + floordiv((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*1024)) + (ax0_ax1_fused_2_1*256)) + (ax0_ax1_fused_3_1*8)) + ax0_ax1_fused_4_1), 128)))
                          bind(v1_1, ((j_0*128) + floormod((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*1024)) + (ax0_ax1_fused_2_1*256)) + (ax0_ax1_fused_3_1*8)) + ax0_ax1_fused_4_1), 128)))
                          tir.reads([B[v0_1, v1_1]])
                          tir.writes([B_shared[v0_1, v1_1]])
                          tir.attrs({"buffer_dim_align": [[0, 0, 32, 8]]})
                          B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                      }
                    }
                  }
                }
              }
              for (i_1_1_0: int32, 0, 8) {
                for (j_1_1_0: int32, 0, 2) {
                  for (k_1_0: int32, 0, 2) {
                    block([1024, 1024], "A_shared_warp_o") as [v0_o, v1_o] {
                      bind(v0_o, (((i_0*16) + (i_1_0*8)) + i_1_1_0))
                      bind(v1_o, ((k_0*2) + k_1_0))
                      tir.reads([A_shared[(v0_o*16):((v0_o*16) + 16), (v1_o*16):((v1_o*16) + 16)]])
                      tir.writes([A_shared_warp[v0_o, v1_o, 0:32, 0:8]])
                      warp = match_buffer(A_shared_warp[v0_o, v1_o, 0:32, 0:8])
                      shared = match_buffer(A_shared[(v0_o*16):((v0_o*16) + 16), (v1_o*16):((v1_o*16) + 16)])
                      attr [IterVar(tx: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                      @tir.ptx_ldmatrix(False, 4, ".b16", warp_1: Pointer(warp float16), (elem_offset: int32 + (8*tx)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_1: Pointer(shared float16), elem_offset_1: int32, (shared_s0: int32*16), 1, dtype=handle), ((shared_s0*floormod(tx, 16)) + (8*floordiv(tx, 16))), dtype=float16)
                    block([1024, 1024], "B_shared_warp_o") as [v0_o_1, v1_o_1] {
                      bind(v0_o_1, ((k_0*2) + k_1_0))
                      bind(v1_o_1, (((j_0*8) + (j_1_0*2)) + j_1_1_0))
                      tir.reads([B_shared[(v0_o_1*16):((v0_o_1*16) + 16), (v1_o_1*16):((v1_o_1*16) + 16)]])
                      tir.writes([B_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8]])
                      warp_2 = match_buffer(B_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8])
                      shared_2 = match_buffer(B_shared[(v0_o_1*16):((v0_o_1*16) + 16), (v1_o_1*16):((v1_o_1*16) + 16)])
                      attr [IterVar(tx_1: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                      @tir.ptx_ldmatrix(True, 4, ".b16", warp_3: Pointer(warp float16), (elem_offset_2: int32 + (8*tx_1)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_3: Pointer(shared float16), elem_offset_3: int32, (shared_s0_1: int32*16), 1, dtype=handle), ((shared_s0_1*floormod(tx_1, 16)) + (8*floordiv(tx_1, 16))), dtype=float16)
                    block([1024, 1024, tir.reduce_axis(0, 1024)], "B_update_o") as [vi_o, vj_o, vk_o] {
                      bind(vi_o, (((i_0*16) + (i_1_0*8)) + i_1_1_0))
                      bind(vj_o, (((j_0*8) + (j_1_0*2)) + j_1_1_0))
                      bind(vk_o, ((k_0*2) + k_1_0))
                      tir.reads([C_warp[vi_o, vj_o, 0:32, 0:8], A_shared_warp[vi_o, vk_o, 0:32, 0:8], B_shared_warp[vk_o, vj_o, 0:32, 0:8]])
                      tir.writes([C_warp[vi_o, vj_o, 0:32, 0:8]])
                      A_2 = match_buffer(A_shared_warp[vi_o, vk_o, 0:32, 0:8])
                      B_2 = match_buffer(B_shared_warp[vk_o, vj_o, 0:32, 0:8])
                      C_2 = match_buffer(C_warp[vi_o, vj_o, 0:32, 0:8])
                      attr [IterVar(tx_2: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32 {
                        @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3: Pointer(warp float16), (elem_offset_4: int32 + (tx_2*8)), B_3: Pointer(warp float16), (elem_offset_5: int32 + (tx_2*8)), C_3: Pointer(warp float16), (elem_offset_6: int32 + (tx_2*8)), False, dtype=float16)
                        @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3, (elem_offset_4 + (tx_2*8)), B_3, ((elem_offset_5 + (tx_2*8)) + floordiv(8, 2)), C_3, ((elem_offset_6 + (tx_2*8)) + floordiv(8, 2)), False, dtype=float16)
                      }
                  }
                }
              }
            }
            for (ax0_0: int32, 0, 8) {
              for (ax1_0: int32, 0, 2) {
                for (ax0_1: int32, 0, 16) {
                  for (ax1_1: int32, 0, 16) {
                    block([16384, 16384], "C_warp") as [v0_2, v1_2] {
                      bind(v0_2, ((((i_0*256) + (i_1_0*128)) + (ax0_0*16)) + ax0_1))
                      bind(v1_2, ((((j_0*128) + (j_1_0*32)) + (ax1_0*16)) + ax1_1))
                      tir.reads([C_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 8), 2)), (((floordiv(floormod(v1_2, 16), 8)*4) + (floordiv(floormod(v0_2, 16), 8)*2)) + floormod(v1_2, 2))]])
                      tir.writes([C[v0_2, v1_2]])
                      C[v0_2, v1_2] = C_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 8), 2)), (((floordiv(floormod(v1_2, 16), 8)*4) + (floordiv(floormod(v0_2, 16), 8)*2)) + floormod(v1_2, 2))]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}