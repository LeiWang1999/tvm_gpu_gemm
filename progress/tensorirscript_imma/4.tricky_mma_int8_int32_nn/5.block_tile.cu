#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16, 8, 16, 32], []),
             B: Buffer(B_1: Pointer(global int8), int8, [8, 16, 32, 16], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16, 16, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(int8[16, 8, 16, 32])
    A_shared_warp = alloc_buffer(int8[16, 8, 16, 32])
    B_shared = alloc_buffer(int8[8, 16, 32, 16])
    B_shared_warp = alloc_buffer(int8[8, 16, 32, 16])
    C_warp = alloc_buffer(int32[16, 16, 16, 16])
     {
      for (ax0: int32, 0, 8) {
        for (ax1: int32, 0, 16) {
          for (ax2: int32, 0, 32) {
            for (ax3: int32, 0, 16) {
              block([8, 16, 32, 16], "B_shared") as [v0, v1, v2, v3] {
                bind(v0, ax0)
                bind(v1, ax1)
                bind(v2, ax2)
                bind(v3, ax3)
                tir.reads([B[v0, v1, v2, v3]])
                tir.writes([B_shared[v0, v1, v2, v3]])
                B_shared[v0, v1, v2, v3] = B[v0, v1, v2, v3]
            }
          }
        }
      }
      for (ax0_1: int32, 0, 16) {
        for (ax1_1: int32, 0, 8) {
          for (ax2_1: int32, 0, 16) {
            for (ax3_1: int32, 0, 32) {
              block([16, 8, 16, 32], "A_shared") as [v0_1, v1_1, v2_1, v3_1] {
                bind(v0_1, ax0_1)
                bind(v1_1, ax1_1)
                bind(v2_1, ax2_1)
                bind(v3_1, ax3_1)
                tir.reads([A[v0_1, v1_1, v2_1, v3_1]])
                tir.writes([A_shared[v0_1, v1_1, v2_1, v3_1]])
                A_shared[v0_1, v1_1, v2_1, v3_1] = A[v0_1, v1_1, v2_1, v3_1]
            }
          }
        }
      }
      for (ax0_2: int32, 0, 16) {
        for (ax1_2: int32, 0, 8) {
          for (ax2_2: int32, 0, 16) {
            for (ax3_2: int32, 0, 32) {
              block([16, 8, 16, 32], "A_shared_warp") as [v0_2, v1_2, v2_2, v3_2] {
                bind(v0_2, ax0_2)
                bind(v1_2, ax1_2)
                bind(v2_2, ax2_2)
                bind(v3_2, ax3_2)
                tir.reads([A_shared[v0_2, v1_2, v2_2, v3_2]])
                tir.writes([A_shared_warp[v0_2, v1_2, v2_2, v3_2]])
                A_shared_warp[v0_2, v1_2, v2_2, v3_2] = A_shared[v0_2, v1_2, v2_2, v3_2]
            }
          }
        }
      }
      for (ax0_3: int32, 0, 8) {
        for (ax1_3: int32, 0, 16) {
          for (ax2_3: int32, 0, 32) {
            for (ax3_3: int32, 0, 16) {
              block([8, 16, 32, 16], "B_shared_warp") as [v0_3, v1_3, v2_3, v3_3] {
                bind(v0_3, ax0_3)
                bind(v1_3, ax1_3)
                bind(v2_3, ax2_3)
                bind(v3_3, ax3_3)
                tir.reads([B_shared[v0_3, v1_3, v2_3, v3_3]])
                tir.writes([B_shared_warp[v0_3, v1_3, v2_3, v3_3]])
                B_shared_warp[v0_3, v1_3, v2_3, v3_3] = B_shared[v0_3, v1_3, v2_3, v3_3]
            }
          }
        }
      }
      for (ii_0: int32, 0, 4) {
        for (jj_0: int32, 0, 1) {
          for (ii_1: int32, 0, 2) {
            for (jj_1: int32, 0, 2) {
              for (kk_0: int32, 0, 4) {
                for (kk_1: int32, 0, 2) {
                  for (ii_2: int32, 0, 2) {
                    for (jj_2: int32, 0, 8) {
                      for (i: int32, 0, 16) {
                        for (j: int32, 0, 16) {
                          for (k: int32, 0, 32) {
                            block([16, 16, tir.reduce_axis(0, 8), 16, 16, tir.reduce_axis(0, 32)], "B") as [vii, vjj, vkk, vi, vj, vk] {
                              bind(vii, (((ii_0*4) + (ii_1*2)) + ii_2))
                              bind(vjj, (((jj_0*16) + (jj_1*8)) + jj_2))
                              bind(vkk, ((kk_0*2) + kk_1))
                              bind(vi, i)
                              bind(vj, j)
                              bind(vk, k)
                              tir.reads([A_shared_warp[vii, vkk, vi, vk], B_shared_warp[vkk, vjj, vk, vj]])
                              tir.writes([C_warp[vii, vjj, vi, vj]])
                              with init() {
                                C_warp[vii, vjj, vi, vj] = 0
                              }
                              C_warp[vii, vjj, vi, vj] = (C_warp[vii, vjj, vi, vj] + (cast(int32, A_shared_warp[vii, vkk, vi, vk])*cast(int32, B_shared_warp[vkk, vjj, vk, vj])))
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_4: int32, 0, 16) {
        for (ax1_4: int32, 0, 16) {
          for (ax2_4: int32, 0, 16) {
            for (ax3_4: int32, 0, 16) {
              block([16, 16, 16, 16], "C_warp") as [v0_4, v1_4, v2_4, v3_4] {
                bind(v0_4, ax0_4)
                bind(v1_4, ax1_4)
                bind(v2_4, ax2_4)
                bind(v3_4, ax3_4)
                tir.reads([C_warp[v0_4, v1_4, v2_4, v3_4]])
                tir.writes([C[v0_4, v1_4, v2_4, v3_4]])
                C[v0_4, v1_4, v2_4, v3_4] = C_warp[v0_4, v1_4, v2_4, v3_4]
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}