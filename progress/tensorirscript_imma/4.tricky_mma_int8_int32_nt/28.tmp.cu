
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  int C_warp[128];
  __shared__ signed char A_shared[16384];
  __shared__ signed char B_shared[4096];
  signed char A_shared_warp[64];
  signed char B_shared_warp[64];
  for (int ii_2_init = 0; ii_2_init < 4; ++ii_2_init) {
    for (int jj_2_init = 0; jj_2_init < 4; ++jj_2_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[((ii_2_init * 32) + (jj_2_init * 8)) + i] = 0.0;}
;
    }
  }

  const int MAX_BLOCK_N = 16;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int kk_0 = 0; kk_0 < 256; ++kk_0) {
    __syncthreads();
    for (int ax0_ax1_ax2_ax3_fused_2 = 0; ax0_ax1_ax2_ax3_fused_2 < 8; ++ax0_ax1_ax2_ax3_fused_2) {
      *(int4*)(A_shared + (((((int)threadIdx.y) * 4096) + (ax0_ax1_ax2_ax3_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A + (((((((((int)blockIdx.x) * 4194304) + (((int)threadIdx.y) * 1048576)) + ((ax0_ax1_ax2_ax3_fused_2 >> 1) * 262144)) + (kk_0 * 1024)) + ((ax0_ax1_ax2_ax3_fused_2 & 1) * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((int)threadIdx.x) >> 4) * 16)));
    }
    for (int ax0_ax1_ax2_ax3_fused_2_1 = 0; ax0_ax1_ax2_ax3_fused_2_1 < 2; ++ax0_ax1_ax2_ax3_fused_2_1) {
      *(int4*)(B_shared + (((((int)threadIdx.y) * 1024) + (ax0_ax1_ax2_ax3_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B + (((((((((int)blockIdx.y) * 1048576) + (((int)threadIdx.y) * 262144)) + (kk_0 * 1024)) + (ax0_ax1_ax2_ax3_fused_2_1 * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + (((((int)threadIdx.x) & 15) >> 3) * 16)));
    }
    __syncthreads();
    for (int kk_1 = 0; kk_1 < 2; ++kk_1) {
      for (int ax0 = 0; ax0 < 4; ++ax0) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[(((((int)threadIdx.y) * 4096) + (ax0 * 1024)) + (kk_1 * 512))])) + (((int)threadIdx.x) * 16)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp + (ax0 * 16)))[0]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 16)))[1]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 16)))[2]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 16)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax0_1 = 0; ax0_1 < 4; ++ax0_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_shared[((ax0_1 * 1024) + (kk_1 * 512))])) + (((int)threadIdx.x) * 16)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 16)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 16)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 16)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 16)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ii_2 = 0; ii_2 < 4; ++ii_2) {
        for (int jj_2 = 0; jj_2 < 4; ++jj_2) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[0]), "=r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[1]), "=r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[2]), "=r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[3])
      : "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[0]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[1]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[2]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[3]), "r"(((unsigned *)(B_shared_warp + (jj_2 * 16)))[0]), "r"(((unsigned *)(B_shared_warp + (jj_2 * 16)))[1]), "r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[0]), "r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[1]), "r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[2]), "r"(((int *)(C_warp + ((ii_2 * 32) + (jj_2 * 8))))[3]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[0]), "=r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[1]), "=r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[2]), "=r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[3])
      : "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[0]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[1]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[2]), "r"(((unsigned *)(A_shared_warp + (ii_2 * 16)))[3]), "r"(((unsigned *)(B_shared_warp + ((jj_2 * 16) + 8)))[0]), "r"(((unsigned *)(B_shared_warp + ((jj_2 * 16) + 8)))[1]), "r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[0]), "r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[1]), "r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[2]), "r"(((int *)(C_warp + (((ii_2 * 32) + (jj_2 * 8)) + 4)))[3]));
  }
        }
      }
    }
  }
  for (int ax0_2 = 0; ax0_2 < 4; ++ax0_2) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      for (int local_id = 0; local_id < 8; ++local_id) {
(&(C[(((((((int)blockIdx.x) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_2 * 262144)) + (((int)blockIdx.y) * 1024)) + (ax1 * 256))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))] = C_warp[((ax0_2 * 32) + (ax1 * 8)) + local_id];
}
;
    }
  }
}

