#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ A_global, signed char* __restrict__ B_global, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_global_shared[2048];
  __shared__ signed char B_global_shared[8192];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> B_global_shared_wmma_matrix_b[8];
  for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 8; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 8) + j_0_2_init)], 0.000000e+00f);
    }
  }
  for (int k_0_0 = 0; k_0_0 < 512; ++k_0_0) {
    __syncthreads();
    *(int4*)(A_global_shared + (((((int)threadIdx.y) * 1024) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A_global + (((((((int)blockIdx.y) * 1048576) + (((int)threadIdx.y) * 524288)) + (((int)threadIdx.z) * 262144)) + (k_0_0 * 512)) + (((int)threadIdx.x) * 16)));
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 4; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
      *(int4*)(B_global_shared + ((((((int)threadIdx.y) * 4096) + (((int)threadIdx.z) * 2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B_global + ((((((((int)blockIdx.z) * 4194304) + (((int)threadIdx.y) * 2097152)) + (((int)threadIdx.z) * 1048576)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 262144)) + (k_0_0 * 512)) + (((int)threadIdx.x) * 16)));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a[ax0_0], (&(A_global_shared[(((((int)threadIdx.y) * 1024) + (ax0_0 * 512)) + (k_0_1 * 256))])), 16);
      }
      for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1) {
        nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b[ax0_0_1], (&(B_global_shared[(((((int)threadIdx.z) * 4096) + (ax0_0_1 * 512)) + (k_0_1 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 8; ++j_0_2) {
          nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 8) + j_0_2)], A_global_shared_wmma_matrix_a[i_0_2], B_global_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 8) + j_0_2)]);
        }
      }
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {
    for (int ax1_0 = 0; ax1_0 < 8; ++ax1_0) {
      nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.y) * 1048576) + (((int)threadIdx.y) * 524288)) + (ax0_0_2 * 262144)) + (((int)blockIdx.z) * 256)) + (((int)threadIdx.z) * 128)) + (ax1_0 * 16))])), C_wmma_accumulator[((ax0_0_2 * 8) + ax1_0)], 16384, nvcuda::wmma::mem_row_major);
    }
  }
}

extern "C" __global__ void __launch_bounds__(8) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {
    *(int4*)(B_global + (((((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (ax0_ax1_fused_2 * 128)) + (((int)threadIdx.x) * 16)) >> 18) * 262144) + (((((((int)blockIdx.x) * 32) + (ax0_ax1_fused_2 * 8)) + ((int)threadIdx.x)) & 1023) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + ((((int)blockIdx.x) >> 5) * 16))) = *(int4*)(B + ((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (ax0_ax1_fused_2 * 128)) + (((int)threadIdx.x) * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(8) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {
    *(int4*)(A_global + (((((((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (ax0_ax1_fused_2 * 128)) + (((int)threadIdx.x) * 16)) >> 18) * 262144) + (((((((int)blockIdx.x) * 32) + (ax0_ax1_fused_2 * 8)) + ((int)threadIdx.x)) & 1023) * 256)) + ((((int)blockIdx.y) & 7) * 32)) + ((((int)blockIdx.x) >> 5) * 16))) = *(int4*)(A + ((((((int)blockIdx.y) * 32768) + (((int)blockIdx.x) * 512)) + (ax0_ax1_fused_2 * 128)) + (((int)threadIdx.x) * 16)));
  }
}

