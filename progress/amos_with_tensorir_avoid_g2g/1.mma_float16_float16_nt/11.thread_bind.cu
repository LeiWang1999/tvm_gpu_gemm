#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float16), float16, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    A_shared_warp = alloc_buffer(float16[1024, 1024, 16, 16])
    B_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    B_shared_warp = alloc_buffer(float16[1024, 1024, 16, 16])
    C_warp = alloc_buffer(float16[1024, 1024, 16, 16])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_shared") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_shared") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (ax0_2: int32, 0, 16384) {
        for (ax1_2: int32, 0, 16384) {
          block([16384, 16384], "A_shared_warp") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([A_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
            tir.writes([A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
            A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "B_shared_warp") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([B_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
            tir.writes([B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
            B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
        }
      }
      for (i_0_0: int32, 0, 128) "thread_binding" {
        for (j_0_0_0: int32, 0, 4) "thread_binding" {
          for (j_0_0_1: int32, 0, 16) "thread_binding" {
            for (i_0_1: int32, 0, 4) "thread_binding" {
              for (j_0_1: int32, 0, 1) "thread_binding" {
                for (k_0_0: int32, 0, 512) {
                  for (k_0_1: int32, 0, 2) {
                    for (i_0_2: int32, 0, 2) {
                      for (j_0_2: int32, 0, 16) {
                        for (i_1: int32, 0, 16) {
                          for (j_1: int32, 0, 16) {
                            for (k_1: int32, 0, 16) {
                              block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                                bind(vi, ((((i_0_0*128) + (i_0_1*32)) + (i_0_2*16)) + i_1))
                                bind(vj, (((((j_0_0_0*4096) + (j_0_0_1*256)) + (j_0_1*256)) + (j_0_2*16)) + j_1))
                                bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                                tir.reads([A_shared_warp[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)], B_shared_warp[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)]])
                                tir.writes([C_warp[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                                with init() {
                                  C_warp[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = 0f16
                                }
                                C_warp[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = (C_warp[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] + (A_shared_warp[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)]*B_shared_warp[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)]))
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_4: int32, 0, 16384) {
        for (ax1_4: int32, 0, 16384) {
          block([16384, 16384], "C_warp") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
            tir.writes([C[v0_4, v1_4]])
            C[v0_4, v1_4] = C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}