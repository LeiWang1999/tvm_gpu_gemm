#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float16), float16, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    A_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    B_shared = alloc_buffer(float16[1024, 1024, 16, 16])
    B_shared_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    C_warp = alloc_buffer(float16[1024, 1024, 32, 8])
    for (i_0_0: int32, 0, 128) "thread_binding" {
      for (j_0_0_0: int32, 0, 4) "thread_binding" {
        for (j_0_0_1: int32, 0, 16) "thread_binding" {
          for (i_0_1: int32, 0, 4) "thread_binding" {
            for (j_0_1: int32, 0, 1) "thread_binding" {
              for (i_0_2_init: int32, 0, 2) {
                for (j_0_2_init: int32, 0, 16) {
                  block([1024, 1024], "B_init_o") as [vi_o, vj_o] {
                    bind(vi_o, (((i_0_0*8) + (i_0_1*2)) + i_0_2_init))
                    bind(vj_o, (((j_0_0_0*256) + (j_0_0_1*16)) + j_0_2_init))
                    tir.reads([])
                    tir.writes([C_warp[vi_o, vj_o, 0:32, 0:8]])
                    C_warp_1 = match_buffer(C_warp[vi_o, vj_o, 0:32, 0:8])
                    attr [IterVar(tx: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_fill(8, C_warp_2: Pointer(warp float16), elem_offset: int32, dtype=float16)
                }
              }
              for (k_0_0: int32, 0, 512) {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 4) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 8) "vectorized" {
                          block([16384, 16384], "A_shared") as [v0, v1] {
                            bind(v0, (((i_0_0*128) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                            bind(v1, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                            tir.reads([A[(((floordiv(v0, 16)*16) + (floormod(v0, 8)*2)) + floordiv(floormod(v1, 16), 8)), (((floordiv(v1, 16)*16) + (floordiv(floormod(v0, 16), 8)*8)) + floormod(v1, 8))]])
                            tir.writes([A_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
                            A_shared[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = A[(((floordiv(v0, 16)*16) + (floormod(v0, 8)*2)) + floordiv(floormod(v1, 16), 8)), (((floordiv(v1, 16)*16) + (floordiv(floormod(v0, 16), 8)*8)) + floormod(v1, 8))]
                        }
                      }
                    }
                  }
                }
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 4) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 8) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 8) "vectorized" {
                          block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                            bind(v0_1, (((k_0_0*32) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 4096)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                            bind(v1_1, ((((j_0_0_0*4096) + (j_0_0_1*256)) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 4096), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                            tir.reads([B[(((floordiv(v0_1, 16)*16) + (floormod(v0_1, 8)*2)) + floordiv(floormod(v1_1, 16), 8)), (((floordiv(v1_1, 16)*16) + (floordiv(floormod(v0_1, 16), 8)*8)) + floormod(v1_1, 8))]])
                            tir.writes([B_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
                            B_shared[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = B[(((floordiv(v0_1, 16)*16) + (floormod(v0_1, 8)*2)) + floordiv(floormod(v1_1, 16), 8)), (((floordiv(v1_1, 16)*16) + (floordiv(floormod(v0_1, 16), 8)*8)) + floormod(v1_1, 8))]
                        }
                      }
                    }
                  }
                }
                for (k_0_1: int32, 0, 2) {
                  for (ax0_0: int32, 0, 2) {
                    for (ax1_0: int32, 0, 1) {
                      block([1024, 1024], "A_shared_warp_o") as [v0_o, v1_o] {
                        bind(v0_o, (((i_0_0*8) + (i_0_1*2)) + ax0_0))
                        bind(v1_o, ((k_0_0*2) + k_0_1))
                        tir.reads([A_shared[v0_o, v1_o, 0:16, 0:16]])
                        tir.writes([A_shared_warp[v0_o, v1_o, 0:32, 0:8]])
                        warp = match_buffer(A_shared_warp[v0_o, v1_o, 0:32, 0:8])
                        shared = match_buffer(A_shared[v0_o, v1_o, 0:16, 0:16])
                        attr [IterVar(tx_1: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                        @tir.ptx_ldmatrix(False, 4, ".b16", warp_1: Pointer(warp float16), (elem_offset_1: int32 + (8*tx_1)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_1: Pointer(shared float16), elem_offset_2: int32, (shared_s0: int32*16), 1, dtype=handle), (8*tx_1), dtype=float16)
                    }
                  }
                  for (ax0_0_1: int32, 0, 1) {
                    for (ax1_0_1: int32, 0, 16) {
                      block([1024, 1024], "B_shared_warp_o") as [v0_o_1, v1_o_1] {
                        bind(v0_o_1, ((k_0_0*2) + k_0_1))
                        bind(v1_o_1, (((j_0_0_0*256) + (j_0_0_1*16)) + ax1_0_1))
                        tir.reads([B_shared[v0_o_1, v1_o_1, 0:16, 0:16]])
                        tir.writes([B_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8]])
                        warp_2 = match_buffer(B_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8])
                        shared_2 = match_buffer(B_shared[v0_o_1, v1_o_1, 0:16, 0:16])
                        attr [IterVar(tx_2: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                        @tir.ptx_ldmatrix(True, 4, ".b16", warp_3: Pointer(warp float16), (elem_offset_3: int32 + (8*tx_2)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_3: Pointer(shared float16), elem_offset_4: int32, (shared_s0_1: int32*16), 1, dtype=handle), (8*tx_2), dtype=float16)
                    }
                  }
                  for (i_0_2: int32, 0, 2) {
                    for (j_0_2: int32, 0, 16) {
                      block([1024, 1024, tir.reduce_axis(0, 1024)], "B_update_o") as [vi_o_1, vj_o_1, vk_o] {
                        bind(vi_o_1, (((i_0_0*8) + (i_0_1*2)) + i_0_2))
                        bind(vj_o_1, (((j_0_0_0*256) + (j_0_0_1*16)) + j_0_2))
                        bind(vk_o, ((k_0_0*2) + k_0_1))
                        tir.reads([C_warp[vi_o_1, vj_o_1, 0:32, 0:8], A_shared_warp[vi_o_1, vk_o, 0:32, 0:8], B_shared_warp[vk_o, vj_o_1, 0:32, 0:8]])
                        tir.writes([C_warp[vi_o_1, vj_o_1, 0:32, 0:8]])
                        A_2 = match_buffer(A_shared_warp[vi_o_1, vk_o, 0:32, 0:8])
                        B_2 = match_buffer(B_shared_warp[vk_o, vj_o_1, 0:32, 0:8])
                        C_2 = match_buffer(C_warp[vi_o_1, vj_o_1, 0:32, 0:8])
                        attr [IterVar(tx_3: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32 {
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3: Pointer(warp float16), (elem_offset_5: int32 + (tx_3*8)), B_3: Pointer(warp float16), (elem_offset_6: int32 + (tx_3*8)), C_3: Pointer(warp float16), (elem_offset_7: int32 + (tx_3*8)), False, dtype=float16)
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3, (elem_offset_5 + (tx_3*8)), B_3, ((elem_offset_6 + (tx_3*8)) + floordiv(8, 2)), C_3, ((elem_offset_7 + (tx_3*8)) + floordiv(8, 2)), False, dtype=float16)
                        }
                    }
                  }
                }
              }
              for (ax0_0_2: int32, 0, 2) {
                for (ax1_0_2: int32, 0, 16) {
                  block([1024, 1024], "C_warp_o") as [v0_o_2, v1_o_2] {
                    bind(v0_o_2, (((i_0_0*8) + (i_0_1*2)) + ax0_0_2))
                    bind(v1_o_2, (((j_0_0_0*256) + (j_0_0_1*16)) + ax1_0_2))
                    tir.reads([C_warp[v0_o_2, v1_o_2, 0:32, 0:8]])
                    tir.writes([C[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)]])
                    C_warp_3 = match_buffer(C_warp[v0_o_2, v1_o_2, 0:32, 0:8])
                    C_4 = match_buffer(C[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)])
                    attr [IterVar(tx_4: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_store(16, 16, @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), C_5: Pointer(global float16), elem_offset_8: int32, (C_s0: int32*16), 2, dtype=handle), C_warp_4: Pointer(warp float16), elem_offset_9: int32, C_s0, dtype=float16)
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}