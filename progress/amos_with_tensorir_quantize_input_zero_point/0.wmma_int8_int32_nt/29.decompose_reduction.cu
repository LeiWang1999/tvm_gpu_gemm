#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle, pb: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], []),
             PB: Buffer(PB_1: Pointer(global int32), int32, [16384], [])}
  buffer_map = {a: A, b: B, c: C, pb: PB} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QC = alloc_buffer(int32[16384, 16384])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
    C_local = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 16) "thread_binding" {
          for (j_0_0_1: int32, 0, 16) "thread_binding" {
            for (i_0_1: int32, 0, 1) "thread_binding" {
              for (j_0_1: int32, 0, 4) "thread_binding" {
                for (i_0_2_init: int32, 0, 16) {
                  for (j_0_2_init: int32, 0, 1) {
                    for (i_1_init: int32, 0, 16) {
                      for (j_1_init: int32, 0, 16) {
                        block([16384, 16384], "B_init") as [vi, vj] {
                          bind(vi, ((((i_0_0*256) + (i_0_1*256)) + (i_0_2_init*16)) + i_1_init))
                          bind(vj, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (j_0_2_init*16)) + j_1_init))
                          tir.reads([])
                          tir.writes([QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                          QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = 0
                      }
                    }
                  }
                }
                for (k_0_0: int32, 0, 512) {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 4) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
                              bind(v0_2, (((i_0_0*256) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                              bind(v1_2, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                              tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                          }
                        }
                      }
                    }
                  }
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 4) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 1) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
                              bind(v0_3, (((k_0_0*32) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                              bind(v1_3, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                              tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                          }
                        }
                      }
                    }
                  }
                  for (k_0_1: int32, 0, 2) {
                    for (ax0_0: int32, 0, 16) {
                      for (ax1_0: int32, 0, 1) {
                        for (ax0_1_1: int32, 0, 16) {
                          for (ax1_1_1: int32, 0, 16) {
                            block([16384, 16384], "A_global_shared_wmma.matrix_a") as [v0_4, v1_4] {
                              bind(v0_4, (((i_0_0*256) + (ax0_0*16)) + ax0_1_1))
                              bind(v1_4, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0*16)) + ax1_1_1))
                              tir.reads([A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              tir.writes([A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                          }
                        }
                      }
                    }
                    for (ax0_0_1: int32, 0, 1) {
                      for (ax1_0_1: int32, 0, 1) {
                        for (ax0_1_2: int32, 0, 16) {
                          for (ax1_1_2: int32, 0, 16) {
                            block([16384, 16384], "B_global_shared_wmma.matrix_b") as [v0_5, v1_5] {
                              bind(v0_5, ((((k_0_0*32) + (k_0_1*16)) + (ax0_0_1*16)) + ax0_1_2))
                              bind(v1_5, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (ax1_0_1*16)) + ax1_1_2))
                              tir.reads([B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              tir.writes([B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                          }
                        }
                      }
                    }
                    for (i_0_2: int32, 0, 16) {
                      for (j_0_2: int32, 0, 1) {
                        for (i_1: int32, 0, 16) {
                          for (j_1: int32, 0, 16) {
                            for (k_1: int32, 0, 16) {
                              block([16384, 16384, tir.reduce_axis(0, 16384)], "B_update") as [vi_1, vj_1, vk] {
                                bind(vi_1, ((((i_0_0*256) + (i_0_1*256)) + (i_0_2*16)) + i_1))
                                bind(vj_1, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (j_0_2*16)) + j_1))
                                bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                                tir.reads([QC_wmma.accumulator[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)], A_global_shared_wmma.matrix_a[floordiv(vi_1, 16), floordiv(vk, 16), floormod(vi_1, 16), floormod(vk, 16)], B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj_1, 16), floormod(vk, 16), floormod(vj_1, 16)]])
                                tir.writes([QC_wmma.accumulator[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)]])
                                QC_wmma.accumulator[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)] = (QC_wmma.accumulator[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)] + (cast(int32, A_global_shared_wmma.matrix_a[floordiv(vi_1, 16), floordiv(vk, 16), floormod(vi_1, 16), floormod(vk, 16)])*cast(int32, B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj_1, 16), floormod(vk, 16), floormod(vj_1, 16)])))
                            }
                          }
                        }
                      }
                    }
                  }
                }
                for (ax0_0_2: int32, 0, 16) {
                  for (ax1_0_2: int32, 0, 1) {
                    for (ax0_1_3: int32, 0, 16) {
                      for (ax1_1_3: int32, 0, 16) {
                        block([16384, 16384], "QC_wmma.accumulator") as [v0_6, v1_6] {
                          bind(v0_6, (((i_0_0*256) + (ax0_0_2*16)) + ax0_1_3))
                          bind(v1_6, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (ax1_0_2*16)) + ax1_1_3))
                          tir.reads([QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                          tir.writes([QC[v0_6, v1_6]])
                          QC[v0_6, v1_6] = QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]
                      }
                    }
                    for (ax0_ax1_fused_0: int32, 0, 8) {
                      for (ax0_ax1_fused_1: int32, 0, 1) "thread_binding" {
                        for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                          block([16384, 16384], "C") as [vi_2, vj_2] {
                            bind(vi_2, (((i_0_0*256) + (ax0_0_2*16)) + floordiv((((ax0_ax1_fused_0*32) + (ax0_ax1_fused_1*32)) + ax0_ax1_fused_2), 16)))
                            bind(vj_2, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + floormod((((ax0_ax1_fused_0*32) + (ax0_ax1_fused_1*32)) + ax0_ax1_fused_2), 16)))
                            tir.reads([QC[vi_2, vj_2], PB[vj_2]])
                            tir.writes([C_local[vi_2, vj_2]])
                            C_local[vi_2, vj_2] = ((QC[vi_2, vj_2] + 12) + PB[vj_2])
                        }
                      }
                    }
                    for (ax0_ax1_fused_0_1: int32, 0, 8) {
                      for (ax0_ax1_fused_1_1: int32, 0, 1) "thread_binding" {
                        for (ax0_ax1_fused_2_1: int32, 0, 32) "thread_binding" {
                          block([16384, 16384], "C_local") as [v0_7, v1_7] {
                            bind(v0_7, (((i_0_0*256) + (ax0_0_2*16)) + floordiv((((ax0_ax1_fused_0_1*32) + (ax0_ax1_fused_1_1*32)) + ax0_ax1_fused_2_1), 16)))
                            bind(v1_7, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + floormod((((ax0_ax1_fused_0_1*32) + (ax0_ax1_fused_1_1*32)) + ax0_ax1_fused_2_1), 16)))
                            tir.reads([C_local[v0_7, v1_7]])
                            tir.writes([C[v0_7, v1_7]])
                            C[v0_7, v1_7] = C_local[v0_7, v1_7]
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}