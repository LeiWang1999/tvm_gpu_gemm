#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle, pb: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], []),
             PB: Buffer(PB_1: Pointer(global int32), int32, [16384], [])}
  buffer_map = {a: A, b: B, c: C, pb: PB} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QC = alloc_buffer(int32[16384, 16384])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
    C_local = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 16) "thread_binding" {
          for (j_0_0_1: int32, 0, 16) "thread_binding" {
            for (i_0_1: int32, 0, 1) "thread_binding" {
              for (j_0_1: int32, 0, 4) "thread_binding" {
                for (i_0_2_init: int32, 0, 16) {
                  for (j_0_2_init: int32, 0, 1) {
                    block([1024, 1024], "B_init_o") as [vi_o, vj_o] {
                      bind(vi_o, (((i_0_0*16) + (i_0_1*16)) + i_0_2_init))
                      bind(vj_o, ((((j_0_0_0*64) + (j_0_0_1*4)) + j_0_1) + j_0_2_init))
                      tir.reads([])
                      tir.writes([QC_wmma.accumulator[vi_o, vj_o, 0:16, 0:16]])
                      C_2 = match_buffer(QC_wmma.accumulator[vi_o, vj_o, 0:16, 0:16])
                      @tir.tvm_fill_fragment(C_3: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
                  }
                }
                for (k_0_0: int32, 0, 512) {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 4) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
                              bind(v0_2, (((i_0_0*256) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                              bind(v1_2, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                              tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                          }
                        }
                      }
                    }
                  }
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 1) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 4) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 1) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
                              bind(v0_3, (((k_0_0*32) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                              bind(v1_3, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                              tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                          }
                        }
                      }
                    }
                  }
                  for (k_0_1: int32, 0, 2) {
                    for (ax0_0: int32, 0, 16) {
                      for (ax1_0: int32, 0, 1) {
                        block([1024, 1024], "A_global_shared_wmma.matrix_a_o") as [v0_o, v1_o] {
                          bind(v0_o, ((i_0_0*16) + ax0_0))
                          bind(v1_o, (((k_0_0*2) + k_0_1) + ax1_0))
                          tir.reads([A_global_shared[v0_o, v1_o, 0:16, 0:16]])
                          tir.writes([A_global_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16]])
                          A_2 = match_buffer(A_global_shared[v0_o, v1_o, 0:16, 0:16])
                          C_4 = match_buffer(A_global_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16])
                          @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_a int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_3: Pointer(shared int8), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                      }
                    }
                    for (ax0_0_1: int32, 0, 1) {
                      for (ax1_0_1: int32, 0, 1) {
                        block([1024, 1024], "B_global_shared_wmma.matrix_b_o") as [v0_o_1, v1_o_1] {
                          bind(v0_o_1, (((k_0_0*2) + k_0_1) + ax0_0_1))
                          bind(v1_o_1, ((((j_0_0_0*64) + (j_0_0_1*4)) + j_0_1) + ax1_0_1))
                          tir.reads([B_global_shared[v0_o_1, v1_o_1, 0:16, 0:16]])
                          tir.writes([B_global_shared_wmma.matrix_b[v0_o_1, v1_o_1, 0:16, 0:16]])
                          A_4 = match_buffer(B_global_shared[v0_o_1, v1_o_1, 0:16, 0:16])
                          C_6 = match_buffer(B_global_shared_wmma.matrix_b[v0_o_1, v1_o_1, 0:16, 0:16])
                          @tir.tvm_load_matrix_sync(C_7: Pointer(wmma.matrix_b int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_3: int32, C_s0_2: int32), 16)*floordiv(C_s0_2, 16)) + floordiv(floormod(elem_offset_3, C_s0_2), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_5: Pointer(shared int8), elem_offset_4: int32, (A_s0_1: int32*16), 1, dtype=handle), A_s0_1, "row_major", dtype=handle)
                      }
                    }
                    for (i_0_2: int32, 0, 16) {
                      for (j_0_2: int32, 0, 1) {
                        block([1024, 1024, tir.reduce_axis(0, 1024)], "B_update_o") as [vi_o_1, vj_o_1, vk_o] {
                          bind(vi_o_1, (((i_0_0*16) + (i_0_1*16)) + i_0_2))
                          bind(vj_o_1, ((((j_0_0_0*64) + (j_0_0_1*4)) + j_0_1) + j_0_2))
                          bind(vk_o, ((k_0_0*2) + k_0_1))
                          tir.reads([QC_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16], A_global_shared_wmma.matrix_a[vi_o_1, vk_o, 0:16, 0:16], B_global_shared_wmma.matrix_b[vk_o, vj_o_1, 0:16, 0:16]])
                          tir.writes([QC_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16]])
                          A_6 = match_buffer(A_global_shared_wmma.matrix_a[vi_o_1, vk_o, 0:16, 0:16])
                          B_2 = match_buffer(B_global_shared_wmma.matrix_b[vk_o, vj_o_1, 0:16, 0:16])
                          C_8 = match_buffer(QC_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16])
                          @tir.tvm_mma_sync(C_9: Pointer(wmma.accumulator int32), ((floordiv(floordiv(elem_offset_5: int32, C_s0_3: int32), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), A_7: Pointer(wmma.matrix_a int8), ((floordiv(floordiv(elem_offset_6: int32, A_s0_2: int32), 16)*floordiv(A_s0_2, 16)) + floordiv(floormod(elem_offset_6, A_s0_2), 16)), B_3: Pointer(wmma.matrix_b int8), ((floordiv(floordiv(elem_offset_7: int32, B_s0: int32), 16)*floordiv(B_s0, 16)) + floordiv(floormod(elem_offset_7, B_s0), 16)), C_9, ((floordiv(floordiv(elem_offset_5, C_s0_3), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), dtype=handle)
                      }
                    }
                  }
                }
                for (ax0_0_2: int32, 0, 16) {
                  for (ax1_0_2: int32, 0, 1) {
                    block([1024, 1024], "QC_wmma.accumulator_o") as [v0_o_2, v1_o_2] {
                      bind(v0_o_2, ((i_0_0*16) + ax0_0_2))
                      bind(v1_o_2, ((((j_0_0_0*64) + (j_0_0_1*4)) + j_0_1) + ax1_0_2))
                      tir.reads([QC_wmma.accumulator[v0_o_2, v1_o_2, 0:16, 0:16]])
                      tir.writes([QC[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)]])
                      A_8 = match_buffer(QC_wmma.accumulator[v0_o_2, v1_o_2, 0:16, 0:16])
                      C_10 = match_buffer(QC[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)])
                      @tir.tvm_store_matrix_sync(A_9: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset_8: int32, A_s0_3: int32), 16)*floordiv(A_s0_3, 16)) + floordiv(floormod(elem_offset_8, A_s0_3), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int32), C_11: Pointer(shared int32), elem_offset_9: int32, (C_s0_4: int32*16), 2, dtype=handle), C_s0_4, "row_major", dtype=handle)
                    for (ax0_2: int32, 0, 16) {
                      for (ax1_2: int32, 0, 16) {
                        block([16384, 16384], "C") as [vi, vj] {
                          bind(vi, (((i_0_0*256) + (ax0_0_2*16)) + ax0_2))
                          bind(vj, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + ax1_2))
                          tir.reads([QC[vi, vj], PB[vj]])
                          tir.writes([C_local[vi, vj]])
                          C_local[vi, vj] = ((QC[vi, vj] + 12) + PB[vj])
                      }
                    }
                    for (ax0_3: int32, 0, 16) {
                      for (ax1_3: int32, 0, 16) {
                        block([16384, 16384], "C_local") as [v0_4, v1_4] {
                          bind(v0_4, (((i_0_0*256) + (ax0_0_2*16)) + ax0_3))
                          bind(v1_4, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + ax1_3))
                          tir.reads([C_local[v0_4, v1_4]])
                          tir.writes([C[v0_4, v1_4]])
                          C[v0_4, v1_4] = C_local[v0_4, v1_4]
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}