#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle, pb: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], []),
             PB: Buffer(PB_1: Pointer(global int32), int32, [16384], [])}
  buffer_map = {a: A, b: B, c: C, pb: PB} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QC = alloc_buffer(int32[16384, 16384])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
    C_local = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 16) "thread_binding" {
          for (j_0_0_1: int32, 0, 16) "thread_binding" {
            for (i_0_1: int32, 0, 1) "thread_binding" {
              for (j_0_1: int32, 0, 4) "thread_binding" {
                for (k_0_0: int32, 0, 512) {
                  for (ax0_0: int32, 0, 16) {
                    for (ax1_0: int32, 0, 2) {
                      for (ax0_1_1: int32, 0, 16) {
                        for (ax1_1_1: int32, 0, 16) {
                          block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
                            bind(v0_2, (((i_0_0*256) + (ax0_0*16)) + ax0_1_1))
                            bind(v1_2, (((k_0_0*32) + (ax1_0*16)) + ax1_1_1))
                            tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                        }
                      }
                    }
                  }
                  for (ax0_0_1: int32, 0, 2) {
                    for (ax1_0_1: int32, 0, 4) {
                      for (ax0_1_2: int32, 0, 16) {
                        for (ax1_1_2: int32, 0, 16) {
                          block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
                            bind(v0_3, (((k_0_0*32) + (ax0_0_1*16)) + ax0_1_2))
                            bind(v1_3, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (ax1_0_1*16)) + ax1_1_2))
                            tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                            tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                            B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                        }
                      }
                    }
                  }
                  for (k_0_1: int32, 0, 2) {
                    for (ax0_0_2: int32, 0, 16) {
                      for (ax1_0_2: int32, 0, 1) {
                        for (ax0_1_3: int32, 0, 16) {
                          for (ax1_1_3: int32, 0, 16) {
                            block([16384, 16384], "A_global_shared_wmma.matrix_a") as [v0_4, v1_4] {
                              bind(v0_4, (((i_0_0*256) + (ax0_0_2*16)) + ax0_1_3))
                              bind(v1_4, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0_2*16)) + ax1_1_3))
                              tir.reads([A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              tir.writes([A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              A_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = A_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                          }
                        }
                      }
                    }
                    for (ax0_0_3: int32, 0, 1) {
                      for (ax1_0_3: int32, 0, 1) {
                        for (ax0_1_4: int32, 0, 16) {
                          for (ax1_1_4: int32, 0, 16) {
                            block([16384, 16384], "B_global_shared_wmma.matrix_b") as [v0_5, v1_5] {
                              bind(v0_5, ((((k_0_0*32) + (k_0_1*16)) + (ax0_0_3*16)) + ax0_1_4))
                              bind(v1_5, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (ax1_0_3*16)) + ax1_1_4))
                              tir.reads([B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              tir.writes([B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              B_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = B_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                          }
                        }
                      }
                    }
                    for (i_0_2: int32, 0, 16) {
                      for (j_0_2: int32, 0, 1) {
                        for (i_1: int32, 0, 16) {
                          for (j_1: int32, 0, 16) {
                            for (k_1: int32, 0, 16) {
                              block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                                bind(vi, ((((i_0_0*256) + (i_0_1*256)) + (i_0_2*16)) + i_1))
                                bind(vj, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (j_0_2*16)) + j_1))
                                bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                                tir.reads([A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)], B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)]])
                                tir.writes([QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                                with init() {
                                  QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = 0
                                }
                                QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = (QC_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] + (cast(int32, A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)])*cast(int32, B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)])))
                            }
                          }
                        }
                      }
                    }
                  }
                }
                for (ax0_0_4: int32, 0, 16) {
                  for (ax1_0_4: int32, 0, 1) {
                    for (ax0_1_5: int32, 0, 16) {
                      for (ax1_1_5: int32, 0, 16) {
                        block([16384, 16384], "QC_wmma.accumulator") as [v0_6, v1_6] {
                          bind(v0_6, (((i_0_0*256) + (ax0_0_4*16)) + ax0_1_5))
                          bind(v1_6, (((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + (ax1_0_4*16)) + ax1_1_5))
                          tir.reads([QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                          tir.writes([QC[v0_6, v1_6]])
                          QC[v0_6, v1_6] = QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]
                      }
                    }
                    for (ax0_ax1_fused_0: int32, 0, 8) {
                      for (ax0_ax1_fused_1: int32, 0, 1) "thread_binding" {
                        for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                          block([16384, 16384], "C") as [vi_1, vj_1] {
                            bind(vi_1, (((i_0_0*256) + (ax0_0_4*16)) + floordiv((((ax0_ax1_fused_0*32) + (ax0_ax1_fused_1*32)) + ax0_ax1_fused_2), 16)))
                            bind(vj_1, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + floormod((((ax0_ax1_fused_0*32) + (ax0_ax1_fused_1*32)) + ax0_ax1_fused_2), 16)))
                            tir.reads([QC[vi_1, vj_1], PB[vj_1]])
                            tir.writes([C_local[vi_1, vj_1]])
                            C_local[vi_1, vj_1] = ((QC[vi_1, vj_1] + 12) + PB[vj_1])
                        }
                      }
                    }
                    for (ax0_ax1_fused_0_1: int32, 0, 8) {
                      for (ax0_ax1_fused_1_1: int32, 0, 1) "thread_binding" {
                        for (ax0_ax1_fused_2_1: int32, 0, 32) "thread_binding" {
                          block([16384, 16384], "C_local") as [v0_7, v1_7] {
                            bind(v0_7, (((i_0_0*256) + (ax0_0_4*16)) + floordiv((((ax0_ax1_fused_0_1*32) + (ax0_ax1_fused_1_1*32)) + ax0_ax1_fused_2_1), 16)))
                            bind(v1_7, ((((j_0_0_0*1024) + (j_0_0_1*64)) + (j_0_1*16)) + floormod((((ax0_ax1_fused_0_1*32) + (ax0_ax1_fused_1_1*32)) + ax0_ax1_fused_2_1), 16)))
                            tir.reads([C_local[v0_7, v1_7]])
                            tir.writes([C[v0_7, v1_7]])
                            C[v0_7, v1_7] = C_local[v0_7, v1_7]
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}