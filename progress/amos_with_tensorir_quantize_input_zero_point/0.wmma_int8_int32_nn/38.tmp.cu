#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ A_global, signed char* __restrict__ B_global, int* __restrict__ PB, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> QC_wmma_accumulator[16];
  __shared__ signed char A_global_shared[8192];
  __shared__ signed char B_global_shared[2048];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a[16];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> B_global_shared_wmma_matrix_b[1];
  int C_local[240];
  for (int i_0_2_init = 0; i_0_2_init < 16; ++i_0_2_init) {
    nvcuda::wmma::fill_fragment(QC_wmma_accumulator[i_0_2_init], 0.000000e+00f);
  }
  for (int k_0_0 = 0; k_0_0 < 512; ++k_0_0) {
    __syncthreads();
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 4; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
      *(int4*)(A_global_shared + (((((int)threadIdx.z) * 2048) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A_global + (((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.z) * 1048576)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 262144)) + (k_0_0 * 512)) + (((int)threadIdx.x) * 16)));
    }
    *(int4*)(B_global_shared + ((((int)threadIdx.z) * 512) + (((int)threadIdx.x) * 16))) = *(int4*)(B_global + ((((((k_0_0 * 524288) + ((((int)threadIdx.z) >> 1) * 262144)) + (((int)blockIdx.z) * 16384)) + (((int)blockIdx.x) * 1024)) + ((((int)threadIdx.z) & 1) * 512)) + (((int)threadIdx.x) * 16)));
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 16; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a[ax0_0], (&(A_global_shared[((ax0_0 * 512) + (k_0_1 * 256))])), 16);
      }
      nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b[0], (&(B_global_shared[((k_0_1 * 1024) + (((int)threadIdx.z) * 256))])), 16);
      for (int i_0_2 = 0; i_0_2 < 16; ++i_0_2) {
        nvcuda::wmma::mma_sync(QC_wmma_accumulator[i_0_2], A_global_shared_wmma_matrix_a[i_0_2], B_global_shared_wmma_matrix_b[0], QC_wmma_accumulator[i_0_2]);
      }
    }
  }
  for (int ax0_0_1 = 0; ax0_0_1 < 16; ++ax0_0_1) {
    __syncthreads();
    nvcuda::wmma::store_matrix_sync((&(((int*)A_global_shared)[(((int)threadIdx.z) * 16)])), QC_wmma_accumulator[ax0_0_1], 64, nvcuda::wmma::mem_row_major);
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 8; ++ax0_ax1_fused_0) {
      C_local[((ax0_ax1_fused_0 * 32) + (((int)threadIdx.x) & 15))] = ((((int*)A_global_shared)[((((ax0_ax1_fused_0 * 128) + ((((int)threadIdx.x) >> 4) * 64)) + (((int)threadIdx.z) * 16)) + (((int)threadIdx.x) & 15))] + PB[((((((int)blockIdx.z) * 1024) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16)) + (((int)threadIdx.x) & 15))]) + 12);
    }
    for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 8; ++ax0_ax1_fused_0_1) {
      C[((((((((((int)blockIdx.y) * 4194304) + (ax0_0_1 * 262144)) + (ax0_ax1_fused_0_1 * 32768)) + ((((int)threadIdx.x) >> 4) * 16384)) + (((int)blockIdx.z) * 1024)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16)) + (((int)threadIdx.x) & 15))] = C_local[((ax0_ax1_fused_0_1 * 32) + (((int)threadIdx.x) & 15))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(B_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(B + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(A_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(A + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

