#include "hip/hip_runtime.h"
#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int8), int8, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QA = alloc_buffer(int8[16384, 16384])
    QB = alloc_buffer(int8[16384, 16384])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_local = alloc_buffer(int8[16384, 16384])
    QA_local_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_local_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_local = alloc_buffer(int8[16384, 16384])
    QB_local_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_local_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_shared = alloc_buffer(int32[16384, 16384])
    QC_shared_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i: int32, 0, 16384) {
        for (j: int32, 0, 16384) {
          block([16384, 16384], "Quantize_A") as [vi, vj] {
            bind(vi, i)
            bind(vj, j)
            tir.reads([A_global[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
            tir.writes([QA[vi, vj]])
            QA[vi, vj] = cast(int8, (@tir.round((cast(float32, A_global[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)])*0.5f32), dtype=float32) - 0f32))
        }
      }
      for (i_1: int32, 0, 16384) {
        for (j_1: int32, 0, 16384) {
          block([16384, 16384], "Quantize_B") as [vi_1, vj_1] {
            bind(vi_1, i_1)
            bind(vj_1, j_1)
            tir.reads([B_global[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)]])
            tir.writes([QB[vi_1, vj_1]])
            QB[vi_1, vj_1] = cast(int8, (@tir.round((cast(float32, B_global[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)])*0.1f32), dtype=float32) - 0f32))
        }
      }
      for (ax0_2: int32, 0, 16384) {
        for (ax1_2: int32, 0, 16384) {
          block([16384, 16384], "QA_local") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([QA[v0_2, v1_2]])
            tir.writes([QA_local[v0_2, v1_2]])
            QA_local[v0_2, v1_2] = QA[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "QA_local_shared") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([QA_local[v0_3, v1_3]])
            tir.writes([QA_local_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
            QA_local_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = QA_local[v0_3, v1_3]
        }
      }
      for (ax0_4: int32, 0, 16384) {
        for (ax1_4: int32, 0, 16384) {
          block([16384, 16384], "QA_local_shared_wmma.matrix_a") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([QA_local_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
            tir.writes([QA_local_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
            QA_local_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = QA_local_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
        }
      }
      for (ax0_5: int32, 0, 16384) {
        for (ax1_5: int32, 0, 16384) {
          block([16384, 16384], "QB_local") as [v0_5, v1_5] {
            bind(v0_5, ax0_5)
            bind(v1_5, ax1_5)
            tir.reads([QB[v0_5, v1_5]])
            tir.writes([QB_local[v0_5, v1_5]])
            QB_local[v0_5, v1_5] = QB[v0_5, v1_5]
        }
      }
      for (ax0_6: int32, 0, 16384) {
        for (ax1_6: int32, 0, 16384) {
          block([16384, 16384], "QB_local_shared") as [v0_6, v1_6] {
            bind(v0_6, ax0_6)
            bind(v1_6, ax1_6)
            tir.reads([QB_local[v0_6, v1_6]])
            tir.writes([QB_local_shared[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
            QB_local_shared[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)] = QB_local[v0_6, v1_6]
        }
      }
      for (ax0_7: int32, 0, 16384) {
        for (ax1_7: int32, 0, 16384) {
          block([16384, 16384], "QB_local_shared_wmma.matrix_b") as [v0_7, v1_7] {
            bind(v0_7, ax0_7)
            bind(v1_7, ax1_7)
            tir.reads([QB_local_shared[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)]])
            tir.writes([QB_local_shared_wmma.matrix_b[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)]])
            QB_local_shared_wmma.matrix_b[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)] = QB_local_shared[floordiv(v0_7, 16), floordiv(v1_7, 16), floormod(v0_7, 16), floormod(v1_7, 16)]
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 8) "thread_binding" {
          for (j_0_0_1: int32, 0, 32) "thread_binding" {
            for (i_0_1: int32, 0, 4) "thread_binding" {
              for (j_0_1: int32, 0, 1) "thread_binding" {
                for (k_0_0: int32, 0, 256) {
                  for (k_0_1: int32, 0, 4) {
                    for (i_0_2: int32, 0, 4) {
                      for (j_0_2: int32, 0, 4) {
                        for (i_1_1: int32, 0, 16) {
                          for (j_1_1: int32, 0, 16) {
                            for (k_1: int32, 0, 16) {
                              block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi_2, vj_2, vk] {
                                bind(vi_2, ((((i_0_0*256) + (i_0_1*64)) + (i_0_2*16)) + i_1_1))
                                bind(vj_2, (((((j_0_0_0*2048) + (j_0_0_1*64)) + (j_0_1*64)) + (j_0_2*16)) + j_1_1))
                                bind(vk, (((k_0_0*64) + (k_0_1*16)) + k_1))
                                tir.reads([QA_local_shared_wmma.matrix_a[floordiv(vi_2, 16), floordiv(vk, 16), floormod(vi_2, 16), floormod(vk, 16)], QB_local_shared_wmma.matrix_b[floordiv(vj_2, 16), floordiv(vk, 16), floormod(vj_2, 16), floormod(vk, 16)]])
                                tir.writes([QC_shared_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)]])
                                with init() {
                                  QC_shared_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)] = 0
                                }
                                QC_shared_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)] = (QC_shared_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)] + (cast(int32, QA_local_shared_wmma.matrix_a[floordiv(vi_2, 16), floordiv(vk, 16), floormod(vi_2, 16), floormod(vk, 16)])*cast(int32, QB_local_shared_wmma.matrix_b[floordiv(vj_2, 16), floordiv(vk, 16), floormod(vj_2, 16), floormod(vk, 16)])))
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_8: int32, 0, 16384) {
        for (ax1_8: int32, 0, 16384) {
          block([16384, 16384], "QC_shared_wmma.accumulator") as [v0_8, v1_8] {
            bind(v0_8, ax0_8)
            bind(v1_8, ax1_8)
            tir.reads([QC_shared_wmma.accumulator[floordiv(v0_8, 16), floordiv(v1_8, 16), floormod(v0_8, 16), floormod(v1_8, 16)]])
            tir.writes([QC_shared[v0_8, v1_8]])
            QC_shared[v0_8, v1_8] = QC_shared_wmma.accumulator[floordiv(v0_8, 16), floordiv(v1_8, 16), floormod(v0_8, 16), floormod(v1_8, 16)]
        }
      }
      for (ax0_9: int32, 0, 16384) {
        for (ax1_9: int32, 0, 16384) {
          block([16384, 16384], "QC_shared") as [v0_9, v1_9] {
            bind(v0_9, ax0_9)
            bind(v1_9, ax1_9)
            tir.reads([QC_shared[v0_9, v1_9]])
            tir.writes([C[v0_9, v1_9]])
            C[v0_9, v1_9] = cast(int8, ((cast(float32, QC_shared[v0_9, v1_9]) / 0.01f32) + 0f32))
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}