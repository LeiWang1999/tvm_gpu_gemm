#include "hip/hip_runtime.h"
#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int8), int8, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QA = alloc_buffer(int8[16384, 16384])
    QB = alloc_buffer(int8[16384, 16384])
    QC = alloc_buffer(int32[16384, 16384])
    A_global = alloc_buffer(int8[16384, 16384])
    QA_local = alloc_buffer(int8[16384, 16384])
    QA_local_shared = alloc_buffer(int8[16384, 16384])
    QA_local_shared_wmma.matrix_a = alloc_buffer(int8[16384, 16384])
    B_global = alloc_buffer(int8[16384, 16384])
    QB_local = alloc_buffer(int8[16384, 16384])
    QB_local_shared = alloc_buffer(int8[16384, 16384])
    QB_local_shared_wmma.matrix_b = alloc_buffer(int8[16384, 16384])
    QC_shared = alloc_buffer(int32[16384, 16384])
    QC_shared_wmma.accumulator = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[v0, v1]])
            B_global[v0, v1] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[v0_1, v1_1]])
            A_global[v0_1, v1_1] = A[v0_1, v1_1]
        }
      }
      for (i: int32, 0, 16384) {
        for (j: int32, 0, 16384) {
          block([16384, 16384], "Quantize_A") as [vi, vj] {
            bind(vi, i)
            bind(vj, j)
            tir.reads([A_global[vi, vj]])
            tir.writes([QA[vi, vj]])
            QA[vi, vj] = cast(int8, (@tir.round((cast(float32, A_global[vi, vj])*0.5f32), dtype=float32) - 0f32))
        }
      }
      for (i_1: int32, 0, 16384) {
        for (j_1: int32, 0, 16384) {
          block([16384, 16384], "Quantize_B") as [vi_1, vj_1] {
            bind(vi_1, i_1)
            bind(vj_1, j_1)
            tir.reads([B_global[vi_1, vj_1]])
            tir.writes([QB[vi_1, vj_1]])
            QB[vi_1, vj_1] = cast(int8, (@tir.round((cast(float32, B_global[vi_1, vj_1])*0.1f32), dtype=float32) - 0f32))
        }
      }
      for (ax0_2: int32, 0, 16384) {
        for (ax1_2: int32, 0, 16384) {
          block([16384, 16384], "QA_local") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([QA[v0_2, v1_2]])
            tir.writes([QA_local[v0_2, v1_2]])
            QA_local[v0_2, v1_2] = QA[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "QA_local_shared") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([QA_local[v0_3, v1_3]])
            tir.writes([QA_local_shared[v0_3, v1_3]])
            QA_local_shared[v0_3, v1_3] = QA_local[v0_3, v1_3]
        }
      }
      for (ax0_4: int32, 0, 16384) {
        for (ax1_4: int32, 0, 16384) {
          block([16384, 16384], "QA_local_shared_wmma.matrix_a") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([QA_local_shared[v0_4, v1_4]])
            tir.writes([QA_local_shared_wmma.matrix_a[v0_4, v1_4]])
            QA_local_shared_wmma.matrix_a[v0_4, v1_4] = QA_local_shared[v0_4, v1_4]
        }
      }
      for (ax0_5: int32, 0, 16384) {
        for (ax1_5: int32, 0, 16384) {
          block([16384, 16384], "QB_local") as [v0_5, v1_5] {
            bind(v0_5, ax0_5)
            bind(v1_5, ax1_5)
            tir.reads([QB[v0_5, v1_5]])
            tir.writes([QB_local[v0_5, v1_5]])
            QB_local[v0_5, v1_5] = QB[v0_5, v1_5]
        }
      }
      for (ax0_6: int32, 0, 16384) {
        for (ax1_6: int32, 0, 16384) {
          block([16384, 16384], "QB_local_shared") as [v0_6, v1_6] {
            bind(v0_6, ax0_6)
            bind(v1_6, ax1_6)
            tir.reads([QB_local[v0_6, v1_6]])
            tir.writes([QB_local_shared[v0_6, v1_6]])
            QB_local_shared[v0_6, v1_6] = QB_local[v0_6, v1_6]
        }
      }
      for (ax0_7: int32, 0, 16384) {
        for (ax1_7: int32, 0, 16384) {
          block([16384, 16384], "QB_local_shared_wmma.matrix_b") as [v0_7, v1_7] {
            bind(v0_7, ax0_7)
            bind(v1_7, ax1_7)
            tir.reads([QB_local_shared[v0_7, v1_7]])
            tir.writes([QB_local_shared_wmma.matrix_b[v0_7, v1_7]])
            QB_local_shared_wmma.matrix_b[v0_7, v1_7] = QB_local_shared[v0_7, v1_7]
        }
      }
      for (i_2: int32, 0, 16384) {
        for (j_2: int32, 0, 16384) {
          for (k: int32, 0, 16384) {
            block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi_2, vj_2, vk] {
              bind(vi_2, i_2)
              bind(vj_2, j_2)
              bind(vk, k)
              tir.reads([QA_local_shared_wmma.matrix_a[vi_2, vk], QB_local_shared_wmma.matrix_b[vj_2, vk]])
              tir.writes([QC_shared_wmma.accumulator[vi_2, vj_2]])
              with init() {
                QC_shared_wmma.accumulator[vi_2, vj_2] = 0
              }
              QC_shared_wmma.accumulator[vi_2, vj_2] = (QC_shared_wmma.accumulator[vi_2, vj_2] + (cast(int32, QA_local_shared_wmma.matrix_a[vi_2, vk])*cast(int32, QB_local_shared_wmma.matrix_b[vj_2, vk])))
          }
        }
      }
      for (ax0_8: int32, 0, 16384) {
        for (ax1_8: int32, 0, 16384) {
          block([16384, 16384], "QC_shared_wmma.accumulator") as [v0_8, v1_8] {
            bind(v0_8, ax0_8)
            bind(v1_8, ax1_8)
            tir.reads([QC_shared_wmma.accumulator[v0_8, v1_8]])
            tir.writes([QC_shared[v0_8, v1_8]])
            QC_shared[v0_8, v1_8] = QC_shared_wmma.accumulator[v0_8, v1_8]
        }
      }
      for (ax0_9: int32, 0, 16384) {
        for (ax1_9: int32, 0, 16384) {
          block([16384, 16384], "QC_shared") as [v0_9, v1_9] {
            bind(v0_9, ax0_9)
            bind(v1_9, ax1_9)
            tir.reads([QC_shared[v0_9, v1_9]])
            tir.writes([QC[v0_9, v1_9]])
            QC[v0_9, v1_9] = QC_shared[v0_9, v1_9]
        }
      }
      for (i_3: int32, 0, 16384) {
        for (j_3: int32, 0, 16384) {
          block([16384, 16384], "DeQuantize_C") as [vi_3, vj_3] {
            bind(vi_3, i_3)
            bind(vj_3, j_3)
            tir.reads([QC[vi_3, vj_3]])
            tir.writes([C[vi_3, vj_3]])
            C[vi_3, vj_3] = cast(int8, ((cast(float32, QC[vi_3, vj_3]) / 0.01f32) + 0f32))
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}