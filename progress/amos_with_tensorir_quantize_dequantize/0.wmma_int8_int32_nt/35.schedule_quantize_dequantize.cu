#include "hip/hip_runtime.h"
#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int8), int8, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_local = alloc_buffer(int8[16384, 16384])
    QA_local_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_local_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_local = alloc_buffer(int8[16384, 16384])
    QB_local_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_local_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_shared = alloc_buffer(int32[16384, 16384])
    QC_shared_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
     {
      for (ax0_ax1_fused_0: int32, 0, 4) "thread_binding" {
        for (ax0_ax1_fused_1: int32, 0, 2048) "thread_binding" {
          for (ax0_ax1_fused_2: int32, 0, 1) "thread_binding" {
            for (ax0_ax1_fused_3: int32, 0, 128) "thread_binding" {
              for (ax0_ax1_fused_4: int32, 0, 8) "thread_binding" {
                for (ax0_ax1_fused_5: int32, 0, 2) {
                  for (ax0_ax1_fused_6: int32, 0, 16) "vectorized" {
                    block([16384, 16384], "B_global") as [v0, v1] {
                      bind(v0, floordiv((((((((ax0_ax1_fused_0*67108864) + (ax0_ax1_fused_1*32768)) + (ax0_ax1_fused_2*32768)) + (ax0_ax1_fused_3*256)) + (ax0_ax1_fused_4*32)) + (ax0_ax1_fused_5*16)) + ax0_ax1_fused_6), 16384))
                      bind(v1, floormod((((((((ax0_ax1_fused_0*67108864) + (ax0_ax1_fused_1*32768)) + (ax0_ax1_fused_2*32768)) + (ax0_ax1_fused_3*256)) + (ax0_ax1_fused_4*32)) + (ax0_ax1_fused_5*16)) + ax0_ax1_fused_6), 16384))
                      tir.reads([B[v0, v1]])
                      tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
                      B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_ax1_fused_0_1: int32, 0, 4) "thread_binding" {
        for (ax0_ax1_fused_1_1: int32, 0, 2048) "thread_binding" {
          for (ax0_ax1_fused_2_1: int32, 0, 1) "thread_binding" {
            for (ax0_ax1_fused_3_1: int32, 0, 128) "thread_binding" {
              for (ax0_ax1_fused_4_1: int32, 0, 8) "thread_binding" {
                for (ax0_ax1_fused_5_1: int32, 0, 2) {
                  for (ax0_ax1_fused_6_1: int32, 0, 16) "vectorized" {
                    block([16384, 16384], "A_global") as [v0_1, v1_1] {
                      bind(v0_1, floordiv((((((((ax0_ax1_fused_0_1*67108864) + (ax0_ax1_fused_1_1*32768)) + (ax0_ax1_fused_2_1*32768)) + (ax0_ax1_fused_3_1*256)) + (ax0_ax1_fused_4_1*32)) + (ax0_ax1_fused_5_1*16)) + ax0_ax1_fused_6_1), 16384))
                      bind(v1_1, floormod((((((((ax0_ax1_fused_0_1*67108864) + (ax0_ax1_fused_1_1*32768)) + (ax0_ax1_fused_2_1*32768)) + (ax0_ax1_fused_3_1*256)) + (ax0_ax1_fused_4_1*32)) + (ax0_ax1_fused_5_1*16)) + ax0_ax1_fused_6_1), 16384))
                      tir.reads([A[v0_1, v1_1]])
                      tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
                      A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
                  }
                }
              }
            }
          }
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 8) "thread_binding" {
          for (j_0_0_1: int32, 0, 32) "thread_binding" {
            for (i_0_1: int32, 0, 4) "thread_binding" {
              for (j_0_1: int32, 0, 1) "thread_binding" {
                for (i_0_2_init: int32, 0, 4) {
                  for (j_0_2_init: int32, 0, 4) {
                    block([1024, 1024], "B_init_o") as [vi_o, vj_o] {
                      bind(vi_o, (((i_0_0*16) + (i_0_1*4)) + i_0_2_init))
                      bind(vj_o, ((((j_0_0_0*128) + (j_0_0_1*4)) + (j_0_1*4)) + j_0_2_init))
                      tir.reads([])
                      tir.writes([QC_shared_wmma.accumulator[vi_o, vj_o, 0:16, 0:16]])
                      C_2 = match_buffer(QC_shared_wmma.accumulator[vi_o, vj_o, 0:16, 0:16])
                      @tir.tvm_fill_fragment(C_3: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
                  }
                }
                for (k_0_0: int32, 0, 256) {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 8) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "QA_local") as [v0_2, v1_2] {
                              bind(v0_2, ((((i_0_0*256) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_0*64)) + (floordiv(ax0_0_ax1_0_ax0_1_ax1_1_fused_2, 2)*16)) + floormod(ax0_0_ax1_0_ax0_1_ax1_1_fused_3, 16)))
                              bind(v1_2, (((k_0_0*64) + (floordiv(floormod((((ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 1024), 256)*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4))
                              tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              tir.writes([QA_local[v0_2, v1_2]])
                              QA_local[v0_2, v1_2] = cast(int8, (@tir.round((cast(float32, A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)])*0.5f32), dtype=float32) - 0f32))
                            block([16384, 16384], "QA_local_shared") as [v0_3, v1_3] {
                              bind(v0_3, (((i_0_0*256) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                              bind(v1_3, (((k_0_0*64) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                              tir.reads([QA_local[v0_3, v1_3]])
                              tir.writes([QA_local_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              QA_local_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = QA_local[v0_3, v1_3]
                          }
                        }
                      }
                    }
                  }
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 2) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "QB_local") as [v0_4, v1_4] {
                              bind(v0_4, ((((j_0_0_0*2048) + (j_0_0_1*64)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*16)) + floormod(ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1, 16)))
                              bind(v1_4, ((((k_0_0*64) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*32)) + (floordiv(ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1, 16)*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1))
                              tir.reads([B_global[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              tir.writes([QB_local[v0_4, v1_4]])
                              QB_local[v0_4, v1_4] = cast(int8, (@tir.round((cast(float32, B_global[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)])*0.1f32), dtype=float32) - 0f32))
                            block([16384, 16384], "QB_local_shared") as [v0_5, v1_5] {
                              bind(v0_5, ((((j_0_0_0*2048) + (j_0_0_1*64)) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                              bind(v1_5, (((k_0_0*64) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                              tir.reads([QB_local[v0_5, v1_5]])
                              tir.writes([QB_local_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              QB_local_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = QB_local[v0_5, v1_5]
                          }
                        }
                      }
                    }
                  }
                  for (k_0_1: int32, 0, 4) {
                    for (ax0_0: int32, 0, 4) {
                      for (ax1_0: int32, 0, 1) {
                        block([1024, 1024], "QA_local_shared_wmma.matrix_a_o") as [v0_o, v1_o] {
                          bind(v0_o, (((i_0_0*16) + (i_0_1*4)) + ax0_0))
                          bind(v1_o, (((k_0_0*4) + k_0_1) + ax1_0))
                          tir.reads([QA_local_shared[v0_o, v1_o, 0:16, 0:16]])
                          tir.writes([QA_local_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16]])
                          A_2 = match_buffer(QA_local_shared[v0_o, v1_o, 0:16, 0:16])
                          C_4 = match_buffer(QA_local_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16])
                          @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_a int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_3: Pointer(shared int8), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                      }
                    }
                    for (ax0_0_1: int32, 0, 4) {
                      for (ax1_0_1: int32, 0, 1) {
                        block([1024, 1024], "QB_local_shared_wmma.matrix_b_o") as [v0_o_1, v1_o_1] {
                          bind(v0_o_1, (((j_0_0_0*128) + (j_0_0_1*4)) + ax0_0_1))
                          bind(v1_o_1, (((k_0_0*4) + k_0_1) + ax1_0_1))
                          tir.reads([QB_local_shared[v0_o_1, v1_o_1, 0:16, 0:16]])
                          tir.writes([QB_local_shared_wmma.matrix_b[v0_o_1, v1_o_1, 0:16, 0:16]])
                          A_4 = match_buffer(QB_local_shared[v0_o_1, v1_o_1, 0:16, 0:16])
                          C_6 = match_buffer(QB_local_shared_wmma.matrix_b[v0_o_1, v1_o_1, 0:16, 0:16])
                          @tir.tvm_load_matrix_sync(C_7: Pointer(wmma.matrix_b int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_3: int32, C_s0_2: int32), 16)*floordiv(C_s0_2, 16)) + floordiv(floormod(elem_offset_3, C_s0_2), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_5: Pointer(shared int8), elem_offset_4: int32, (A_s0_1: int32*16), 1, dtype=handle), A_s0_1, "col_major", dtype=handle)
                      }
                    }
                    for (i_0_2: int32, 0, 4) {
                      for (j_0_2: int32, 0, 4) {
                        block([1024, 1024, tir.reduce_axis(0, 1024)], "B_update_o") as [vi_o_1, vj_o_1, vk_o] {
                          bind(vi_o_1, (((i_0_0*16) + (i_0_1*4)) + i_0_2))
                          bind(vj_o_1, ((((j_0_0_0*128) + (j_0_0_1*4)) + (j_0_1*4)) + j_0_2))
                          bind(vk_o, ((k_0_0*4) + k_0_1))
                          tir.reads([QC_shared_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16], QA_local_shared_wmma.matrix_a[vi_o_1, vk_o, 0:16, 0:16], QB_local_shared_wmma.matrix_b[vj_o_1, vk_o, 0:16, 0:16]])
                          tir.writes([QC_shared_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16]])
                          A_6 = match_buffer(QA_local_shared_wmma.matrix_a[vi_o_1, vk_o, 0:16, 0:16])
                          B_2 = match_buffer(QB_local_shared_wmma.matrix_b[vj_o_1, vk_o, 0:16, 0:16])
                          C_8 = match_buffer(QC_shared_wmma.accumulator[vi_o_1, vj_o_1, 0:16, 0:16])
                          @tir.tvm_mma_sync(C_9: Pointer(wmma.accumulator int32), ((floordiv(floordiv(elem_offset_5: int32, C_s0_3: int32), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), A_7: Pointer(wmma.matrix_a int8), ((floordiv(floordiv(elem_offset_6: int32, A_s0_2: int32), 16)*floordiv(A_s0_2, 16)) + floordiv(floormod(elem_offset_6, A_s0_2), 16)), B_3: Pointer(wmma.matrix_b int8), ((floordiv(floordiv(elem_offset_7: int32, B_s0: int32), 16)*floordiv(B_s0, 16)) + floordiv(floormod(elem_offset_7, B_s0), 16)), C_9, ((floordiv(floordiv(elem_offset_5, C_s0_3), 16)*floordiv(C_s0_3, 16)) + floordiv(floormod(elem_offset_5, C_s0_3), 16)), dtype=handle)
                      }
                    }
                  }
                }
                for (ax0_0_2: int32, 0, 4) {
                  for (ax1_0_2: int32, 0, 4) {
                    block([1024, 1024], "QC_shared_wmma.accumulator_o") as [v0_o_2, v1_o_2] {
                      bind(v0_o_2, (((i_0_0*16) + (i_0_1*4)) + ax0_0_2))
                      bind(v1_o_2, (((j_0_0_0*128) + (j_0_0_1*4)) + ax1_0_2))
                      tir.reads([QC_shared_wmma.accumulator[v0_o_2, v1_o_2, 0:16, 0:16]])
                      tir.writes([QC_shared[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)]])
                      A_8 = match_buffer(QC_shared_wmma.accumulator[v0_o_2, v1_o_2, 0:16, 0:16])
                      C_10 = match_buffer(QC_shared[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)])
                      @tir.tvm_store_matrix_sync(A_9: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset_8: int32, A_s0_3: int32), 16)*floordiv(A_s0_3, 16)) + floordiv(floormod(elem_offset_8, A_s0_3), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int32), C_11: Pointer(shared int32), elem_offset_9: int32, (C_s0_4: int32*16), 2, dtype=handle), C_s0_4, "row_major", dtype=handle)
                  }
                }
                for (ax0: int32, 0, 64) {
                  for (ax1: int32, 0, 64) {
                    block([16384, 16384], "QC_shared") as [v0_6, v1_6] {
                      bind(v0_6, (((i_0_0*256) + (i_0_1*64)) + ax0))
                      bind(v1_6, (((j_0_0_0*2048) + (j_0_0_1*64)) + ax1))
                      tir.reads([QC_shared[v0_6, v1_6]])
                      tir.writes([C[v0_6, v1_6]])
                      C[v0_6, v1_6] = cast(int8, ((cast(float32, QC_shared[v0_6, v1_6]) / 0.01f32) + 0f32))
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}