#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ QA, signed char* __restrict__ A_global, signed char* __restrict__ QB, signed char* __restrict__ B_global, int* __restrict__ QC, signed char* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> QC_wmma_accumulator[16];
  __shared__ signed char QA_shared[16384];
  __shared__ signed char QB_shared[4096];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> QA_shared_wmma_matrix_a[4];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> QB_shared_wmma_matrix_b[4];
  for (int ax0 = 0; ax0 < 256; ++ax0) {
    for (int ax1 = 0; ax1 < 16384; ++ax1) {
      QA[(((((int)blockIdx.y) * 4194304) + (ax0 * 16384)) + ax1)] = ((signed char)roundf((((float)A_global[(((((((int)blockIdx.y) * 4194304) + ((ax0 >> 4) * 262144)) + ((ax1 >> 4) * 256)) + ((ax0 & 15) * 16)) + (ax1 & 15))]) * 5.000000e-01f)));
    }
  }
  for (int ax0_1 = 0; ax0_1 < 64; ++ax0_1) {
    for (int ax1_1 = 0; ax1_1 < 16384; ++ax1_1) {
      QB[((((((int)blockIdx.z) * 33554432) + (((int)blockIdx.x) * 1048576)) + (ax0_1 * 16384)) + ax1_1)] = ((signed char)roundf((((float)B_global[((((((((int)blockIdx.z) * 33554432) + (((int)blockIdx.x) * 1048576)) + ((ax0_1 >> 4) * 262144)) + ((ax1_1 >> 4) * 256)) + ((ax0_1 & 15) * 16)) + (ax1_1 & 15))]) * 1.000000e-01f)));
    }
  }
  for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 4; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(QC_wmma_accumulator[((i_0_2_init * 4) + j_0_2_init)], 0.000000e+00f);
    }
  }
  for (int k_0_0 = 0; k_0_0 < 256; ++k_0_0) {
    __syncthreads();
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 8; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
      *(int4*)(QA_shared + (((((int)threadIdx.y) * 4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(QA + (((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + ((ax0_0_ax1_0_ax0_1_ax1_1_fused_2 >> 1) * 262144)) + ((((int)threadIdx.x) & 15) * 16384)) + (k_0_0 * 64)) + ((ax0_0_ax1_0_ax0_1_ax1_1_fused_2 & 1) * 32)) + ((((int)threadIdx.x) >> 4) * 16)));
    }
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1) {
      *(int4*)(QB_shared + (((((int)threadIdx.y) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(QB + (((((((((int)blockIdx.z) * 33554432) + (((int)blockIdx.x) * 1048576)) + (((int)threadIdx.y) * 262144)) + ((((int)threadIdx.x) & 15) * 16384)) + (k_0_0 * 64)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 32)) + ((((int)threadIdx.x) >> 4) * 16)));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(QA_shared_wmma_matrix_a[ax0_0], (&(QA_shared[(((((int)threadIdx.y) * 4096) + (ax0_0 * 1024)) + (k_0_1 * 256))])), 16);
      }
      for (int ax0_0_1 = 0; ax0_0_1 < 4; ++ax0_0_1) {
        nvcuda::wmma::load_matrix_sync(QB_shared_wmma_matrix_b[ax0_0_1], (&(QB_shared[((ax0_0_1 * 1024) + (k_0_1 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 4; ++j_0_2) {
          nvcuda::wmma::mma_sync(QC_wmma_accumulator[((i_0_2 * 4) + j_0_2)], QA_shared_wmma_matrix_a[i_0_2], QB_shared_wmma_matrix_b[j_0_2], QC_wmma_accumulator[((i_0_2 * 4) + j_0_2)]);
        }
      }
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2) {
    for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
      nvcuda::wmma::store_matrix_sync((&(QC[((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_0_2 * 262144)) + (((int)blockIdx.z) * 2048)) + (((int)blockIdx.x) * 64)) + (ax1_0 * 16))])), QC_wmma_accumulator[((ax0_0_2 * 4) + ax1_0)], 16384, nvcuda::wmma::mem_row_major);
    }
  }
  for (int ax0_2 = 0; ax0_2 < 64; ++ax0_2) {
    for (int ax1_2 = 0; ax1_2 < 64; ++ax1_2) {
      C[((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_2 * 16384)) + (((int)blockIdx.z) * 2048)) + (((int)blockIdx.x) * 64)) + ax1_2)] = ((signed char)(((float)QC[((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_2 * 16384)) + (((int)blockIdx.z) * 2048)) + (((int)blockIdx.x) * 64)) + ax1_2)]) * 1.000000e+02f));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(B_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(B + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(A_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(A + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

