#include "hip/hip_runtime.h"
#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int8), int8, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QA = alloc_buffer(int8[16384, 16384])
    QB = alloc_buffer(int8[16384, 16384])
    QC = alloc_buffer(int32[16384, 16384])
     {
      for (i: int32, 0, 16384) {
        for (j: int32, 0, 16384) {
          block([16384, 16384], "Quantize_A") as [vi, vj] {
            bind(vi, i)
            bind(vj, j)
            tir.reads([A[vi, vj]])
            tir.writes([QA[vi, vj]])
            QA[vi, vj] = cast(int8, (@tir.round((cast(float32, A[vi, vj])*0.5f32), dtype=float32) - 0f32))
        }
      }
      for (i_1: int32, 0, 16384) {
        for (j_1: int32, 0, 16384) {
          block([16384, 16384], "Quantize_B") as [vi_1, vj_1] {
            bind(vi_1, i_1)
            bind(vj_1, j_1)
            tir.reads([B[vi_1, vj_1]])
            tir.writes([QB[vi_1, vj_1]])
            QB[vi_1, vj_1] = cast(int8, (@tir.round((cast(float32, B[vi_1, vj_1])*0.1f32), dtype=float32) - 0f32))
        }
      }
      for (i_2: int32, 0, 16384) {
        for (j_2: int32, 0, 16384) {
          for (k: int32, 0, 16384) {
            block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi_2, vj_2, vk] {
              bind(vi_2, i_2)
              bind(vj_2, j_2)
              bind(vk, k)
              tir.reads([QA[vi_2, vk], QB[vj_2, vk]])
              tir.writes([QC[vi_2, vj_2]])
              with init() {
                QC[vi_2, vj_2] = 0
              }
              QC[vi_2, vj_2] = (QC[vi_2, vj_2] + (cast(int32, QA[vi_2, vk])*cast(int32, QB[vj_2, vk])))
          }
        }
      }
      for (i_3: int32, 0, 16384) {
        for (j_3: int32, 0, 16384) {
          block([16384, 16384], "DeQuantize_C") as [vi_3, vj_3] {
            bind(vi_3, i_3)
            bind(vj_3, j_3)
            tir.reads([QC[vi_3, vj_3]])
            tir.writes([C[vi_3, vj_3]])
            C[vi_3, vj_3] = cast(int8, ((cast(float32, QC[vi_3, vj_3]) / 0.01f32) + 0f32))
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}