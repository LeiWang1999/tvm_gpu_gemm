#include "hip/hip_runtime.h"
#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int8), int8, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    QA = alloc_buffer(int8[16384, 16384])
    QB = alloc_buffer(int8[16384, 16384])
    QC = alloc_buffer(int32[16384, 16384])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QA_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    QB_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    QC_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 64) "thread_binding" {
        for (j_0_0_0: int32, 0, 8) "thread_binding" {
          for (j_0_0_1: int32, 0, 32) "thread_binding" {
            for (i_0_1: int32, 0, 4) "thread_binding" {
              for (j_0_1: int32, 0, 1) "thread_binding" {
                for (ax0_2: int32, 0, 256) {
                  for (ax1_2: int32, 0, 16384) {
                    block([16384, 16384], "Quantize_A") as [vi, vj] {
                      bind(vi, ((i_0_0*256) + ax0_2))
                      bind(vj, ax1_2)
                      tir.reads([A_global[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                      tir.writes([QA[vi, vj]])
                      QA[vi, vj] = cast(int8, (@tir.round((cast(float32, A_global[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)])*0.5f32), dtype=float32) - 0f32))
                  }
                }
                for (ax0_3: int32, 0, 64) {
                  for (ax1_3: int32, 0, 16384) {
                    block([16384, 16384], "Quantize_B") as [vi_1, vj_1] {
                      bind(vi_1, (((j_0_0_0*2048) + (j_0_0_1*64)) + ax0_3))
                      bind(vj_1, ax1_3)
                      tir.reads([B_global[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)]])
                      tir.writes([QB[vi_1, vj_1]])
                      QB[vi_1, vj_1] = cast(int8, (@tir.round((cast(float32, B_global[floordiv(vi_1, 16), floordiv(vj_1, 16), floormod(vi_1, 16), floormod(vj_1, 16)])*0.1f32), dtype=float32) - 0f32))
                  }
                }
                for (i_0_2_init: int32, 0, 4) {
                  for (j_0_2_init: int32, 0, 4) {
                    for (i_1_init: int32, 0, 16) {
                      for (j_1_init: int32, 0, 16) {
                        block([16384, 16384], "B_init") as [vi_2, vj_2] {
                          bind(vi_2, ((((i_0_0*256) + (i_0_1*64)) + (i_0_2_init*16)) + i_1_init))
                          bind(vj_2, (((((j_0_0_0*2048) + (j_0_0_1*64)) + (j_0_1*64)) + (j_0_2_init*16)) + j_1_init))
                          tir.reads([])
                          tir.writes([QC_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)]])
                          QC_wmma.accumulator[floordiv(vi_2, 16), floordiv(vj_2, 16), floormod(vi_2, 16), floormod(vj_2, 16)] = 0
                      }
                    }
                  }
                }
                for (k_0_0: int32, 0, 256) {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 8) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "QA_shared") as [v0_2, v1_2] {
                              bind(v0_2, (((i_0_0*256) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                              bind(v1_2, (((k_0_0*64) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*4096)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                              tir.reads([QA[v0_2, v1_2]])
                              tir.writes([QA_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                              QA_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = QA[v0_2, v1_2]
                          }
                        }
                      }
                    }
                  }
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 1) "thread_binding" {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 2) {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                          for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 16) "vectorized" {
                            block([16384, 16384], "QB_shared") as [v0_3, v1_3] {
                              bind(v0_3, ((((j_0_0_0*2048) + (j_0_0_1*64)) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                              bind(v1_3, (((k_0_0*64) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 1024), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                              tir.reads([QB[v0_3, v1_3]])
                              tir.writes([QB_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                              QB_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = QB[v0_3, v1_3]
                          }
                        }
                      }
                    }
                  }
                  for (k_0_1: int32, 0, 4) {
                    for (ax0_0: int32, 0, 4) {
                      for (ax1_0: int32, 0, 1) {
                        for (ax0_1_1: int32, 0, 16) {
                          for (ax1_1_1: int32, 0, 16) {
                            block([16384, 16384], "QA_shared_wmma.matrix_a") as [v0_4, v1_4] {
                              bind(v0_4, ((((i_0_0*256) + (i_0_1*64)) + (ax0_0*16)) + ax0_1_1))
                              bind(v1_4, ((((k_0_0*64) + (k_0_1*16)) + (ax1_0*16)) + ax1_1_1))
                              tir.reads([QA_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              tir.writes([QA_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                              QA_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = QA_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                          }
                        }
                      }
                    }
                    for (ax0_0_1: int32, 0, 4) {
                      for (ax1_0_1: int32, 0, 1) {
                        for (ax0_1_2: int32, 0, 16) {
                          for (ax1_1_2: int32, 0, 16) {
                            block([16384, 16384], "QB_shared_wmma.matrix_b") as [v0_5, v1_5] {
                              bind(v0_5, ((((j_0_0_0*2048) + (j_0_0_1*64)) + (ax0_0_1*16)) + ax0_1_2))
                              bind(v1_5, ((((k_0_0*64) + (k_0_1*16)) + (ax1_0_1*16)) + ax1_1_2))
                              tir.reads([QB_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              tir.writes([QB_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                              QB_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = QB_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                          }
                        }
                      }
                    }
                    for (i_0_2: int32, 0, 4) {
                      for (j_0_2: int32, 0, 4) {
                        for (i_1: int32, 0, 16) {
                          for (j_1: int32, 0, 16) {
                            for (k_1: int32, 0, 16) {
                              block([16384, 16384, tir.reduce_axis(0, 16384)], "B_update") as [vi_3, vj_3, vk] {
                                bind(vi_3, ((((i_0_0*256) + (i_0_1*64)) + (i_0_2*16)) + i_1))
                                bind(vj_3, (((((j_0_0_0*2048) + (j_0_0_1*64)) + (j_0_1*64)) + (j_0_2*16)) + j_1))
                                bind(vk, (((k_0_0*64) + (k_0_1*16)) + k_1))
                                tir.reads([QC_wmma.accumulator[floordiv(vi_3, 16), floordiv(vj_3, 16), floormod(vi_3, 16), floormod(vj_3, 16)], QA_shared_wmma.matrix_a[floordiv(vi_3, 16), floordiv(vk, 16), floormod(vi_3, 16), floormod(vk, 16)], QB_shared_wmma.matrix_b[floordiv(vj_3, 16), floordiv(vk, 16), floormod(vj_3, 16), floormod(vk, 16)]])
                                tir.writes([QC_wmma.accumulator[floordiv(vi_3, 16), floordiv(vj_3, 16), floormod(vi_3, 16), floormod(vj_3, 16)]])
                                QC_wmma.accumulator[floordiv(vi_3, 16), floordiv(vj_3, 16), floormod(vi_3, 16), floormod(vj_3, 16)] = (QC_wmma.accumulator[floordiv(vi_3, 16), floordiv(vj_3, 16), floormod(vi_3, 16), floormod(vj_3, 16)] + (cast(int32, QA_shared_wmma.matrix_a[floordiv(vi_3, 16), floordiv(vk, 16), floormod(vi_3, 16), floormod(vk, 16)])*cast(int32, QB_shared_wmma.matrix_b[floordiv(vj_3, 16), floordiv(vk, 16), floormod(vj_3, 16), floormod(vk, 16)])))
                            }
                          }
                        }
                      }
                    }
                  }
                }
                for (ax0_0_2: int32, 0, 4) {
                  for (ax1_0_2: int32, 0, 4) {
                    for (ax0_1_3: int32, 0, 16) {
                      for (ax1_1_3: int32, 0, 16) {
                        block([16384, 16384], "QC_wmma.accumulator") as [v0_6, v1_6] {
                          bind(v0_6, ((((i_0_0*256) + (i_0_1*64)) + (ax0_0_2*16)) + ax0_1_3))
                          bind(v1_6, ((((j_0_0_0*2048) + (j_0_0_1*64)) + (ax1_0_2*16)) + ax1_1_3))
                          tir.reads([QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                          tir.writes([QC[v0_6, v1_6]])
                          QC[v0_6, v1_6] = QC_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]
                      }
                    }
                  }
                }
                for (ax0_4: int32, 0, 64) {
                  for (ax1_4: int32, 0, 64) {
                    block([16384, 16384], "DeQuantize_C") as [vi_4, vj_4] {
                      bind(vi_4, (((i_0_0*256) + (i_0_1*64)) + ax0_4))
                      bind(vj_4, (((j_0_0_0*2048) + (j_0_0_1*64)) + ax1_4))
                      tir.reads([QC[vi_4, vj_4]])
                      tir.writes([C[vi_4, vj_4]])
                      C[vi_4, vj_4] = cast(int8, ((cast(float32, QC[vi_4, vj_4]) / 0.01f32) + 0f32))
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}