#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float32), float32, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float32), float32, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float32), float32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_local = alloc_buffer(float32[16384, 16384])
    A_local_shared = alloc_buffer(float32[16384, 16384])
    A_local_shared_local = alloc_buffer(float32[16384, 16384])
    B_shared = alloc_buffer(float32[16384, 16384])
    B_shared_local = alloc_buffer(float32[16384, 16384])
    C_local = alloc_buffer(float32[16384, 16384])
    for (i_0: int32, 0, 512) "thread_binding" {
      for (j_0: int32, 0, 64) "thread_binding" {
        for (i_1: int32, 0, 2) "thread_binding" {
          for (j_1: int32, 0, 2) "thread_binding" {
            for (i_2: int32, 0, 4) "thread_binding" {
              for (j_2: int32, 0, 32) "thread_binding" {
                for (k_0: int32, 0, 512) {
                  for (ax0_ax1_0_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_0_fused_2: int32, 0, 32) "thread_binding" {
                      for (ax0_ax1_0_fused_0: int32, 0, 2) {
                        for (ax1_1: int32, 0, 4) "vectorized" {
                          block([16384, 16384], "A_local") as [v0, v1] {
                            bind(v0, ((i_0*32) + floordiv((((ax0_ax1_0_fused_0*128) + (ax0_ax1_0_fused_1*32)) + ax0_ax1_0_fused_2), 8)))
                            bind(v1, (((k_0*32) + (floormod((((ax0_ax1_0_fused_0*128) + (ax0_ax1_0_fused_1*32)) + ax0_ax1_0_fused_2), 8)*4)) + ax1_1))
                            tir.reads([A[v0, v1]])
                            tir.writes([A_local[v0, v1]])
                            A_local[v0, v1] = A[v0, v1]
                        }
                        for (ax0: int32, 0, 4) {
                          block([16384, 16384], "A_local_shared") as [v0_1, v1_1] {
                            bind(v0_1, ((i_0*32) + floordiv((((ax0_ax1_0_fused_0*128) + (ax0_ax1_0_fused_1*32)) + ax0_ax1_0_fused_2), 8)))
                            bind(v1_1, (((k_0*32) + (floormod((((ax0_ax1_0_fused_0*128) + (ax0_ax1_0_fused_1*32)) + ax0_ax1_0_fused_2), 8)*4)) + ax0))
                            tir.reads([A_local[v0_1, v1_1]])
                            tir.writes([A_local_shared[v1_1, v0_1]])
                            A_local_shared[v1_1, v0_1] = A_local[v0_1, v1_1]
                        }
                      }
                    }
                  }
                  for (ax0_ax1_fused_0: int32, 0, 16) {
                    for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
                      for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                        for (ax0_ax1_fused_3: int32, 0, 4) "vectorized" {
                          block([16384, 16384], "B_shared") as [v0_2, v1_2] {
                            bind(v0_2, ((k_0*32) + floordiv(((((ax0_ax1_fused_0*512) + (ax0_ax1_fused_1*128)) + (ax0_ax1_fused_2*4)) + ax0_ax1_fused_3), 256)))
                            bind(v1_2, ((j_0*256) + floormod(((((ax0_ax1_fused_0*512) + (ax0_ax1_fused_1*128)) + (ax0_ax1_fused_2*4)) + ax0_ax1_fused_3), 256)))
                            tir.reads([B[v0_2, v1_2]])
                            tir.writes([B_shared[v0_2, v1_2]])
                            B_shared[v0_2, v1_2] = B[v0_2, v1_2]
                        }
                      }
                    }
                  }
                  for (k_1: int32, 0, 32) {
                    for (ax0_1: int32, 0, 4) {
                      block([16384, 16384], "A_local_shared_local") as [v0_3, v1_3] {
                        bind(v0_3, ((((i_0*32) + (i_1*16)) + (i_2*4)) + ax0_1))
                        bind(v1_3, ((k_0*32) + k_1))
                        tir.reads([A_local_shared[v1_3, v0_3]])
                        tir.writes([A_local_shared_local[v0_3, v1_3]])
                        A_local_shared_local[v0_3, v1_3] = A_local_shared[v1_3, v0_3]
                    }
                    for (ax0_2: int32, 0, 4) {
                      block([16384, 16384], "B_shared_local") as [v0_4, v1_4] {
                        bind(v0_4, ((k_0*32) + k_1))
                        bind(v1_4, ((((j_0*256) + (j_1*128)) + (j_2*4)) + ax0_2))
                        tir.reads([B_shared[v0_4, v1_4]])
                        tir.writes([B_shared_local[v0_4, v1_4]])
                        B_shared_local[v0_4, v1_4] = B_shared[v0_4, v1_4]
                    }
                    for (i_3: int32, 0, 4) {
                      for (j_3: int32, 0, 4) {
                        block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                          bind(vi, ((((i_0*32) + (i_1*16)) + (i_2*4)) + i_3))
                          bind(vj, ((((j_0*256) + (j_1*128)) + (j_2*4)) + j_3))
                          bind(vk, ((k_0*32) + k_1))
                          tir.reads([A_local_shared_local[vi, vk], B_shared_local[vk, vj]])
                          tir.writes([C_local[vi, vj]])
                          with init() {
                            C_local[vi, vj] = 0f32
                          }
                          C_local[vi, vj] = (C_local[vi, vj] + (A_local_shared_local[vi, vk]*B_shared_local[vk, vj]))
                      }
                    }
                  }
                }
                for (ax0_3: int32, 0, 4) {
                  for (ax1: int32, 0, 4) {
                    block([16384, 16384], "C_local") as [v0_5, v1_5] {
                      bind(v0_5, ((((i_0*32) + (i_1*16)) + (i_2*4)) + ax0_3))
                      bind(v1_5, ((((j_0*256) + (j_1*128)) + (j_2*4)) + ax1))
                      tir.reads([C_local[v0_5, v1_5]])
                      tir.writes([C[v0_5, v1_5]])
                      C[v0_5, v1_5] = C_local[v0_5, v1_5]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}