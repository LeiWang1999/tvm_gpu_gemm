
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  float C_local[64];
  __shared__ float A_local_shared[1024];
  float A_local[4];
  __shared__ float4 B_shared[2048];
  float A_local_shared_local[8];
  float B_shared_local[8];
  for (int i_3_init = 0; i_3_init < 4; ++i_3_init) {
    for (int j_3_init = 0; j_3_init < 4; ++j_3_init) {
      C_local[((i_3_init * 4) + j_3_init)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 32)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 16)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int k_0 = 0; k_0 < 512; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_0_fused_0 = 0; ax0_ax1_0_fused_0 < 2; ++ax0_ax1_0_fused_0) {
      *(float4*)(A_local + 0) = *(float4*)(A + ((((((((int)blockIdx.y) * 524288) + (ax0_ax1_0_fused_0 * 262144)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (k_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)));
      for (int ax0 = 0; ax0 < 4; ++ax0) {
        A_local_shared[((((((((int)threadIdx.x) & 7) * 128) + (ax0 * 32)) + (ax0_ax1_0_fused_0 * 16)) + (((int)threadIdx.y) * 4)) + (((int)threadIdx.x) >> 3))] = A_local[ax0];
      }
    }
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 16; ++ax0_ax1_fused_0) {
      B_shared[(((ax0_ax1_fused_0 * 128) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x))] = *(float4*)(B + ((((((k_0 * 524288) + (ax0_ax1_fused_0 * 32768)) + ((((int)threadIdx.y) >> 1) * 16384)) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.y) & 1) * 128)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 32; ++k_1) {
      *(float4*)(A_local_shared_local + 0) = *(float4*)(A_local_shared + ((k_1 * 32) + (((int)threadIdx.y) * 4)));
      *(float4*)(A_local_shared_local + 4) = *(float4*)(A_local_shared + (((k_1 * 32) + (((int)threadIdx.y) * 4)) + 16));
      *(float4*)(B_shared_local + 0) = B_shared[((k_1 * 64) + ((int)threadIdx.x))];
      *(float4*)(B_shared_local + 4) = B_shared[(((k_1 * 64) + ((int)threadIdx.x)) + 32)];
      for (int i_3 = 0; i_3 < 4; ++i_3) {
        for (int j_3 = 0; j_3 < 4; ++j_3) {
          C_local[((i_3 * 4) + j_3)] = (C_local[((i_3 * 4) + j_3)] + (A_local_shared_local[i_3] * B_shared_local[j_3]));
          C_local[(((i_3 * 4) + j_3) + 32)] = (C_local[(((i_3 * 4) + j_3) + 32)] + (A_local_shared_local[(i_3 + 4)] * B_shared_local[j_3]));
          C_local[(((i_3 * 4) + j_3) + 16)] = (C_local[(((i_3 * 4) + j_3) + 16)] + (A_local_shared_local[i_3] * B_shared_local[(j_3 + 4)]));
          C_local[(((i_3 * 4) + j_3) + 48)] = (C_local[(((i_3 * 4) + j_3) + 48)] + (A_local_shared_local[(i_3 + 4)] * B_shared_local[(j_3 + 4)]));
        }
      }
    }
  }
  for (int ax0_1 = 0; ax0_1 < 4; ++ax0_1) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      C[((((((((int)blockIdx.y) * 524288) + (((int)threadIdx.y) * 65536)) + (ax0_1 * 16384)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 4)) + ax1)] = C_local[((ax0_1 * 4) + ax1)];
      C[(((((((((int)blockIdx.y) * 524288) + (((int)threadIdx.y) * 65536)) + (ax0_1 * 16384)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 4)) + ax1) + 262144)] = C_local[(((ax0_1 * 4) + ax1) + 32)];
      C[(((((((((int)blockIdx.y) * 524288) + (((int)threadIdx.y) * 65536)) + (ax0_1 * 16384)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 4)) + ax1) + 128)] = C_local[(((ax0_1 * 4) + ax1) + 16)];
      C[(((((((((int)blockIdx.y) * 524288) + (((int)threadIdx.y) * 65536)) + (ax0_1 * 16384)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 4)) + ax1) + 262272)] = C_local[(((ax0_1 * 4) + ax1) + 48)];
    }
  }
}

