#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {AT: Buffer(AT_1: Pointer(global float32), float32, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global float32), float32, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global float32), float32, [16384, 16384], [])}
  buffer_map = {a: AT, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    AT_shared = alloc_buffer(float32[16384, 16384])
    AT_shared_local = alloc_buffer(float32[16384, 16384])
    B_shared = alloc_buffer(float32[16384, 16384])
    B_shared_local = alloc_buffer(float32[16384, 16384])
    C_local = alloc_buffer(float32[16384, 16384])
    for (i_0: int32, 0, 128) "thread_binding" {
      for (j_0: int32, 0, 128) "thread_binding" {
        for (i_1: int32, 0, 2) "thread_binding" {
          for (j_1: int32, 0, 2) "thread_binding" {
            for (i_2: int32, 0, 16) "thread_binding" {
              for (j_2: int32, 0, 16) "thread_binding" {
                for (k_0: int32, 0, 1024) {
                  for (ax0_ax1_fused_0: int32, 0, 2) {
                    for (ax0_ax1_fused_1: int32, 0, 16) "thread_binding" {
                      for (ax0_ax1_fused_2: int32, 0, 16) "thread_binding" {
                        for (ax0_ax1_fused_3: int32, 0, 4) "vectorized" {
                          block([16384, 16384], "AT_shared") as [v0, v1] {
                            bind(v0, ((k_0*16) + floordiv(((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*64)) + (ax0_ax1_fused_2*4)) + ax0_ax1_fused_3), 128)))
                            bind(v1, ((i_0*128) + floormod(((((ax0_ax1_fused_0*1024) + (ax0_ax1_fused_1*64)) + (ax0_ax1_fused_2*4)) + ax0_ax1_fused_3), 128)))
                            tir.reads([AT[v0, v1]])
                            tir.writes([AT_shared[v0, v1]])
                            AT_shared[v0, v1] = AT[v0, v1]
                        }
                      }
                    }
                  }
                  for (ax0: int32, 0, 16) {
                    for (ax1: int32, 0, 128) {
                      block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                        bind(v0_1, ((k_0*16) + ax0))
                        bind(v1_1, ((j_0*128) + ax1))
                        tir.reads([B[v0_1, v1_1]])
                        tir.writes([B_shared[v0_1, v1_1]])
                        B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                    }
                  }
                  for (k_1: int32, 0, 16) {
                    for (ax0_1: int32, 0, 4) {
                      block([16384, 16384], "AT_shared_local") as [v0_2, v1_2] {
                        bind(v0_2, ((k_0*16) + k_1))
                        bind(v1_2, ((((i_0*128) + (i_1*64)) + (i_2*4)) + ax0_1))
                        tir.reads([AT_shared[v0_2, v1_2]])
                        tir.writes([AT_shared_local[v0_2, v1_2]])
                        AT_shared_local[v0_2, v1_2] = AT_shared[v0_2, v1_2]
                    }
                    for (ax0_2: int32, 0, 4) {
                      block([16384, 16384], "B_shared_local") as [v0_3, v1_3] {
                        bind(v0_3, ((k_0*16) + k_1))
                        bind(v1_3, ((((j_0*128) + (j_1*64)) + (j_2*4)) + ax0_2))
                        tir.reads([B_shared[v0_3, v1_3]])
                        tir.writes([B_shared_local[v0_3, v1_3]])
                        B_shared_local[v0_3, v1_3] = B_shared[v0_3, v1_3]
                    }
                    for (i_3: int32, 0, 4) {
                      for (j_3: int32, 0, 4) {
                        block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                          bind(vi, ((((i_0*128) + (i_1*64)) + (i_2*4)) + i_3))
                          bind(vj, ((((j_0*128) + (j_1*64)) + (j_2*4)) + j_3))
                          bind(vk, ((k_0*16) + k_1))
                          tir.reads([AT_shared_local[vk, vi], B_shared_local[vk, vj]])
                          tir.writes([C_local[vi, vj]])
                          with init() {
                            C_local[vi, vj] = 0f32
                          }
                          C_local[vi, vj] = (C_local[vi, vj] + (AT_shared_local[vk, vi]*B_shared_local[vk, vj]))
                      }
                    }
                  }
                }
                for (ax0_3: int32, 0, 4) {
                  for (ax1_1: int32, 0, 4) {
                    block([16384, 16384], "C_local") as [v0_4, v1_4] {
                      bind(v0_4, ((((i_0*128) + (i_1*64)) + (i_2*4)) + ax0_3))
                      bind(v1_4, ((((j_0*128) + (j_1*64)) + (j_2*4)) + ax1_1))
                      tir.reads([C_local[v0_4, v1_4]])
                      tir.writes([C[v0_4, v1_4]])
                      C[v0_4, v1_4] = C_local[v0_4, v1_4]
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}