
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) main_kernel0(float* __restrict__ AT, float* __restrict__ B, float* __restrict__ C) {
  float C_local[64];
  __shared__ float4 AT_shared[512];
  __shared__ float4 B_shared[512];
  float AT_shared_local[8];
  float B_shared_local[8];
  for (int i_3_init = 0; i_3_init < 4; ++i_3_init) {
    for (int j_3_init = 0; j_3_init < 4; ++j_3_init) {
      C_local[((i_3_init * 4) + j_3_init)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 32)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 16)] = 0.000000e+00f;
      C_local[(((i_3_init * 4) + j_3_init) + 48)] = 0.000000e+00f;
    }
  }

  const int MAX_BLOCK_N = 1;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int k_0 = 0; k_0 < 1024; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
      AT_shared[(((ax0_ax1_fused_0 * 256) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = *(float4*)(AT + ((((((k_0 * 262144) + (ax0_ax1_fused_0 * 131072)) + ((((int)threadIdx.y) >> 1) * 16384)) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.y) & 1) * 64)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 2; ++ax0_ax1_fused_0_1) {
      B_shared[(((ax0_ax1_fused_0_1 * 256) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = *(float4*)(B + ((((((k_0 * 262144) + (ax0_ax1_fused_0_1 * 131072)) + ((((int)threadIdx.y) >> 1) * 16384)) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.y) & 1) * 64)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 16; ++k_1) {
      *(float4*)(AT_shared_local + 0) = AT_shared[((k_1 * 32) + ((int)threadIdx.y))];
      *(float4*)(AT_shared_local + 4) = AT_shared[(((k_1 * 32) + ((int)threadIdx.y)) + 16)];
      *(float4*)(B_shared_local + 0) = B_shared[((k_1 * 32) + ((int)threadIdx.x))];
      *(float4*)(B_shared_local + 4) = B_shared[(((k_1 * 32) + ((int)threadIdx.x)) + 16)];
      for (int i_3 = 0; i_3 < 4; ++i_3) {
        for (int j_3 = 0; j_3 < 4; ++j_3) {
          C_local[((i_3 * 4) + j_3)] = (C_local[((i_3 * 4) + j_3)] + (AT_shared_local[i_3] * B_shared_local[j_3]));
          C_local[(((i_3 * 4) + j_3) + 32)] = (C_local[(((i_3 * 4) + j_3) + 32)] + (AT_shared_local[(i_3 + 4)] * B_shared_local[j_3]));
          C_local[(((i_3 * 4) + j_3) + 16)] = (C_local[(((i_3 * 4) + j_3) + 16)] + (AT_shared_local[i_3] * B_shared_local[(j_3 + 4)]));
          C_local[(((i_3 * 4) + j_3) + 48)] = (C_local[(((i_3 * 4) + j_3) + 48)] + (AT_shared_local[(i_3 + 4)] * B_shared_local[(j_3 + 4)]));
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 4; ++ax0) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      C[((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.y) * 65536)) + (ax0 * 16384)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 4)) + ax1)] = C_local[((ax0 * 4) + ax1)];
      C[(((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.y) * 65536)) + (ax0 * 16384)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 4)) + ax1) + 1048576)] = C_local[(((ax0 * 4) + ax1) + 32)];
      C[(((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.y) * 65536)) + (ax0 * 16384)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 4)) + ax1) + 64)] = C_local[(((ax0 * 4) + ax1) + 16)];
      C[(((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.y) * 65536)) + (ax0 * 16384)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 4)) + ax1) + 1048640)] = C_local[(((ax0 * 4) + ax1) + 48)];
    }
  }
}

