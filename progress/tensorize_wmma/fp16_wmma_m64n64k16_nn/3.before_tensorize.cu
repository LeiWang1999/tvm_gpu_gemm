@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [4096], []),
             B: Buffer(B_2: Pointer(float16), float16, [4096], []),
             C: Buffer(C_2: Pointer(float32), float32, [4096], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [64, 64], []), B_1: B_3: Buffer(B_2, float16, [64, 64], []), C_1: C_3: Buffer(C_2, float32, [64, 64], [])} {
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 1;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [4096]), storage_scope = wmma.accumulator;
  allocate(A.shared: Pointer(shared float16), float16, [1024]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float16), float16, [1024]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [1024]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [1024]), storage_scope = wmma.matrix_b;
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 1;
  attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 1;
  attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 1 {
    for (ii.c.outer.init: int32, 0, 4) {
      for (jj.c.outer.init: int32, 0, 4) {
        for (ii.c.inner.init: int32, 0, 16) {
          for (jj.c.inner.init: int32, 0, 16) {
            C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [4096], [], scope="wmma.accumulator")[((((ii.c.outer.init*1024) + (ii.c.inner.init*64)) + (jj.c.outer.init*16)) + jj.c.inner.init)] = 0f32
          }
        }
      }
    }
    for (rk.outer.outer: int32, 0, 4) {
      for (ax0.ax1.fused.outer.outer.outer.outer: int32, 0, 4) {
        attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
        A.shared_1: Buffer(A.shared, float16, [1024], [], scope="shared")[ramp(((ax0.ax1.fused.outer.outer.outer.outer*256) + (threadIdx.x*8)), 1, 8)] = A[ramp(((((ax0.ax1.fused.outer.outer.outer.outer*1024) + (floordiv(threadIdx.x, 2)*64)) + (rk.outer.outer*16)) + (floormod(threadIdx.x, 2)*8)), 1, 8)]
      }
      for (ax0.ax1.fused.outer.outer.outer.outer_1: int32, 0, 4) {
        let cse_var_1: int32 = (ax0.ax1.fused.outer.outer.outer.outer_1*256)
        attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
        B.shared_1: Buffer(B.shared, float16, [1024], [], scope="shared")[ramp((cse_var_1 + (threadIdx.x*8)), 1, 8)] = B[ramp((((rk.outer.outer*1024) + cse_var_1) + (threadIdx.x*8)), 1, 8)]
      }
      for (ax0.outer: int32, 0, 4) {
        for (ax0.inner: int32, 0, 16) {
          for (ax1.inner: int32, 0, 16) {
            let cse_var_2: int32 = (((ax0.outer*256) + (ax0.inner*16)) + ax1.inner)
            A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [1024], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
          }
        }
      }
      for (ax1.outer: int32, 0, 4) {
        for (ax0.inner_1: int32, 0, 16) {
          for (ax1.inner_1: int32, 0, 16) {
            let cse_var_3: int32 = (((ax0.inner_1*64) + (ax1.outer*16)) + ax1.inner_1)
            B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [1024], [], scope="wmma.matrix_b")[cse_var_3] = B.shared_1[cse_var_3]
          }
        }
      }
      for (ii.c.outer: int32, 0, 4) {
        for (jj.c.outer: int32, 0, 4) {
          for (ii.c.inner: int32, 0, 16) {
            for (jj.c.inner: int32, 0, 16) {
              for (rk.inner: int32, 0, 16) {
                let cse_var_5: int32 = (jj.c.outer*16)
                let cse_var_4: int32 = ((((ii.c.outer*1024) + (ii.c.inner*64)) + cse_var_5) + jj.c.inner)
                C.wmma.accumulator_1[cse_var_4] = (C.wmma.accumulator_1[cse_var_4] + (cast(float32, A.shared.wmma.matrix_a_1[(((ii.c.outer*256) + (ii.c.inner*16)) + rk.inner)])*cast(float32, B.shared.wmma.matrix_b_1[(((rk.inner*64) + cse_var_5) + jj.c.inner)])))
              }
            }
          }
        }
      }
    }
    for (ii.inner.outer.inner: int32, 0, 4) {
      for (jj.inner.outer.inner: int32, 0, 4) {
        for (ii.inner.inner: int32, 0, 16) {
          for (jj.inner.inner: int32, 0, 16) {
            let cse_var_6: int32 = ((((ii.inner.outer.inner*1024) + (ii.inner.inner*64)) + (jj.inner.outer.inner*16)) + jj.inner.inner)
            C[cse_var_6] = C.wmma.accumulator_1[cse_var_6]
          }
        }
      }
    }
  }
}

