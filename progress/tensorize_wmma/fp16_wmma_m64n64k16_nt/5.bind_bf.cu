@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [4096], []),
             B: Buffer(B_2: Pointer(float16), float16, [8192], []),
             C: Buffer(C_2: Pointer(float32), float32, [32768], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [128, 32], []), B_1: B_3: Buffer(B_2, float16, [256, 32], []), C_1: C_3: Buffer(C_2, float32, [128, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [2048]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float16), float16, [2048]), storage_scope = shared {
    for (ax0: int32, 0, 64) {
      for (ax1: int32, 0, 32) {
        let cse_var_1: int32 = (ax0*32)
        A.shared_1: Buffer(A.shared, float16, [2048], [], scope="shared")[(cse_var_1 + ax1)] = A[(((blockIdx.y: int32*2048) + cse_var_1) + ax1)]
      }
    }
    for (ax0_1: int32, 0, 64) {
      for (ax1_1: int32, 0, 32) {
        let cse_var_2: int32 = (ax0_1*32)
        B.shared_1: Buffer(B.shared, float16, [2048], [], scope="shared")[(cse_var_2 + ax1_1)] = B[(((blockIdx.x: int32*2048) + cse_var_2) + ax1_1)]
      }
    }
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 2;
    allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [4096]), storage_scope = wmma.accumulator;
    allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [1024]), storage_scope = wmma.matrix_a;
    allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [1024]), storage_scope = wmma.matrix_b;
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 4;
    attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 1;
    attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 1 {
      for (ii.c.outer.init: int32, 0, 4) {
        for (jj.c.outer.init: int32, 0, 4) {
          for (ii.c.inner.init: int32, 0, 16) {
            for (jj.c.inner.init: int32, 0, 16) {
              C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [4096], [], scope="wmma.accumulator")[((((ii.c.outer.init*1024) + (ii.c.inner.init*64)) + (jj.c.outer.init*16)) + jj.c.inner.init)] = 0f32
            }
          }
        }
      }
      for (rk.outer.inner: int32, 0, 2) {
        for (ax0.outer: int32, 0, 4) {
          for (ax0.inner: int32, 0, 16) {
            for (ax1.inner: int32, 0, 16) {
              A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [1024], [], scope="wmma.matrix_a")[(((ax0.outer*256) + (ax0.inner*16)) + ax1.inner)] = A.shared_1[((((ax0.outer*512) + (ax0.inner*32)) + (rk.outer.inner*16)) + ax1.inner)]
            }
          }
        }
        for (ax0.outer_1: int32, 0, 4) {
          for (ax0.inner_1: int32, 0, 16) {
            for (ax1.inner_1: int32, 0, 16) {
              B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [1024], [], scope="wmma.matrix_b")[(((ax0.outer_1*256) + (ax0.inner_1*16)) + ax1.inner_1)] = B.shared_1[((((ax0.outer_1*512) + (ax0.inner_1*32)) + (rk.outer.inner*16)) + ax1.inner_1)]
            }
          }
        }
        for (ii.c.outer: int32, 0, 4) {
          for (jj.c.outer: int32, 0, 4) {
            for (ii.c.inner: int32, 0, 16) {
              for (jj.c.inner: int32, 0, 16) {
                for (rk.inner: int32, 0, 16) {
                  let cse_var_3: int32 = ((((ii.c.outer*1024) + (ii.c.inner*64)) + (jj.c.outer*16)) + jj.c.inner)
                  C.wmma.accumulator_1[cse_var_3] = (C.wmma.accumulator_1[cse_var_3] + (cast(float32, A.shared.wmma.matrix_a_1[(((ii.c.outer*256) + (ii.c.inner*16)) + rk.inner)])*cast(float32, B.shared.wmma.matrix_b_1[(((jj.c.outer*256) + (jj.c.inner*16)) + rk.inner)])))
                }
              }
            }
          }
        }
      }
      for (ii.inner.outer.inner: int32, 0, 4) {
        for (jj.inner.outer.inner: int32, 0, 4) {
          for (ii.inner.inner: int32, 0, 16) {
            for (jj.inner.inner: int32, 0, 16) {
              let cse_var_4: int32 = (jj.inner.outer.inner*16)
              C[((((((blockIdx.y*16384) + (ii.inner.outer.inner*4096)) + (ii.inner.inner*256)) + (blockIdx.x*64)) + cse_var_4) + jj.inner.inner)] = C.wmma.accumulator_1[((((ii.inner.outer.inner*1024) + (ii.inner.inner*64)) + cse_var_4) + jj.inner.inner)]
            }
          }
        }
      }
    }
  }
}

