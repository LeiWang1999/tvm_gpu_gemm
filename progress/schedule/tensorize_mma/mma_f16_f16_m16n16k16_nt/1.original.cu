#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1, 1, 16, 16], []),
             B: Buffer(B_1: Pointer(global float16), float16, [1, 1, 16, 16], []),
             C: Buffer(C_1: Pointer(global float16), float16, [1, 1, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    for (ii: int32, 0, 1) {
      for (jj: int32, 0, 1) {
        for (kk: int32, 0, 1) {
          for (i: int32, 0, 16) {
            for (j: int32, 0, 16) {
              for (k: int32, 0, 16) {
                block([1, 1, tir.reduce_axis(0, 1), 16, 16, tir.reduce_axis(0, 16)], "B") as [vii, vjj, vkk, vi, vj, vk] {
                  bind(vii, ii)
                  bind(vjj, jj)
                  bind(vkk, kk)
                  bind(vi, i)
                  bind(vj, j)
                  bind(vk, k)
                  tir.reads([A[vii, vkk, vi, vk], B[vjj, vkk, vj, vk]])
                  tir.writes([C[vii, vjj, vi, vj]])
                  with init() {
                    C[vii, vjj, vi, vj] = 0f16
                  }
                  C[vii, vjj, vi, vj] = (C[vii, vjj, vi, vj] + (A[vii, vkk, vi, vk]*B[vjj, vkk, vj, vk]))
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}