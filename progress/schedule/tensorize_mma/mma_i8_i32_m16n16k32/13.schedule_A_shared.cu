#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [1, 1, 16, 32], []),
             B: Buffer(B_1: Pointer(global int8), int8, [1, 1, 16, 32], []),
             C: Buffer(C_1: Pointer(global int32), int32, [1, 1, 16, 16], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_shared = alloc_buffer(int8[1, 1, 16, 32])
    A_shared_warp = alloc_buffer(int8[1, 1, 16, 32])
    B_shared = alloc_buffer(int8[1, 1, 16, 32])
    B_shared_warp = alloc_buffer(int8[1, 1, 16, 32])
    B_shared_warp_warp = alloc_buffer(int8[1, 1, 16, 32])
    C_warp = alloc_buffer(int32[1, 1, 16, 16])
    for (ii: int32, 0, 1) "thread_binding" {
      for (jj: int32, 0, 1) "thread_binding" {
        for (ax0_ax1_fused_0: int32, 0, 1) "thread_binding" {
          for (ax0_ax1_fused_1: int32, 0, 1) "thread_binding" {
            for (ax0_ax1_fused_2: int32, 0, 1) {
              for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                for (ax0_ax1_fused_4: int32, 0, 16) "vectorized" {
                  block([1, 1, 16, 32], "A_shared") as [v0, v1, v2, v3] {
                    bind(v0, 0)
                    bind(v1, 0)
                    bind(v2, floordiv((((((ax0_ax1_fused_0*512) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*512)) + (ax0_ax1_fused_3*16)) + ax0_ax1_fused_4), 32))
                    bind(v3, floormod((((((ax0_ax1_fused_0*512) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*512)) + (ax0_ax1_fused_3*16)) + ax0_ax1_fused_4), 32))
                    tir.reads([A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 16)), ((floordiv(v2, 8)*16) + floormod(v3, 16))]])
                    tir.writes([A_shared[v0, v1, v2, v3]])
                    A_shared[v0, v1, v2, v3] = A[v0, v1, ((floormod(v2, 8)*2) + floordiv(v3, 16)), ((floordiv(v2, 8)*16) + floormod(v3, 16))]
                }
              }
            }
          }
        }
        for (ax0: int32, 0, 16) {
          for (ax1: int32, 0, 32) {
            block([1, 1, 16, 32], "A_shared_warp") as [v0_1, v1_1, v2_1, v3_1] {
              bind(v0_1, 0)
              bind(v1_1, 0)
              bind(v2_1, ax0)
              bind(v3_1, ax1)
              tir.reads([A_shared[v0_1, v1_1, v2_1, v3_1]])
              tir.writes([A_shared_warp[v0_1, v1_1, v2_1, v3_1]])
              A_shared_warp[v0_1, v1_1, v2_1, v3_1] = A_shared[v0_1, v1_1, v2_1, v3_1]
          }
        }
        for (ax0_1: int32, 0, 16) {
          for (ax1_1: int32, 0, 32) {
            block([1, 1, 16, 32], "B_shared") as [v0_2, v1_2, v2_2, v3_2] {
              bind(v0_2, 0)
              bind(v1_2, 0)
              bind(v2_2, ax0_1)
              bind(v3_2, ax1_1)
              tir.reads([B[v0_2, v1_2, ((floormod(v2_2, 8)*2) + floordiv(v3_2, 16)), ((floordiv(v2_2, 8)*16) + floormod(v3_2, 16))]])
              tir.writes([B_shared[v0_2, v1_2, v2_2, v3_2]])
              B_shared[v0_2, v1_2, v2_2, v3_2] = B[v0_2, v1_2, ((floormod(v2_2, 8)*2) + floordiv(v3_2, 16)), ((floordiv(v2_2, 8)*16) + floormod(v3_2, 16))]
          }
        }
        for (ax0_2: int32, 0, 16) {
          for (ax1_2: int32, 0, 32) {
            block([1, 1, 16, 32], "B_shared_warp") as [v0_3, v1_3, v2_3, v3_3] {
              bind(v0_3, 0)
              bind(v1_3, 0)
              bind(v2_3, ax0_2)
              bind(v3_3, ax1_2)
              tir.reads([B_shared[v0_3, v1_3, v2_3, v3_3]])
              tir.writes([B_shared_warp[v0_3, v1_3, v2_3, v3_3]])
              B_shared_warp[v0_3, v1_3, v2_3, v3_3] = B_shared[v0_3, v1_3, v2_3, v3_3]
          }
        }
        for (ax0_3: int32, 0, 16) {
          for (ax1_3: int32, 0, 32) {
            block([1, 1, 16, 32], "B_shared_warp_warp") as [v0_4, v1_4, v2_4, v3_4] {
              bind(v0_4, 0)
              bind(v1_4, 0)
              bind(v2_4, ax0_3)
              bind(v3_4, ax1_3)
              tir.reads([B_shared_warp[v0_4, v1_4, v2_4, v3_4]])
              tir.writes([B_shared_warp_warp[v0_4, v1_4, v2_4, v3_4]])
              B_shared_warp_warp[v0_4, v1_4, v2_4, v3_4] = B_shared_warp[v0_4, v1_4, v2_4, v3_4]
          }
        }
        for (kk: int32, 0, 1) {
          for (i: int32, 0, 16) {
            for (j: int32, 0, 16) {
              for (k: int32, 0, 32) {
                block([1, 1, tir.reduce_axis(0, 1), 16, 16, tir.reduce_axis(0, 32)], "B") as [vii, vjj, vkk, vi, vj, vk] {
                  bind(vii, ii)
                  bind(vjj, jj)
                  bind(vkk, kk)
                  bind(vi, i)
                  bind(vj, j)
                  bind(vk, k)
                  tir.reads([A_shared_warp[vii, vkk, vi, vk], B_shared_warp_warp[vjj, vkk, vj, vk]])
                  tir.writes([C_warp[vii, vjj, vi, vj]])
                  with init() {
                    C_warp[vii, vjj, vi, vj] = 0
                  }
                  C_warp[vii, vjj, vi, vj] = (C_warp[vii, vjj, vi, vj] + (cast(int32, A_shared_warp[vii, vkk, vi, vk])*cast(int32, B_shared_warp_warp[vjj, vkk, vj, vk])))
              }
            }
          }
        }
        for (ax0_4: int32, 0, 16) {
          for (ax1_4: int32, 0, 16) {
            block([1, 1, 16, 16], "C_warp") as [v0_5, v1_5, v2_5, v3_5] {
              bind(v0_5, 0)
              bind(v1_5, 0)
              bind(v2_5, ax0_4)
              bind(v3_5, ax1_4)
              tir.reads([C_warp[v0_5, v1_5, v2_5, v3_5]])
              tir.writes([C[v0_5, v1_5, v2_5, v3_5]])
              C[v0_5, v1_5, v2_5, v3_5] = C_warp[v0_5, v1_5, v2_5, v3_5]
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}