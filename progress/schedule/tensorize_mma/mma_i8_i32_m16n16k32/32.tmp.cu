
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  __shared__ signed char A_shared[512];
  signed char A_shared_warp[16];
  signed char B_shared_warp[16];
  int C_warp[8];
  *(int4*)(A_shared + (((int)threadIdx.x) * 16)) = *(int4*)(A + (((((int)threadIdx.x) & 15) * 32) + ((((int)threadIdx.x) >> 4) * 16)));
  __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[0])) + (((int)threadIdx.x) * 16)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
      : "r"(addr)
    );
  }
  __syncthreads();
  *(int4*)(A_shared + (((int)threadIdx.x) * 16)) = *(int4*)(B + (((((int)threadIdx.x) & 15) * 32) + ((((int)threadIdx.x) >> 4) * 16)));
  __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[0])) + (((int)threadIdx.x) * 16)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_shared_warp + 0))[0]), "=r"(((unsigned *)(B_shared_warp + 0))[1]), "=r"(((unsigned *)(B_shared_warp + 0))[2]), "=r"(((unsigned *)(B_shared_warp + 0))[3])
      : "r"(addr)
    );
  }
  for (int i = 0; i < 8; ++i) {
C_warp[0 + i] = 0.0;}
;

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + 0))[0]), "=r"(((int *)(C_warp + 0))[1]), "=r"(((int *)(C_warp + 0))[2]), "=r"(((int *)(C_warp + 0))[3])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + 0))[0]), "r"(((unsigned *)(B_shared_warp + 0))[1]), "r"(((int *)(C_warp + 0))[0]), "r"(((int *)(C_warp + 0))[1]), "r"(((int *)(C_warp + 0))[2]), "r"(((int *)(C_warp + 0))[3]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=r"(((int *)(C_warp + 4))[0]), "=r"(((int *)(C_warp + 4))[1]), "=r"(((int *)(C_warp + 4))[2]), "=r"(((int *)(C_warp + 4))[3])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + 8))[0]), "r"(((unsigned *)(B_shared_warp + 8))[1]), "r"(((int *)(C_warp + 4))[0]), "r"(((int *)(C_warp + 4))[1]), "r"(((int *)(C_warp + 4))[2]), "r"(((int *)(C_warp + 4))[3]));
  }
  for (int local_id = 0; local_id < 8; ++local_id) {
(&(C[0]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))] = C_warp[0 + local_id];
}
;
}

