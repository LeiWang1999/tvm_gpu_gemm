#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global float16), float16, [1024, 16384], []),
             B: Buffer(B_1: Pointer(global float16), float16, [16384, 1024], []),
             C: Buffer(C_1: Pointer(global float16), float16, [1024, 1024], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(float16[64, 1024, 16, 16])
    A_global_shared = alloc_buffer(float16[64, 1024, 16, 16])
    A_global_shared_warp = alloc_buffer(float16[64, 1024, 32, 8])
    B_global = alloc_buffer(float16[1024, 64, 16, 16])
    B_global_shared = alloc_buffer(float16[1024, 64, 16, 16])
    B_global_shared_warp = alloc_buffer(float16[1024, 64, 32, 8])
    C_warp = alloc_buffer(float16[64, 64, 32, 8])
     {
      for (ax0_ax1_fused_0: int32, 0, 8192) "thread_binding" {
        for (ax0_ax1_fused_1: int32, 0, 32) "thread_binding" {
          for (ax0_ax1_fused_2: int32, 0, 1) "thread_binding" {
            for (ax0_ax1_fused_3: int32, 0, 1) "thread_binding" {
              for (ax0_ax1_fused_4: int32, 0, 8) "thread_binding" {
                for (ax0_ax1_fused_5: int32, 0, 1) {
                  for (ax0_ax1_fused_6: int32, 0, 8) "vectorized" {
                    block([16384, 1024], "B_global") as [v0, v1] {
                      bind(v0, floordiv((((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*64)) + (ax0_ax1_fused_2*64)) + (ax0_ax1_fused_3*64)) + (ax0_ax1_fused_4*8)) + (ax0_ax1_fused_5*8)) + ax0_ax1_fused_6), 1024))
                      bind(v1, floormod((((((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*64)) + (ax0_ax1_fused_2*64)) + (ax0_ax1_fused_3*64)) + (ax0_ax1_fused_4*8)) + (ax0_ax1_fused_5*8)) + ax0_ax1_fused_6), 1024))
                      tir.reads([B[v0, v1]])
                      tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), ((floormod(v0, 8)*2) + floordiv(floormod(v1, 16), 8)), ((floordiv(floormod(v0, 16), 8)*8) + floormod(v1, 8))]])
                      B_global[floordiv(v0, 16), floordiv(v1, 16), ((floormod(v0, 8)*2) + floordiv(floormod(v1, 16), 8)), ((floordiv(floormod(v0, 16), 8)*8) + floormod(v1, 8))] = B[v0, v1]
                  }
                }
              }
            }
          }
        }
      }
      for (ax0_ax1_fused_0_1: int32, 0, 8192) "thread_binding" {
        for (ax0_ax1_fused_1_1: int32, 0, 32) "thread_binding" {
          for (ax0_ax1_fused_2_1: int32, 0, 1) "thread_binding" {
            for (ax0_ax1_fused_3_1: int32, 0, 1) "thread_binding" {
              for (ax0_ax1_fused_4_1: int32, 0, 8) "thread_binding" {
                for (ax0_ax1_fused_5_1: int32, 0, 1) {
                  for (ax0_ax1_fused_6_1: int32, 0, 8) "vectorized" {
                    block([1024, 16384], "A_global") as [v0_1, v1_1] {
                      bind(v0_1, floordiv((((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*64)) + (ax0_ax1_fused_2_1*64)) + (ax0_ax1_fused_3_1*64)) + (ax0_ax1_fused_4_1*8)) + (ax0_ax1_fused_5_1*8)) + ax0_ax1_fused_6_1), 16384))
                      bind(v1_1, floormod((((((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*64)) + (ax0_ax1_fused_2_1*64)) + (ax0_ax1_fused_3_1*64)) + (ax0_ax1_fused_4_1*8)) + (ax0_ax1_fused_5_1*8)) + ax0_ax1_fused_6_1), 16384))
                      tir.reads([A[v0_1, v1_1]])
                      tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), ((floormod(v0_1, 8)*2) + floordiv(floormod(v1_1, 16), 8)), ((floordiv(floormod(v0_1, 16), 8)*8) + floormod(v1_1, 8))]])
                      A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), ((floormod(v0_1, 8)*2) + floordiv(floormod(v1_1, 16), 8)), ((floordiv(floormod(v0_1, 16), 8)*8) + floormod(v1_1, 8))] = A[v0_1, v1_1]
                  }
                }
              }
            }
          }
        }
      }
      for (i_0_0: int32, 0, 8) "thread_binding" {
        for (j_0_0: int32, 0, 4) "thread_binding" {
          for (i_0_1: int32, 0, 1) "thread_binding" {
            for (j_0_1: int32, 0, 4) "thread_binding" {
              for (i_0_2_init: int32, 0, 8) {
                for (j_0_2_init: int32, 0, 4) {
                  block([64, 64], "B_init_o") as [vi_o, vj_o] {
                    bind(vi_o, (((i_0_0*8) + (i_0_1*8)) + i_0_2_init))
                    bind(vj_o, (((j_0_0*16) + (j_0_1*4)) + j_0_2_init))
                    tir.reads([])
                    tir.writes([C_warp[vi_o, vj_o, 0:32, 0:8]])
                    C_warp_1 = match_buffer(C_warp[vi_o, vj_o, 0:32, 0:8])
                    attr [IterVar(tx: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_fill(8, C_warp_2: Pointer(warp float16), elem_offset: int32, dtype=float16)
                }
              }
              for (k_0_0: int32, 0, 512) {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 4) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 8) "vectorized" {
                          block([1024, 16384], "A_global_shared") as [v0_2, v1_2] {
                            bind(v0_2, (((i_0_0*128) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                            bind(v1_2, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                            tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*2) + floordiv(floormod(v1_2, 16), 8)), ((floordiv(floormod(v0_2, 16), 8)*8) + floormod(v1_2, 8))]])
                            tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), ((floormod(v0_2, 8)*2) + floordiv(floormod(v1_2, 16), 8)), ((floordiv(floormod(v0_2, 16), 8)*8) + floormod(v1_2, 8))]
                        }
                      }
                    }
                  }
                }
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 1) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 8) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 8) "vectorized" {
                          block([16384, 1024], "B_global_shared") as [v0_3, v1_3] {
                            bind(v0_3, (((k_0_0*32) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 4096)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                            bind(v1_3, (((j_0_0*256) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 4096), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*8192) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*2048)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*256)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*8)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                            tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), ((floormod(v0_3, 8)*2) + floordiv(floormod(v1_3, 16), 8)), ((floordiv(floormod(v0_3, 16), 8)*8) + floormod(v1_3, 8))]])
                            tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                            B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), ((floormod(v0_3, 8)*2) + floordiv(floormod(v1_3, 16), 8)), ((floordiv(floormod(v0_3, 16), 8)*8) + floormod(v1_3, 8))]
                        }
                      }
                    }
                  }
                }
                for (k_0_1: int32, 0, 2) {
                  for (ax0_0: int32, 0, 8) {
                    for (ax1_0: int32, 0, 1) {
                      block([64, 1024], "A_global_shared_warp_o") as [v0_o, v1_o] {
                        bind(v0_o, ((i_0_0*8) + ax0_0))
                        bind(v1_o, (((k_0_0*2) + k_0_1) + ax1_0))
                        tir.reads([A_global_shared[v0_o, v1_o, 0:16, 0:16]])
                        tir.writes([A_global_shared_warp[v0_o, v1_o, 0:32, 0:8]])
                        warp = match_buffer(A_global_shared_warp[v0_o, v1_o, 0:32, 0:8])
                        shared = match_buffer(A_global_shared[v0_o, v1_o, 0:16, 0:16])
                        attr [IterVar(tx_1: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                        @tir.ptx_ldmatrix(False, 4, ".b16", warp_1: Pointer(warp float16), (elem_offset_1: int32 + (8*tx_1)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_1: Pointer(shared float16), elem_offset_2: int32, (shared_s0: int32*16), 1, dtype=handle), (8*tx_1), dtype=float16)
                    }
                  }
                  for (ax0_0_1: int32, 0, 1) {
                    for (ax1_0_1: int32, 0, 4) {
                      block([1024, 64], "B_global_shared_warp_o") as [v0_o_1, v1_o_1] {
                        bind(v0_o_1, (((k_0_0*2) + k_0_1) + ax0_0_1))
                        bind(v1_o_1, (((j_0_0*16) + (j_0_1*4)) + ax1_0_1))
                        tir.reads([B_global_shared[v0_o_1, v1_o_1, 0:16, 0:16]])
                        tir.writes([B_global_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8]])
                        warp_2 = match_buffer(B_global_shared_warp[v0_o_1, v1_o_1, 0:32, 0:8])
                        shared_2 = match_buffer(B_global_shared[v0_o_1, v1_o_1, 0:16, 0:16])
                        attr [IterVar(tx_2: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                        @tir.ptx_ldmatrix(True, 4, ".b16", warp_3: Pointer(warp float16), (elem_offset_3: int32 + (8*tx_2)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), shared_3: Pointer(shared float16), elem_offset_4: int32, (shared_s0_1: int32*16), 1, dtype=handle), (8*tx_2), dtype=float16)
                    }
                  }
                  for (i_0_2: int32, 0, 8) {
                    for (j_0_2: int32, 0, 4) {
                      block([64, 64, tir.reduce_axis(0, 1024)], "B_update_o") as [vi_o_1, vj_o_1, vk_o] {
                        bind(vi_o_1, (((i_0_0*8) + (i_0_1*8)) + i_0_2))
                        bind(vj_o_1, (((j_0_0*16) + (j_0_1*4)) + j_0_2))
                        bind(vk_o, ((k_0_0*2) + k_0_1))
                        tir.reads([C_warp[vi_o_1, vj_o_1, 0:32, 0:8], A_global_shared_warp[vi_o_1, vk_o, 0:32, 0:8], B_global_shared_warp[vk_o, vj_o_1, 0:32, 0:8]])
                        tir.writes([C_warp[vi_o_1, vj_o_1, 0:32, 0:8]])
                        A_2 = match_buffer(A_global_shared_warp[vi_o_1, vk_o, 0:32, 0:8])
                        B_2 = match_buffer(B_global_shared_warp[vk_o, vj_o_1, 0:32, 0:8])
                        C_2 = match_buffer(C_warp[vi_o_1, vj_o_1, 0:32, 0:8])
                        attr [IterVar(tx_3: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32 {
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3: Pointer(warp float16), (elem_offset_5: int32 + (tx_3*8)), B_3: Pointer(warp float16), (elem_offset_6: int32 + (tx_3*8)), C_3: Pointer(warp float16), (elem_offset_7: int32 + (tx_3*8)), False, dtype=float16)
                          @tir.ptx_mma("m16n8k16", "row", "col", "fp16", "fp16", "fp16", A_3, (elem_offset_5 + (tx_3*8)), B_3, ((elem_offset_6 + (tx_3*8)) + floordiv(8, 2)), C_3, ((elem_offset_7 + (tx_3*8)) + floordiv(8, 2)), False, dtype=float16)
                        }
                    }
                  }
                }
              }
              for (ax0_0_2: int32, 0, 8) {
                for (ax1_0_2: int32, 0, 4) {
                  block([64, 64], "C_warp_o") as [v0_o_2, v1_o_2] {
                    bind(v0_o_2, ((i_0_0*8) + ax0_0_2))
                    bind(v1_o_2, (((j_0_0*16) + (j_0_1*4)) + ax1_0_2))
                    tir.reads([C_warp[v0_o_2, v1_o_2, 0:32, 0:8]])
                    tir.writes([C[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)]])
                    C_warp_3 = match_buffer(C_warp[v0_o_2, v1_o_2, 0:32, 0:8])
                    C_4 = match_buffer(C[(v0_o_2*16):((v0_o_2*16) + 16), (v1_o_2*16):((v1_o_2*16) + 16)])
                    attr [IterVar(tx_4: int32, [0:32], "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
                    @tir.mma_store(16, 16, @tir.tvm_access_ptr(@tir.type_annotation(, dtype=float16), C_5: Pointer(global float16), elem_offset_8: int32, (C_s0: int32*16), 2, dtype=handle), C_warp_4: Pointer(warp float16), elem_offset_9: int32, C_s0, dtype=float16)
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}