#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ APad_global, signed char* __restrict__ BPad_global, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> CPad_shared_wmma_accumulator[16];
  __shared__ signed char APad_global_shared[4096];
  __shared__ signed char BPad_global_shared[4096];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> APad_global_shared_wmma_matrix_a[8];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> BPad_global_shared_wmma_matrix_b[2];
  __shared__ int CPad_shared[2048];
  for (int i_0_2_init = 0; i_0_2_init < 8; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 2; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(CPad_shared_wmma_accumulator[((i_0_2_init * 2) + j_0_2_init)], 0.000000e+00f);
    }
  }

  const int MAX_BLOCK_N = 1;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int k_0_0 = 0; k_0_0 < 20; ++k_0_0) {
    __syncthreads();
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
      *(int4*)(APad_global_shared + (((((int)threadIdx.z) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(APad_global + (((((((int)blockIdx.y) * 81920) + (((int)threadIdx.z) * 20480)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 10240)) + (k_0_0 * 512)) + (((int)threadIdx.x) * 16)));
    }
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1) {
      *(int4*)(BPad_global_shared + (((((int)threadIdx.z) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(BPad_global + ((((k_0_0 * 4096) + (((int)threadIdx.z) * 1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16)));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 8; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(APad_global_shared_wmma_matrix_a[ax0_0], (&(APad_global_shared[((ax0_0 * 512) + (k_0_1 * 256))])), 16);
      }
      for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) {
        nvcuda::wmma::load_matrix_sync(BPad_global_shared_wmma_matrix_b[ax1_0], (&(BPad_global_shared[(((k_0_1 * 2048) + (((int)threadIdx.z) * 512)) + (ax1_0 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 8; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 2; ++j_0_2) {
          nvcuda::wmma::mma_sync(CPad_shared_wmma_accumulator[((i_0_2 * 2) + j_0_2)], APad_global_shared_wmma_matrix_a[i_0_2], BPad_global_shared_wmma_matrix_b[j_0_2], CPad_shared_wmma_accumulator[((i_0_2 * 2) + j_0_2)]);
        }
      }
    }
  }
  for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1) {
    __syncthreads();
    for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
      nvcuda::wmma::store_matrix_sync((&(CPad_shared[((((int)threadIdx.z) * 32) + (ax1_0_1 * 16))])), CPad_shared_wmma_accumulator[((ax0_0_1 * 2) + ax1_0_1)], 128, nvcuda::wmma::mem_row_major);
    }
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {
      C[(((((((int)blockIdx.y) * 8192) + (ax0_0_1 * 1024)) + (ax0_ax1_fused_0 * 256)) + (((int)threadIdx.z) * 96)) + ((int)threadIdx.x))] = CPad_shared[(((ax0_ax1_fused_0 * 512) + (((int)threadIdx.z) * 160)) + ((int)threadIdx.x))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ APad_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_6_s = 0; ax0_ax1_fused_6_s < 16; ++ax0_ax1_fused_6_s) {
    if ((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) < 2048000) {
      APad_global[((((((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) / 10240) * 10240) + ((((((((int)blockIdx.y) * 2097152) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) % 40) * 256)) + ((((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) % 10240) / 640) * 16)) + ax0_ax1_fused_6_s)] = ((((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) < 2006976) && (((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) % 640) < 576)) ? A[((((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) / 640) * 576) + ((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) % 640))] : (signed char)0);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ BPad_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_6_s = 0; ax0_ax1_fused_6_s < 16; ++ax0_ax1_fused_6_s) {
    if ((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) < 81920) {
      BPad_global[((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + ((((int)threadIdx.y) >> 4) * 2048)) + (((int)threadIdx.x) * 256)) + ((((int)threadIdx.y) & 15) * 16)) + ax0_ax1_fused_6_s)] = ((((((((((int)blockIdx.y) * 33554432) + (((int)blockIdx.x) * 16384)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s) < 73664) && (((int)threadIdx.x) < 4)) ? B[(((((((int)blockIdx.y) * 16777216) + (((int)blockIdx.x) * 8192)) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_6_s)] : (signed char)0);
    }
  }
}

