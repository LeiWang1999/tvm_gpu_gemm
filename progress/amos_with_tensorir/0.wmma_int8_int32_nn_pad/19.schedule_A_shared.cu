#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [3136, 576], []),
             B: Buffer(B_1: Pointer(global int8), int8, [576, 64], []),
             C: Buffer(C_1: Pointer(global int32), int32, [3136, 64], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    APad_global = alloc_buffer(int8[200, 40, 16, 16])
    APad_global_shared = alloc_buffer(int8[200, 40, 16, 16])
    APad_global_shared_wmma.matrix_a = alloc_buffer(int8[200, 40, 16, 16])
    BPad_global = alloc_buffer(int8[40, 8, 16, 16])
    BPad_global_shared = alloc_buffer(int8[40, 8, 16, 16])
    BPad_global_shared_wmma.matrix_b = alloc_buffer(int8[40, 8, 16, 16])
    CPad_shared = alloc_buffer(int32[3200, 128])
    CPad_shared_wmma.accumulator = alloc_buffer(int32[200, 8, 16, 16])
     {
      for (ax0: int32, 0, 3200) {
        for (ax1: int32, 0, 640) {
          block([3200, 640], "APad_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([A[v0, v1]])
            tir.writes([APad_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            APad_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = @tir.if_then_else(((v0 < 3136) && (v1 < 576)), A[v0, v1], 0i8, dtype=int8)
        }
      }
      for (ax0_1: int32, 0, 640) {
        for (ax1_1: int32, 0, 128) {
          block([640, 128], "BPad_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([B[v0_1, v1_1]])
            tir.writes([BPad_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            BPad_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = @tir.if_then_else(((v0_1 < 576) && (v1_1 < 64)), B[v0_1, v1_1], 0i8, dtype=int8)
        }
      }
      for (i_0_0: int32, 0, 25) "thread_binding" {
        for (j_0_0: int32, 0, 1) "thread_binding" {
          for (i_0_1: int32, 0, 1) "thread_binding" {
            for (j_0_1: int32, 0, 4) "thread_binding" {
              for (k_0_0: int32, 0, 20) {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 2) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                          block([3200, 640], "APad_global_shared") as [v0_2, v1_2] {
                            bind(v0_2, (((i_0_0*128) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                            bind(v1_2, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                            tir.reads([APad_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            tir.writes([APad_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            APad_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = APad_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                        }
                      }
                    }
                  }
                }
                for (ax0_0: int32, 0, 2) {
                  for (ax1_0: int32, 0, 8) {
                    for (ax0_1_1: int32, 0, 16) {
                      for (ax1_1_1: int32, 0, 16) {
                        block([640, 128], "BPad_global_shared") as [v0_3, v1_3] {
                          bind(v0_3, (((k_0_0*32) + (ax0_0*16)) + ax0_1_1))
                          bind(v1_3, ((ax1_0*16) + ax1_1_1))
                          tir.reads([BPad_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                          tir.writes([BPad_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                          BPad_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = BPad_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                      }
                    }
                  }
                }
                for (k_0_1: int32, 0, 2) {
                  for (ax0_0_1: int32, 0, 8) {
                    for (ax1_0_1: int32, 0, 1) {
                      for (ax0_1_2: int32, 0, 16) {
                        for (ax1_1_2: int32, 0, 16) {
                          block([3200, 640], "APad_global_shared_wmma.matrix_a") as [v0_4, v1_4] {
                            bind(v0_4, (((i_0_0*128) + (ax0_0_1*16)) + ax0_1_2))
                            bind(v1_4, ((((k_0_0*32) + (k_0_1*16)) + (ax1_0_1*16)) + ax1_1_2))
                            tir.reads([APad_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            tir.writes([APad_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            APad_global_shared_wmma.matrix_a[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = APad_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                        }
                      }
                    }
                  }
                  for (ax0_0_2: int32, 0, 1) {
                    for (ax1_0_2: int32, 0, 2) {
                      for (ax0_1_3: int32, 0, 16) {
                        for (ax1_1_3: int32, 0, 16) {
                          block([640, 128], "BPad_global_shared_wmma.matrix_b") as [v0_5, v1_5] {
                            bind(v0_5, ((((k_0_0*32) + (k_0_1*16)) + (ax0_0_2*16)) + ax0_1_3))
                            bind(v1_5, (((j_0_1*32) + (ax1_0_2*16)) + ax1_1_3))
                            tir.reads([BPad_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                            tir.writes([BPad_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                            BPad_global_shared_wmma.matrix_b[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)] = BPad_global_shared[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                        }
                      }
                    }
                  }
                  for (i_0_2: int32, 0, 8) {
                    for (j_0_2: int32, 0, 2) {
                      for (i_1: int32, 0, 16) {
                        for (j_1: int32, 0, 16) {
                          for (k_1: int32, 0, 16) {
                            block([3200, 128, tir.reduce_axis(0, 640)], "B") as [vi, vj, vk] {
                              bind(vi, ((((i_0_0*128) + (i_0_1*128)) + (i_0_2*16)) + i_1))
                              bind(vj, ((((j_0_0*128) + (j_0_1*32)) + (j_0_2*16)) + j_1))
                              bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                              tir.reads([APad_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)], BPad_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)]])
                              tir.writes([CPad_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                              with init() {
                                CPad_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = 0
                              }
                              CPad_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = (CPad_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] + (cast(int32, APad_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)])*cast(int32, BPad_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)])))
                          }
                        }
                      }
                    }
                  }
                }
              }
              for (ax0_0_3: int32, 0, 8) {
                for (ax1_0_3: int32, 0, 2) {
                  for (ax0_1_4: int32, 0, 16) {
                    for (ax1_1_4: int32, 0, 16) {
                      block([3200, 128], "CPad_shared_wmma.accumulator") as [v0_6, v1_6] {
                        bind(v0_6, (((i_0_0*128) + (ax0_0_3*16)) + ax0_1_4))
                        bind(v1_6, (((j_0_1*32) + (ax1_0_3*16)) + ax1_1_4))
                        tir.reads([CPad_shared_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]])
                        tir.writes([CPad_shared[v0_6, v1_6]])
                        CPad_shared[v0_6, v1_6] = CPad_shared_wmma.accumulator[floordiv(v0_6, 16), floordiv(v1_6, 16), floormod(v0_6, 16), floormod(v1_6, 16)]
                    }
                  }
                }
                for (ax0_ax1_fused_0: int32, 0, 4) {
                  for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_2: int32, 0, 1) "thread_binding" {
                      for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                        block([3200, 128], "CPad_shared") as [v0_7, v1_7] {
                          bind(v0_7, (((i_0_0*128) + (ax0_0_3*16)) + floordiv(((((ax0_ax1_fused_0*128) + (ax0_ax1_fused_1*32)) + (ax0_ax1_fused_2*32)) + ax0_ax1_fused_3), 32)))
                          bind(v1_7, ((j_0_1*32) + floormod(((((ax0_ax1_fused_0*128) + (ax0_ax1_fused_1*32)) + (ax0_ax1_fused_2*32)) + ax0_ax1_fused_3), 32)))
                          tir.reads([CPad_shared[v0_7, v1_7]])
                          tir.writes([C[v0_7, v1_7]])
                          C[v0_7, v1_7] = CPad_shared[v0_7, v1_7]
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}