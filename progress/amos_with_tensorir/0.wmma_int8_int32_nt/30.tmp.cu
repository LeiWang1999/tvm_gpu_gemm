#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ A_global, signed char* __restrict__ B_global, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_global_shared[8192];
  __shared__ signed char B_global_shared[8192];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a[8];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> B_global_shared_wmma_matrix_b[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a_1[8];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> B_global_shared_wmma_matrix_b_1[2];
  for (int i_0_2_init = 0; i_0_2_init < 8; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 2; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 2) + j_0_2_init)], 0.000000e+00f);
    }
  }
  for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
    *(int4*)(A_global_shared + (((((int)threadIdx.z) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A_global + ((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.z) * 524288)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 262144)) + (((int)threadIdx.x) * 16)));
  }
  for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1) {
    *(int4*)(B_global_shared + (((((int)threadIdx.z) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B_global + (((((((int)threadIdx.z) >> 1) * 262144) + (((int)blockIdx.x) * 2048)) + (((((((int)threadIdx.z) * 4) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4)) & 7) * 256)) + ((((int)threadIdx.x) & 15) * 16)));
  }

  const int MAX_BLOCK_N = 1;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int k_0_0 = 0; k_0_0 < 511; ++k_0_0) {
    __syncthreads();
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_2 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_2) {
      *(int4*)(A_global_shared + ((((((k_0_0 + 1) & 1) * 4096) + (((int)threadIdx.z) * 1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A_global + ((((((((int)blockIdx.y) * 2097152) + (((int)threadIdx.z) * 524288)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_2 * 262144)) + (k_0_0 * 512)) + (((int)threadIdx.x) * 16)) + 512));
    }
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_3 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_3 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_3) {
      *(int4*)(B_global_shared + ((((((k_0_0 + 1) & 1) * 4096) + (((int)threadIdx.z) * 1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_3 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B_global + ((((((k_0_0 * 524288) + ((((int)threadIdx.z) >> 1) * 262144)) + (((int)blockIdx.x) * 2048)) + (((((((int)threadIdx.z) * 4) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_3 * 2)) + (((int)threadIdx.x) >> 4)) & 7) * 256)) + ((((int)threadIdx.x) & 15) * 16)) + 524288));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 8; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a[ax0_0], (&(A_global_shared[((((k_0_0 & 1) * 4096) + (ax0_0 * 512)) + (k_0_1 * 256))])), 16);
      }
      for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) {
        nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b[ax1_0], (&(B_global_shared[(((((k_0_0 & 1) * 4096) + (k_0_1 * 2048)) + (((int)threadIdx.z) * 512)) + (ax1_0 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 8; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 2; ++j_0_2) {
          nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 2) + j_0_2)], A_global_shared_wmma_matrix_a[i_0_2], B_global_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 2) + j_0_2)]);
        }
      }
    }
  }
  for (int k_0_1_1 = 0; k_0_1_1 < 2; ++k_0_1_1) {
    for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1) {
      nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a_1[ax0_0_1], (&(A_global_shared[(((ax0_0_1 * 512) + (k_0_1_1 * 256)) + 4096)])), 16);
    }
    for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
      nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b_1[ax1_0_1], (&(B_global_shared[((((k_0_1_1 * 2048) + (((int)threadIdx.z) * 512)) + (ax1_0_1 * 256)) + 4096)])), 16);
    }
    for (int i_0_2_1 = 0; i_0_2_1 < 8; ++i_0_2_1) {
      for (int j_0_2_1 = 0; j_0_2_1 < 2; ++j_0_2_1) {
        nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2_1 * 2) + j_0_2_1)], A_global_shared_wmma_matrix_a_1[i_0_2_1], B_global_shared_wmma_matrix_b_1[j_0_2_1], C_wmma_accumulator[((i_0_2_1 * 2) + j_0_2_1)]);
      }
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 8; ++ax0_0_2) {
    for (int ax1_0_2 = 0; ax1_0_2 < 2; ++ax1_0_2) {
      nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 2097152) + (ax0_0_2 * 262144)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.z) * 32)) + (ax1_0_2 * 16))])), C_wmma_accumulator[((ax0_0_2 * 2) + ax1_0_2)], 16384, nvcuda::wmma::mem_row_major);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(B_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(B + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(A_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(A + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

