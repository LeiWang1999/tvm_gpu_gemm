#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(int8[16384, 16384])
    A_global_shared = alloc_buffer(int8[16384, 16384])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[16384, 16384])
    B_global = alloc_buffer(int8[16384, 16384])
    B_global_shared = alloc_buffer(int8[16384, 16384])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[16384, 16384])
    C_wmma.accumulator = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[v0, v1]])
            B_global[v0, v1] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[v0_1, v1_1]])
            A_global[v0_1, v1_1] = A[v0_1, v1_1]
        }
      }
      for (ax0_2: int32, 0, 16384) {
        for (ax1_2: int32, 0, 16384) {
          block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([A_global[v0_2, v1_2]])
            tir.writes([A_global_shared[v0_2, v1_2]])
            A_global_shared[v0_2, v1_2] = A_global[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "A_global_shared_wmma.matrix_a") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([A_global_shared[v0_3, v1_3]])
            tir.writes([A_global_shared_wmma.matrix_a[v0_3, v1_3]])
            A_global_shared_wmma.matrix_a[v0_3, v1_3] = A_global_shared[v0_3, v1_3]
        }
      }
      for (ax0_4: int32, 0, 16384) {
        for (ax1_4: int32, 0, 16384) {
          block([16384, 16384], "B_global_shared") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([B_global[v0_4, v1_4]])
            tir.writes([B_global_shared[v0_4, v1_4]])
            B_global_shared[v0_4, v1_4] = B_global[v0_4, v1_4]
        }
      }
      for (ax0_5: int32, 0, 16384) {
        for (ax1_5: int32, 0, 16384) {
          block([16384, 16384], "B_global_shared_wmma.matrix_b") as [v0_5, v1_5] {
            bind(v0_5, ax0_5)
            bind(v1_5, ax1_5)
            tir.reads([B_global_shared[v0_5, v1_5]])
            tir.writes([B_global_shared_wmma.matrix_b[v0_5, v1_5]])
            B_global_shared_wmma.matrix_b[v0_5, v1_5] = B_global_shared[v0_5, v1_5]
        }
      }
      for (i: int32, 0, 16384) {
        for (j: int32, 0, 16384) {
          for (k: int32, 0, 16384) {
            block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
              bind(vi, i)
              bind(vj, j)
              bind(vk, k)
              tir.reads([A_global_shared_wmma.matrix_a[vi, vk], B_global_shared_wmma.matrix_b[vk, vj]])
              tir.writes([C_wmma.accumulator[vi, vj]])
              with init() {
                C_wmma.accumulator[vi, vj] = 0
              }
              C_wmma.accumulator[vi, vj] = (C_wmma.accumulator[vi, vj] + (cast(int32, A_global_shared_wmma.matrix_a[vi, vk])*cast(int32, B_global_shared_wmma.matrix_b[vk, vj])))
          }
        }
      }
      for (ax0_6: int32, 0, 16384) {
        for (ax1_6: int32, 0, 16384) {
          block([16384, 16384], "C_wmma.accumulator") as [v0_6, v1_6] {
            bind(v0_6, ax0_6)
            bind(v1_6, ax1_6)
            tir.reads([C_wmma.accumulator[v0_6, v1_6]])
            tir.writes([C[v0_6, v1_6]])
            C[v0_6, v1_6] = C_wmma.accumulator[v0_6, v1_6]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}