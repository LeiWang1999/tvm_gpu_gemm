#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    A_global_shared_wmma.matrix_a = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared = alloc_buffer(int8[1024, 1024, 16, 16])
    B_global_shared_wmma.matrix_b = alloc_buffer(int8[1024, 1024, 16, 16])
    C_shared = alloc_buffer(int32[16384, 16384])
    C_shared_wmma.accumulator = alloc_buffer(int32[1024, 1024, 16, 16])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)]])
            B_global[floordiv(v0, 16), floordiv(v1, 16), floormod(v0, 16), floormod(v1, 16)] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)]])
            A_global[floordiv(v0_1, 16), floordiv(v1_1, 16), floormod(v0_1, 16), floormod(v1_1, 16)] = A[v0_1, v1_1]
        }
      }
      for (i_0_0: int32, 0, 128) "thread_binding" {
        for (j_0_0: int32, 0, 128) "thread_binding" {
          for (i_0_1: int32, 0, 1) "thread_binding" {
            for (j_0_1: int32, 0, 4) "thread_binding" {
              for (i_0_2_init: int32, 0, 8) {
                for (j_0_2_init: int32, 0, 2) {
                  block([1024, 1024], "B_init_o") as [vi_o, vj_o] {
                    bind(vi_o, (((i_0_0*8) + (i_0_1*8)) + i_0_2_init))
                    bind(vj_o, (((j_0_0*8) + (j_0_1*2)) + j_0_2_init))
                    tir.reads([])
                    tir.writes([C_shared_wmma.accumulator[vi_o, vj_o, 0:16, 0:16]])
                    C_2 = match_buffer(C_shared_wmma.accumulator[vi_o, vj_o, 0:16, 0:16])
                    @tir.tvm_fill_fragment(C_3: Pointer(wmma.accumulator int32), 16, 16, 16, ((floordiv(floordiv(elem_offset: int32, C_s0: int32), 16)*floordiv(C_s0, 16)) + floordiv(floormod(elem_offset, C_s0), 16)), 0f32, dtype=handle)
                }
              }
              for (k_0_0: int32, 0, 512) {
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0: int32, 0, 1) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2: int32, 0, 2) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4: int32, 0, 16) "vectorized" {
                          block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
                            bind(v0_2, (((i_0_0*128) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 256), 16)))
                            bind(v1_2, (((k_0_0*32) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 512), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4), 16)))
                            tir.reads([A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            tir.writes([A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]])
                            A_global_shared[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)] = A_global[floordiv(v0_2, 16), floordiv(v1_2, 16), floormod(v0_2, 16), floormod(v1_2, 16)]
                        }
                      }
                    }
                  }
                }
                for (ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1: int32, 0, 1) "thread_binding" {
                  for (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1: int32, 0, 4) "thread_binding" {
                    for (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1: int32, 0, 2) {
                      for (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1: int32, 0, 32) "thread_binding" {
                        for (ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1: int32, 0, 16) "vectorized" {
                          block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
                            bind(v0_3, (((k_0_0*32) + (floordiv((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 2048)*16)) + floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 256), 16)))
                            bind(v1_3, (((j_0_0*128) + (floordiv(floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 2048), 256)*16)) + floormod((((((ax0_0_ax1_0_ax0_1_ax1_1_fused_0_1*4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_1_1*1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1*512)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_3_1*16)) + ax0_0_ax1_0_ax0_1_ax1_1_fused_4_1), 16)))
                            tir.reads([B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                            tir.writes([B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]])
                            B_global_shared[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)] = B_global[floordiv(v0_3, 16), floordiv(v1_3, 16), floormod(v0_3, 16), floormod(v1_3, 16)]
                        }
                      }
                    }
                  }
                }
                for (k_0_1: int32, 0, 2) {
                  for (ax0_0: int32, 0, 8) {
                    for (ax1_0: int32, 0, 1) {
                      block([1024, 1024], "A_global_shared_wmma.matrix_a_o") as [v0_o, v1_o] {
                        bind(v0_o, ((i_0_0*8) + ax0_0))
                        bind(v1_o, (((k_0_0*2) + k_0_1) + ax1_0))
                        tir.reads([A_global_shared[v0_o, v1_o, 0:16, 0:16]])
                        tir.writes([A_global_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16]])
                        A_2 = match_buffer(A_global_shared[v0_o, v1_o, 0:16, 0:16])
                        C_4 = match_buffer(A_global_shared_wmma.matrix_a[v0_o, v1_o, 0:16, 0:16])
                        @tir.tvm_load_matrix_sync(C_5: Pointer(wmma.matrix_a int8), 16, 16, 16, ((floordiv(floordiv(elem_offset_1: int32, C_s0_1: int32), 16)*floordiv(C_s0_1, 16)) + floordiv(floormod(elem_offset_1, C_s0_1), 16)), @tir.tvm_access_ptr(@tir.type_annotation(, dtype=int8), A_3: Pointer(shared int8), elem_offset_2: int32, (A_s0: int32*16), 1, dtype=handle), A_s0, "row_major", dtype=handle)
                    }
                  }
                  for (ax0_0_1: int32, 0, 1) {
                    for (ax1_0_1: int32, 0, 2) {
                      for (ax0_1_1: int32, 0, 16) {
                        for (ax1_1_1: int32, 0, 16) {
                          block([16384, 16384], "B_global_shared_wmma.matrix_b") as [v0_4, v1_4] {
                            bind(v0_4, ((((k_0_0*32) + (k_0_1*16)) + (ax0_0_1*16)) + ax0_1_1))
                            bind(v1_4, ((((j_0_0*128) + (j_0_1*32)) + (ax1_0_1*16)) + ax1_1_1))
                            tir.reads([B_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            tir.writes([B_global_shared_wmma.matrix_b[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]])
                            B_global_shared_wmma.matrix_b[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)] = B_global_shared[floordiv(v0_4, 16), floordiv(v1_4, 16), floormod(v0_4, 16), floormod(v1_4, 16)]
                        }
                      }
                    }
                  }
                  for (i_0_2: int32, 0, 8) {
                    for (j_0_2: int32, 0, 2) {
                      for (i_1: int32, 0, 16) {
                        for (j_1: int32, 0, 16) {
                          for (k_1: int32, 0, 16) {
                            block([16384, 16384, tir.reduce_axis(0, 16384)], "B_update") as [vi, vj, vk] {
                              bind(vi, ((((i_0_0*128) + (i_0_1*128)) + (i_0_2*16)) + i_1))
                              bind(vj, ((((j_0_0*128) + (j_0_1*32)) + (j_0_2*16)) + j_1))
                              bind(vk, (((k_0_0*32) + (k_0_1*16)) + k_1))
                              tir.reads([C_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)], A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)], B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)]])
                              tir.writes([C_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)]])
                              C_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] = (C_shared_wmma.accumulator[floordiv(vi, 16), floordiv(vj, 16), floormod(vi, 16), floormod(vj, 16)] + (cast(int32, A_global_shared_wmma.matrix_a[floordiv(vi, 16), floordiv(vk, 16), floormod(vi, 16), floormod(vk, 16)])*cast(int32, B_global_shared_wmma.matrix_b[floordiv(vk, 16), floordiv(vj, 16), floormod(vk, 16), floormod(vj, 16)])))
                          }
                        }
                      }
                    }
                  }
                }
              }
              for (ax0_0_2: int32, 0, 8) {
                for (ax1_0_2: int32, 0, 2) {
                  for (ax0_1_2: int32, 0, 16) {
                    for (ax1_1_2: int32, 0, 16) {
                      block([16384, 16384], "C_shared_wmma.accumulator") as [v0_5, v1_5] {
                        bind(v0_5, (((i_0_0*128) + (ax0_0_2*16)) + ax0_1_2))
                        bind(v1_5, ((((j_0_0*128) + (j_0_1*32)) + (ax1_0_2*16)) + ax1_1_2))
                        tir.reads([C_shared_wmma.accumulator[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]])
                        tir.writes([C_shared[v0_5, v1_5]])
                        C_shared[v0_5, v1_5] = C_shared_wmma.accumulator[floordiv(v0_5, 16), floordiv(v1_5, 16), floormod(v0_5, 16), floormod(v1_5, 16)]
                    }
                  }
                }
                for (ax0_ax1_fused_0: int32, 0, 4) {
                  for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
                    for (ax0_ax1_fused_2: int32, 0, 1) "thread_binding" {
                      for (ax0_ax1_fused_3: int32, 0, 32) "thread_binding" {
                        block([16384, 16384], "C_shared") as [v0_6, v1_6] {
                          bind(v0_6, (((i_0_0*128) + (ax0_0_2*16)) + floordiv(((((ax0_ax1_fused_0*128) + (ax0_ax1_fused_1*32)) + (ax0_ax1_fused_2*32)) + ax0_ax1_fused_3), 32)))
                          bind(v1_6, (((j_0_0*128) + (j_0_1*32)) + floormod(((((ax0_ax1_fused_0*128) + (ax0_ax1_fused_1*32)) + (ax0_ax1_fused_2*32)) + ax0_ax1_fused_3), 32)))
                          tir.reads([C_shared[v0_6, v1_6]])
                          tir.writes([C[v0_6, v1_6]])
                          C[v0_6, v1_6] = C_shared[v0_6, v1_6]
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}