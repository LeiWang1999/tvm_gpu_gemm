#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    for (i: int32, 0, 16384) {
      for (j: int32, 0, 16384) {
        for (k: int32, 0, 16384) {
          block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
            bind(vi, i)
            bind(vj, j)
            bind(vk, k)
            tir.reads([A[vi, vk], B[vk, vj]])
            tir.writes([C[vi, vj]])
            with init() {
              C[vi, vj] = 0
            }
            C[vi, vj] = (C[vi, vj] + (cast(int32, A[vi, vk])*cast(int32, B[vk, vj])))
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}