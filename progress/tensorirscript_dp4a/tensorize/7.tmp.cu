
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ C) {
  signed char C_local[64];
  __shared__ int A_shared[1024];
  __shared__ signed char B_shared[4096];
  signed char A_shared_local[32];
  signed char B_shared_local[32];
  for (int j_1_1_0_1_init = 0; j_1_1_0_1_init < 2; ++j_1_1_0_1_init) {
    for (int i_1_1_0_1_init = 0; i_1_1_0_1_init < 2; ++i_1_1_0_1_init) {
      *(float*)(C_local + ((i_1_1_0_1_init * 2) + j_1_1_0_1_init)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 16)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 32)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 48)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 4)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 20)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 36)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 52)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 8)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 24)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 40)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 56)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 12)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 28)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 44)) = 0.000000e+00f;
      *(float*)(C_local + (((i_1_1_0_1_init * 2) + j_1_1_0_1_init) + 60)) = 0.000000e+00f;
    }
  }
  for (int k_0 = 0; k_0 < 512; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {
      A_shared[(((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_2)] = *(int*)(A + ((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.y) * 131072)) + ((((int)threadIdx.x) >> 1) * 16384)) + (k_0 * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ax0_ax1_fused_2 * 4)));
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 4; ++ax0_ax1_fused_2_1) {
      *(int*)(B_shared + (((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 16)) + (ax0_ax1_fused_2_1 * 4))) = *(int*)(B + ((((((k_0 * 524288) + (((int)threadIdx.y) * 32768)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + (ax0_ax1_fused_2_1 * 4)));
    }
    __syncthreads();
    for (int k_1_1_0 = 0; k_1_1_0 < 8; ++k_1_1_0) {
      for (int ax0 = 0; ax0 < 2; ++ax0) {
        *(int*)(A_shared_local + (ax0 * 4)) = A_shared[(((((int)threadIdx.x) * 16) + (ax0 * 8)) + k_1_1_0)];
        *(int*)(A_shared_local + ((ax0 * 4) + 8)) = A_shared[((((((int)threadIdx.x) * 16) + (ax0 * 8)) + k_1_1_0) + 256)];
        *(int*)(A_shared_local + ((ax0 * 4) + 16)) = A_shared[((((((int)threadIdx.x) * 16) + (ax0 * 8)) + k_1_1_0) + 512)];
        *(int*)(A_shared_local + ((ax0 * 4) + 24)) = A_shared[((((((int)threadIdx.x) * 16) + (ax0 * 8)) + k_1_1_0) + 768)];
      }
      for (int ax0_1 = 0; ax0_1 < 4; ++ax0_1) {
        *(char2*)(B_shared_local + (ax0_1 * 2)) = *(char2*)(B_shared + (((k_1_1_0 * 512) + (ax0_1 * 128)) + (((int)threadIdx.y) * 2)));
        *(char2*)(B_shared_local + ((ax0_1 * 2) + 8)) = *(char2*)(B_shared + ((((k_1_1_0 * 512) + (ax0_1 * 128)) + (((int)threadIdx.y) * 2)) + 32));
        *(char2*)(B_shared_local + ((ax0_1 * 2) + 16)) = *(char2*)(B_shared + ((((k_1_1_0 * 512) + (ax0_1 * 128)) + (((int)threadIdx.y) * 2)) + 64));
        *(char2*)(B_shared_local + ((ax0_1 * 2) + 24)) = *(char2*)(B_shared + ((((k_1_1_0 * 512) + (ax0_1 * 128)) + (((int)threadIdx.y) * 2)) + 96));
      }
      for (int j_1_1_0_1 = 0; j_1_1_0_1 < 2; ++j_1_1_0_1) {
        for (int i_1_1_0_1 = 0; i_1_1_0_1 < 2; ++i_1_1_0_1) {
          for (int k_1_1_1 = 0; k_1_1_1 < 4; ++k_1_1_1) {
            C_local[((i_1_1_0_1 * 2) + j_1_1_0_1)] = (C_local[((i_1_1_0_1 * 2) + j_1_1_0_1)] + (A_shared_local[((i_1_1_0_1 * 4) + k_1_1_1)] * B_shared_local[((k_1_1_1 * 2) + j_1_1_0_1)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 16)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 16)] + (A_shared_local[((i_1_1_0_1 * 4) + k_1_1_1)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 8)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 32)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 32)] + (A_shared_local[((i_1_1_0_1 * 4) + k_1_1_1)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 16)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 48)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 48)] + (A_shared_local[((i_1_1_0_1 * 4) + k_1_1_1)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 24)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 4)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 4)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 8)] * B_shared_local[((k_1_1_1 * 2) + j_1_1_0_1)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 20)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 20)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 8)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 8)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 36)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 36)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 8)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 16)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 52)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 52)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 8)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 24)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 8)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 8)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 16)] * B_shared_local[((k_1_1_1 * 2) + j_1_1_0_1)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 24)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 24)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 16)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 8)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 40)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 40)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 16)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 16)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 56)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 56)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 16)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 24)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 12)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 12)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 24)] * B_shared_local[((k_1_1_1 * 2) + j_1_1_0_1)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 28)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 28)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 24)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 8)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 44)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 44)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 24)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 16)]));
            C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 60)] = (C_local[(((i_1_1_0_1 * 2) + j_1_1_0_1) + 60)] + (A_shared_local[(((i_1_1_0_1 * 4) + k_1_1_1) + 24)] * B_shared_local[(((k_1_1_1 * 2) + j_1_1_0_1) + 24)]));
          }
        }
      }
    }
  }
  for (int ax0_2 = 0; ax0_2 < 2; ++ax0_2) {
    for (int ax1 = 0; ax1 < 2; ++ax1) {
      C[((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1)] = C_local[((ax0_2 * 2) + ax1)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 32)] = C_local[(((ax0_2 * 2) + ax1) + 16)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 64)] = C_local[(((ax0_2 * 2) + ax1) + 32)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 96)] = C_local[(((ax0_2 * 2) + ax1) + 48)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 524288)] = C_local[(((ax0_2 * 2) + ax1) + 4)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 524320)] = C_local[(((ax0_2 * 2) + ax1) + 20)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 524352)] = C_local[(((ax0_2 * 2) + ax1) + 36)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 524384)] = C_local[(((ax0_2 * 2) + ax1) + 52)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1048576)] = C_local[(((ax0_2 * 2) + ax1) + 8)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1048608)] = C_local[(((ax0_2 * 2) + ax1) + 24)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1048640)] = C_local[(((ax0_2 * 2) + ax1) + 40)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1048672)] = C_local[(((ax0_2 * 2) + ax1) + 56)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1572864)] = C_local[(((ax0_2 * 2) + ax1) + 12)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1572896)] = C_local[(((ax0_2 * 2) + ax1) + 28)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1572928)] = C_local[(((ax0_2 * 2) + ax1) + 44)];
      C[(((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.x) * 32768)) + (ax0_2 * 16384)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 2)) + ax1) + 1572960)] = C_local[(((ax0_2 * 2) + ax1) + 60)];
    }
  }
}

