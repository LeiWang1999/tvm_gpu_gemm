# from tvm.script import tir as T
@T.prim_func
def func(A: T.Buffer[(16384, 16384), "int8"], B: T.Buffer[(16384, 16384), "int8"], C: T.Buffer[(16384, 16384), "int8"]):
    # function attr dict
    T.func_attr({"tir.noalias": True, "global_symbol": "main"})
    # body
    # with T.block("root")
    A_shared = T.alloc_buffer([16384, 16384], dtype="int8", scope="shared")
    A_shared_local = T.alloc_buffer([16384, 16384], dtype="int8", scope="local")
    B_shared = T.alloc_buffer([16384, 16384], dtype="int8", scope="shared")
    B_shared_local = T.alloc_buffer([16384, 16384], dtype="int8", scope="local")
    C_local = T.alloc_buffer([16384, 16384], dtype="int8", scope="local")
    for ax0, ax1 in T.grid(16384, 16384):
        with T.block("B_shared"):
            v0, v1 = T.axis.remap("SS", [ax0, ax1])
            T.reads(B[v0, v1])
            T.writes(B_shared[v0, v1])
            B_shared[v0, v1] = B[v0, v1]
    for ax0, ax1 in T.grid(16384, 16384):
        with T.block("A_shared"):
            v0, v1 = T.axis.remap("SS", [ax0, ax1])
            T.reads(A[v0, v1])
            T.writes(A_shared[v0, v1])
            A_shared[v0, v1] = A[v0, v1]
    for ax0, ax1 in T.grid(16384, 16384):
        with T.block("A_shared_local"):
            v0, v1 = T.axis.remap("SS", [ax0, ax1])
            T.reads(A_shared[v0, v1])
            T.writes(A_shared_local[v0, v1])
            A_shared_local[v0, v1] = A_shared[v0, v1]
    for ax0, ax1 in T.grid(16384, 16384):
        with T.block("B_shared_local"):
            v0, v1 = T.axis.remap("SS", [ax0, ax1])
            T.reads(B_shared[v0, v1])
            T.writes(B_shared_local[v0, v1])
            B_shared_local[v0, v1] = B_shared[v0, v1]
    for j_0 in T.thread_binding(128, thread="blockIdx.y"):
        for i_0 in T.thread_binding(128, thread="blockIdx.x"):
            for j_1_0 in T.thread_binding(4, thread="vthread.y"):
                for i_1_0 in T.thread_binding(4, thread="vthread.x"):
                    for j_1_1_0_0 in T.thread_binding(16, thread="threadIdx.y"):
                        for i_1_1_0_0 in T.thread_binding(16, thread="threadIdx.x"):
                            for k_0, k_1_0, k_1_1_0, j_1_1_0_1, i_1_1_0_1, j_1_1_1, i_1_1_1, k_1_1_1 in T.grid(512, 1, 8, 2, 2, 1, 1, 4):
                                with T.block("B"):
                                    vi = T.axis.spatial(16384, i_1_1_1 + i_0 * 128 + i_1_0 * 32 + i_1_1_0_0 * 2 + i_1_1_0_1)
                                    vj = T.axis.spatial(16384, j_0 * 128 + j_1_0 * 32 + j_1_1_0_0 * 2 + j_1_1_0_1 + j_1_1_1)
                                    vk = T.axis.reduce(16384, k_0 * 32 + k_1_0 * 32 + k_1_1_0 * 4 + k_1_1_1)
                                    T.reads(A_shared_local[vi, vk], B_shared_local[vk, vj])
                                    T.writes(C_local[vi, vj])
                                    with T.init():
                                        C_local[vi, vj] = T.float32(0)
                                    C_local[vi, vj] = C_local[vi, vj] + A_shared_local[vi, vk] * B_shared_local[vk, vj]
                            for ax0, ax1 in T.grid(2, 2):
                                with T.block("C_local"):
                                    v0 = T.axis.spatial(16384, i_0 * 128 + i_1_0 * 32 + i_1_1_0_0 * 2 + ax0)
                                    v1 = T.axis.spatial(16384, j_0 * 128 + j_1_0 * 32 + j_1_1_0_0 * 2 + ax1)
                                    T.reads(C_local[v0, v1])
                                    T.writes(C[v0, v1])
                                    C[v0, v1] = C_local[v0, v1]
