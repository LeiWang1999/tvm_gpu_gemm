#[version = "0.0.5"]
@main = primfn(A_handle: handle, B_handle: handle, C_handle: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {A_handle: A, B_handle: B, C_handle: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    PA = alloc_buffer(int32[16384])
    for (i_0: int32, 0, 128) "thread_binding" {
      for (j_0: int32, 0, 64) "thread_binding" {
        for (k_0: int32, 0, 256) {
          for (i_1: int32, 0, 128) {
            for (j_1: int32, 0, 256) {
              for (k_1: int32, 0, 64) {
                block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
                  bind(vi, ((i_0*128) + i_1))
                  bind(vj, ((j_0*256) + j_1))
                  bind(vk, ((k_0*64) + k_1))
                  tir.reads([A[vi, vk], B[vj, vk]])
                  tir.writes([C[vi, vj]])
                  with init() {
                    C[vi, vj] = 0
                  }
                  C[vi, vj] = (C[vi, vj] + (cast(int32, A[vi, vk])*cast(int32, B[vj, vk])))
              }
            }
          }
          for (ax0: int32, 0, 128) {
            for (ax1: int32, 0, 64) {
              block([16384, tir.reduce_axis(0, 16384)], "Pre_compute_A") as [vi_1, vk_1] {
                bind(vi_1, ((i_0*128) + ax0))
                bind(vk_1, ((k_0*64) + ax1))
                tir.reads([A[vi_1, vk_1]])
                tir.writes([PA[vi_1]])
                with init() {
                  PA[vi_1] = 0
                }
                PA[vi_1] = (PA[vi_1] + (1*cast(int32, A[vi_1, vk_1])))
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}