#[version = "0.0.5"]
@main = primfn(A_handle: handle, B_handle: handle, C_handle: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {A_handle: A, B_handle: B, C_handle: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    PA = alloc_buffer(int32[16384])
    A_shared = alloc_buffer(int8[16384, 16384])
    B_shared = alloc_buffer(int8[16384, 16384])
    A_shared_warp = alloc_buffer(int8[1024, 512, 32, 16])
    B_shared_warp = alloc_buffer(int8[1024, 512, 32, 16])
    C_warp = alloc_buffer(int32[1024, 1024, 32, 8])
    for (i_0: int32, 0, 128) "thread_binding" {
      for (j_0: int32, 0, 64) "thread_binding" {
        for (ax0_0_init: int32, 0, 4) "thread_binding" {
          for (ax0_1_init: int32, 0, 32) "thread_binding" {
            block([16384], "Pre_compute_A_init") as [vi] {
              bind(vi, (((i_0*128) + (ax0_0_init*32)) + ax0_1_init))
              tir.reads([])
              tir.writes([PA[vi]])
              PA[vi] = 0
          }
        }
        for (i_1_0_init: int32, 0, 2) "thread_binding" {
          for (j_1_0_init: int32, 0, 4) "thread_binding" {
            for (i_1_1_0_init: int32, 0, 4) {
              for (j_1_1_0_init: int32, 0, 4) {
                for (i_1_1_1_init: int32, 0, 16) {
                  for (j_1_1_1_init: int32, 0, 16) {
                    block([16384, 16384], "B_init") as [vi_1, vj] {
                      bind(vi_1, ((((i_0*128) + (i_1_0_init*64)) + (i_1_1_0_init*16)) + i_1_1_1_init))
                      bind(vj, ((((j_0*256) + (j_1_0_init*64)) + (j_1_1_0_init*16)) + j_1_1_1_init))
                      tir.reads([])
                      tir.writes([C_warp[floordiv(vi_1, 16), floordiv(vj, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj, 2))]])
                      C_warp[floordiv(vi_1, 16), floordiv(vj, 16), ((floormod(vi_1, 8)*4) + floordiv(floormod(vj, 8), 2)), (((floordiv(floormod(vj, 16), 8)*4) + (floordiv(floormod(vi_1, 16), 8)*2)) + floormod(vj, 2))] = 0
                  }
                }
              }
            }
          }
        }
        for (k_0: int32, 0, 256) {
          for (ax0_ax1_fused_0: int32, 0, 4) {
            for (ax0_ax1_fused_1: int32, 0, 4) "thread_binding" {
              for (ax0_ax1_fused_2: int32, 0, 32) "thread_binding" {
                for (ax0_ax1_fused_3: int32, 0, 16) "vectorized" {
                  block([16384, 16384], "A_shared") as [v0, v1] {
                    bind(v0, ((i_0*128) + floordiv(((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*16)) + ax0_ax1_fused_3), 64)))
                    bind(v1, ((k_0*64) + floormod(((((ax0_ax1_fused_0*2048) + (ax0_ax1_fused_1*512)) + (ax0_ax1_fused_2*16)) + ax0_ax1_fused_3), 64)))
                    tir.reads([A[v0, v1]])
                    tir.writes([A_shared[v0, v1]])
                    tir.attrs({"buffer_dim_align": [[0, 0, 32, 0]]})
                    A_shared[v0, v1] = A[v0, v1]
                }
              }
            }
          }
          for (ax0_ax1_fused_0_1: int32, 0, 8) {
            for (ax0_ax1_fused_1_1: int32, 0, 4) "thread_binding" {
              for (ax0_ax1_fused_2_1: int32, 0, 32) "thread_binding" {
                for (ax0_ax1_fused_3_1: int32, 0, 16) "vectorized" {
                  block([16384, 16384], "B_shared") as [v0_1, v1_1] {
                    bind(v0_1, ((j_0*256) + floordiv(((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*512)) + (ax0_ax1_fused_2_1*16)) + ax0_ax1_fused_3_1), 64)))
                    bind(v1_1, ((k_0*64) + floormod(((((ax0_ax1_fused_0_1*2048) + (ax0_ax1_fused_1_1*512)) + (ax0_ax1_fused_2_1*16)) + ax0_ax1_fused_3_1), 64)))
                    tir.reads([B[v0_1, v1_1]])
                    tir.writes([B_shared[v0_1, v1_1]])
                    tir.attrs({"buffer_dim_align": [[0, 0, 32, 0]]})
                    B_shared[v0_1, v1_1] = B[v0_1, v1_1]
                }
              }
            }
          }
          for (i_1_0: int32, 0, 2) "thread_binding" {
            for (j_1_0: int32, 0, 4) "thread_binding" {
              for (i_1_1_0: int32, 0, 4) {
                for (j_1_1_0: int32, 0, 4) {
                  for (k_1_0: int32, 0, 2) {
                    for (ax0: int32, 0, 16) {
                      for (ax1: int32, 0, 32) {
                        block([16384, 16384], "A_shared_warp") as [v0_2, v1_2] {
                          bind(v0_2, ((((i_0*128) + (i_1_0*64)) + (i_1_1_0*16)) + ax0))
                          bind(v1_2, (((k_0*64) + (k_1_0*32)) + ax1))
                          tir.reads([A_shared[v0_2, v1_2]])
                          tir.writes([A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 32), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 16), 4)), (((floordiv(floormod(v1_2, 32), 16)*8) + (floordiv(floormod(v0_2, 16), 8)*4)) + floormod(v1_2, 4))]])
                          A_shared_warp[floordiv(v0_2, 16), floordiv(v1_2, 32), ((floormod(v0_2, 8)*4) + floordiv(floormod(v1_2, 16), 4)), (((floordiv(floormod(v1_2, 32), 16)*8) + (floordiv(floormod(v0_2, 16), 8)*4)) + floormod(v1_2, 4))] = A_shared[v0_2, v1_2]
                      }
                    }
                    for (ax0_1: int32, 0, 16) {
                      for (ax1_1: int32, 0, 32) {
                        block([16384, 16384], "B_shared_warp") as [v0_3, v1_3] {
                          bind(v0_3, ((((j_0*256) + (j_1_0*64)) + (j_1_1_0*16)) + ax0_1))
                          bind(v1_3, (((k_0*64) + (k_1_0*32)) + ax1_1))
                          tir.reads([B_shared[v0_3, v1_3]])
                          tir.writes([B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 32), ((floormod(v0_3, 8)*4) + floordiv(floormod(v1_3, 16), 4)), (((floordiv(floormod(v1_3, 32), 16)*8) + (floordiv(floormod(v0_3, 16), 8)*4)) + floormod(v1_3, 4))]])
                          B_shared_warp[floordiv(v0_3, 16), floordiv(v1_3, 32), ((floormod(v0_3, 8)*4) + floordiv(floormod(v1_3, 16), 4)), (((floordiv(floormod(v1_3, 32), 16)*8) + (floordiv(floormod(v0_3, 16), 8)*4)) + floormod(v1_3, 4))] = B_shared[v0_3, v1_3]
                      }
                    }
                    for (i_1_1_1: int32, 0, 16) {
                      for (j_1_1_1: int32, 0, 16) {
                        for (k_1_1: int32, 0, 32) {
                          block([16384, 16384, tir.reduce_axis(0, 16384)], "B_update") as [vi_2, vj_1, vk] {
                            bind(vi_2, ((((i_0*128) + (i_1_0*64)) + (i_1_1_0*16)) + i_1_1_1))
                            bind(vj_1, ((((j_0*256) + (j_1_0*64)) + (j_1_1_0*16)) + j_1_1_1))
                            bind(vk, (((k_0*64) + (k_1_0*32)) + k_1_1))
                            tir.reads([C_warp[floordiv(vi_2, 16), floordiv(vj_1, 16), ((floormod(vi_2, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_2, 16), 8)*2)) + floormod(vj_1, 2))], A_shared_warp[floordiv(vi_2, 16), floordiv(vk, 32), ((floormod(vi_2, 8)*4) + floordiv(floormod(vk, 16), 4)), (((floordiv(floormod(vk, 32), 16)*8) + (floordiv(floormod(vi_2, 16), 8)*4)) + floormod(vk, 4))], B_shared_warp[floordiv(vj_1, 16), floordiv(vk, 32), ((floormod(vj_1, 8)*4) + floordiv(floormod(vk, 16), 4)), (((floordiv(floormod(vk, 32), 16)*8) + (floordiv(floormod(vj_1, 16), 8)*4)) + floormod(vk, 4))]])
                            tir.writes([C_warp[floordiv(vi_2, 16), floordiv(vj_1, 16), ((floormod(vi_2, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_2, 16), 8)*2)) + floormod(vj_1, 2))]])
                            C_warp[floordiv(vi_2, 16), floordiv(vj_1, 16), ((floormod(vi_2, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_2, 16), 8)*2)) + floormod(vj_1, 2))] = (C_warp[floordiv(vi_2, 16), floordiv(vj_1, 16), ((floormod(vi_2, 8)*4) + floordiv(floormod(vj_1, 8), 2)), (((floordiv(floormod(vj_1, 16), 8)*4) + (floordiv(floormod(vi_2, 16), 8)*2)) + floormod(vj_1, 2))] + (cast(int32, A_shared_warp[floordiv(vi_2, 16), floordiv(vk, 32), ((floormod(vi_2, 8)*4) + floordiv(floormod(vk, 16), 4)), (((floordiv(floormod(vk, 32), 16)*8) + (floordiv(floormod(vi_2, 16), 8)*4)) + floormod(vk, 4))])*cast(int32, B_shared_warp[floordiv(vj_1, 16), floordiv(vk, 32), ((floormod(vj_1, 8)*4) + floordiv(floormod(vk, 16), 4)), (((floordiv(floormod(vk, 32), 16)*8) + (floordiv(floormod(vj_1, 16), 8)*4)) + floormod(vk, 4))])))
                        }
                      }
                    }
                  }
                }
              }
              for (ax0_0: int32, 0, 4) {
                for (ax1_0: int32, 0, 4) {
                  for (ax0_1_1: int32, 0, 16) {
                    for (ax1_1_1: int32, 0, 16) {
                      block([16384, 16384], "C_warp") as [v0_4, v1_4] {
                        bind(v0_4, ((((i_0*128) + (i_1_0*64)) + (ax0_0*16)) + ax0_1_1))
                        bind(v1_4, ((((j_0*256) + (j_1_0*64)) + (ax1_0*16)) + ax1_1_1))
                        tir.reads([C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), ((floormod(v0_4, 8)*4) + floordiv(floormod(v1_4, 8), 2)), (((floordiv(floormod(v1_4, 16), 8)*4) + (floordiv(floormod(v0_4, 16), 8)*2)) + floormod(v1_4, 2))]])
                        tir.writes([C[v0_4, v1_4]])
                        C[v0_4, v1_4] = C_warp[floordiv(v0_4, 16), floordiv(v1_4, 16), ((floormod(v0_4, 8)*4) + floordiv(floormod(v1_4, 8), 2)), (((floordiv(floormod(v1_4, 16), 8)*4) + (floordiv(floormod(v0_4, 16), 8)*2)) + floormod(v1_4, 2))]
                    }
                  }
                }
              }
            }
          }
          for (ax0_0_1: int32, 0, 4) "thread_binding" {
            for (ax0_1_2: int32, 0, 32) "thread_binding" {
              for (ax1_0_1: int32, 0, 2) "thread_binding" {
                for (ax1_1_2: int32, 0, 32) {
                  block([16384, tir.reduce_axis(0, 16384)], "Pre_compute_A_update") as [vi_3, vk_1] {
                    bind(vi_3, (((i_0*128) + (ax0_0_1*32)) + ax0_1_2))
                    bind(vk_1, (((k_0*64) + (ax1_0_1*32)) + ax1_1_2))
                    tir.reads([PA[vi_3], A_shared[vi_3, vk_1]])
                    tir.writes([PA[vi_3]])
                    PA[vi_3] = (PA[vi_3] + (1*cast(int32, A_shared[vi_3, vk_1])))
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}