#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel2(signed char* __restrict__ A_global, signed char* __restrict__ B_global, int* __restrict__ PA, int* __restrict__ QC) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> QC_wmma_accumulator[16];
  __shared__ signed char A_global_shared[16384];
  __shared__ signed char B_global_shared[4096];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_global_shared_wmma_matrix_a[4];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> B_global_shared_wmma_matrix_b[4];
  for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init) {
    for (int j_0_2_init = 0; j_0_2_init < 4; ++j_0_2_init) {
      nvcuda::wmma::fill_fragment(QC_wmma_accumulator[((i_0_2_init * 4) + j_0_2_init)], 0.000000e+00f);
    }
  }

  const int MAX_BLOCK_N = 32;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int k_0_0 = 0; k_0_0 < 256; ++k_0_0) {
    __syncthreads();
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2 < 8; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2) {
      *(int4*)(A_global_shared + (((((int)threadIdx.y) * 4096) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A_global + ((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + ((ax0_0_ax1_0_ax0_1_ax1_1_fused_2 >> 1) * 262144)) + (k_0_0 * 1024)) + ((ax0_0_ax1_0_ax0_1_ax1_1_fused_2 & 1) * 512)) + (((int)threadIdx.x) * 16)));
    }
    for (int ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 = 0; ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 < 2; ++ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1) {
      *(int4*)(B_global_shared + (((((int)threadIdx.y) * 1024) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B_global + (((((((int)blockIdx.x) * 1048576) + (((int)threadIdx.y) * 262144)) + (k_0_0 * 1024)) + (ax0_0_ax1_0_ax0_1_ax1_1_fused_2_1 * 512)) + (((int)threadIdx.x) * 16)));
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(A_global_shared_wmma_matrix_a[ax0_0], (&(A_global_shared[(((((int)threadIdx.y) * 4096) + (ax0_0 * 1024)) + (k_0_1 * 256))])), 16);
      }
      for (int ax0_0_1 = 0; ax0_0_1 < 4; ++ax0_0_1) {
        nvcuda::wmma::load_matrix_sync(B_global_shared_wmma_matrix_b[ax0_0_1], (&(B_global_shared[((ax0_0_1 * 1024) + (k_0_1 * 256))])), 16);
      }
      for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2) {
        for (int j_0_2 = 0; j_0_2 < 4; ++j_0_2) {
          nvcuda::wmma::mma_sync(QC_wmma_accumulator[((i_0_2 * 4) + j_0_2)], A_global_shared_wmma_matrix_a[i_0_2], B_global_shared_wmma_matrix_b[j_0_2], QC_wmma_accumulator[((i_0_2 * 4) + j_0_2)]);
        }
      }
      for (int ax0_1 = 0; ax0_1 < 256; ++ax0_1) {
        for (int ax1 = 0; ax1 < 16384; ++ax1) {
          if (ax1 == 0) {
            PA[((((int)blockIdx.y) * 256) + ax0_1)] = 0;
          }
          PA[((((int)blockIdx.y) * 256) + ax0_1)] = ((((int)A_global[(((((((int)blockIdx.y) * 4194304) + ((ax0_1 >> 4) * 262144)) + ((ax1 >> 4) * 256)) + ((ax0_1 & 15) * 16)) + (ax1 & 15))]) * 4) + PA[((((int)blockIdx.y) * 256) + ax0_1)]);
        }
      }
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2) {
    for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
      nvcuda::wmma::store_matrix_sync((&(QC[(((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 1048576)) + (ax0_0_2 * 262144)) + (((int)blockIdx.x) * 64)) + (ax1_0 * 16))])), QC_wmma_accumulator[((ax0_0_2 * 4) + ax1_0)], 16384, nvcuda::wmma::mem_row_major);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel0(signed char* __restrict__ B_global, signed char* __restrict__ B) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(B_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(B + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

extern "C" __global__ void __launch_bounds__(1024) main_kernel1(signed char* __restrict__ A_global, signed char* __restrict__ A) {
  for (int ax0_ax1_fused_5 = 0; ax0_ax1_fused_5 < 2; ++ax0_ax1_fused_5) {
    *(int4*)(A_global + (((((((int)blockIdx.y) * 67108864) + ((((((((int)blockIdx.x) * 32768) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)) >> 18) * 262144)) + (((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_5) & 1023) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.y) >> 6) * 16))) = *(int4*)(A + (((((((int)blockIdx.y) * 67108864) + (((int)blockIdx.x) * 32768)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.x) * 32)) + (ax0_ax1_fused_5 * 16)));
  }
}

