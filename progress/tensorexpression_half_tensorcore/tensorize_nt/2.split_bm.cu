@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [8192], []),
             B: Buffer(B_2: Pointer(float16), float16, [16384], []),
             C: Buffer(C_2: Pointer(float32), float32, [32768], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [128, 64], []), B_1: B_3: Buffer(B_2, float16, [256, 64], []), C_1: C_3: Buffer(C_2, float32, [128, 256], [])} {
  allocate(A.shared: Pointer(shared float16), float16, [16384]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [8192]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [16384]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [32768]), storage_scope = wmma.accumulator;
  allocate(C.wmma.accumulator.shared: Pointer(shared float32), float32, [32768]), storage_scope = shared {
    for (ax0: int32, 0, 128) {
      for (ax1: int32, 0, 64) {
        let cse_var_1: int32 = ((ax0*64) + ax1)
        A.shared_1: Buffer(A.shared, float16, [8192], [], scope="shared")[cse_var_1] = A[cse_var_1]
      }
    }
    for (ax0_1: int32, 0, 128) {
      for (ax1_1: int32, 0, 64) {
        let cse_var_2: int32 = ((ax0_1*64) + ax1_1)
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [8192], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
      }
    }
    for (ax0_2: int32, 0, 256) {
      for (ax1_2: int32, 0, 64) {
        let cse_var_3: int32 = ((ax0_2*64) + ax1_2)
        A.shared_2: Buffer(A.shared, float16, [16384], [], scope="shared")[cse_var_3] = B[cse_var_3]
      }
    }
    for (ax0_3: int32, 0, 256) {
      for (ax1_3: int32, 0, 64) {
        let cse_var_4: int32 = ((ax0_3*64) + ax1_3)
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [16384], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
      }
    }
    for (ii.c: int32, 0, 128) {
      for (jj.c: int32, 0, 256) {
        C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [32768], [], scope="wmma.accumulator")[((ii.c*256) + jj.c)] = 0f32
        for (rk: int32, 0, 64) {
          let cse_var_5: int32 = ((ii.c*256) + jj.c)
          C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(float32, A.shared.wmma.matrix_a_1[((ii.c*64) + rk)])*cast(float32, B.shared.wmma.matrix_b_1[((jj.c*64) + rk)])))
        }
      }
    }
    for (ax0_4: int32, 0, 128) {
      for (ax1_4: int32, 0, 256) {
        let cse_var_6: int32 = ((ax0_4*256) + ax1_4)
        C.wmma.accumulator.shared_1: Buffer(C.wmma.accumulator.shared, float32, [32768], [], scope="shared")[cse_var_6] = C.wmma.accumulator_1[cse_var_6]
      }
    }
    for (ii.outer: int32, 0, 8) {
      for (ii.inner: int32, 0, 16) {
        for (jj: int32, 0, 256) {
          let cse_var_7: int32 = (((ii.outer*4096) + (ii.inner*256)) + jj)
          C[cse_var_7] = C.wmma.accumulator.shared_1[cse_var_7]
        }
      }
    }
  }
}

