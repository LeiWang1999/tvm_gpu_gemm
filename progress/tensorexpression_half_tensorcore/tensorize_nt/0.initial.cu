@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [8192], []),
             B: Buffer(B_2: Pointer(float16), float16, [16384], []),
             C: Buffer(C_2: Pointer(float32), float32, [32768], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [128, 64], []), B_1: B_3: Buffer(B_2, float16, [256, 64], []), C_1: C_3: Buffer(C_2, float32, [128, 256], [])} {
  for (ii: int32, 0, 128) {
    for (jj: int32, 0, 256) {
      C[((ii*256) + jj)] = 0f32
      for (rk: int32, 0, 64) {
        let cse_var_1: int32 = ((ii*256) + jj)
        C[cse_var_1] = (C[cse_var_1] + (cast(float32, A[((ii*64) + rk)])*cast(float32, B[((jj*64) + rk)])))
      }
    }
  }
}

