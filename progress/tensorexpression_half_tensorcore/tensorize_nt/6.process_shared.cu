@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [8192], []),
             B: Buffer(B_2: Pointer(float16), float16, [16384], []),
             C: Buffer(C_2: Pointer(float32), float32, [32768], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, float16, [128, 64], []), B_1: B_3: Buffer(B_2, float16, [256, 64], []), C_1: C_3: Buffer(C_2, float32, [128, 256], [])} {
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 8;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [256]), storage_scope = wmma.accumulator;
  allocate(A.shared: Pointer(shared float16), float16, [256]), storage_scope = shared;
  allocate(B.shared: Pointer(shared float16), float16, [256]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [256]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [256]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator.shared: Pointer(shared float32), float32, [256]), storage_scope = shared;
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 16 {
    attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 1;
    attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 1 {
      for (ii.c.inner.init: int32, 0, 16) {
        for (jj.c.inner.init: int32, 0, 16) {
          C.wmma.accumulator_1: Buffer(C.wmma.accumulator, float32, [256], [], scope="wmma.accumulator")[((ii.c.inner.init*16) + jj.c.inner.init)] = 0f32
        }
      }
      for (rk.outer.outer: int32, 0, 4) {
        let cse_var_1: int32 = (rk.outer.outer*16)
         {
          attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
          A.shared_1: Buffer(A.shared, float16, [256], [], scope="shared")[ramp((threadIdx.x*8), 1, 8)] = A[ramp(((((blockIdx.y*1024) + (floordiv(threadIdx.x, 2)*64)) + cse_var_1) + (floormod(threadIdx.x, 2)*8)), 1, 8)]
          attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
          B.shared_1: Buffer(B.shared, float16, [256], [], scope="shared")[ramp((threadIdx.x*8), 1, 8)] = B[ramp(((((blockIdx.x*1024) + (floordiv(threadIdx.x, 2)*64)) + cse_var_1) + (floormod(threadIdx.x, 2)*8)), 1, 8)]
          for (ax0.inner: int32, 0, 16) {
            for (ax1.inner: int32, 0, 16) {
              let cse_var_2: int32 = ((ax0.inner*16) + ax1.inner)
              A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, float16, [256], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
            }
          }
          for (ax0.inner_1: int32, 0, 16) {
            for (ax1.inner_1: int32, 0, 16) {
              let cse_var_3: int32 = ((ax0.inner_1*16) + ax1.inner_1)
              B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, float16, [256], [], scope="wmma.matrix_b")[cse_var_3] = B.shared_1[cse_var_3]
            }
          }
          for (ii.c.inner: int32, 0, 16) {
            for (jj.c.inner: int32, 0, 16) {
              for (rk.inner: int32, 0, 16) {
                let cse_var_5: int32 = (ii.c.inner*16)
                let cse_var_4: int32 = (cse_var_5 + jj.c.inner)
                C.wmma.accumulator_1[cse_var_4] = (C.wmma.accumulator_1[cse_var_4] + (cast(float32, A.shared.wmma.matrix_a_1[(cse_var_5 + rk.inner)])*cast(float32, B.shared.wmma.matrix_b_1[((jj.c.inner*16) + rk.inner)])))
              }
            }
          }
        }
      }
      for (ax0.inner_2: int32, 0, 16) {
        for (ax1.inner_2: int32, 0, 16) {
          let cse_var_6: int32 = ((ax0.inner_2*16) + ax1.inner_2)
          C.wmma.accumulator.shared_1: Buffer(C.wmma.accumulator.shared, float32, [256], [], scope="shared")[cse_var_6] = C.wmma.accumulator_1[cse_var_6]
        }
      }
    }
    attr [IterVar(threadIdx.z, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 1;
    attr [IterVar(threadIdx.y, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 1;
    attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
    C[ramp(((((blockIdx.y*4096) + (floordiv(threadIdx.x, 2)*256)) + (blockIdx.x*16)) + (floormod(threadIdx.x, 2)*8)), 1, 8)] = C.wmma.accumulator.shared_1[ramp((threadIdx.x*8), 1, 8)]
  }
}

