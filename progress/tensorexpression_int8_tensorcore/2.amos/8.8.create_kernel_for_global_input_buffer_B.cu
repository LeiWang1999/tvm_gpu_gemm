@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main", "from_legacy_te_schedule": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.global: Pointer(global int8), int8, [268435456i64]), storage_scope = global;
  allocate(A.global.shared: Pointer(shared int8), int8, [268435456i64]), storage_scope = shared;
  allocate(A.global.shared.local: Pointer(local int8), int8, [268435456]), storage_scope = local;
  allocate(B.global.shared.local: Pointer(local int8), int8, [268435456]), storage_scope = local;
  allocate(C.local: Pointer(local int32), int32, [268435456]), storage_scope = local;
  allocate(C.local.global: Pointer(global int32), int32, [268435456]), storage_scope = global {
    attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 1024;
    attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 32;
    attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
    for (axis2.axis3.fused.outer: int32, 0, 16) {
      A.global_1: Buffer(A.global, int8, [268435456], [])[ramp(((((blockIdx.x*262144) + (threadIdx.y*8192)) + (threadIdx.x*256)) + (axis2.axis3.fused.outer*16)), 1, 16)] = A[ramp(((((blockIdx.x*262144) + (axis2.axis3.fused.outer*16384)) + (threadIdx.y*512)) + (threadIdx.x*16)), 1, 16)]
    }
    for (axis0: int32, 0, 1024) {
      for (axis1: int32, 0, 1024) {
        for (axis2: int32, 0, 16) {
          for (axis3: int32, 0, 16) {
            let cse_var_1: int32 = ((((axis0*262144) + (axis1*256)) + (axis2*16)) + axis3)
            A.global.shared_1: Buffer(A.global.shared, int8, [268435456], [], scope="shared")[cse_var_1] = A.global_1[cse_var_1]
          }
        }
      }
    }
    for (axis0_1: int32, 0, 1024) {
      for (axis1_1: int32, 0, 1024) {
        for (axis2_1: int32, 0, 16) {
          for (axis3_1: int32, 0, 16) {
            let cse_var_2: int32 = ((((axis0_1*262144) + (axis1_1*256)) + (axis2_1*16)) + axis3_1)
            A.global.shared.local_1: Buffer(A.global.shared.local, int8, [268435456], [], scope="local")[cse_var_2] = A.global.shared_1[cse_var_2]
          }
        }
      }
    }
    attr [IterVar(blockIdx.x_1: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 1024;
    attr [IterVar(threadIdx.y_1: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 32;
    attr [IterVar(threadIdx.x_1: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
    for (axis2.axis3.fused.outer_1: int32, 0, 16) {
      A.global_2: Buffer(A.global, int8, [268435456], [])[ramp(((((blockIdx.x_1*262144) + (threadIdx.y_1*8192)) + (threadIdx.x_1*256)) + (axis2.axis3.fused.outer_1*16)), 1, 16)] = B[ramp(((((blockIdx.x_1*262144) + (axis2.axis3.fused.outer_1*16384)) + (threadIdx.y_1*512)) + (threadIdx.x_1*16)), 1, 16)]
    }
    for (axis0_2: int32, 0, 1024) {
      for (axis1_2: int32, 0, 1024) {
        for (axis2_2: int32, 0, 16) {
          for (axis3_2: int32, 0, 16) {
            let cse_var_3: int32 = ((((axis0_2*262144) + (axis1_2*256)) + (axis2_2*16)) + axis3_2)
            A.global.shared_2: Buffer(A.global.shared, int8, [268435456], [], scope="shared")[cse_var_3] = A.global_2[cse_var_3]
          }
        }
      }
    }
    for (axis0_3: int32, 0, 1024) {
      for (axis1_3: int32, 0, 1024) {
        for (axis2_3: int32, 0, 16) {
          for (axis3_3: int32, 0, 16) {
            let cse_var_4: int32 = ((((axis0_3*262144) + (axis1_3*256)) + (axis2_3*16)) + axis3_3)
            B.global.shared.local_1: Buffer(B.global.shared.local, int8, [268435456], [], scope="local")[cse_var_4] = A.global.shared_2[cse_var_4]
          }
        }
      }
    }
    for (axis0_4: int32, 0, 1024) {
      for (axis1_4: int32, 0, 1024) {
        for (axis2_4: int32, 0, 16) {
          for (axis3_4: int32, 0, 16) {
            C.local_1: Buffer(C.local, int32, [268435456], [], scope="local")[((((axis0_4*262144) + (axis1_4*256)) + (axis2_4*16)) + axis3_4)] = 0
            for (rk.outer: int32, 0, 1024) {
              for (rk.inner: int32, 0, 16) {
                let cse_var_8: int32 = (axis0_4*262144)
                let cse_var_7: int32 = (axis2_4*16)
                let cse_var_6: int32 = (rk.outer*256)
                let cse_var_5: int32 = (((cse_var_8 + (axis1_4*256)) + cse_var_7) + axis3_4)
                C.local_1[cse_var_5] = (C.local_1[cse_var_5] + (cast(int32, A.global.shared.local_1[(((cse_var_8 + cse_var_6) + cse_var_7) + rk.inner)])*cast(int32, B.global.shared.local_1[((((axis1_4*262144) + cse_var_6) + (axis3_4*16)) + rk.inner)])))
              }
            }
          }
        }
      }
    }
    for (axis0_5: int32, 0, 1024) {
      for (axis1_5: int32, 0, 1024) {
        for (axis2_5: int32, 0, 16) {
          for (axis3_5: int32, 0, 16) {
            let cse_var_9: int32 = ((((axis0_5*262144) + (axis1_5*256)) + (axis2_5*16)) + axis3_5)
            C.local.global_1: Buffer(C.local.global, int32, [268435456], [])[cse_var_9] = C.local_1[cse_var_9]
          }
        }
      }
    }
    for (ii: int32, 0, 16384) {
      for (jj: int32, 0, 16384) {
        C[((ii*16384) + jj)] = C.local.global_1[((((floordiv(ii, 16)*262144) + (floordiv(jj, 16)*256)) + (floormod(ii, 16)*16)) + floormod(jj, 16))]
      }
    }
  }
}

