@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main", "from_legacy_te_schedule": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.global: Pointer(global int8), int8, [268435456i64]), storage_scope = global;
  allocate(A.global.shared: Pointer(shared int8), int8, [268435456i64]), storage_scope = shared;
  allocate(A.global.shared.local: Pointer(local int8), int8, [268435456]), storage_scope = local;
  allocate(B.global.shared.local: Pointer(local int8), int8, [268435456]), storage_scope = local;
  allocate(C.local: Pointer(local int32), int32, [268435456]), storage_scope = local;
  allocate(C.local.global: Pointer(global int32), int32, [268435456]), storage_scope = global {
    for (axis0: int32, 0, 1024) {
      for (axis1: int32, 0, 1024) {
        for (axis2: int32, 0, 16) {
          for (axis3: int32, 0, 16) {
            let cse_var_1: int32 = (axis0*262144)
            A.global_1: Buffer(A.global, int8, [268435456], [])[(((cse_var_1 + (axis1*256)) + (axis2*16)) + axis3)] = A[(((cse_var_1 + (axis2*16384)) + (axis1*16)) + axis3)]
          }
        }
      }
    }
    for (axis0_1: int32, 0, 1024) {
      for (axis1_1: int32, 0, 1024) {
        for (axis2_1: int32, 0, 16) {
          for (axis3_1: int32, 0, 16) {
            let cse_var_2: int32 = ((((axis0_1*262144) + (axis1_1*256)) + (axis2_1*16)) + axis3_1)
            A.global.shared_1: Buffer(A.global.shared, int8, [268435456], [], scope="shared")[cse_var_2] = A.global_1[cse_var_2]
          }
        }
      }
    }
    for (axis0_2: int32, 0, 1024) {
      for (axis1_2: int32, 0, 1024) {
        for (axis2_2: int32, 0, 16) {
          for (axis3_2: int32, 0, 16) {
            let cse_var_3: int32 = ((((axis0_2*262144) + (axis1_2*256)) + (axis2_2*16)) + axis3_2)
            A.global.shared.local_1: Buffer(A.global.shared.local, int8, [268435456], [], scope="local")[cse_var_3] = A.global.shared_1[cse_var_3]
          }
        }
      }
    }
    for (axis0_3: int32, 0, 1024) {
      for (axis1_3: int32, 0, 1024) {
        for (axis2_3: int32, 0, 16) {
          for (axis3_3: int32, 0, 16) {
            let cse_var_4: int32 = (axis0_3*262144)
            A.global_2: Buffer(A.global, int8, [268435456], [])[(((cse_var_4 + (axis1_3*256)) + (axis2_3*16)) + axis3_3)] = B[(((cse_var_4 + (axis2_3*16384)) + (axis1_3*16)) + axis3_3)]
          }
        }
      }
    }
    for (axis0_4: int32, 0, 1024) {
      for (axis1_4: int32, 0, 1024) {
        for (axis2_4: int32, 0, 16) {
          for (axis3_4: int32, 0, 16) {
            let cse_var_5: int32 = ((((axis0_4*262144) + (axis1_4*256)) + (axis2_4*16)) + axis3_4)
            A.global.shared_2: Buffer(A.global.shared, int8, [268435456], [], scope="shared")[cse_var_5] = A.global_2[cse_var_5]
          }
        }
      }
    }
    for (axis0_5: int32, 0, 1024) {
      for (axis1_5: int32, 0, 1024) {
        for (axis2_5: int32, 0, 16) {
          for (axis3_5: int32, 0, 16) {
            let cse_var_6: int32 = ((((axis0_5*262144) + (axis1_5*256)) + (axis2_5*16)) + axis3_5)
            B.global.shared.local_1: Buffer(B.global.shared.local, int8, [268435456], [], scope="local")[cse_var_6] = A.global.shared_2[cse_var_6]
          }
        }
      }
    }
    for (axis0_6: int32, 0, 1024) {
      for (axis1_6: int32, 0, 1024) {
        for (axis2_6: int32, 0, 16) {
          for (axis3_6: int32, 0, 16) {
            C.local_1: Buffer(C.local, int32, [268435456], [], scope="local")[((((axis0_6*262144) + (axis1_6*256)) + (axis2_6*16)) + axis3_6)] = 0
            for (rk.outer: int32, 0, 1024) {
              for (rk.inner: int32, 0, 16) {
                let cse_var_10: int32 = (axis0_6*262144)
                let cse_var_9: int32 = (axis2_6*16)
                let cse_var_8: int32 = (rk.outer*256)
                let cse_var_7: int32 = (((cse_var_10 + (axis1_6*256)) + cse_var_9) + axis3_6)
                C.local_1[cse_var_7] = (C.local_1[cse_var_7] + (cast(int32, A.global.shared.local_1[(((cse_var_10 + cse_var_8) + cse_var_9) + rk.inner)])*cast(int32, B.global.shared.local_1[((((axis1_6*262144) + cse_var_8) + (axis3_6*16)) + rk.inner)])))
              }
            }
          }
        }
      }
    }
    for (ax0: int32, 0, 16384) {
      for (ax1: int32, 0, 16384) {
        C.local.global_1: Buffer(C.local.global, int32, [268435456], [])[((ax0*16384) + ax1)] = C.local_1[((((floordiv(ax0, 16)*262144) + (floordiv(ax1, 16)*256)) + (floormod(ax0, 16)*16)) + floormod(ax1, 16))]
      }
    }
    for (ii: int32, 0, 16384) {
      for (jj: int32, 0, 16384) {
        let cse_var_11: int32 = ((ii*16384) + jj)
        C[cse_var_11] = C.local.global_1[cse_var_11]
      }
    }
  }
}

