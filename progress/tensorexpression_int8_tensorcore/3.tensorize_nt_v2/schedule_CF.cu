@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main", "from_legacy_te_schedule": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.shared: Pointer(shared int8), int8, [262144]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a int8), int8, [262144]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b int8), int8, [262144]), storage_scope = wmma.matrix_b {
    for (axis1: int32, 0, 1024) {
      for (axis2: int32, 0, 16) {
        for (axis3: int32, 0, 16) {
          A.shared_1: Buffer(A.shared, int8, [262144], [], scope="shared")[(((axis1*256) + (axis2*16)) + axis3)] = A[((((floordiv(blockIdx.x: int32, 4)*262144) + (axis2*16384)) + (axis1*16)) + axis3)]
        }
      }
    }
    for (ax0: int32, 0, 16) {
      for (ax1: int32, 0, 16384) {
        A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, int8, [262144], [], scope="wmma.matrix_a")[((ax0*16384) + ax1)] = A.shared_1[(((floordiv(ax1, 16)*256) + (ax0*16)) + floormod(ax1, 16))]
      }
    }
    for (axis1_1: int32, 0, 1024) {
      for (axis2_1: int32, 0, 16) {
        for (axis3_1: int32, 0, 16) {
          A.shared_2: Buffer(A.shared, int8, [262144], [], scope="shared")[(((axis1_1*256) + (axis2_1*16)) + axis3_1)] = B[((((blockIdx.y: int32*262144) + (axis2_1*16384)) + (axis1_1*16)) + axis3_1)]
        }
      }
    }
    for (ax0_1: int32, 0, 16) {
      for (ax1_1: int32, 0, 16384) {
        B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, int8, [262144], [], scope="wmma.matrix_b")[((ax0_1*16384) + ax1_1)] = A.shared_2[(((floordiv(ax1_1, 16)*256) + (ax0_1*16)) + floormod(ax1_1, 16))]
      }
    }
    attr [IterVar(blockIdx.x, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 4096;
    allocate(C.wmma.accumulator: Pointer(wmma.accumulator int32), int32, [16]), storage_scope = wmma.accumulator;
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 1024;
    attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
    attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 2 {
      for (axis2.init: int32, 0, 16) {
        for (axis3.init: int32, 0, 16) {
          C.wmma.accumulator_1: Buffer(C.wmma.accumulator, int32, [16], [], scope="wmma.accumulator", align=64)[(((((floordiv(((axis2.init - (threadIdx.y*2)) - (blockIdx.x*4)), 16)*16) + (floordiv(blockIdx.x, 4)*16)) + (floormod((((threadIdx.y*14) + (blockIdx.x*12)) + axis2.init), 16)*8)) + axis3.init) - (threadIdx.z*8))] = 0
        }
      }
      for (rk.outer.outer: int32, 0, 256) {
        for (rk.outer.inner: int32, 0, 4) {
          for (axis2_2: int32, 0, 16) {
            for (axis3_2: int32, 0, 16) {
              for (rk.inner: int32, 0, 16) {
                let cse_var_2: int32 = (rk.outer.outer*64)
                let cse_var_1: int32 = (rk.outer.inner*16)
                C.wmma.accumulator_1[(((((floordiv(((axis2_2 - (threadIdx.y*2)) - (blockIdx.x*4)), 16)*16) + (floordiv(blockIdx.x, 4)*16)) + (floormod((((threadIdx.y*14) + (blockIdx.x*12)) + axis2_2), 16)*8)) + axis3_2) - (threadIdx.z*8))] = (C.wmma.accumulator_1[(((((floordiv(((axis2_2 - (threadIdx.y*2)) - (blockIdx.x*4)), 16)*16) + (floordiv(blockIdx.x, 4)*16)) + (floormod((((threadIdx.y*14) + (blockIdx.x*12)) + axis2_2), 16)*8)) + axis3_2) - (threadIdx.z*8))] + (cast(int32, A.shared.wmma.matrix_a_1[((((axis2_2*16384) + cse_var_2) + cse_var_1) + rk.inner)])*cast(int32, B.shared.wmma.matrix_b_1[((((axis3_2*16384) + cse_var_2) + cse_var_1) + rk.inner)])))
              }
            }
          }
        }
      }
      for (ii.inner: int32, 0, 2) {
        for (jj.inner: int32, 0, 8) {
          C[((((((blockIdx.x*65536) + (threadIdx.y*32768)) + (ii.inner*16384)) + (blockIdx.y*16)) + (threadIdx.z*8)) + jj.inner)] = C.wmma.accumulator_1[((ii.inner*8) + jj.inner)]
        }
      }
    }
  }
}

