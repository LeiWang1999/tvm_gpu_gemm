#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_shared[4096];
  __shared__ signed char B_shared[8192];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> B_shared_wmma_matrix_b[4];
  for (int i_c_init = 0; i_c_init < 4; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 4; ++j_c_init) {
      nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_c_init * 4) + j_c_init)], 0.000000e+00f);
    }
  }
  for (int k1_outer = 0; k1_outer < 512; ++k1_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_inner = 0; ax0_ax1_fused_inner_inner < 2; ++ax0_ax1_fused_inner_inner) {
      *(int2*)(A_shared + ((((((int)threadIdx.y) * 2048) + (((int)threadIdx.z) * 512)) + (ax0_ax1_fused_inner_inner * 256)) + (((int)threadIdx.x) * 8))) = *(int2*)(A + ((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.y) * 1048576)) + (((int)threadIdx.z) * 262144)) + (k1_outer * 512)) + (ax0_ax1_fused_inner_inner * 256)) + (((int)threadIdx.x) * 8)));
    }
    for (int ax0_ax1_fused_inner_inner1 = 0; ax0_ax1_fused_inner_inner1 < 4; ++ax0_ax1_fused_inner_inner1) {
      *(int2*)(B_shared + ((((((int)threadIdx.y) * 4096) + (((int)threadIdx.z) * 1024)) + (ax0_ax1_fused_inner_inner1 * 256)) + (((int)threadIdx.x) * 8))) = *(int2*)(B + ((((((k1_outer * 524288) + (((int)threadIdx.y) * 262144)) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.z) * 1024)) + (ax0_ax1_fused_inner_inner1 * 256)) + (((int)threadIdx.x) * 8)));
    }
    __syncthreads();
    for (int k1_inner = 0; k1_inner < 2; ++k1_inner) {
      for (int ax0 = 0; ax0 < 4; ++ax0) {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0], (&(A_shared[(((((int)threadIdx.y) * 2048) + (ax0 * 512)) + (k1_inner * 256))])), 16);
      }
      for (int ax1 = 0; ax1 < 4; ++ax1) {
        nvcuda::wmma::load_matrix_sync(B_shared_wmma_matrix_b[ax1], (&(B_shared[(((k1_inner * 4096) + (((int)threadIdx.z) * 1024)) + (ax1 * 256))])), 16);
      }
      for (int i_c = 0; i_c < 4; ++i_c) {
        for (int j_c = 0; j_c < 4; ++j_c) {
          nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_c * 4) + j_c)], A_shared_wmma_matrix_a[i_c], B_shared_wmma_matrix_b[j_c], C_wmma_accumulator[((i_c * 4) + j_c)]);
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.x) * 2097152) + (((int)threadIdx.y) * 1048576)) + (i_inner * 262144)) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.z) * 1024)) + (j_inner * 256))])), C_wmma_accumulator[((i_inner * 4) + j_inner)], 16, nvcuda::wmma::mem_row_major);
    }
  }
}

