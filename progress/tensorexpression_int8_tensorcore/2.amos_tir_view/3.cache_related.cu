#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [16384, 16384], []),
             B: Buffer(B_1: Pointer(global int8), int8, [16384, 16384], []),
             C: Buffer(C_1: Pointer(global int32), int32, [16384, 16384], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_global = alloc_buffer(int8[16384, 16384])
    A_global_shared = alloc_buffer(int8[16384, 16384])
    B_global = alloc_buffer(int8[16384, 16384])
    B_global_shared = alloc_buffer(int8[16384, 16384])
    C_global = alloc_buffer(int32[16384, 16384])
     {
      for (ax0: int32, 0, 16384) {
        for (ax1: int32, 0, 16384) {
          block([16384, 16384], "B_global") as [v0, v1] {
            bind(v0, ax0)
            bind(v1, ax1)
            tir.reads([B[v0, v1]])
            tir.writes([B_global[v0, v1]])
            B_global[v0, v1] = B[v0, v1]
        }
      }
      for (ax0_1: int32, 0, 16384) {
        for (ax1_1: int32, 0, 16384) {
          block([16384, 16384], "A_global") as [v0_1, v1_1] {
            bind(v0_1, ax0_1)
            bind(v1_1, ax1_1)
            tir.reads([A[v0_1, v1_1]])
            tir.writes([A_global[v0_1, v1_1]])
            A_global[v0_1, v1_1] = A[v0_1, v1_1]
        }
      }
      for (ax0_2: int32, 0, 16384) {
        for (ax1_2: int32, 0, 16384) {
          block([16384, 16384], "A_global_shared") as [v0_2, v1_2] {
            bind(v0_2, ax0_2)
            bind(v1_2, ax1_2)
            tir.reads([A_global[v0_2, v1_2]])
            tir.writes([A_global_shared[v0_2, v1_2]])
            A_global_shared[v0_2, v1_2] = A_global[v0_2, v1_2]
        }
      }
      for (ax0_3: int32, 0, 16384) {
        for (ax1_3: int32, 0, 16384) {
          block([16384, 16384], "B_global_shared") as [v0_3, v1_3] {
            bind(v0_3, ax0_3)
            bind(v1_3, ax1_3)
            tir.reads([B_global[v0_3, v1_3]])
            tir.writes([B_global_shared[v0_3, v1_3]])
            B_global_shared[v0_3, v1_3] = B_global[v0_3, v1_3]
        }
      }
      for (i: int32, 0, 16384) {
        for (j: int32, 0, 16384) {
          for (k: int32, 0, 16384) {
            block([16384, 16384, tir.reduce_axis(0, 16384)], "B") as [vi, vj, vk] {
              bind(vi, i)
              bind(vj, j)
              bind(vk, k)
              tir.reads([A_global_shared[vi, vk], B_global_shared[vj, vk]])
              tir.writes([C_global[vi, vj]])
              with init() {
                C_global[vi, vj] = 0
              }
              C_global[vi, vj] = (C_global[vi, vj] + (cast(int32, A_global_shared[vi, vk])*cast(int32, B_global_shared[vj, vk])))
          }
        }
      }
      for (ax0_4: int32, 0, 16384) {
        for (ax1_4: int32, 0, 16384) {
          block([16384, 16384], "C_global") as [v0_4, v1_4] {
            bind(v0_4, ax0_4)
            bind(v1_4, ax1_4)
            tir.reads([C_global[v0_4, v1_4]])
            tir.writes([C[v0_4, v1_4]])
            C[v0_4, v1_4] = C_global[v0_4, v1_4]
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}