#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#include <hip/hip_runtime.h>
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> C_wmma_accumulator[16];
  __shared__ signed char A_shared[4096];
  __shared__ signed char B_shared[16384];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> B_shared_wmma_matrix_b[8];
  for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
      nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_c_init * 8) + j_c_init)], 0.000000e+00f);
    }
  }
  for (int k1_outer = 0; k1_outer < 256; ++k1_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer) {
      *(int4*)(A_shared + ((((((int)threadIdx.y) * 2048) + (((int)threadIdx.z) * 1024)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(A + ((((((((int)blockIdx.x) * 1048576) + (((int)threadIdx.y) * 524288)) + (((int)threadIdx.z) * 262144)) + (k1_outer * 1024)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer * 512)) + (((int)threadIdx.x) * 16)));
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1) {
      *(int4*)(B_shared + ((((((int)threadIdx.y) * 8192) + (((int)threadIdx.z) * 4096)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1 * 512)) + (((int)threadIdx.x) * 16))) = *(int4*)(B + (((((((((int)blockIdx.y) * 4194304) + (((int)threadIdx.y) * 2097152)) + (((int)threadIdx.z) * 1048576)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1 >> 1) * 262144)) + (k1_outer * 1024)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_outer_1 & 1) * 512)) + (((int)threadIdx.x) * 16)));
    }
    __syncthreads();
    for (int k1_inner = 0; k1_inner < 4; ++k1_inner) {
      for (int ax0 = 0; ax0 < 2; ++ax0) {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0], (&(A_shared[(((((int)threadIdx.y) * 2048) + (ax0 * 1024)) + (k1_inner * 256))])), 16);
      }
      for (int ax0_1 = 0; ax0_1 < 8; ++ax0_1) {
        nvcuda::wmma::load_matrix_sync(B_shared_wmma_matrix_b[ax0_1], (&(B_shared[(((((int)threadIdx.z) * 8192) + (ax0_1 * 1024)) + (k1_inner * 256))])), 16);
      }
      for (int i_c = 0; i_c < 2; ++i_c) {
        for (int j_c = 0; j_c < 8; ++j_c) {
          nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_c * 8) + j_c)], A_shared_wmma_matrix_a[i_c], B_shared_wmma_matrix_b[j_c], C_wmma_accumulator[((i_c * 8) + j_c)]);
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.x) * 1048576) + (((int)threadIdx.y) * 524288)) + (i_inner * 262144)) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.z) * 2048)) + (j_inner * 256))])), C_wmma_accumulator[((i_inner * 8) + j_inner)], 16, nvcuda::wmma::mem_row_major);
    }
  }
}

