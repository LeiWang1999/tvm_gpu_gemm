@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [1024, 1024, 16, 16], []), B_1: B_3: Buffer(B_2, int8, [1024, 1024, 16, 16], []), C_1: C_3: Buffer(C_2, int32, [1024, 1024, 16, 16], [])} {
  allocate(A.shared: Pointer(shared int8), int8, [268435456]), storage_scope = shared;
  allocate(A.shared.wmma.matrix_a: Pointer(wmma.matrix_a int8), int8, [268435456]), storage_scope = wmma.matrix_a;
  allocate(B.shared.wmma.matrix_b: Pointer(wmma.matrix_b int8), int8, [268435456]), storage_scope = wmma.matrix_b;
  allocate(C.wmma.accumulator: Pointer(wmma.accumulator int32), int32, [268435456]), storage_scope = wmma.accumulator {
    for (ax0: int32, 0, 1024) {
      for (ax1: int32, 0, 1024) {
        for (ax2: int32, 0, 16) {
          for (ax3: int32, 0, 16) {
            let cse_var_1: int32 = ((((ax0*262144) + (ax1*256)) + (ax2*16)) + ax3)
            A.shared_1: Buffer(A.shared, int8, [268435456], [], scope="shared")[cse_var_1] = A[cse_var_1]
          }
        }
      }
    }
    for (ax0_1: int32, 0, 1024) {
      for (ax1_1: int32, 0, 1024) {
        for (ax2_1: int32, 0, 16) {
          for (ax3_1: int32, 0, 16) {
            let cse_var_2: int32 = ((((ax0_1*262144) + (ax1_1*256)) + (ax2_1*16)) + ax3_1)
            A.shared.wmma.matrix_a_1: Buffer(A.shared.wmma.matrix_a, int8, [268435456], [], scope="wmma.matrix_a")[cse_var_2] = A.shared_1[cse_var_2]
          }
        }
      }
    }
    for (ax0_2: int32, 0, 1024) {
      for (ax1_2: int32, 0, 1024) {
        for (ax2_2: int32, 0, 16) {
          for (ax3_2: int32, 0, 16) {
            let cse_var_3: int32 = ((((ax0_2*262144) + (ax1_2*256)) + (ax2_2*16)) + ax3_2)
            A.shared_2: Buffer(A.shared, int8, [268435456], [], scope="shared")[cse_var_3] = B[cse_var_3]
          }
        }
      }
    }
    for (ax0_3: int32, 0, 1024) {
      for (ax1_3: int32, 0, 1024) {
        for (ax2_3: int32, 0, 16) {
          for (ax3_3: int32, 0, 16) {
            let cse_var_4: int32 = ((((ax0_3*262144) + (ax1_3*256)) + (ax2_3*16)) + ax3_3)
            B.shared.wmma.matrix_b_1: Buffer(B.shared.wmma.matrix_b, int8, [268435456], [], scope="wmma.matrix_b")[cse_var_4] = A.shared_2[cse_var_4]
          }
        }
      }
    }
    for (i.c: int32, 0, 1024) {
      for (j.c: int32, 0, 1024) {
        for (ii.c: int32, 0, 16) {
          for (jj.c: int32, 0, 16) {
            C.wmma.accumulator_1: Buffer(C.wmma.accumulator, int32, [268435456], [], scope="wmma.accumulator")[((((i.c*262144) + (j.c*256)) + (ii.c*16)) + jj.c)] = 0
            for (k1: int32, 0, 1024) {
              for (k2: int32, 0, 16) {
                let cse_var_8: int32 = (i.c*262144)
                let cse_var_7: int32 = (ii.c*16)
                let cse_var_6: int32 = (k1*256)
                let cse_var_5: int32 = (((cse_var_8 + (j.c*256)) + cse_var_7) + jj.c)
                C.wmma.accumulator_1[cse_var_5] = (C.wmma.accumulator_1[cse_var_5] + (cast(int32, A.shared.wmma.matrix_a_1[(((cse_var_8 + cse_var_6) + cse_var_7) + k2)])*cast(int32, B.shared.wmma.matrix_b_1[((((j.c*262144) + cse_var_6) + (jj.c*16)) + k2)])))
              }
            }
          }
        }
      }
    }
    for (i: int32, 0, 1024) {
      for (j: int32, 0, 1024) {
        for (ii: int32, 0, 16) {
          for (jj: int32, 0, 16) {
            let cse_var_9: int32 = ((((i*262144) + (j*256)) + (ii*16)) + jj)
            C[cse_var_9] = C.wmma.accumulator_1[cse_var_9]
          }
        }
      }
    }
  }
}

