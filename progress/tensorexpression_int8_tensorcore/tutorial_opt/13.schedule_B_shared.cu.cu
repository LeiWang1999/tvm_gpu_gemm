#[version = "0.0.5"]
@main = primfn(A_1: handle, W_1: handle, Conv_1: handle) -> ()
  attr = {"from_legacy_te_schedule": True, "global_symbol": "main", "tir.noalias": True}
  buffers = {A: Buffer(A_2: Pointer(float16), float16, [16, 14, 14, 16, 16, 16], []),
             W: Buffer(W_2: Pointer(float16), float16, [3, 3, 16, 32, 16, 16], []),
             Conv: Buffer(Conv_2: Pointer(float32), float32, [16, 14, 14, 32, 16, 16], [])}
  buffer_map = {A_1: A, W_1: W, Conv_1: Conv} {
  attr [IterVar(blockIdx.z: int32, (nullptr), "ThreadIndex", "blockIdx.z")] "thread_extent" = 196;
  allocate(Conv.wmma.accumulator: Pointer(wmma.accumulator float32), float32, [2048]), storage_scope = wmma.accumulator;
  allocate(Apad.shared: Pointer(shared float16), float16, [12288]), storage_scope = shared;
  allocate(W.shared: Pointer(shared float16), float16, [12288]), storage_scope = shared;
  allocate(Apad.shared.wmma.matrix_a: Pointer(wmma.matrix_a float16), float16, [512]), storage_scope = wmma.matrix_a;
  allocate(W.shared.wmma.matrix_b: Pointer(wmma.matrix_b float16), float16, [1024]), storage_scope = wmma.matrix_b;
  attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 2;
  attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 4;
  attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 4;
  attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 2 {
    for (n.c.init: int32, 0, 2) {
      for (o.c.init: int32, 0, 4) {
        for (nn.c.init: int32, 0, 16) {
          for (oo.c.init: int32, 0, 16) {
            Conv.wmma.accumulator_1: Buffer(Conv.wmma.accumulator, float32, [2048], [], scope="wmma.accumulator")[((((n.c.init*1024) + (o.c.init*256)) + (nn.c.init*16)) + oo.c.init)] = 0f32
          }
        }
      }
    }
    for (ic.outer: int32, 0, 8) {
      for (kh: int32, 0, 3) {
        for (ax2: int32, 0, 3) {
          for (ax3: int32, 0, 2) {
            for (ax4.ax5.fused.outer: int32, 0, 8) {
              let cse_var_2: int32 = (ax3*256)
              let cse_var_1: int32 = (ax4.ax5.fused.outer*32)
              attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
              Apad.shared_1: Buffer(Apad.shared, float16, [12288], [], scope="shared")[((((((threadIdx.y*3072) + (threadIdx.z*1536)) + (ax2*512)) + cse_var_2) + cse_var_1) + threadIdx.x)] = @tir.if_then_else(((((1 <= (floordiv(blockIdx.z, 14) + kh)) && ((floordiv(blockIdx.z, 14) + kh) < 15)) && (1 <= (ax2 + floormod(blockIdx.z, 14)))) && ((ax2 + floormod(blockIdx.z, 14)) < 15)), A_3: Buffer(A_2, float16, [12845056], [])[(((((((((((blockIdx.x*6422528) + (threadIdx.y*1605632)) + (threadIdx.z*802816)) + (kh*57344)) + (blockIdx.z*4096)) + (ax2*4096)) + (ic.outer*512)) + cse_var_2) + cse_var_1) + threadIdx.x) - 61440)], 0f16, dtype=float16)
            }
          }
        }
        for (ax1: int32, 0, 3) {
          for (ax2_1: int32, 0, 2) {
            attr [IterVar(threadIdx.x, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
            W.shared_1: Buffer(W.shared, float16, [12288], [], scope="shared")[ramp((((((ax1*4096) + (ax2_1*2048)) + (threadIdx.y*512)) + (threadIdx.z*256)) + (threadIdx.x*8)), 1, 8)] = W_3: Buffer(W_2, float16, [1179648], [])[ramp(((((((((kh*393216) + (ax1*131072)) + (ic.outer*16384)) + (ax2_1*8192)) + (blockIdx.y*2048)) + (threadIdx.y*512)) + (threadIdx.z*256)) + (threadIdx.x*8)), 1, 8)]
          }
        }
        for (ic.inner: int32, 0, 2) {
          for (kw: int32, 0, 3) {
            for (ax0: int32, 0, 2) {
              for (ax4: int32, 0, 16) {
                for (ax5: int32, 0, 16) {
                  let cse_var_3: int32 = (ax4*16)
                  Apad.shared.wmma.matrix_a_1: Buffer(Apad.shared.wmma.matrix_a, float16, [512], [], scope="wmma.matrix_a")[(((ax0*256) + cse_var_3) + ax5)] = Apad.shared_1[((((((threadIdx.y*3072) + (ax0*1536)) + (kw*512)) + (ic.inner*256)) + cse_var_3) + ax5)]
                }
              }
            }
            for (ax3_1: int32, 0, 4) {
              for (ax4_1: int32, 0, 16) {
                for (ax5_1: int32, 0, 16) {
                  let cse_var_5: int32 = (ax3_1*256)
                  let cse_var_4: int32 = (ax4_1*16)
                  W.shared.wmma.matrix_b_1: Buffer(W.shared.wmma.matrix_b, float16, [1024], [], scope="wmma.matrix_b")[((cse_var_5 + cse_var_4) + ax5_1)] = W.shared_1[((((((kw*4096) + (ic.inner*2048)) + (threadIdx.z*1024)) + cse_var_5) + cse_var_4) + ax5_1)]
                }
              }
            }
            for (n.c: int32, 0, 2) {
              for (o.c: int32, 0, 4) {
                for (nn.c: int32, 0, 16) {
                  for (oo.c: int32, 0, 16) {
                    for (ii: int32, 0, 16) {
                      let cse_var_8: int32 = (o.c*256)
                      let cse_var_7: int32 = (nn.c*16)
                      let cse_var_6: int32 = ((((n.c*1024) + cse_var_8) + cse_var_7) + oo.c)
                      Conv.wmma.accumulator_1[cse_var_6] = (Conv.wmma.accumulator_1[cse_var_6] + (cast(float32, Apad.shared.wmma.matrix_a_1[(((n.c*256) + cse_var_7) + ii)])*cast(float32, W.shared.wmma.matrix_b_1[((cse_var_8 + (ii*16)) + oo.c)])))
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
    for (n.inner: int32, 0, 2) {
      for (o.inner: int32, 0, 4) {
        for (nn: int32, 0, 16) {
          for (oo: int32, 0, 16) {
            let cse_var_10: int32 = (o.inner*256)
            let cse_var_9: int32 = (nn*16)
            Conv_3: Buffer(Conv_2, float32, [25690112], [])[(((((((((blockIdx.x*12845056) + (threadIdx.y*3211264)) + (n.inner*1605632)) + (blockIdx.z*8192)) + (blockIdx.y*2048)) + (threadIdx.z*1024)) + cse_var_10) + cse_var_9) + oo)] = Conv.wmma.accumulator_1[((((n.inner*1024) + cse_var_10) + cse_var_9) + oo)]
          }
        }
      }
    }
  }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3, 4]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}