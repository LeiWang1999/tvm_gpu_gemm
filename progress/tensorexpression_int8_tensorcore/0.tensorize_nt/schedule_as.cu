@main = primfn(A_1: handle, B_1: handle, C_1: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main", "from_legacy_te_schedule": True}
  buffers = {A: Buffer(A_2: Pointer(int8), int8, [268435456], []),
             B: Buffer(B_2: Pointer(int8), int8, [268435456], []),
             C: Buffer(C_2: Pointer(int32), int32, [268435456], [])}
  buffer_map = {A_1: A, B_1: B, C_1: C}
  preflattened_buffer_map = {A_1: A_3: Buffer(A_2, int8, [16384, 16384], []), B_1: B_3: Buffer(B_2, int8, [16384, 16384], []), C_1: C_3: Buffer(C_2, int32, [16384, 16384], [])} {
  allocate(A.global: Pointer(global int8), int8, [268435456]), storage_scope = global;
  allocate(B.global: Pointer(global int8), int8, [4194304]), storage_scope = global;
  allocate(B.global.shared: Pointer(shared int8), int8, [4194304]), storage_scope = shared;
  allocate(C.wmma.accumulator.global: Pointer(global int32), int32, [268435456]), storage_scope = global {
    for (axis0: int32, 0, 1024) {
      for (axis1: int32, 0, 1024) {
        for (axis2: int32, 0, 16) {
          for (axis3: int32, 0, 16) {
            let cse_var_1: int32 = (axis0*262144)
            A.global_1: Buffer(A.global, int8, [268435456], [])[(((cse_var_1 + (axis1*256)) + (axis2*16)) + axis3)] = A[(((cse_var_1 + (axis2*16384)) + (axis1*16)) + axis3)]
          }
        }
      }
    }
    if @tir.likely((blockIdx.y: int32 < 64), dtype=bool) {
      for (axis0.idx: int32, 0, 16) {
        for (axis1_1: int32, 0, 1024) {
          for (axis2_1: int32, 0, 16) {
            if @tir.likely((0 <= (((blockIdx.y*256) + (axis0.idx*16)) + axis2_1)), dtype=bool) {
              for (axis3_1: int32, 0, 16) {
                let cse_var_2: int32 = (axis0.idx*262144)
                B.global_1: Buffer(B.global, int8, [4194304], [])[(((cse_var_2 + (axis1_1*256)) + (axis2_1*16)) + axis3_1)] = B[(((((blockIdx.y*4194304) + cse_var_2) + (axis2_1*16384)) + (axis1_1*16)) + axis3_1)]
              }
            }
          }
        }
      }
    }
    for (ax0: int32, 0, 256) {
      for (ax1: int32, 0, 16384) {
        B.global.shared_1: Buffer(B.global.shared, int8, [4194304], [], scope="shared")[((ax0*16384) + ax1)] = B.global_1[((((floordiv(ax0, 16)*262144) + (floordiv(ax1, 16)*256)) + (floormod(ax0, 16)*16)) + floormod(ax1, 16))]
      }
    }
    attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 256;
    allocate(C.wmma.accumulator: Pointer(wmma.accumulator int32), int32, [4096]), storage_scope = wmma.accumulator;
    allocate(A.global.shared: Pointer(shared int8), int8, [4096]), storage_scope = shared;
    allocate(A.global.shared.wmma.matrix_a: Pointer(wmma.matrix_a int8), int8, [512]), storage_scope = wmma.matrix_a;
    allocate(B.global.shared.wmma.matrix_b: Pointer(wmma.matrix_b int8), int8, [2048]), storage_scope = wmma.matrix_b;
    attr [IterVar(blockIdx.y, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 64;
    attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 2;
    attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 2 {
      for (ii.c.outer.init: int32, 0, 2) {
        for (jj.c.outer.init: int32, 0, 8) {
          for (ii.c.inner.init: int32, 0, 16) {
            for (jj.c.inner.init: int32, 0, 16) {
              C.wmma.accumulator_1: Buffer(C.wmma.accumulator, int32, [4096], [], scope="wmma.accumulator")[((((ii.c.outer.init*2048) + (ii.c.inner.init*128)) + (jj.c.outer.init*16)) + jj.c.inner.init)] = 0
            }
          }
        }
      }
      for (rk.outer.outer: int32, 0, 256) {
        for (ax0.ax1.fused.inner.inner.outer: int32, 0, 2) {
          attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
          A.global.shared_1: Buffer(A.global.shared, int8, [4096], [], scope="shared")[ramp(((((threadIdx.y*2048) + (threadIdx.z*1024)) + (ax0.ax1.fused.inner.inner.outer*512)) + (threadIdx.x*16)), 1, 16)] = A.global_1[ramp((((((((blockIdx.x*1048576) + (threadIdx.y*524288)) + (threadIdx.z*262144)) + (rk.outer.outer*1024)) + (floormod(threadIdx.x, 4)*256)) + (ax0.ax1.fused.inner.inner.outer*128)) + (floordiv(threadIdx.x, 4)*16)), 1, 16)]
        }
        for (rk.outer.inner: int32, 0, 4) {
          for (ax0.outer: int32, 0, 2) {
            for (ax0.inner: int32, 0, 16) {
              for (ax1.inner: int32, 0, 16) {
                A.global.shared.wmma.matrix_a_1: Buffer(A.global.shared.wmma.matrix_a, int8, [512], [], scope="wmma.matrix_a")[(((ax0.outer*256) + (ax0.inner*16)) + ax1.inner)] = A.global.shared_1[(((((threadIdx.y*2048) + (ax0.outer*1024)) + (ax0.inner*64)) + (rk.outer.inner*16)) + ax1.inner)]
              }
            }
          }
          for (ax0.outer_1: int32, 0, 8) {
            for (ax0.inner_1: int32, 0, 16) {
              for (ax1.inner_1: int32, 0, 16) {
                B.global.shared.wmma.matrix_b_1: Buffer(B.global.shared.wmma.matrix_b, int8, [2048], [], scope="wmma.matrix_b")[(((ax0.outer_1*256) + (ax0.inner_1*16)) + ax1.inner_1)] = B.global.shared_1[((((((threadIdx.z*2097152) + (ax0.outer_1*262144)) + (ax0.inner_1*16384)) + (rk.outer.outer*64)) + (rk.outer.inner*16)) + ax1.inner_1)]
              }
            }
          }
          for (ii.c.outer: int32, 0, 2) {
            for (jj.c.outer: int32, 0, 8) {
              for (ii.c.inner: int32, 0, 16) {
                for (jj.c.inner: int32, 0, 16) {
                  for (rk.inner: int32, 0, 16) {
                    let cse_var_3: int32 = ((((ii.c.outer*2048) + (ii.c.inner*128)) + (jj.c.outer*16)) + jj.c.inner)
                    C.wmma.accumulator_1[cse_var_3] = (C.wmma.accumulator_1[cse_var_3] + (cast(int32, A.global.shared.wmma.matrix_a_1[(((ii.c.outer*256) + (ii.c.inner*16)) + rk.inner)])*cast(int32, B.global.shared.wmma.matrix_b_1[(((jj.c.outer*256) + (jj.c.inner*16)) + rk.inner)])))
                  }
                }
              }
            }
          }
        }
      }
      for (axis0.inner: int32, 0, 2) {
        for (axis1.inner: int32, 0, 8) {
          for (axis2_2: int32, 0, 16) {
            for (axis3_2: int32, 0, 16) {
              C.wmma.accumulator.global_1: Buffer(C.wmma.accumulator.global, int32, [268435456], [])[((((((((blockIdx.x*1048576) + (threadIdx.y*524288)) + (axis0.inner*262144)) + (blockIdx.y*4096)) + (threadIdx.z*2048)) + (axis1.inner*256)) + (axis2_2*16)) + axis3_2)] = C.wmma.accumulator_1[((((axis0.inner*2048) + (axis2_2*128)) + (axis1.inner*16)) + axis3_2)]
            }
          }
        }
      }
    }
    for (ii: int32, 0, 16384) {
      for (jj: int32, 0, 16384) {
        C[((ii*16384) + jj)] = C.wmma.accumulator.global_1[((((floordiv(ii, 16)*262144) + (floordiv(jj, 16)*256)) + (floormod(ii, 16)*16)) + floormod(jj, 16))]
      }
    }
  }
}

