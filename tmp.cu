
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel0(float* __restrict__ C, float* __restrict__ A, float* __restrict__ B) {
  C[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 1024; ++k) {
    C[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (C[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (A[((k * 1024) + ((int)threadIdx.x))] * B[((k * 1024) + ((int)blockIdx.x))]));
  }
}

