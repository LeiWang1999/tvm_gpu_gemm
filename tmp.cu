
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  float C_local[1];
  C_local[0] = 0.000000e+00f;
  for (int k = 0; k < 16384; ++k) {
    C_local[0] = (C_local[0] + (A[(((k * 16384) + (((int)blockIdx.y) * 32)) + ((int)threadIdx.y))] * B[(((k * 16384) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x))]));
  }
  C[((((((int)blockIdx.x) * 524288) + (((int)threadIdx.x) * 16384)) + (((int)blockIdx.y) * 32)) + ((int)threadIdx.y))] = C_local[0];
}

