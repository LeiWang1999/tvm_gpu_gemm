
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif


#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) main_kernel0(signed char* __restrict__ A, signed char* __restrict__ B, int* __restrict__ C) {
  int in_thread_C[1];
  signed char A_local[12];
  signed char B_local[12];
  int red_buf0[1];
  for (int i_2 = 0; i_2 < 3; ++i_2) {
    in_thread_C[0] = 0;
    for (int k_0 = 0; k_0 < 2; ++k_0) {
      for (int ax0_ax1_fused_1_s = 0; ax0_ax1_fused_1_s < 12; ++ax0_ax1_fused_1_s) {
        if (((k_0 * 3) + (((((int)threadIdx.x) * 3) + (ax0_ax1_fused_1_s >> 2)) >> 1)) < 4) {
          A_local[ax0_ax1_fused_1_s] = A[(((((((((int)blockIdx.x) * 12288) + (((int)threadIdx.z) * 1536)) + (i_2 * 512)) + (((int)threadIdx.y) * 32)) + (k_0 * 24)) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_1_s)];
        }
      }
      for (int ax0_ax1_fused_1_s_1 = 0; ax0_ax1_fused_1_s_1 < 12; ++ax0_ax1_fused_1_s_1) {
        if (((k_0 * 3) + (((((int)threadIdx.x) * 3) + (ax0_ax1_fused_1_s_1 >> 2)) >> 1)) < 4) {
          B_local[ax0_ax1_fused_1_s_1] = B[(((k_0 * 24) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_1_s_1)];
        }
      }
      for (int k_2 = 0; k_2 < 3; ++k_2) {
        for (int k_3 = 0; k_3 < 4; ++k_3) {
          if (((((k_0 * 24) + (((int)threadIdx.x) * 12)) + (k_2 * 4)) + k_3) < 32) {
            in_thread_C[0] = (in_thread_C[0] + (((int)A_local[((k_2 * 4) + k_3)]) * ((int)B_local[((k_2 * 4) + k_3)])));
          }
        }
      }
    }
    uint mask[1];
    int t0[1];
    red_buf0[0] = in_thread_C[0];
    mask[0] = (__activemask() & ((uint)(3 << ((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 2)))));
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], ((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 2)), 32);
    C[((((((int)blockIdx.x) * 384) + (((int)threadIdx.z) * 48)) + (i_2 * 16)) + ((int)threadIdx.y))] = red_buf0[0];
  }
}

