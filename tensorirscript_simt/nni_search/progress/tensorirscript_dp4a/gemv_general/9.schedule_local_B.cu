#[version = "0.0.5"]
@main = primfn(a: handle, b: handle, c: handle) -> ()
  attr = {"tir.noalias": True, "global_symbol": "main"}
  buffers = {A: Buffer(A_1: Pointer(global int8), int8, [18966528, 32], []),
             B: Buffer(B_1: Pointer(global int8), int8, [1, 32], []),
             C: Buffer(C_1: Pointer(global int32), int32, [18966528, 1], [])}
  buffer_map = {a: A, b: B, c: C} {
  block([], "root") {
    tir.reads([])
    tir.writes([])
    A_local = alloc_buffer(int8[18966528, 32])
    B_local = alloc_buffer(int8[1, 32])
    for (i_0: int32, 0, 49392) "thread_binding" {
      for (i_1: int32, 0, 8) "thread_binding" {
        for (i_2: int32, 0, 3) {
          for (i_3: int32, 0, 16) "thread_binding" {
            for (j: int32, 0, 1) {
              for (k_0: int32, 0, 2) {
                for (k_1: int32, 0, 2) "thread_binding" {
                  for (ax0_ax1_fused_0: int32, 0, 1) {
                    for (ax0_ax1_fused_1: int32, 0, 12) "vectorized" {
                      block([18966528, 32], "A_local") as [v0, v1] {
                        where(((((k_0*24) + (k_1*12)) + floormod(((ax0_ax1_fused_0*12) + ax0_ax1_fused_1), 12)) < 32))
                        bind(v0, ((((i_0*384) + (i_1*48)) + (i_2*16)) + i_3))
                        bind(v1, ((((k_0*24) + (k_1*12)) + (ax0_ax1_fused_0*12)) + ax0_ax1_fused_1))
                        tir.reads([A[v0, v1]])
                        tir.writes([A_local[v0, v1]])
                        A_local[v0, v1] = A[v0, v1]
                    }
                  }
                  for (ax0_ax1_fused_0_1: int32, 0, 1) {
                    for (ax0_ax1_fused_1_1: int32, 0, 12) "vectorized" {
                      block([1, 32], "B_local") as [v0_1, v1_1] {
                        where(((((k_0*24) + (k_1*12)) + floormod(((ax0_ax1_fused_0_1*12) + ax0_ax1_fused_1_1), 12)) < 32))
                        bind(v0_1, 0)
                        bind(v1_1, ((((k_0*24) + (k_1*12)) + (ax0_ax1_fused_0_1*12)) + ax0_ax1_fused_1_1))
                        tir.reads([B[v0_1, v1_1]])
                        tir.writes([B_local[v0_1, v1_1]])
                        B_local[v0_1, v1_1] = B[v0_1, v1_1]
                    }
                  }
                  for (k_2: int32, 0, 3) {
                    for (k_3: int32, 0, 4) {
                      block([18966528, 1, tir.reduce_axis(0, 32)], "B") as [vi, vj, vk] {
                        where((((((((k_0*2) + k_1)*3) + k_2)*4) + k_3) < 32))
                        bind(vi, ((((i_0*384) + (i_1*48)) + (i_2*16)) + i_3))
                        bind(vj, j)
                        bind(vk, ((((k_0*24) + (k_1*12)) + (k_2*4)) + k_3))
                        tir.reads([A_local[vi, vk], B_local[vj, vk]])
                        tir.writes([C[vi, vj]])
                        with init() {
                          C[vi, vj] = 0
                        }
                        C[vi, vj] = (C[vi, vj] + (cast(int32, A_local[vi, vk])*cast(int32, B_local[vj, vk])))
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
}

#[metadata]
{
  "root": 1, 
  "nodes": [
    {
      "type_key": ""
    }, 
    {
      "type_key": "Map", 
      "keys": [
        "IntImm"
      ], 
      "data": [2]
    }, 
    {
      "type_key": "Array", 
      "data": [3]
    }, 
    {
      "type_key": "IntImm", 
      "attrs": {
        "dtype": "bool", 
        "span": "0", 
        "value": "1"
      }
    }
  ], 
  "b64ndarrays": [], 
  "attrs": {"tvm_version": "0.11.dev0"}
}